
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[2];
  __shared__ float pad_temp_shared[18];
  __shared__ float kernel_shared[9216];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 18) {
      pad_temp_shared[((int)threadIdx.x)] = (((1 <= (((((int)blockIdx.x) / 7) * 2) + ((((int)threadIdx.x) % 9) / 3))) && (1 <= (((((int)blockIdx.x) % 7) * 2) + (((int)threadIdx.x) % 3)))) ? data[(((((((rc_outer_outer * 392) + ((((int)threadIdx.x) / 9) * 196)) + ((((int)blockIdx.x) / 7) * 28)) + (((((int)threadIdx.x) % 9) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) % 3)) - 15)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 18) * 2304) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) + 256) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) + 512) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) + 768) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) + 1024) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) + 1280) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) + 1536) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) + 1792) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) + 2048) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((int)threadIdx.x) / 18) * 2304) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18)) + 294912)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) + 2560) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)threadIdx.x) + 2816) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) + 3072) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)threadIdx.x) + 3328) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) + 3584) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)threadIdx.x) + 3840) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) + 4096) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)threadIdx.x) + 4352) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((int)threadIdx.x) / 18) * 2304) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18)) + 589824)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[(((((((int)threadIdx.x) + 4864) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)threadIdx.x) + 5120) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[(((((((int)threadIdx.x) + 5376) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)threadIdx.x) + 5632) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[(((((((int)threadIdx.x) + 5888) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)threadIdx.x) + 6144) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[(((((((int)threadIdx.x) + 6400) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)threadIdx.x) + 6656) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[(((((((int)threadIdx.x) / 18) * 2304) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18)) + 884736)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)threadIdx.x) + 7168) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[(((((((int)threadIdx.x) + 7424) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)threadIdx.x) + 7680) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[(((((((int)threadIdx.x) + 7936) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 8192)] = kernel[(((((((int)threadIdx.x) + 8192) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 8448)] = kernel[(((((((int)threadIdx.x) + 8448) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 8704)] = kernel[(((((((int)threadIdx.x) + 8704) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 8960)] = kernel[(((((((int)threadIdx.x) + 8960) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.x) * 36)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.x) * 36) + 18)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[1] * kernel_shared[((((int)threadIdx.x) * 36) + 1)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[1] * kernel_shared[((((int)threadIdx.x) * 36) + 19)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.x) * 36) + 2)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.x) * 36) + 20)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[3] * kernel_shared[((((int)threadIdx.x) * 36) + 3)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[3] * kernel_shared[((((int)threadIdx.x) * 36) + 21)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.x) * 36) + 4)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.x) * 36) + 22)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[5] * kernel_shared[((((int)threadIdx.x) * 36) + 5)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[5] * kernel_shared[((((int)threadIdx.x) * 36) + 23)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.x) * 36) + 6)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.x) * 36) + 24)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[7] * kernel_shared[((((int)threadIdx.x) * 36) + 7)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[7] * kernel_shared[((((int)threadIdx.x) * 36) + 25)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.x) * 36) + 8)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.x) * 36) + 26)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.x) * 36) + 9)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.x) * 36) + 27)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[10] * kernel_shared[((((int)threadIdx.x) * 36) + 10)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[10] * kernel_shared[((((int)threadIdx.x) * 36) + 28)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.x) * 36) + 11)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.x) * 36) + 29)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[12] * kernel_shared[((((int)threadIdx.x) * 36) + 12)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[12] * kernel_shared[((((int)threadIdx.x) * 36) + 30)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.x) * 36) + 13)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.x) * 36) + 31)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[14] * kernel_shared[((((int)threadIdx.x) * 36) + 14)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[14] * kernel_shared[((((int)threadIdx.x) * 36) + 32)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.x) * 36) + 15)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.x) * 36) + 33)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[16] * kernel_shared[((((int)threadIdx.x) * 36) + 16)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[16] * kernel_shared[((((int)threadIdx.x) * 36) + 34)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.x) * 36) + 17)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.x) * 36) + 35)]));
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    conv2d_nchw[(((((int)threadIdx.x) * 98) + (ff_inner * 49)) + ((int)blockIdx.x))] = conv2d_nchw_local[ff_inner];
  }
}


