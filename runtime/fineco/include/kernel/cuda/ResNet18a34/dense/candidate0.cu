
#include <hip/hip_runtime.h>
#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(50) candidate0(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[1];
  __shared__ float data_shared[196];
  __shared__ float weight_shared[9800];
  T_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 49) {
      *(float4*)(data_shared + (((int)threadIdx.x) * 4)) = *(float4*)(data + ((k_outer_outer * 196) + (((int)threadIdx.x) * 4)));
    }
    *(float4*)(weight_shared + (((int)threadIdx.x) * 4)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((int)threadIdx.x) / 49) * 25088)) + (k_outer_outer * 196)) + ((((int)threadIdx.x) % 49) * 4)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 4) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 8) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 12) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 16) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 1000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 20) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 1200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 24) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 1400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 28) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 1600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 32) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 1800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 36) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 2000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 40) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 2200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 44) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 2400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 48) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 2600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 52) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 2800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 56) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 3000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 60) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 3200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 64) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 3400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 68) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 3600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 72) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 3800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 76) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 4000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 80) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 4200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 84) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 4400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 88) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 4600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 92) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 4800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 96) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 5000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 100) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 5200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 104) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 5400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 108) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 5600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 112) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 5800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 116) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 6000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 120) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 6200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 124) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 6400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 128) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 6600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 132) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 6800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 136) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 7000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 140) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 7200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 144) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 7400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 148) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 7600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 152) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 7800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 156) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 8000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 8000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 160) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 8200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 8200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 164) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 8400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 8400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 168) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 8600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 8600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 172) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 8800)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 8800) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 176) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 9000)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 9000) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 180) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 9200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 9200) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 184) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 9400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 9400) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 188) % 196)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 9600)) = *(float4*)(weight + ((((((int)blockIdx.x) * 1254400) + ((((((int)threadIdx.x) * 4) + 9600) / 196) * 25088)) + (k_outer_outer * 196)) + (((((int)threadIdx.x) * 4) + 192) % 196)));
    __syncthreads();
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[0] * weight_shared[(((int)threadIdx.x) * 196)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[1] * weight_shared[((((int)threadIdx.x) * 196) + 1)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[2] * weight_shared[((((int)threadIdx.x) * 196) + 2)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[3] * weight_shared[((((int)threadIdx.x) * 196) + 3)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[4] * weight_shared[((((int)threadIdx.x) * 196) + 4)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[5] * weight_shared[((((int)threadIdx.x) * 196) + 5)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[6] * weight_shared[((((int)threadIdx.x) * 196) + 6)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[7] * weight_shared[((((int)threadIdx.x) * 196) + 7)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[8] * weight_shared[((((int)threadIdx.x) * 196) + 8)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[9] * weight_shared[((((int)threadIdx.x) * 196) + 9)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[10] * weight_shared[((((int)threadIdx.x) * 196) + 10)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[11] * weight_shared[((((int)threadIdx.x) * 196) + 11)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[12] * weight_shared[((((int)threadIdx.x) * 196) + 12)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[13] * weight_shared[((((int)threadIdx.x) * 196) + 13)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[14] * weight_shared[((((int)threadIdx.x) * 196) + 14)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[15] * weight_shared[((((int)threadIdx.x) * 196) + 15)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[16] * weight_shared[((((int)threadIdx.x) * 196) + 16)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[17] * weight_shared[((((int)threadIdx.x) * 196) + 17)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[18] * weight_shared[((((int)threadIdx.x) * 196) + 18)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[19] * weight_shared[((((int)threadIdx.x) * 196) + 19)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[20] * weight_shared[((((int)threadIdx.x) * 196) + 20)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[21] * weight_shared[((((int)threadIdx.x) * 196) + 21)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[22] * weight_shared[((((int)threadIdx.x) * 196) + 22)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[23] * weight_shared[((((int)threadIdx.x) * 196) + 23)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[24] * weight_shared[((((int)threadIdx.x) * 196) + 24)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[25] * weight_shared[((((int)threadIdx.x) * 196) + 25)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[26] * weight_shared[((((int)threadIdx.x) * 196) + 26)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[27] * weight_shared[((((int)threadIdx.x) * 196) + 27)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[28] * weight_shared[((((int)threadIdx.x) * 196) + 28)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[29] * weight_shared[((((int)threadIdx.x) * 196) + 29)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[30] * weight_shared[((((int)threadIdx.x) * 196) + 30)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[31] * weight_shared[((((int)threadIdx.x) * 196) + 31)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[32] * weight_shared[((((int)threadIdx.x) * 196) + 32)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[33] * weight_shared[((((int)threadIdx.x) * 196) + 33)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[34] * weight_shared[((((int)threadIdx.x) * 196) + 34)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[35] * weight_shared[((((int)threadIdx.x) * 196) + 35)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[36] * weight_shared[((((int)threadIdx.x) * 196) + 36)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[37] * weight_shared[((((int)threadIdx.x) * 196) + 37)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[38] * weight_shared[((((int)threadIdx.x) * 196) + 38)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[39] * weight_shared[((((int)threadIdx.x) * 196) + 39)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[40] * weight_shared[((((int)threadIdx.x) * 196) + 40)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[41] * weight_shared[((((int)threadIdx.x) * 196) + 41)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[42] * weight_shared[((((int)threadIdx.x) * 196) + 42)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[43] * weight_shared[((((int)threadIdx.x) * 196) + 43)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[44] * weight_shared[((((int)threadIdx.x) * 196) + 44)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[45] * weight_shared[((((int)threadIdx.x) * 196) + 45)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[46] * weight_shared[((((int)threadIdx.x) * 196) + 46)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[47] * weight_shared[((((int)threadIdx.x) * 196) + 47)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[48] * weight_shared[((((int)threadIdx.x) * 196) + 48)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[49] * weight_shared[((((int)threadIdx.x) * 196) + 49)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[50] * weight_shared[((((int)threadIdx.x) * 196) + 50)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[51] * weight_shared[((((int)threadIdx.x) * 196) + 51)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[52] * weight_shared[((((int)threadIdx.x) * 196) + 52)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[53] * weight_shared[((((int)threadIdx.x) * 196) + 53)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[54] * weight_shared[((((int)threadIdx.x) * 196) + 54)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[55] * weight_shared[((((int)threadIdx.x) * 196) + 55)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[56] * weight_shared[((((int)threadIdx.x) * 196) + 56)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[57] * weight_shared[((((int)threadIdx.x) * 196) + 57)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[58] * weight_shared[((((int)threadIdx.x) * 196) + 58)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[59] * weight_shared[((((int)threadIdx.x) * 196) + 59)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[60] * weight_shared[((((int)threadIdx.x) * 196) + 60)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[61] * weight_shared[((((int)threadIdx.x) * 196) + 61)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[62] * weight_shared[((((int)threadIdx.x) * 196) + 62)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[63] * weight_shared[((((int)threadIdx.x) * 196) + 63)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[64] * weight_shared[((((int)threadIdx.x) * 196) + 64)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[65] * weight_shared[((((int)threadIdx.x) * 196) + 65)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[66] * weight_shared[((((int)threadIdx.x) * 196) + 66)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[67] * weight_shared[((((int)threadIdx.x) * 196) + 67)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[68] * weight_shared[((((int)threadIdx.x) * 196) + 68)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[69] * weight_shared[((((int)threadIdx.x) * 196) + 69)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[70] * weight_shared[((((int)threadIdx.x) * 196) + 70)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[71] * weight_shared[((((int)threadIdx.x) * 196) + 71)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[72] * weight_shared[((((int)threadIdx.x) * 196) + 72)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[73] * weight_shared[((((int)threadIdx.x) * 196) + 73)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[74] * weight_shared[((((int)threadIdx.x) * 196) + 74)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[75] * weight_shared[((((int)threadIdx.x) * 196) + 75)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[76] * weight_shared[((((int)threadIdx.x) * 196) + 76)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[77] * weight_shared[((((int)threadIdx.x) * 196) + 77)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[78] * weight_shared[((((int)threadIdx.x) * 196) + 78)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[79] * weight_shared[((((int)threadIdx.x) * 196) + 79)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[80] * weight_shared[((((int)threadIdx.x) * 196) + 80)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[81] * weight_shared[((((int)threadIdx.x) * 196) + 81)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[82] * weight_shared[((((int)threadIdx.x) * 196) + 82)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[83] * weight_shared[((((int)threadIdx.x) * 196) + 83)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[84] * weight_shared[((((int)threadIdx.x) * 196) + 84)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[85] * weight_shared[((((int)threadIdx.x) * 196) + 85)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[86] * weight_shared[((((int)threadIdx.x) * 196) + 86)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[87] * weight_shared[((((int)threadIdx.x) * 196) + 87)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[88] * weight_shared[((((int)threadIdx.x) * 196) + 88)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[89] * weight_shared[((((int)threadIdx.x) * 196) + 89)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[90] * weight_shared[((((int)threadIdx.x) * 196) + 90)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[91] * weight_shared[((((int)threadIdx.x) * 196) + 91)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[92] * weight_shared[((((int)threadIdx.x) * 196) + 92)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[93] * weight_shared[((((int)threadIdx.x) * 196) + 93)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[94] * weight_shared[((((int)threadIdx.x) * 196) + 94)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[95] * weight_shared[((((int)threadIdx.x) * 196) + 95)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[96] * weight_shared[((((int)threadIdx.x) * 196) + 96)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[97] * weight_shared[((((int)threadIdx.x) * 196) + 97)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[98] * weight_shared[((((int)threadIdx.x) * 196) + 98)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[99] * weight_shared[((((int)threadIdx.x) * 196) + 99)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[100] * weight_shared[((((int)threadIdx.x) * 196) + 100)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[101] * weight_shared[((((int)threadIdx.x) * 196) + 101)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[102] * weight_shared[((((int)threadIdx.x) * 196) + 102)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[103] * weight_shared[((((int)threadIdx.x) * 196) + 103)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[104] * weight_shared[((((int)threadIdx.x) * 196) + 104)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[105] * weight_shared[((((int)threadIdx.x) * 196) + 105)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[106] * weight_shared[((((int)threadIdx.x) * 196) + 106)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[107] * weight_shared[((((int)threadIdx.x) * 196) + 107)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[108] * weight_shared[((((int)threadIdx.x) * 196) + 108)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[109] * weight_shared[((((int)threadIdx.x) * 196) + 109)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[110] * weight_shared[((((int)threadIdx.x) * 196) + 110)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[111] * weight_shared[((((int)threadIdx.x) * 196) + 111)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[112] * weight_shared[((((int)threadIdx.x) * 196) + 112)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[113] * weight_shared[((((int)threadIdx.x) * 196) + 113)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[114] * weight_shared[((((int)threadIdx.x) * 196) + 114)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[115] * weight_shared[((((int)threadIdx.x) * 196) + 115)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[116] * weight_shared[((((int)threadIdx.x) * 196) + 116)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[117] * weight_shared[((((int)threadIdx.x) * 196) + 117)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[118] * weight_shared[((((int)threadIdx.x) * 196) + 118)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[119] * weight_shared[((((int)threadIdx.x) * 196) + 119)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[120] * weight_shared[((((int)threadIdx.x) * 196) + 120)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[121] * weight_shared[((((int)threadIdx.x) * 196) + 121)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[122] * weight_shared[((((int)threadIdx.x) * 196) + 122)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[123] * weight_shared[((((int)threadIdx.x) * 196) + 123)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[124] * weight_shared[((((int)threadIdx.x) * 196) + 124)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[125] * weight_shared[((((int)threadIdx.x) * 196) + 125)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[126] * weight_shared[((((int)threadIdx.x) * 196) + 126)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[127] * weight_shared[((((int)threadIdx.x) * 196) + 127)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[128] * weight_shared[((((int)threadIdx.x) * 196) + 128)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[129] * weight_shared[((((int)threadIdx.x) * 196) + 129)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[130] * weight_shared[((((int)threadIdx.x) * 196) + 130)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[131] * weight_shared[((((int)threadIdx.x) * 196) + 131)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[132] * weight_shared[((((int)threadIdx.x) * 196) + 132)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[133] * weight_shared[((((int)threadIdx.x) * 196) + 133)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[134] * weight_shared[((((int)threadIdx.x) * 196) + 134)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[135] * weight_shared[((((int)threadIdx.x) * 196) + 135)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[136] * weight_shared[((((int)threadIdx.x) * 196) + 136)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[137] * weight_shared[((((int)threadIdx.x) * 196) + 137)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[138] * weight_shared[((((int)threadIdx.x) * 196) + 138)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[139] * weight_shared[((((int)threadIdx.x) * 196) + 139)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[140] * weight_shared[((((int)threadIdx.x) * 196) + 140)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[141] * weight_shared[((((int)threadIdx.x) * 196) + 141)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[142] * weight_shared[((((int)threadIdx.x) * 196) + 142)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[143] * weight_shared[((((int)threadIdx.x) * 196) + 143)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[144] * weight_shared[((((int)threadIdx.x) * 196) + 144)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[145] * weight_shared[((((int)threadIdx.x) * 196) + 145)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[146] * weight_shared[((((int)threadIdx.x) * 196) + 146)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[147] * weight_shared[((((int)threadIdx.x) * 196) + 147)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[148] * weight_shared[((((int)threadIdx.x) * 196) + 148)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[149] * weight_shared[((((int)threadIdx.x) * 196) + 149)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[150] * weight_shared[((((int)threadIdx.x) * 196) + 150)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[151] * weight_shared[((((int)threadIdx.x) * 196) + 151)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[152] * weight_shared[((((int)threadIdx.x) * 196) + 152)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[153] * weight_shared[((((int)threadIdx.x) * 196) + 153)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[154] * weight_shared[((((int)threadIdx.x) * 196) + 154)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[155] * weight_shared[((((int)threadIdx.x) * 196) + 155)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[156] * weight_shared[((((int)threadIdx.x) * 196) + 156)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[157] * weight_shared[((((int)threadIdx.x) * 196) + 157)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[158] * weight_shared[((((int)threadIdx.x) * 196) + 158)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[159] * weight_shared[((((int)threadIdx.x) * 196) + 159)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[160] * weight_shared[((((int)threadIdx.x) * 196) + 160)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[161] * weight_shared[((((int)threadIdx.x) * 196) + 161)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[162] * weight_shared[((((int)threadIdx.x) * 196) + 162)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[163] * weight_shared[((((int)threadIdx.x) * 196) + 163)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[164] * weight_shared[((((int)threadIdx.x) * 196) + 164)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[165] * weight_shared[((((int)threadIdx.x) * 196) + 165)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[166] * weight_shared[((((int)threadIdx.x) * 196) + 166)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[167] * weight_shared[((((int)threadIdx.x) * 196) + 167)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[168] * weight_shared[((((int)threadIdx.x) * 196) + 168)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[169] * weight_shared[((((int)threadIdx.x) * 196) + 169)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[170] * weight_shared[((((int)threadIdx.x) * 196) + 170)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[171] * weight_shared[((((int)threadIdx.x) * 196) + 171)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[172] * weight_shared[((((int)threadIdx.x) * 196) + 172)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[173] * weight_shared[((((int)threadIdx.x) * 196) + 173)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[174] * weight_shared[((((int)threadIdx.x) * 196) + 174)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[175] * weight_shared[((((int)threadIdx.x) * 196) + 175)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[176] * weight_shared[((((int)threadIdx.x) * 196) + 176)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[177] * weight_shared[((((int)threadIdx.x) * 196) + 177)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[178] * weight_shared[((((int)threadIdx.x) * 196) + 178)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[179] * weight_shared[((((int)threadIdx.x) * 196) + 179)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[180] * weight_shared[((((int)threadIdx.x) * 196) + 180)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[181] * weight_shared[((((int)threadIdx.x) * 196) + 181)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[182] * weight_shared[((((int)threadIdx.x) * 196) + 182)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[183] * weight_shared[((((int)threadIdx.x) * 196) + 183)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[184] * weight_shared[((((int)threadIdx.x) * 196) + 184)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[185] * weight_shared[((((int)threadIdx.x) * 196) + 185)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[186] * weight_shared[((((int)threadIdx.x) * 196) + 186)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[187] * weight_shared[((((int)threadIdx.x) * 196) + 187)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[188] * weight_shared[((((int)threadIdx.x) * 196) + 188)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[189] * weight_shared[((((int)threadIdx.x) * 196) + 189)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[190] * weight_shared[((((int)threadIdx.x) * 196) + 190)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[191] * weight_shared[((((int)threadIdx.x) * 196) + 191)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[192] * weight_shared[((((int)threadIdx.x) * 196) + 192)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[193] * weight_shared[((((int)threadIdx.x) * 196) + 193)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[194] * weight_shared[((((int)threadIdx.x) * 196) + 194)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[195] * weight_shared[((((int)threadIdx.x) * 196) + 195)]));
  }
  compute[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))]);
}


