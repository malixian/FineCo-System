
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[2320];
  __shared__ float kernel_shared[4608];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x) * 6)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + (((((int)threadIdx.x) * 6) % 145) / 29))) && (1 <= ((((int)threadIdx.x) * 6) % 29))) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) * 6) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + ((((((int)threadIdx.x) * 6) % 145) / 29) * 28)) + ((((int)threadIdx.x) * 6) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 1)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 1) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 1) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 1) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 1) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 1) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 2)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 2) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 2) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 2) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 2) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 2) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 3)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 3) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 3) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 3) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 3) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 3) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 4)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 4) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 4) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 4) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 4) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 4) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 5)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 5) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 5) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 5) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 5) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 5) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 672)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 92) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 5) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 672) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 92) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 5) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 673)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 93) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 6) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 673) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 93) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 6) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 674)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 94) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 7) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 674) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 94) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 7) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 675)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 95) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 8) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 675) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 95) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 8) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 676)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 96) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 9) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 676) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 96) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 9) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 677)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 97) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 10) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 677) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 97) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 10) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 1344)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 39) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 10) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 1344) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 39) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 10) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 1345)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 40) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 11) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 1345) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 40) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 11) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 1346)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 41) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 12) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 1346) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 41) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 12) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 1347)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 42) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 13) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 1347) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 42) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 13) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 1348)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 43) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 14) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 1348) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 43) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 14) % 29)) - 29)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 6) + 1349)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 44) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 15) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 1349) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 44) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 15) % 29)) - 29)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 51) {
      pad_temp_shared[((((int)threadIdx.x) * 6) + 2016)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 131) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 15) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 2016) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 131) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 15) % 29)) - 29)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 51) {
      pad_temp_shared[((((int)threadIdx.x) * 6) + 2017)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 132) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 16) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 2017) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 132) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 16) % 29)) - 29)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 51) {
      pad_temp_shared[((((int)threadIdx.x) * 6) + 2018)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 133) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 17) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 2018) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 133) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 17) % 29)) - 29)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 51) {
      pad_temp_shared[((((int)threadIdx.x) * 6) + 2019)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 134) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 18) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 2019) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 134) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 18) % 29)) - 29)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 50) {
      pad_temp_shared[((((int)threadIdx.x) * 6) + 2020)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 135) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 19) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 2020) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 135) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 19) % 29)) - 29)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 50) {
      pad_temp_shared[((((int)threadIdx.x) * 6) + 2021)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((((int)threadIdx.x) * 6) + 136) % 145) / 29))) && (1 <= (((((int)threadIdx.x) * 6) + 20) % 29))) ? data[((((((rc_outer_outer * 12544) + ((((((int)threadIdx.x) * 6) + 2021) / 145) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((((int)threadIdx.x) * 6) + 136) % 145) / 29) * 28)) + (((((int)threadIdx.x) * 6) + 20) % 29)) - 29)] : 0.000000e+00f);
    }
    kernel_shared[(((int)threadIdx.x) * 2)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 72) * 2))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1)] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 72) * 2)) + 1)];
    kernel_shared[((((int)threadIdx.x) * 2) + 224)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 112) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 80) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 225)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 112) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 81) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 448)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 224) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 16) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 449)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 224) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 17) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 672)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 336) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 96) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 673)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 336) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 97) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 896)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 448) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 32) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 897)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 448) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 33) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1120)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 560) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 112) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1121)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 560) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 113) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1344)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 672) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 48) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1345)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 672) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 49) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1568)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 784) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 128) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1569)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 784) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 129) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1792)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 896) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 64) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 1793)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 896) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 65) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 2016)] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 72) * 2)) + 16128)];
    kernel_shared[((((int)threadIdx.x) * 2) + 2017)] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 72) * 2)) + 16129)];
    kernel_shared[((((int)threadIdx.x) * 2) + 2240)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1120) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 80) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 2241)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1120) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 81) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 2464)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1232) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 16) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 2465)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1232) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 17) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 2688)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1344) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 96) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 2689)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1344) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 97) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 2912)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1456) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 32) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 2913)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1456) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 33) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 3136)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1568) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 112) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 3137)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1568) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 113) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 3360)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1680) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 48) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 3361)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1680) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 49) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 3584)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1792) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 128) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 3585)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1792) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 129) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 3808)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1904) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 64) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 3809)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 1904) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 65) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 4032)] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 72) * 2)) + 32256)];
    kernel_shared[((((int)threadIdx.x) * 2) + 4033)] = kernel[((((((((int)blockIdx.x) / 7) * 36864) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 72) * 2)) + 32257)];
    kernel_shared[((((int)threadIdx.x) * 2) + 4256)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 2128) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 80) % 144))];
    kernel_shared[((((int)threadIdx.x) * 2) + 4257)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 2128) / 72) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) * 2) + 81) % 144))];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[((((int)threadIdx.x) * 2) + 4480)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 2240) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) * 2) + 16))];
    }
    if (((int)threadIdx.x) < 64) {
      kernel_shared[((((int)threadIdx.x) * 2) + 4481)] = kernel[(((((((int)blockIdx.x) / 7) * 36864) + (((((int)threadIdx.x) + 2240) / 72) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) * 2) + 17))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 58)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1152)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 58)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1152)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2304)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 58)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2304)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 3456)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 58)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 3456)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 59)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1153)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 59)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1153)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2305)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 59)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2305)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 3457)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 59)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 3457)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 60)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1154)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 60)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1154)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2306)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 60)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2306)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 3458)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (rc_inner * 145)) + (ry_inner * 29)) + ((((int)threadIdx.x) % 14) * 2)) + 60)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 3458)]));
        }
      }
    }
  }
  conv2d_nchw[(((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14))] = conv2d_nchw_local[0];
  conv2d_nchw[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14)) + 14)] = conv2d_nchw_local[1];
  conv2d_nchw[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14)) + 1568)] = conv2d_nchw_local[2];
  conv2d_nchw[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14)) + 1582)] = conv2d_nchw_local[3];
  conv2d_nchw[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14)) + 3136)] = conv2d_nchw_local[4];
  conv2d_nchw[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14)) + 3150)] = conv2d_nchw_local[5];
  conv2d_nchw[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14)) + 4704)] = conv2d_nchw_local[6];
  conv2d_nchw[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 14) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14)) + 4718)] = conv2d_nchw_local[7];
}


