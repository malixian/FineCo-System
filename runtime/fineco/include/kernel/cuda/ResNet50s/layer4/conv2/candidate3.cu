
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[7];
  __shared__ float pad_temp_shared[2048];
  __shared__ float kernel_shared[576];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((16 <= ((int)threadIdx.x)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[((((rc_outer_outer * 1568) + ((((int)threadIdx.x) >> 4) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 14) & 15)) && ((((((int)threadIdx.x) >> 4) + 14) & 15) < 15)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 224) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 14) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 12) & 15)) && ((((((int)threadIdx.x) >> 4) + 12) & 15) < 15)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 448) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 12) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 672)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 10) & 15)) && ((((((int)threadIdx.x) >> 4) + 10) & 15) < 15)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 672) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 10) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 8) & 15)) && ((((((int)threadIdx.x) >> 4) + 8) & 15) < 15)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 896) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 8) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 6) & 15)) && ((((((int)threadIdx.x) >> 4) + 6) & 15) < 15)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1120) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 6) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 4) & 15)) && ((((((int)threadIdx.x) >> 4) + 4) & 15) < 15)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1344) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 4) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = ((((((int)threadIdx.x) < 208) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1568) >> 8) * 196)) + (((((int)threadIdx.x) >> 4) + 2) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = ((((16 <= ((int)threadIdx.x)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[((((rc_outer_outer * 1568) + ((((int)threadIdx.x) >> 4) * 14)) + (((int)threadIdx.x) & 15)) + 1357)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 32) {
      pad_temp_shared[(((int)threadIdx.x) + 2016)] = ((((((int)threadIdx.x) < 16) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[(((rc_outer_outer * 1568) + ((int)threadIdx.x)) + 1553)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) * 18432) + ((((int)threadIdx.x) / 72) * 2304)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((int)blockIdx.x) * 18432) + (((((int)threadIdx.x) + 224) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    if (((int)threadIdx.x) < 128) {
      kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((int)blockIdx.x) * 18432) + (((((int)threadIdx.x) + 448) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 16)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 32)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 48)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 64)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 80)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 96)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 16)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 3)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 32)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 3)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 48)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 3)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 64)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 3)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 80)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 3)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 96)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 3)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 3)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 32)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 6)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 48)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 6)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 64)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 6)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 80)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 6)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 96)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 6)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 6)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 256)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + rx_outer_inner) + (((int)threadIdx.x) % 14)) + 128)] * kernel_shared[((((((((int)threadIdx.x) / 28) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + rx_outer_inner) + 6)]));
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
    conv2d_nchw[((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 14) * 98)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14))] = conv2d_nchw_local[yy_inner];
  }
}


