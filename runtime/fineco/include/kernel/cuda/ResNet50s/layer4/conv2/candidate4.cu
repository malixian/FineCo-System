
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[2];
  __shared__ float pad_temp_shared[64];
  __shared__ float kernel_shared[9216];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) / 7) * 2) + ((((int)threadIdx.x) & 15) >> 2))) && ((((((int)blockIdx.x) / 7) * 2) + ((((int)threadIdx.x) & 15) >> 2)) < 15)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + (((int)threadIdx.x) & 3)))) && ((((((int)blockIdx.x) % 7) * 2) + (((int)threadIdx.x) & 3)) < 15)) ? data[(((((((rc_outer_outer * 784) + ((((int)threadIdx.x) >> 4) * 196)) + ((((int)blockIdx.x) / 7) * 28)) + (((((int)threadIdx.x) & 15) >> 2) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 3)) - 15)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 36) * 2304) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) + 512) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) + 1024) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 16) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) + 1536) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 24) % 36))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) + 2048) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 32) % 36))];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) + 2560) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 4) % 36))];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) + 3072) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 12) % 36))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) + 3584) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 20) % 36))];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) + 4096) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 28) % 36))];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((int)threadIdx.x) / 36) * 2304) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36)) + 294912)];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)threadIdx.x) + 5120) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)threadIdx.x) + 5632) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 16) % 36))];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)threadIdx.x) + 6144) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 24) % 36))];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)threadIdx.x) + 6656) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 32) % 36))];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)threadIdx.x) + 7168) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 4) % 36))];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)threadIdx.x) + 7680) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 12) % 36))];
    kernel_shared[(((int)threadIdx.x) + 8192)] = kernel[(((((((int)threadIdx.x) + 8192) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 20) % 36))];
    kernel_shared[(((int)threadIdx.x) + 8704)] = kernel[(((((((int)threadIdx.x) + 8704) / 36) * 2304) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 28) % 36))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4))] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9))]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 4)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 3)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 6)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9))]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 5)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 3)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 9)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 6)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 1)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 1)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 5)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 4)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 9)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 7)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 2)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 1)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 6)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 4)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 10)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 7)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 2)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 2)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 6)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 5)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 10)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 8)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 3)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 2)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 7)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 5)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 4)) + 11)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 36) + (rc_outer_inner * 9)) + 8)]));
    }
  }
  for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
    conv2d_nchw[((((((((int)threadIdx.x) >> 1) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner)] = conv2d_nchw_local[xx_inner];
  }
}


