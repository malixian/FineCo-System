
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate6(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[3136];
  __shared__ float kernel_shared[256];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[(((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x))];
    pad_temp_shared[(((int)threadIdx.x) + 196)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 392)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 588)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 980)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 3920)];
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 1372)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 5488)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 1764)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 7056)];
    pad_temp_shared[(((int)threadIdx.x) + 1960)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 7840)];
    pad_temp_shared[(((int)threadIdx.x) + 2156)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 8624)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 2548)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 10192)];
    pad_temp_shared[(((int)threadIdx.x) + 2744)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 10976)];
    pad_temp_shared[(((int)threadIdx.x) + 2940)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 11760)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    if (((int)threadIdx.x) < 60) {
      kernel_shared[(((int)threadIdx.x) + 196)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + (((((int)threadIdx.x) + 196) >> 4) * 512)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 4) & 15))];
    }
    __syncthreads();
    for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
      for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[((rc_inner * 196) + ((int)threadIdx.x))] * kernel_shared[((ff_c_outer_inner * 32) + rc_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 8)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 8)] + (pad_temp_shared[((rc_inner * 196) + ((int)threadIdx.x))] * kernel_shared[(((ff_c_outer_inner * 32) + rc_inner) + 128)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[((rc_inner * 196) + ((int)threadIdx.x))] * kernel_shared[(((ff_c_outer_inner * 32) + rc_inner) + 16)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 9)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 9)] + (pad_temp_shared[((rc_inner * 196) + ((int)threadIdx.x))] * kernel_shared[(((ff_c_outer_inner * 32) + rc_inner) + 144)]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    conv2d_nchw[(((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 6272)] = conv2d_nchw_local[(ff_inner + 8)];
  }
}


