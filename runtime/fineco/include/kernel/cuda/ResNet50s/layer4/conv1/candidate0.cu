
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[1024];
  __shared__ float kernel_shared[8192];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
      conv2d_nchw_local[((ff_c_outer_inner_init * 2) + ff_c_inner_init)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + ff_c_inner_init) + 4)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      *(float2*)(pad_temp_shared + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + (((int)threadIdx.x) * 2))) = *(float2*)(data + (((((((rc_outer_outer * 50176) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 25088)) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 7) >> 1) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 2)));
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + ((((((((int)blockIdx.x) / 49) * 65536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8192)) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)));
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
          for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
            conv2d_nchw_local[((ff_c_outer_inner * 2) + ff_c_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + ff_c_inner)] + (pad_temp_shared[(((rc_outer_inner * 64) + (rc_inner * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 256) + (ff_c_outer_inner * 128)) + (ff_c_inner * 64)) + (rc_outer_inner * 4)) + rc_inner)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 2) + ff_c_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + ff_c_inner) + 4)] + (pad_temp_shared[((((rc_outer_inner * 64) + (rc_inner * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 256) + (ff_c_outer_inner * 128)) + (ff_c_inner * 64)) + (rc_outer_inner * 4)) + rc_inner)]));
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 3136)) + (ff_inner * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 7) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[(((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 3136)) + (ff_inner * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 7) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 56)] = conv2d_nchw_local[(ff_inner + 4)];
  }
}


