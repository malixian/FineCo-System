
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[4];
  __shared__ float pad_temp_shared[64];
  __shared__ float kernel_shared[4096];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) >> 2) * 196)) + ((((int)blockIdx.x) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16384) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_inner * 4) + ((((int)threadIdx.x) & 1) * 2))] * kernel_shared[(((((int)threadIdx.x) >> 1) * 16) + rc_inner)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_inner * 4) + ((((int)threadIdx.x) & 1) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 16) + rc_inner)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_inner * 4) + ((((int)threadIdx.x) & 1) * 2))] * kernel_shared[((((((int)threadIdx.x) >> 1) * 16) + rc_inner) + 2048)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_inner * 4) + ((((int)threadIdx.x) & 1) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) >> 1) * 16) + rc_inner) + 2048)]));
    }
  }
  conv2d_nchw[(((((((int)threadIdx.x) >> 1) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2))] = conv2d_nchw_local[0];
  conv2d_nchw[((((((((int)threadIdx.x) >> 1) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + 1)] = conv2d_nchw_local[1];
  conv2d_nchw[((((((((int)threadIdx.x) >> 1) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + 25088)] = conv2d_nchw_local[2];
  conv2d_nchw[((((((((int)threadIdx.x) >> 1) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)threadIdx.x) & 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + 25089)] = conv2d_nchw_local[3];
}


