
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[32];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 1568) + ((((int)threadIdx.x) >> 2) * 196)) + ((((int)blockIdx.x) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 16384)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 32768)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 49152)];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 65536)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 81920)];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 98304)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 114688)];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 131072)];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 147456)];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 163840)];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 180224)];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 196608)];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 212992)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 229376)];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)threadIdx.x) >> 3) * 256) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 245760)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.x) * 16)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[0] * kernel_shared[((((int)threadIdx.x) * 16) + 8)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.x) * 16) + 1)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.x) * 16) + 9)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[1] * kernel_shared[(((int)threadIdx.x) * 16)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[1] * kernel_shared[((((int)threadIdx.x) * 16) + 8)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[5] * kernel_shared[((((int)threadIdx.x) * 16) + 1)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[5] * kernel_shared[((((int)threadIdx.x) * 16) + 9)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[2] * kernel_shared[(((int)threadIdx.x) * 16)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.x) * 16) + 8)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.x) * 16) + 1)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.x) * 16) + 9)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[3] * kernel_shared[(((int)threadIdx.x) * 16)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[3] * kernel_shared[((((int)threadIdx.x) * 16) + 8)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[7] * kernel_shared[((((int)threadIdx.x) * 16) + 1)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[7] * kernel_shared[((((int)threadIdx.x) * 16) + 9)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.x) * 16) + 2)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.x) * 16) + 10)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[12] * kernel_shared[((((int)threadIdx.x) * 16) + 3)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[12] * kernel_shared[((((int)threadIdx.x) * 16) + 11)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.x) * 16) + 2)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.x) * 16) + 10)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.x) * 16) + 3)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.x) * 16) + 11)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[10] * kernel_shared[((((int)threadIdx.x) * 16) + 2)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[10] * kernel_shared[((((int)threadIdx.x) * 16) + 10)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[14] * kernel_shared[((((int)threadIdx.x) * 16) + 3)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[14] * kernel_shared[((((int)threadIdx.x) * 16) + 11)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.x) * 16) + 2)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.x) * 16) + 10)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.x) * 16) + 3)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.x) * 16) + 11)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[16] * kernel_shared[((((int)threadIdx.x) * 16) + 4)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[16] * kernel_shared[((((int)threadIdx.x) * 16) + 12)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[20] * kernel_shared[((((int)threadIdx.x) * 16) + 5)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[20] * kernel_shared[((((int)threadIdx.x) * 16) + 13)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.x) * 16) + 4)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.x) * 16) + 12)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[21] * kernel_shared[((((int)threadIdx.x) * 16) + 5)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[21] * kernel_shared[((((int)threadIdx.x) * 16) + 13)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[18] * kernel_shared[((((int)threadIdx.x) * 16) + 4)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[18] * kernel_shared[((((int)threadIdx.x) * 16) + 12)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[22] * kernel_shared[((((int)threadIdx.x) * 16) + 5)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[22] * kernel_shared[((((int)threadIdx.x) * 16) + 13)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[19] * kernel_shared[((((int)threadIdx.x) * 16) + 4)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[19] * kernel_shared[((((int)threadIdx.x) * 16) + 12)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[23] * kernel_shared[((((int)threadIdx.x) * 16) + 5)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[23] * kernel_shared[((((int)threadIdx.x) * 16) + 13)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[24] * kernel_shared[((((int)threadIdx.x) * 16) + 6)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[24] * kernel_shared[((((int)threadIdx.x) * 16) + 14)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[28] * kernel_shared[((((int)threadIdx.x) * 16) + 7)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[28] * kernel_shared[((((int)threadIdx.x) * 16) + 15)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[25] * kernel_shared[((((int)threadIdx.x) * 16) + 6)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[25] * kernel_shared[((((int)threadIdx.x) * 16) + 14)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[29] * kernel_shared[((((int)threadIdx.x) * 16) + 7)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[29] * kernel_shared[((((int)threadIdx.x) * 16) + 15)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[26] * kernel_shared[((((int)threadIdx.x) * 16) + 6)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[26] * kernel_shared[((((int)threadIdx.x) * 16) + 14)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[30] * kernel_shared[((((int)threadIdx.x) * 16) + 7)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[30] * kernel_shared[((((int)threadIdx.x) * 16) + 15)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[27] * kernel_shared[((((int)threadIdx.x) * 16) + 6)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[27] * kernel_shared[((((int)threadIdx.x) * 16) + 14)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[31] * kernel_shared[((((int)threadIdx.x) * 16) + 7)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[31] * kernel_shared[((((int)threadIdx.x) * 16) + 15)]));
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        conv2d_nchw[((((((((int)threadIdx.x) * 392) + (ff_inner * 196)) + ((((int)blockIdx.x) / 7) * 28)) + (yy_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 4) + (yy_inner * 2)) + xx_inner)];
      }
    }
  }
}


