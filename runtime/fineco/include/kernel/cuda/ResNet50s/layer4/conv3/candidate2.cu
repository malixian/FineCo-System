
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[32];
  __shared__ float pad_temp_shared[896];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[(((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[(((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 4704)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5376)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8960)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12544)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 16128)];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 17920)];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 19712)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 21504)];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 23296)];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 25088)];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 26880)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 30464)];
    kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 32256)];
    kernel_shared[(((int)threadIdx.x) + 4256)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 34048)];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 35840)];
    kernel_shared[(((int)threadIdx.x) + 4704)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 37632)];
    kernel_shared[(((int)threadIdx.x) + 4928)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 39424)];
    kernel_shared[(((int)threadIdx.x) + 5152)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 41216)];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 43008)];
    kernel_shared[(((int)threadIdx.x) + 5600)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 44800)];
    kernel_shared[(((int)threadIdx.x) + 5824)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 46592)];
    kernel_shared[(((int)threadIdx.x) + 6048)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 48384)];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 50176)];
    kernel_shared[(((int)threadIdx.x) + 6496)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 51968)];
    kernel_shared[(((int)threadIdx.x) + 6720)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 53760)];
    kernel_shared[(((int)threadIdx.x) + 6944)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 55552)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 7392)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 59136)];
    kernel_shared[(((int)threadIdx.x) + 7616)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 60928)];
    kernel_shared[(((int)threadIdx.x) + 7840)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 62720)];
    if (((int)threadIdx.x) < 128) {
      kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[((((((((int)blockIdx.x) / 7) * 65536) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 64512)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 14) * 64) + rc_outer_inner)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 1024)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 2048)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 3072)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 4096)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 5120)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 6144)]));
      conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 7168)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 14) * 64) + rc_outer_inner)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 1024)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 2048)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 3072)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 4096)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 5120)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 6144)]));
      conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 7168)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 1056)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 2080)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 3104)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 4128)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 5152)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 6176)]));
      conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[((rc_outer_inner * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 7200)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 1056)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 2080)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 3104)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 4128)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 5152)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 6176)]));
      conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((rc_outer_inner * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 7200)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[(((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14)) + 6272)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 4)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14)) + 12544)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 8)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14)) + 18816)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 12)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14)) + 25088)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 16)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14)) + 31360)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 20)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14)) + 37632)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 24)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14)) + 43904)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 28)];
    }
  }
}


