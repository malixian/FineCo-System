
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[2048];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    *(float2*)(pad_temp_shared + (((int)threadIdx.x) * 2)) = *(float2*)(data + ((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 392)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 784));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 784)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 1568));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 1176)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 2352));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 1568)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 3136));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 1960)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 3920));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 2352)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 4704));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 2744)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 5488));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 3136)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 6272));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 3528)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 7056));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 3920)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 7840));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 4312)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 8624));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 4704)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 9408));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 5096)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 10192));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 5488)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 10976));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 5880)) = *(float2*)(data + (((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 49) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 11760));
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 196)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 196) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 4) & 63))];
    kernel_shared[(((int)threadIdx.x) + 392)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 392) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 8) & 63))];
    kernel_shared[(((int)threadIdx.x) + 588)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 588) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 12) & 63))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 784) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 16) & 63))];
    kernel_shared[(((int)threadIdx.x) + 980)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 980) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 20) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1176)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 1176) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 24) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1372)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 1372) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 28) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 1568) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1764)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 1764) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 36) & 63))];
    if (((int)threadIdx.x) < 88) {
      kernel_shared[(((int)threadIdx.x) + 1960)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + (((((int)threadIdx.x) + 1960) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 8; ++ff_c_outer_inner) {
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2))] * kernel_shared[((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32))]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 98)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 1)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 196)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 2)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 294)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 3)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 392)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 4)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 490)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 5)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 588)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 6)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 686)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 7)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 784)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 8)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 882)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 9)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 980)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 10)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1078)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 11)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1176)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 12)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1274)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 13)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1372)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 14)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1470)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 15)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1568)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 16)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1666)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 17)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1764)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 18)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1862)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 19)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1960)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 20)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2058)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 21)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2156)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 22)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2254)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 23)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2352)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 24)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2450)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 25)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2548)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 26)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2646)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 27)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2744)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 28)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2842)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 29)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2940)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 30)]));
        conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 3038)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 31)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32))]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 99)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 1)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 197)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 2)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 295)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 393)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 4)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 491)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 5)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 589)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 687)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 7)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 785)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 8)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 883)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 9)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 981)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 10)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1079)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 11)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1177)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 12)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1275)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 13)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1373)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 14)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1471)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 15)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1569)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 16)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1667)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 17)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1765)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 18)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1863)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 19)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 1961)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 20)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2059)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 21)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2157)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 22)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2255)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 23)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2353)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 24)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2451)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 25)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2549)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 26)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2647)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 27)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2745)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 28)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2843)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 29)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 2941)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 30)]));
        conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((rc_outer_inner * 3136) + ((((int)threadIdx.x) % 49) * 2)) + 3039)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 32)) + 31)]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      conv2d_nchw[(((((((((int)blockIdx.x) >> 1) * 6272) + ((((int)threadIdx.x) / 49) * 1568)) + (ff_inner * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + xx_inner)] = conv2d_nchw_local[((ff_inner * 2) + xx_inner)];
    }
  }
}


