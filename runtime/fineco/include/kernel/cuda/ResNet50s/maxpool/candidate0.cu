
#include <hip/hip_runtime.h>
#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) candidate0(float* __restrict__ tensor, float* __restrict__ data) {
  tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], (((1 <= ((((((((int)blockIdx.x) % 98) * 4) + (((int)threadIdx.x) >> 3)) / 7) * 2) + rv0)) && (1 <= (((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 56) * 2) + rv1))) ? data[((((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) / 7) * 224) + (rv0 * 112)) + ((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 56) * 2)) + rv1) - 113)] : -3.402823e+38f));
    }
  }
}