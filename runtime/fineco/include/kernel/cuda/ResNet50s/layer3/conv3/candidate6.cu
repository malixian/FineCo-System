
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate6(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[28];
  __shared__ float pad_temp_shared[1568];
  __shared__ float kernel_shared[4096];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 49) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + (((((int)threadIdx.x) % 49) / 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 224) / 49) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) / 7) + 4) % 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 448) / 49) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) / 7) + 1) % 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 672) / 49) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) / 7) + 5) % 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 896) / 49) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) / 7) + 2) % 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1120) / 49) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) / 7) + 6) % 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1344) / 49) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) / 7) + 3) % 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 896)];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2688)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4480)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5376)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6272)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8064)];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8960)];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 9856)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 11648)];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12544)];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 13440)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 15232)];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[((((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 16128)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 7))] * kernel_shared[(((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 7)] * kernel_shared[(((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 21)] * kernel_shared[(((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 28)] * kernel_shared[(((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 35)] * kernel_shared[(((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 42)] * kernel_shared[(((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 32)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 7)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 32)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 32)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 21)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 32)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 32)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 35)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 32)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 32)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 64)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 7)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 64)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 64)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 21)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 64)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 64)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 35)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 64)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 64)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 96)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 7)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 96)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 96)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 21)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 96)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 96)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 35)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 96)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 96)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 49)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 63)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 77)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 91)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 49)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 33)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 33)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 63)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 33)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 33)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 77)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 33)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 33)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 91)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 33)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 49)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 65)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 65)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 63)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 65)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 65)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 77)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 65)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 65)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 91)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 65)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 49)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 97)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 97)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 63)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 97)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 97)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 77)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 97)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 97)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 91)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 97)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 105)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 119)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 133)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 34)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 105)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 34)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 34)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 119)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 34)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 34)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 133)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 34)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 34)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 66)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 105)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 66)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 66)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 119)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 66)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 66)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 133)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 66)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 66)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 98)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 105)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 98)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 98)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 119)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 98)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 98)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 133)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 98)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 98)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 147)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 161)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 175)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 189)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 147)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 35)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 35)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 161)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 35)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 35)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 175)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 35)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 35)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 189)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 35)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 147)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 67)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 67)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 161)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 67)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 67)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 175)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 67)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 67)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 189)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 67)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 147)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 99)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 99)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 161)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 99)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 99)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 175)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 99)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 99)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 7)) + 189)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 4)) + 99)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      conv2d_nchw[((((((((((int)blockIdx.x) >> 4) * 100352) + ((((int)threadIdx.x) / 7) * 3136)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + (yy_inner * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))] = conv2d_nchw_local[((ff_inner * 7) + yy_inner)];
    }
  }
}


