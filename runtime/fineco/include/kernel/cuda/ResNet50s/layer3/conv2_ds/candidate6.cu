
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate6(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[6960];
  __shared__ float kernel_shared[2304];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((int)threadIdx.x) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + (((int)threadIdx.x) % 15)))) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((int)threadIdx.x) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 196)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 1) % 15))) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + (((((int)threadIdx.x) + 196) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 392)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 392) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 2) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 392) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 392) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 588)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 3) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 588) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 153) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 784)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 349) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 4) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 784) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 349) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 980)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 5) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 980) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 110) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 306) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 6) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1176) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 306) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1372)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 7) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1372) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 67) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 263) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 8) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1568) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 263) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1764)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 9) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1764) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 24) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1960)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 10) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1960) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 220) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2156)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 416) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 11) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2156) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 416) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 12) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2352) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 177) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2548)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 373) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 13) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2548) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 373) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2744)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 14) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2744) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 134) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2940)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) / 15) + 22) % 29))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + (((int)threadIdx.x) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2940) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) / 15) + 22) % 29) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 1) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 3136) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 91) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3332)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 287) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 2) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 3332) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 287) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3528)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 3) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 3528) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 48) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3724)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 244) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 4) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 3724) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 244) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3920)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 5) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 5) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 3920) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 5) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 4116)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 6) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 4116) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 201) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 4312)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 397) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 7) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 4312) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 397) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 4508)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 8) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 4508) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 158) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 354) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 9) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 4704) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 354) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 4900)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 10) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 4900) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 115) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 5096)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 311) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 11) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 5096) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 311) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 5292)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 12) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 5292) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 72) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 5488)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 268) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 13) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 5488) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 268) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 5684)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 14) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 5684) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 29) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 5880)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + (((int)threadIdx.x) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 5880) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + (((((int)threadIdx.x) / 15) + 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 6076)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 421) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 1) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 6076) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 421) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 6272)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 2) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 6272) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 182) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 6468)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 378) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 3) % 15)))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 6468) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 378) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 6664)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 4) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 6664) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 139) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 100) {
      pad_temp_shared[(((int)threadIdx.x) + 6860)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 5) % 15))) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 6860) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 335) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 57)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + ((((int)threadIdx.x) / 144) * 1152)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144))];
    kernel_shared[(((int)threadIdx.x) + 196)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 196) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 52) % 144))];
    kernel_shared[(((int)threadIdx.x) + 392)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 392) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 104) % 144))];
    kernel_shared[(((int)threadIdx.x) + 588)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 588) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 12) % 144))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 784) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 980)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 980) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 116) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1176)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1176) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 24) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1372)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1372) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 76) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1568) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1764)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1764) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 36) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1960)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1960) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 88) % 144))];
    if (((int)threadIdx.x) < 148) {
      kernel_shared[(((int)threadIdx.x) + 2156)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 2156) / 144) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 140) % 144))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1152)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 144)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1296)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 288)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1440)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 432)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1584)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1153)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 145)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1297)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 289)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1441)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 433)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1585)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1154)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 146)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1298)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 290)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1442)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 434)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((((((rc_outer_inner * 3480) + (rc_inner * 435)) + (((((int)threadIdx.x) % 98) / 7) * 30)) + (ry_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 576) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_inner * 3)) + 1586)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    conv2d_nchw[((((((((((int)blockIdx.x) >> 3) * 12544) + ((((int)threadIdx.x) / 98) * 3136)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + (((((int)threadIdx.x) % 98) / 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[(((((((((((int)blockIdx.x) >> 3) * 12544) + ((((int)threadIdx.x) / 98) * 3136)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + (((((int)threadIdx.x) % 98) / 7) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7)) + 6272)] = conv2d_nchw_local[(ff_inner + 4)];
  }
}


