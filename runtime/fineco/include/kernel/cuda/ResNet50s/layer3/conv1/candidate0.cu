
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[32];
  __shared__ float pad_temp_shared[2048];
  __shared__ float kernel_shared[2048];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 384)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 512)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 640)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 31360)];
    pad_temp_shared[(((int)threadIdx.x) + 768)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 37632)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 43904)];
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 1152)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 56448)];
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 62720)];
    pad_temp_shared[(((int)threadIdx.x) + 1408)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 68992)];
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 75264)];
    pad_temp_shared[(((int)threadIdx.x) + 1664)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 81536)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 87808)];
    pad_temp_shared[(((int)threadIdx.x) + 1920)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 94080)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1024)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3072)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5120)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 9216)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 10240)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 11264)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 13312)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[((((((((int)blockIdx.x) / 49) * 16384) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 15360)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
        conv2d_nchw_local[(ff_c_outer_inner * 4)] = (conv2d_nchw_local[(ff_c_outer_inner * 4)] + (pad_temp_shared[(((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4))]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 16)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4))]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4))]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 17)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4))]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] + (pad_temp_shared[(((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 32)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 18)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 18)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 32)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 32)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 19)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 19)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 32)]));
        conv2d_nchw_local[(ff_c_outer_inner * 4)] = (conv2d_nchw_local[(ff_c_outer_inner * 4)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 1)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 16)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 1)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 1)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 17)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 1)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 33)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 18)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 18)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 33)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 33)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 19)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 19)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 33)]));
        conv2d_nchw_local[(ff_c_outer_inner * 4)] = (conv2d_nchw_local[(ff_c_outer_inner * 4)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 128)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 2)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 16)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 160)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 2)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 136)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 2)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 17)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 168)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 2)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 128)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 34)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 18)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 18)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 160)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 34)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 136)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 34)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 19)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 19)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 168)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 34)]));
        conv2d_nchw_local[(ff_c_outer_inner * 4)] = (conv2d_nchw_local[(ff_c_outer_inner * 4)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 192)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 16)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 224)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 200)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 17)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 232)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 192)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 35)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 18)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 18)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 224)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 35)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 200)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 35)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 19)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 19)] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 232)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 256) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 4)) + 35)]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[(((((((((((int)blockIdx.x) / 49) * 200704) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
      conv2d_nchw[((((((((((((int)blockIdx.x) / 49) * 200704) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 224)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 16)];
    }
  }
}


