
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate6(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[32];
  __shared__ float pad_temp_shared[3136];
  __shared__ float kernel_shared[512];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[(((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 196)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 392)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 588)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 980)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 15680)];
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 1372)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 21952)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 1764)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 28224)];
    pad_temp_shared[(((int)threadIdx.x) + 1960)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 31360)];
    pad_temp_shared[(((int)threadIdx.x) + 2156)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 34496)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 37632)];
    pad_temp_shared[(((int)threadIdx.x) + 2548)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 40768)];
    pad_temp_shared[(((int)threadIdx.x) + 2744)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 43904)];
    pad_temp_shared[(((int)threadIdx.x) + 2940)] = data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 47040)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 4) * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_shared[(((int)threadIdx.x) + 196)] = kernel[(((((((int)blockIdx.x) >> 4) * 8192) + (((((int)threadIdx.x) + 196) >> 4) * 256)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 4) & 15))];
    if (((int)threadIdx.x) < 120) {
      kernel_shared[(((int)threadIdx.x) + 392)] = kernel[(((((((int)blockIdx.x) >> 4) * 8192) + (((((int)threadIdx.x) + 392) >> 4) * 256)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15))];
    }
    __syncthreads();
    for (int ff_c_outer_inner = 0; ff_c_outer_inner < 8; ++ff_c_outer_inner) {
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32))]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32))]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 16)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 16)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 1)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 1)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 17)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 17)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 392)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 2)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 406)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 2)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 392)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 18)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 406)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 18)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 588)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 3)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 602)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 3)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 588)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 19)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 602)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 19)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 784)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 4)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 798)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 4)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 784)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 20)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 798)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 20)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 980)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 5)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 994)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 5)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 980)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 21)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 994)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 21)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1176)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 6)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1190)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 6)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1176)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 22)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1190)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 22)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1372)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 7)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1386)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 7)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1372)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 23)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1386)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 23)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1568)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 8)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1582)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 8)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1568)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 24)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1582)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 24)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1764)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 9)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1778)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 9)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1764)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 25)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1778)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 25)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1960)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 10)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1974)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 10)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1960)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 26)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 1974)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 26)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2156)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 11)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2170)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 11)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2156)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 27)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2170)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 27)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2352)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 12)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2366)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 12)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2352)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 28)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2366)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 28)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2548)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 13)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2562)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 13)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2548)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 29)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2562)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 29)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2744)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 14)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2758)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 14)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2744)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 30)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2758)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 30)]));
      conv2d_nchw_local[(ff_c_outer_inner * 2)] = (conv2d_nchw_local[(ff_c_outer_inner * 2)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2940)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 15)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 16)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2954)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 15)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 1)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2940)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 31)]));
      conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + 17)] + (pad_temp_shared[(((((((int)threadIdx.x) % 98) / 14) * 28) + (((int)threadIdx.x) % 14)) + 2954)] * kernel_shared[((((((int)threadIdx.x) / 98) * 256) + (ff_c_outer_inner * 32)) + 31)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 16; ++ff_inner) {
    conv2d_nchw[((((((((((int)blockIdx.x) >> 4) * 100352) + ((((int)threadIdx.x) / 98) * 50176)) + (ff_inner * 3136)) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + (((((int)threadIdx.x) % 98) / 14) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 14))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[(((((((((((int)blockIdx.x) >> 4) * 100352) + ((((int)threadIdx.x) / 98) * 50176)) + (ff_inner * 3136)) + (((((int)blockIdx.x) & 15) >> 1) * 392)) + (((((int)threadIdx.x) % 98) / 14) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 14)) + 14)] = conv2d_nchw_local[(ff_inner + 16)];
  }
}


