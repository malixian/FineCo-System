
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(392) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[3136];
  __shared__ float kernel_shared[128];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[(((rc_outer_outer * 6272) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x))];
    pad_temp_shared[(((int)threadIdx.x) + 392)] = data[((((rc_outer_outer * 6272) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = data[((((rc_outer_outer * 6272) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = data[((((rc_outer_outer * 6272) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[((((rc_outer_outer * 6272) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 1960)] = data[((((rc_outer_outer * 6272) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 3920)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = data[((((rc_outer_outer * 6272) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 2744)] = data[((((rc_outer_outer * 6272) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 5488)];
    if (((int)threadIdx.x) < 128) {
      kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + ((((int)threadIdx.x) >> 3) * 512)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[rc_outer_inner]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 32)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 64)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 96)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 8)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 40)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 72)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 104)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 16)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 48)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 80)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 112)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 24)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 56)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 88)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[((rc_outer_inner * 392) + ((int)threadIdx.x))] * kernel_shared[(rc_outer_inner + 120)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    conv2d_nchw[(((((((int)blockIdx.x) >> 1) * 12544) + (ff_inner * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[((((((((int)blockIdx.x) >> 1) * 12544) + (ff_inner * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 3136)] = conv2d_nchw_local[(ff_inner + 4)];
    conv2d_nchw[((((((((int)blockIdx.x) >> 1) * 12544) + (ff_inner * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 6272)] = conv2d_nchw_local[(ff_inner + 8)];
    conv2d_nchw[((((((((int)blockIdx.x) >> 1) * 12544) + (ff_inner * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((int)threadIdx.x)) + 9408)] = conv2d_nchw_local[(ff_inner + 12)];
  }
}


