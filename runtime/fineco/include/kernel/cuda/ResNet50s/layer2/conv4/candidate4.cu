
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[4096];
  __shared__ float kernel_shared[4096];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 2; ++yy_c_outer_inner_init) {
      conv2d_nchw_local[((ff_c_outer_inner_init * 2) + yy_c_outer_inner_init)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + yy_c_outer_inner_init) + 4)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x))] = data[(((((((rc_outer_outer * 200704) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 25088)) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 512) + ((int)threadIdx.x))] = kernel[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2048) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int yy_c_outer_inner = 0; yy_c_outer_inner < 2; ++yy_c_outer_inner) {
          for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
            conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 1024) + (rc_inner * 64)) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (yy_c_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 128) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 16)) + rc_inner)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 1024) + (rc_inner * 64)) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (yy_c_outer_inner * 8)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 128) + (ff_c_outer_inner * 64)) + (rc_outer_inner * 16)) + rc_inner)]));
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[((((((((((int)threadIdx.x) >> 4) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 4) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 224)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 4)];
    }
  }
}


