
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[28];
  __shared__ float pad_temp_shared[3584];
  __shared__ float kernel_shared[2048];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 7; ++yy_c_outer_inner_init) {
      conv2d_nchw_local[((ff_c_outer_inner_init * 7) + yy_c_outer_inner_init)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 7) + yy_c_outer_inner_init) + 14)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 14; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x))] = data[((((((rc_outer_outer * 100352) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + (((int)threadIdx.x) >> 4)) / 7) * 3136)) + ((((int)blockIdx.x) / 7) * 784)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + (((int)threadIdx.x) >> 3)) % 14) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 256) + ((int)threadIdx.x))] = kernel[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2048) + ((((int)threadIdx.x) >> 5) * 256)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int yy_c_outer_inner = 0; yy_c_outer_inner < 7; ++yy_c_outer_inner) {
          for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
            conv2d_nchw_local[((ff_c_outer_inner * 7) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 224) + (rc_inner * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (yy_c_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 64) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 2)) + rc_inner)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 7) + yy_c_outer_inner) + 14)] = (conv2d_nchw_local[(((ff_c_outer_inner * 7) + yy_c_outer_inner) + 14)] + (pad_temp_shared[(((((rc_outer_inner * 224) + (rc_inner * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (yy_c_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 64) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 2)) + rc_inner) + 1024)]));
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      conv2d_nchw[((((((((((int)threadIdx.x) >> 4) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 784)) + (((((int)threadIdx.x) & 15) >> 3) * 392)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = conv2d_nchw_local[((ff_inner * 7) + yy_inner)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 4) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 784)) + (((((int)threadIdx.x) & 15) >> 3) * 392)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 100352)] = conv2d_nchw_local[(((ff_inner * 7) + yy_inner) + 14)];
    }
  }
}


