
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(896) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[9280];
  __shared__ float kernel_shared[2304];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) % 580) / 58))) && ((((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) % 580) / 58)) < 57)) && (1 <= (((int)threadIdx.x) % 58))) && ((((int)threadIdx.x) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + ((((int)threadIdx.x) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((int)threadIdx.x) % 580) / 58) * 56)) + (((int)threadIdx.x) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 158) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 158) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 26) % 58))) && (((((int)threadIdx.x) + 26) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 896) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 158) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 26) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 26) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 26) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 52) % 58))) && (((((int)threadIdx.x) + 52) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1792) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 26) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 52) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 184) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 184) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 20) % 58))) && (((((int)threadIdx.x) + 20) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2688) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 184) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 20) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 52) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 52) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 46) % 58))) && (((((int)threadIdx.x) + 46) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 3584) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 52) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 46) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 4480)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 210) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 210) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 14) % 58))) && (((((int)threadIdx.x) + 14) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 4480) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 210) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 14) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 5376)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 78) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 78) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 40) % 58))) && (((((int)threadIdx.x) + 40) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 5376) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 78) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 40) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 6272)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 236) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 236) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 8) % 58))) && (((((int)threadIdx.x) + 8) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 6272) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 236) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 8) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 7168)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 104) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 104) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 34) % 58))) && (((((int)threadIdx.x) + 34) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 7168) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 104) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 34) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 8064)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 262) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 262) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 2) % 58))) && (((((int)threadIdx.x) + 2) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 8064) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 262) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 2) % 58)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 320) {
      pad_temp_shared[(((int)threadIdx.x) + 8960)] = (((((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 130) % 290) / 29)) < 57) && (1 <= ((((int)threadIdx.x) + 28) % 58))) && (((((int)threadIdx.x) + 28) % 58) < 57)) ? data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 8960) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 130) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 28) % 58)) - 57)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 9216) + ((((int)threadIdx.x) / 144) * 576)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)blockIdx.x) / 7) * 9216) + (((((int)threadIdx.x) + 896) / 144) * 576)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    if (((int)threadIdx.x) < 512) {
      kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) / 7) * 9216) + (((((int)threadIdx.x) + 1792) / 144) * 576)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int xx_c_outer_inner = 0; xx_c_outer_inner < 4; ++xx_c_outer_inner) {
          conv2d_nchw_local[xx_c_outer_inner] = (conv2d_nchw_local[xx_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[(xx_c_outer_inner + 4)] = (conv2d_nchw_local[(xx_c_outer_inner + 4)] + (pad_temp_shared[(((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 1152)]));
          conv2d_nchw_local[xx_c_outer_inner] = (conv2d_nchw_local[xx_c_outer_inner] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[(xx_c_outer_inner + 4)] = (conv2d_nchw_local[(xx_c_outer_inner + 4)] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 1153)]));
          conv2d_nchw_local[xx_c_outer_inner] = (conv2d_nchw_local[xx_c_outer_inner] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[(xx_c_outer_inner + 4)] = (conv2d_nchw_local[(xx_c_outer_inner + 4)] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 1154)]));
          conv2d_nchw_local[xx_c_outer_inner] = (conv2d_nchw_local[xx_c_outer_inner] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 580)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 9)]));
          conv2d_nchw_local[(xx_c_outer_inner + 4)] = (conv2d_nchw_local[(xx_c_outer_inner + 4)] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 580)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 1161)]));
          conv2d_nchw_local[xx_c_outer_inner] = (conv2d_nchw_local[xx_c_outer_inner] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 581)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 10)]));
          conv2d_nchw_local[(xx_c_outer_inner + 4)] = (conv2d_nchw_local[(xx_c_outer_inner + 4)] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 581)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 1162)]));
          conv2d_nchw_local[xx_c_outer_inner] = (conv2d_nchw_local[xx_c_outer_inner] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 582)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 11)]));
          conv2d_nchw_local[(xx_c_outer_inner + 4)] = (conv2d_nchw_local[(xx_c_outer_inner + 4)] + (pad_temp_shared[((((((rc_outer_inner * 1160) + (((((int)threadIdx.x) % 112) / 14) * 58)) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 14) * 4)) + xx_c_outer_inner) + 582)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 144) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + 1163)]));
        }
      }
    }
  }
  for (int xx_inner = 0; xx_inner < 4; ++xx_inner) {
    conv2d_nchw[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + ((((int)threadIdx.x) % 112) * 4)) + xx_inner)] = conv2d_nchw_local[xx_inner];
    conv2d_nchw[(((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + ((((int)threadIdx.x) % 112) * 4)) + xx_inner) + 25088)] = conv2d_nchw_local[(xx_inner + 4)];
  }
}


