
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[2560];
  __shared__ float kernel_shared[3072];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 20; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x))] = (((((1 <= ((((((int)blockIdx.x) % 49) / 7) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) % 10))) && (((((((int)blockIdx.x) % 49) / 7) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) % 10)) < 57)) && (1 <= ((((((int)blockIdx.x) % 7) * 8) + rx_outer_outer) + (((int)threadIdx.x) & 7)))) && (((((((int)blockIdx.x) % 7) * 8) + rx_outer_outer) + (((int)threadIdx.x) & 7)) < 57)) ? data[((((((((rc_outer_outer * 100352) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) >> 4)) / 5) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) % 10) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + rx_outer_outer) + (((int)threadIdx.x) & 7)) - 57)] : 0.000000e+00f);
      }
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
        kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 49) * 18432) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 4) + (((int)threadIdx.x) >> 5)) / 3) * 576)) + (rc_outer_outer * 288)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) % 96) * 3)) + rx_outer_outer)];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
            conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 56)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner) + 96)]));
            conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner) + 96)]));
            conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner) + 96)]));
            conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner) + 96)]));
            conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner) + 96)]));
            conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner) + 96)]));
            conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner) + 96)]));
            conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((((rc_outer_inner * 1280) + (rc_inner * 80)) + (ry_inner * 8)) + (((int)threadIdx.x) & 7)) + 56)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 48)) + (rc_inner * 3)) + ry_inner) + 96)]));
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    conv2d_nchw[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 6272)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 6272)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 56)] = conv2d_nchw_local[(ff_inner + 2)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 6272)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 112)] = conv2d_nchw_local[(ff_inner + 4)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 6272)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 168)] = conv2d_nchw_local[(ff_inner + 6)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 6272)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 224)] = conv2d_nchw_local[(ff_inner + 8)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 6272)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 280)] = conv2d_nchw_local[(ff_inner + 10)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 6272)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 336)] = conv2d_nchw_local[(ff_inner + 12)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 3) * 6272)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 392)] = conv2d_nchw_local[(ff_inner + 14)];
  }
}


