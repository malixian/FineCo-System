
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[64];
  __shared__ float pad_temp_shared[1792];
  __shared__ float kernel_shared[4096];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 16; ++ff_c_outer_inner_init) {
    conv2d_nchw_local[(ff_c_outer_inner_init * 4)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 4) + 2)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 4) + 1)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 4) + 3)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 50176) + ((((int)threadIdx.x) / 112) * 3136)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((((rc_outer_outer * 50176) + ((((int)threadIdx.x) / 112) * 3136)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) % 112)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((rc_outer_outer * 50176) + ((((int)threadIdx.x) / 112) * 3136)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) % 112)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[(((((rc_outer_outer * 50176) + ((((int)threadIdx.x) / 112) * 3136)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) % 112)) + 37632)];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 5376)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 8960)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 12544)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 14336)];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[(((((((int)threadIdx.x) >> 4) * 64) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 16128)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 16; ++ff_c_outer_inner) {
        conv2d_nchw_local[(ff_c_outer_inner * 4)] = (conv2d_nchw_local[(ff_c_outer_inner * 4)] + (pad_temp_shared[((rc_outer_inner * 448) + (((int)threadIdx.x) % 56))] * kernel_shared[((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4))]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] + (pad_temp_shared[((rc_outer_inner * 448) + (((int)threadIdx.x) % 56))] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 16)]));
        conv2d_nchw_local[(ff_c_outer_inner * 4)] = (conv2d_nchw_local[(ff_c_outer_inner * 4)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 112)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 1)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 112)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 17)]));
        conv2d_nchw_local[(ff_c_outer_inner * 4)] = (conv2d_nchw_local[(ff_c_outer_inner * 4)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 224)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 2)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 224)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 18)]));
        conv2d_nchw_local[(ff_c_outer_inner * 4)] = (conv2d_nchw_local[(ff_c_outer_inner * 4)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 336)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 2)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 336)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 19)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4))]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 56)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 16)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 168)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 1)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 168)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 17)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 280)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 2)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 280)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 18)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 1)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 392)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + 3)] + (pad_temp_shared[(((rc_outer_inner * 448) + (((int)threadIdx.x) % 56)) + 392)] * kernel_shared[(((((((int)threadIdx.x) / 56) * 512) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 4)) + 19)]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 32; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[((((((((int)threadIdx.x) / 56) * 100352) + (ff_inner * 3136)) + (((int)blockIdx.x) * 112)) + (yy_inner * 56)) + (((int)threadIdx.x) % 56))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
    }
  }
}


