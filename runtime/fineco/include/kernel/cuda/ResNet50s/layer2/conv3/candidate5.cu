
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate5(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[56];
  __shared__ float pad_temp_shared[3584];
  __shared__ float kernel_shared[4096];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 100352) + ((((int)threadIdx.x) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((int)threadIdx.x) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 256) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 32) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 512)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 512) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 64) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 768)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 768) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 96) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1024) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 16) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1280) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 48) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1536) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 80) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((rc_outer_outer * 100352) + ((((int)threadIdx.x) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((int)threadIdx.x) % 112)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 2048)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2048) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 32) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 2304)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2304) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 64) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 2560)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2560) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 96) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 2816)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2816) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 16) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 3072)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3072) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 48) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 3328)] = data[((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3328) / 112) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + ((((int)threadIdx.x) + 80) % 112))];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 512)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1024)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1536)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2560)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3072)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4608)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5120)];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5632)];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144)];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6656)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[((((((((int)blockIdx.x) / 28) * 8192) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7680)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
      conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
      conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
      conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
      conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 160)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
      conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
      conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
      conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
      conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[((((rc_outer_inner * 112) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 224)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    conv2d_nchw[(((((((((int)blockIdx.x) / 28) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[((((((((((int)blockIdx.x) / 28) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 8)] = conv2d_nchw_local[(ff_inner + 8)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 28) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 16)] = conv2d_nchw_local[(ff_inner + 16)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 28) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 24)] = conv2d_nchw_local[(ff_inner + 24)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 28) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 32)] = conv2d_nchw_local[(ff_inner + 32)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 28) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 40)] = conv2d_nchw_local[(ff_inner + 40)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 28) * 401408) + ((((int)threadIdx.x) >> 4) * 25088)) + (ff_inner * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) & 15) >> 3) * 56)) + (((int)threadIdx.x) & 7)) + 48)] = conv2d_nchw_local[(ff_inner + 48)];
  }
}


