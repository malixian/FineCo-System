
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[32];
  __shared__ float pad_temp_shared[1024];
  __shared__ float kernel_shared[2048];
  for (int ff_c_inner_init = 0; ff_c_inner_init < 4; ++ff_c_inner_init) {
    for (int xx_c_inner_init = 0; xx_c_inner_init < 2; ++xx_c_inner_init) {
      conv2d_nchw_local[((ff_c_inner_init * 2) + xx_c_inner_init)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 2) + xx_c_inner_init) + 8)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 2) + xx_c_inner_init) + 16)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 2) + xx_c_inner_init) + 24)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[(((((((rc_outer_outer * 100352) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 6272)) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64) + ((int)threadIdx.x))] = kernel[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        for (int ff_c_inner = 0; ff_c_inner < 4; ++ff_c_inner) {
          for (int xx_c_inner = 0; xx_c_inner < 2; ++xx_c_inner) {
            conv2d_nchw_local[((ff_c_inner * 2) + xx_c_inner)] = (conv2d_nchw_local[((ff_c_inner * 2) + xx_c_inner)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (rc_inner * 32)) + (((((int)threadIdx.x) & 7) >> 1) * 8)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_inner)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (ff_c_inner * 32)) + (rc_outer_inner * 4)) + rc_inner)]));
            conv2d_nchw_local[(((ff_c_inner * 2) + xx_c_inner) + 8)] = (conv2d_nchw_local[(((ff_c_inner * 2) + xx_c_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (rc_inner * 32)) + (((((int)threadIdx.x) & 7) >> 1) * 8)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_inner) + 4)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (ff_c_inner * 32)) + (rc_outer_inner * 4)) + rc_inner)]));
            conv2d_nchw_local[(((ff_c_inner * 2) + xx_c_inner) + 16)] = (conv2d_nchw_local[(((ff_c_inner * 2) + xx_c_inner) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (rc_inner * 32)) + (((((int)threadIdx.x) & 7) >> 1) * 8)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_inner)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 128) + (ff_c_inner * 32)) + (rc_outer_inner * 4)) + rc_inner) + 1024)]));
            conv2d_nchw_local[(((ff_c_inner * 2) + xx_c_inner) + 24)] = (conv2d_nchw_local[(((ff_c_inner * 2) + xx_c_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (rc_inner * 32)) + (((((int)threadIdx.x) & 7) >> 1) * 8)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_inner) + 4)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 128) + (ff_c_inner * 32)) + (rc_outer_inner * 4)) + rc_inner) + 1024)]));
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 7) >> 1) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 1) * 2)) + xx_inner)] = conv2d_nchw_local[((ff_inner * 2) + xx_inner)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 3) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 7) >> 1) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 1) * 2)) + xx_inner) + 4)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 8)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 3) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 7) >> 1) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 1) * 2)) + xx_inner) + 100352)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 16)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 3) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 7) >> 1) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 1) * 2)) + xx_inner) + 100356)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 24)];
    }
  }
}


