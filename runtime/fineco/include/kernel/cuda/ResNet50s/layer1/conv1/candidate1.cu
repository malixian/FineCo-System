
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[224];
  __shared__ float pad_temp_shared[3861];
  __shared__ float kernel_shared[3136];
  for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 7; ++yy_c_outer_inner_init) {
    for (int xx_c_outer_inner_init = 0; xx_c_outer_inner_init < 2; ++xx_c_outer_inner_init) {
      conv2d_nchw_local[((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2))] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 56)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 112)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 168)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 1)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 57)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 113)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 169)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 4)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 60)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 116)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 172)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 5)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 61)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 117)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_outer_inner_init * 8) + (xx_c_outer_inner_init * 2)) + 173)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 3; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) < 3861) {
        pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x))] = (((((3 <= (((((int)blockIdx.x) >> 1) * 28) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) / 117))) && ((((((int)blockIdx.x) >> 1) * 28) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) / 117)) < 227)) && (3 <= (((((int)blockIdx.x) & 1) * 112) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) % 117)))) && ((((((int)blockIdx.x) & 1) * 112) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) % 117)) < 227)) ? data[((((((rc_outer_outer * 50176) + ((((int)blockIdx.x) >> 1) * 6272)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) / 117) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) % 117)) - 675)] : 0.000000e+00f);
      }
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 49) * 147) + (rc_outer_outer * 49)) + (((int)threadIdx.x) % 49))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)threadIdx.x) + 224) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 4) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)threadIdx.x) + 448) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 1) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((((int)threadIdx.x) + 672) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 5) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)threadIdx.x) + 896) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 2) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((((int)threadIdx.x) + 1120) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 6) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)threadIdx.x) + 1344) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 3) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)threadIdx.x) / 49) * 147) + (rc_outer_outer * 49)) + (((int)threadIdx.x) % 49)) + 4704)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)threadIdx.x) + 1792) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 4) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)threadIdx.x) + 2016) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 1) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)threadIdx.x) + 2240) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 5) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[((((((((int)threadIdx.x) + 2464) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 2) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((((int)threadIdx.x) + 2688) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 6) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[((((((((int)threadIdx.x) + 2912) / 49) * 147) + (rc_outer_outer * 49)) + ((((((int)threadIdx.x) / 7) + 3) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    __syncthreads();
    for (int rx_outer_inner = 0; rx_outer_inner < 7; ++rx_outer_inner) {
      for (int yy_c_outer_inner = 0; yy_c_outer_inner < 7; ++yy_c_outer_inner) {
        for (int xx_c_outer_inner = 0; xx_c_outer_inner < 2; ++xx_c_outer_inner) {
          for (int ry_inner = 0; ry_inner < 7; ++ry_inner) {
            conv2d_nchw_local[((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2))] = (conv2d_nchw_local[((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2))] + (pad_temp_shared[(((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner)] * kernel_shared[((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 56)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 56)] + (pad_temp_shared[(((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 784)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 112)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 112)] + (pad_temp_shared[(((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 1568)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 168)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 168)] + (pad_temp_shared[(((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 2352)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 1)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 2)] * kernel_shared[((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 57)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 57)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 784)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 113)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 113)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 1568)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 169)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 169)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 2352)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 4)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 234)] * kernel_shared[((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 60)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 60)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 234)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 784)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 116)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 116)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 234)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 1568)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 172)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 172)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 234)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 2352)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 5)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 5)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 236)] * kernel_shared[((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 61)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 61)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 236)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 784)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 117)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 117)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 236)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 1568)]));
            conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 173)] = (conv2d_nchw_local[(((yy_c_outer_inner * 8) + (xx_c_outer_inner * 2)) + 173)] + (pad_temp_shared[((((((yy_c_outer_inner * 468) + (ry_inner * 117)) + ((((int)threadIdx.x) % 14) * 8)) + (xx_c_outer_inner * 4)) + rx_outer_inner) + 236)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 49) + (ry_inner * 7)) + rx_outer_inner) + 2352)]));
          }
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 14; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 4; ++xx_inner) {
      conv2d_nchw[(((((((((int)threadIdx.x) / 14) * 12544) + ((((int)blockIdx.x) >> 1) * 1568)) + (yy_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) % 14) * 4)) + xx_inner)] = conv2d_nchw_local[((yy_inner * 4) + xx_inner)];
      conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + ((((int)blockIdx.x) >> 1) * 1568)) + (yy_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) % 14) * 4)) + xx_inner) + 200704)] = conv2d_nchw_local[(((yy_inner * 4) + xx_inner) + 56)];
      conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + ((((int)blockIdx.x) >> 1) * 1568)) + (yy_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) % 14) * 4)) + xx_inner) + 401408)] = conv2d_nchw_local[(((yy_inner * 4) + xx_inner) + 112)];
      conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + ((((int)blockIdx.x) >> 1) * 1568)) + (yy_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) % 14) * 4)) + xx_inner) + 602112)] = conv2d_nchw_local[(((yy_inner * 4) + xx_inner) + 168)];
    }
  }
}


