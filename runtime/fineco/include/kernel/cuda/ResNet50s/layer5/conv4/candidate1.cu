
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[14];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[4096];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((rc_outer_outer * 6272) + ((int)threadIdx.x))];
    pad_temp_shared[(((int)threadIdx.x) + 112)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 112)];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 224)];
    pad_temp_shared[(((int)threadIdx.x) + 336)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 336)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 448)];
    pad_temp_shared[(((int)threadIdx.x) + 560)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 560)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 672)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 896)];
    pad_temp_shared[(((int)threadIdx.x) + 1008)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1008)];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1120)];
    pad_temp_shared[(((int)threadIdx.x) + 1232)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1232)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1344)];
    pad_temp_shared[(((int)threadIdx.x) + 1456)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1456)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 1680)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1680)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1792)];
    pad_temp_shared[(((int)threadIdx.x) + 1904)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1904)];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2016)];
    pad_temp_shared[(((int)threadIdx.x) + 2128)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2128)];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2240)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2464)];
    pad_temp_shared[(((int)threadIdx.x) + 2576)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2576)];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2688)];
    pad_temp_shared[(((int)threadIdx.x) + 2800)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2800)];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2912)];
    pad_temp_shared[(((int)threadIdx.x) + 3024)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3024)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 3248)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3248)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3360)];
    pad_temp_shared[(((int)threadIdx.x) + 3472)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3472)];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3584)];
    pad_temp_shared[(((int)threadIdx.x) + 3696)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3696)];
    pad_temp_shared[(((int)threadIdx.x) + 3808)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3808)];
    pad_temp_shared[(((int)threadIdx.x) + 3920)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3920)];
    pad_temp_shared[(((int)threadIdx.x) + 4032)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4032)];
    pad_temp_shared[(((int)threadIdx.x) + 4144)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4144)];
    pad_temp_shared[(((int)threadIdx.x) + 4256)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4256)];
    pad_temp_shared[(((int)threadIdx.x) + 4368)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4368)];
    pad_temp_shared[(((int)threadIdx.x) + 4480)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4480)];
    pad_temp_shared[(((int)threadIdx.x) + 4592)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4592)];
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 4816)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4816)];
    pad_temp_shared[(((int)threadIdx.x) + 4928)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4928)];
    pad_temp_shared[(((int)threadIdx.x) + 5040)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5040)];
    pad_temp_shared[(((int)threadIdx.x) + 5152)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5152)];
    pad_temp_shared[(((int)threadIdx.x) + 5264)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5264)];
    pad_temp_shared[(((int)threadIdx.x) + 5376)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5376)];
    pad_temp_shared[(((int)threadIdx.x) + 5488)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5488)];
    pad_temp_shared[(((int)threadIdx.x) + 5600)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5600)];
    pad_temp_shared[(((int)threadIdx.x) + 5712)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5712)];
    pad_temp_shared[(((int)threadIdx.x) + 5824)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5824)];
    pad_temp_shared[(((int)threadIdx.x) + 5936)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5936)];
    pad_temp_shared[(((int)threadIdx.x) + 6048)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 6048)];
    pad_temp_shared[(((int)threadIdx.x) + 6160)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 6160)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) * 65536) + (rc_outer_outer * 128)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 112) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 224) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 336) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 448) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 560)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 560) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 672) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 784) >> 7) * 2048)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((int)blockIdx.x) * 65536) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 1008) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 1120) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1232)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 1232) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 1344) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1456)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 1456) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 1568) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1680)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 1680) >> 7) * 2048)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((int)blockIdx.x) * 65536) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 1904)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 1904) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 2016) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2128)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 2128) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 2240) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2352)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 2352) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 2464) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2576)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 2576) >> 7) * 2048)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((int)blockIdx.x) * 65536) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 43008)];
    kernel_shared[(((int)threadIdx.x) + 2800)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 2800) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 2912) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3024)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 3024) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 3136) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3248)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 3248) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 3360) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3472)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 3472) >> 7) * 2048)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((int)blockIdx.x) * 65536) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 3696)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 3696) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 3808) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3920)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 3920) >> 7) * 2048)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[((((((int)blockIdx.x) * 65536) + (((((int)threadIdx.x) + 4032) >> 7) * 2048)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 64))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int xx_c_outer_inner = 0; xx_c_outer_inner < 7; ++xx_c_outer_inner) {
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[(((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32))]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 49)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 98)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 147)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 3)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 196)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 4)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 245)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 5)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 294)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 6)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 343)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 7)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 392)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 8)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 441)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 9)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 490)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 10)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 539)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 11)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 588)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 12)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 637)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 13)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 686)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 14)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 735)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 15)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 784)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 16)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 833)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 17)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 882)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 18)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 931)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 19)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 980)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 20)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1029)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 21)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1078)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 22)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1127)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 23)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1176)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 24)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1225)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 25)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1274)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 26)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1323)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 27)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1372)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 28)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1421)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 29)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1470)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 30)]));
          conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_c_outer_inner) + 1519)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 256) + (ff_c_outer_inner * 128)) + (rc_outer_inner * 32)) + 31)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
      conv2d_nchw[(((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 98)) + (ff_inner * 49)) + ((((int)threadIdx.x) % 7) * 7)) + xx_inner)] = conv2d_nchw_local[((ff_inner * 7) + xx_inner)];
    }
  }
}


