
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(48) candidate0(float* __restrict__ Q, float* __restrict__ K, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[40];
  __shared__ float Q_shared[5];
  __shared__ float K_shared[384];
  T_batch_matmul_NT_local[0] = 0.000000e+00f;
  T_batch_matmul_NT_local[5] = 0.000000e+00f;
  T_batch_matmul_NT_local[10] = 0.000000e+00f;
  T_batch_matmul_NT_local[15] = 0.000000e+00f;
  T_batch_matmul_NT_local[20] = 0.000000e+00f;
  T_batch_matmul_NT_local[25] = 0.000000e+00f;
  T_batch_matmul_NT_local[30] = 0.000000e+00f;
  T_batch_matmul_NT_local[35] = 0.000000e+00f;
  T_batch_matmul_NT_local[1] = 0.000000e+00f;
  T_batch_matmul_NT_local[6] = 0.000000e+00f;
  T_batch_matmul_NT_local[11] = 0.000000e+00f;
  T_batch_matmul_NT_local[16] = 0.000000e+00f;
  T_batch_matmul_NT_local[21] = 0.000000e+00f;
  T_batch_matmul_NT_local[26] = 0.000000e+00f;
  T_batch_matmul_NT_local[31] = 0.000000e+00f;
  T_batch_matmul_NT_local[36] = 0.000000e+00f;
  T_batch_matmul_NT_local[2] = 0.000000e+00f;
  T_batch_matmul_NT_local[7] = 0.000000e+00f;
  T_batch_matmul_NT_local[12] = 0.000000e+00f;
  T_batch_matmul_NT_local[17] = 0.000000e+00f;
  T_batch_matmul_NT_local[22] = 0.000000e+00f;
  T_batch_matmul_NT_local[27] = 0.000000e+00f;
  T_batch_matmul_NT_local[32] = 0.000000e+00f;
  T_batch_matmul_NT_local[37] = 0.000000e+00f;
  T_batch_matmul_NT_local[3] = 0.000000e+00f;
  T_batch_matmul_NT_local[8] = 0.000000e+00f;
  T_batch_matmul_NT_local[13] = 0.000000e+00f;
  T_batch_matmul_NT_local[18] = 0.000000e+00f;
  T_batch_matmul_NT_local[23] = 0.000000e+00f;
  T_batch_matmul_NT_local[28] = 0.000000e+00f;
  T_batch_matmul_NT_local[33] = 0.000000e+00f;
  T_batch_matmul_NT_local[38] = 0.000000e+00f;
  T_batch_matmul_NT_local[4] = 0.000000e+00f;
  T_batch_matmul_NT_local[9] = 0.000000e+00f;
  T_batch_matmul_NT_local[14] = 0.000000e+00f;
  T_batch_matmul_NT_local[19] = 0.000000e+00f;
  T_batch_matmul_NT_local[24] = 0.000000e+00f;
  T_batch_matmul_NT_local[29] = 0.000000e+00f;
  T_batch_matmul_NT_local[34] = 0.000000e+00f;
  T_batch_matmul_NT_local[39] = 0.000000e+00f;
  if (((int)threadIdx.x) < 5) {
    Q_shared[((int)threadIdx.x)] = Q[(((int)threadIdx.x) * 5)];
  }
  K_shared[((int)threadIdx.x)] = K[((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5))];
  K_shared[(((int)threadIdx.x) + 48)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 240)];
  K_shared[(((int)threadIdx.x) + 96)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 480)];
  K_shared[(((int)threadIdx.x) + 144)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 720)];
  K_shared[(((int)threadIdx.x) + 192)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 960)];
  K_shared[(((int)threadIdx.x) + 240)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1200)];
  K_shared[(((int)threadIdx.x) + 288)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1440)];
  K_shared[(((int)threadIdx.x) + 336)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1680)];
  __syncthreads();
  T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (Q_shared[0] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[5] = (T_batch_matmul_NT_local[5] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[10] = (T_batch_matmul_NT_local[10] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[15] = (T_batch_matmul_NT_local[15] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[20] = (T_batch_matmul_NT_local[20] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[25] = (T_batch_matmul_NT_local[25] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[30] = (T_batch_matmul_NT_local[30] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[35] = (T_batch_matmul_NT_local[35] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (Q_shared[1] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[6] = (T_batch_matmul_NT_local[6] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[11] = (T_batch_matmul_NT_local[11] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[16] = (T_batch_matmul_NT_local[16] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[21] = (T_batch_matmul_NT_local[21] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[26] = (T_batch_matmul_NT_local[26] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[31] = (T_batch_matmul_NT_local[31] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[36] = (T_batch_matmul_NT_local[36] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (Q_shared[2] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[7] = (T_batch_matmul_NT_local[7] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[12] = (T_batch_matmul_NT_local[12] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[17] = (T_batch_matmul_NT_local[17] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[22] = (T_batch_matmul_NT_local[22] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[27] = (T_batch_matmul_NT_local[27] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[32] = (T_batch_matmul_NT_local[32] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[37] = (T_batch_matmul_NT_local[37] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (Q_shared[3] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[8] = (T_batch_matmul_NT_local[8] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[13] = (T_batch_matmul_NT_local[13] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[18] = (T_batch_matmul_NT_local[18] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[23] = (T_batch_matmul_NT_local[23] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[28] = (T_batch_matmul_NT_local[28] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[33] = (T_batch_matmul_NT_local[33] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[38] = (T_batch_matmul_NT_local[38] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (Q_shared[4] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[9] = (T_batch_matmul_NT_local[9] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[14] = (T_batch_matmul_NT_local[14] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[19] = (T_batch_matmul_NT_local[19] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[24] = (T_batch_matmul_NT_local[24] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[29] = (T_batch_matmul_NT_local[29] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[34] = (T_batch_matmul_NT_local[34] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[39] = (T_batch_matmul_NT_local[39] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 336)]));
  __syncthreads();
  if (((int)threadIdx.x) < 5) {
    Q_shared[((int)threadIdx.x)] = Q[((((int)threadIdx.x) * 5) + 1)];
  }
  K_shared[((int)threadIdx.x)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1)];
  K_shared[(((int)threadIdx.x) + 48)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 241)];
  K_shared[(((int)threadIdx.x) + 96)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 481)];
  K_shared[(((int)threadIdx.x) + 144)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 721)];
  K_shared[(((int)threadIdx.x) + 192)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 961)];
  K_shared[(((int)threadIdx.x) + 240)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1201)];
  K_shared[(((int)threadIdx.x) + 288)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1441)];
  K_shared[(((int)threadIdx.x) + 336)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1681)];
  __syncthreads();
  T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (Q_shared[0] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[5] = (T_batch_matmul_NT_local[5] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[10] = (T_batch_matmul_NT_local[10] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[15] = (T_batch_matmul_NT_local[15] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[20] = (T_batch_matmul_NT_local[20] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[25] = (T_batch_matmul_NT_local[25] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[30] = (T_batch_matmul_NT_local[30] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[35] = (T_batch_matmul_NT_local[35] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (Q_shared[1] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[6] = (T_batch_matmul_NT_local[6] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[11] = (T_batch_matmul_NT_local[11] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[16] = (T_batch_matmul_NT_local[16] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[21] = (T_batch_matmul_NT_local[21] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[26] = (T_batch_matmul_NT_local[26] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[31] = (T_batch_matmul_NT_local[31] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[36] = (T_batch_matmul_NT_local[36] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (Q_shared[2] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[7] = (T_batch_matmul_NT_local[7] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[12] = (T_batch_matmul_NT_local[12] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[17] = (T_batch_matmul_NT_local[17] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[22] = (T_batch_matmul_NT_local[22] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[27] = (T_batch_matmul_NT_local[27] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[32] = (T_batch_matmul_NT_local[32] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[37] = (T_batch_matmul_NT_local[37] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (Q_shared[3] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[8] = (T_batch_matmul_NT_local[8] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[13] = (T_batch_matmul_NT_local[13] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[18] = (T_batch_matmul_NT_local[18] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[23] = (T_batch_matmul_NT_local[23] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[28] = (T_batch_matmul_NT_local[28] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[33] = (T_batch_matmul_NT_local[33] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[38] = (T_batch_matmul_NT_local[38] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (Q_shared[4] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[9] = (T_batch_matmul_NT_local[9] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[14] = (T_batch_matmul_NT_local[14] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[19] = (T_batch_matmul_NT_local[19] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[24] = (T_batch_matmul_NT_local[24] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[29] = (T_batch_matmul_NT_local[29] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[34] = (T_batch_matmul_NT_local[34] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[39] = (T_batch_matmul_NT_local[39] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 336)]));
  __syncthreads();
  if (((int)threadIdx.x) < 5) {
    Q_shared[((int)threadIdx.x)] = Q[((((int)threadIdx.x) * 5) + 2)];
  }
  K_shared[((int)threadIdx.x)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 2)];
  K_shared[(((int)threadIdx.x) + 48)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 242)];
  K_shared[(((int)threadIdx.x) + 96)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 482)];
  K_shared[(((int)threadIdx.x) + 144)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 722)];
  K_shared[(((int)threadIdx.x) + 192)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 962)];
  K_shared[(((int)threadIdx.x) + 240)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1202)];
  K_shared[(((int)threadIdx.x) + 288)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1442)];
  K_shared[(((int)threadIdx.x) + 336)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1682)];
  __syncthreads();
  T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (Q_shared[0] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[5] = (T_batch_matmul_NT_local[5] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[10] = (T_batch_matmul_NT_local[10] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[15] = (T_batch_matmul_NT_local[15] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[20] = (T_batch_matmul_NT_local[20] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[25] = (T_batch_matmul_NT_local[25] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[30] = (T_batch_matmul_NT_local[30] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[35] = (T_batch_matmul_NT_local[35] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (Q_shared[1] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[6] = (T_batch_matmul_NT_local[6] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[11] = (T_batch_matmul_NT_local[11] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[16] = (T_batch_matmul_NT_local[16] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[21] = (T_batch_matmul_NT_local[21] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[26] = (T_batch_matmul_NT_local[26] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[31] = (T_batch_matmul_NT_local[31] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[36] = (T_batch_matmul_NT_local[36] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (Q_shared[2] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[7] = (T_batch_matmul_NT_local[7] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[12] = (T_batch_matmul_NT_local[12] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[17] = (T_batch_matmul_NT_local[17] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[22] = (T_batch_matmul_NT_local[22] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[27] = (T_batch_matmul_NT_local[27] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[32] = (T_batch_matmul_NT_local[32] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[37] = (T_batch_matmul_NT_local[37] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (Q_shared[3] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[8] = (T_batch_matmul_NT_local[8] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[13] = (T_batch_matmul_NT_local[13] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[18] = (T_batch_matmul_NT_local[18] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[23] = (T_batch_matmul_NT_local[23] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[28] = (T_batch_matmul_NT_local[28] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[33] = (T_batch_matmul_NT_local[33] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[38] = (T_batch_matmul_NT_local[38] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (Q_shared[4] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[9] = (T_batch_matmul_NT_local[9] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[14] = (T_batch_matmul_NT_local[14] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[19] = (T_batch_matmul_NT_local[19] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[24] = (T_batch_matmul_NT_local[24] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[29] = (T_batch_matmul_NT_local[29] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[34] = (T_batch_matmul_NT_local[34] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[39] = (T_batch_matmul_NT_local[39] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 336)]));
  __syncthreads();
  if (((int)threadIdx.x) < 5) {
    Q_shared[((int)threadIdx.x)] = Q[((((int)threadIdx.x) * 5) + 3)];
  }
  K_shared[((int)threadIdx.x)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 3)];
  K_shared[(((int)threadIdx.x) + 48)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 243)];
  K_shared[(((int)threadIdx.x) + 96)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 483)];
  K_shared[(((int)threadIdx.x) + 144)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 723)];
  K_shared[(((int)threadIdx.x) + 192)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 963)];
  K_shared[(((int)threadIdx.x) + 240)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1203)];
  K_shared[(((int)threadIdx.x) + 288)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1443)];
  K_shared[(((int)threadIdx.x) + 336)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1683)];
  __syncthreads();
  T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (Q_shared[0] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[5] = (T_batch_matmul_NT_local[5] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[10] = (T_batch_matmul_NT_local[10] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[15] = (T_batch_matmul_NT_local[15] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[20] = (T_batch_matmul_NT_local[20] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[25] = (T_batch_matmul_NT_local[25] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[30] = (T_batch_matmul_NT_local[30] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[35] = (T_batch_matmul_NT_local[35] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (Q_shared[1] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[6] = (T_batch_matmul_NT_local[6] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[11] = (T_batch_matmul_NT_local[11] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[16] = (T_batch_matmul_NT_local[16] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[21] = (T_batch_matmul_NT_local[21] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[26] = (T_batch_matmul_NT_local[26] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[31] = (T_batch_matmul_NT_local[31] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[36] = (T_batch_matmul_NT_local[36] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (Q_shared[2] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[7] = (T_batch_matmul_NT_local[7] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[12] = (T_batch_matmul_NT_local[12] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[17] = (T_batch_matmul_NT_local[17] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[22] = (T_batch_matmul_NT_local[22] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[27] = (T_batch_matmul_NT_local[27] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[32] = (T_batch_matmul_NT_local[32] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[37] = (T_batch_matmul_NT_local[37] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (Q_shared[3] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[8] = (T_batch_matmul_NT_local[8] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[13] = (T_batch_matmul_NT_local[13] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[18] = (T_batch_matmul_NT_local[18] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[23] = (T_batch_matmul_NT_local[23] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[28] = (T_batch_matmul_NT_local[28] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[33] = (T_batch_matmul_NT_local[33] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[38] = (T_batch_matmul_NT_local[38] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (Q_shared[4] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[9] = (T_batch_matmul_NT_local[9] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[14] = (T_batch_matmul_NT_local[14] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[19] = (T_batch_matmul_NT_local[19] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[24] = (T_batch_matmul_NT_local[24] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[29] = (T_batch_matmul_NT_local[29] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[34] = (T_batch_matmul_NT_local[34] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[39] = (T_batch_matmul_NT_local[39] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 336)]));
  __syncthreads();
  if (((int)threadIdx.x) < 5) {
    Q_shared[((int)threadIdx.x)] = Q[((((int)threadIdx.x) * 5) + 4)];
  }
  K_shared[((int)threadIdx.x)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 4)];
  K_shared[(((int)threadIdx.x) + 48)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 244)];
  K_shared[(((int)threadIdx.x) + 96)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 484)];
  K_shared[(((int)threadIdx.x) + 144)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 724)];
  K_shared[(((int)threadIdx.x) + 192)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 964)];
  K_shared[(((int)threadIdx.x) + 240)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1204)];
  K_shared[(((int)threadIdx.x) + 288)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1444)];
  K_shared[(((int)threadIdx.x) + 336)] = K[(((((int)blockIdx.x) * 1920) + (((int)threadIdx.x) * 5)) + 1684)];
  __syncthreads();
  T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (Q_shared[0] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[5] = (T_batch_matmul_NT_local[5] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[10] = (T_batch_matmul_NT_local[10] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[15] = (T_batch_matmul_NT_local[15] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[20] = (T_batch_matmul_NT_local[20] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[25] = (T_batch_matmul_NT_local[25] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[30] = (T_batch_matmul_NT_local[30] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[35] = (T_batch_matmul_NT_local[35] + (Q_shared[0] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (Q_shared[1] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[6] = (T_batch_matmul_NT_local[6] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[11] = (T_batch_matmul_NT_local[11] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[16] = (T_batch_matmul_NT_local[16] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[21] = (T_batch_matmul_NT_local[21] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[26] = (T_batch_matmul_NT_local[26] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[31] = (T_batch_matmul_NT_local[31] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[36] = (T_batch_matmul_NT_local[36] + (Q_shared[1] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (Q_shared[2] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[7] = (T_batch_matmul_NT_local[7] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[12] = (T_batch_matmul_NT_local[12] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[17] = (T_batch_matmul_NT_local[17] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[22] = (T_batch_matmul_NT_local[22] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[27] = (T_batch_matmul_NT_local[27] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[32] = (T_batch_matmul_NT_local[32] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[37] = (T_batch_matmul_NT_local[37] + (Q_shared[2] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (Q_shared[3] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[8] = (T_batch_matmul_NT_local[8] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[13] = (T_batch_matmul_NT_local[13] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[18] = (T_batch_matmul_NT_local[18] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[23] = (T_batch_matmul_NT_local[23] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[28] = (T_batch_matmul_NT_local[28] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[33] = (T_batch_matmul_NT_local[33] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[38] = (T_batch_matmul_NT_local[38] + (Q_shared[3] * K_shared[(((int)threadIdx.x) + 336)]));
  T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (Q_shared[4] * K_shared[((int)threadIdx.x)]));
  T_batch_matmul_NT_local[9] = (T_batch_matmul_NT_local[9] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 48)]));
  T_batch_matmul_NT_local[14] = (T_batch_matmul_NT_local[14] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 96)]));
  T_batch_matmul_NT_local[19] = (T_batch_matmul_NT_local[19] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 144)]));
  T_batch_matmul_NT_local[24] = (T_batch_matmul_NT_local[24] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 192)]));
  T_batch_matmul_NT_local[29] = (T_batch_matmul_NT_local[29] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 240)]));
  T_batch_matmul_NT_local[34] = (T_batch_matmul_NT_local[34] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 288)]));
  T_batch_matmul_NT_local[39] = (T_batch_matmul_NT_local[39] + (Q_shared[4] * K_shared[(((int)threadIdx.x) + 336)]));
  for (int i_inner = 0; i_inner < 5; ++i_inner) {
    T_batch_matmul_NT[(((i_inner * 768) + (((int)blockIdx.x) * 384)) + ((int)threadIdx.x))] = T_batch_matmul_NT_local[i_inner];
    T_batch_matmul_NT[((((i_inner * 768) + (((int)blockIdx.x) * 384)) + ((int)threadIdx.x)) + 48)] = T_batch_matmul_NT_local[(i_inner + 5)];
    T_batch_matmul_NT[((((i_inner * 768) + (((int)blockIdx.x) * 384)) + ((int)threadIdx.x)) + 96)] = T_batch_matmul_NT_local[(i_inner + 10)];
    T_batch_matmul_NT[((((i_inner * 768) + (((int)blockIdx.x) * 384)) + ((int)threadIdx.x)) + 144)] = T_batch_matmul_NT_local[(i_inner + 15)];
    T_batch_matmul_NT[((((i_inner * 768) + (((int)blockIdx.x) * 384)) + ((int)threadIdx.x)) + 192)] = T_batch_matmul_NT_local[(i_inner + 20)];
    T_batch_matmul_NT[((((i_inner * 768) + (((int)blockIdx.x) * 384)) + ((int)threadIdx.x)) + 240)] = T_batch_matmul_NT_local[(i_inner + 25)];
    T_batch_matmul_NT[((((i_inner * 768) + (((int)blockIdx.x) * 384)) + ((int)threadIdx.x)) + 288)] = T_batch_matmul_NT_local[(i_inner + 30)];
    T_batch_matmul_NT[((((i_inner * 768) + (((int)blockIdx.x) * 384)) + ((int)threadIdx.x)) + 336)] = T_batch_matmul_NT_local[(i_inner + 35)];
  }
}


