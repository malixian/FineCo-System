
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(96) candidate1(float* __restrict__ I, float* __restrict__ W, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[5];
  __shared__ float I_shared[40];
  __shared__ float W_shared[768];
  T_batch_matmul_NT_local[0] = 0.000000e+00f;
  T_batch_matmul_NT_local[1] = 0.000000e+00f;
  T_batch_matmul_NT_local[2] = 0.000000e+00f;
  T_batch_matmul_NT_local[3] = 0.000000e+00f;
  T_batch_matmul_NT_local[4] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 96; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 40) {
      I_shared[((int)threadIdx.x)] = I[((((((int)threadIdx.x) >> 3) * 768) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    *(float2*)(W_shared + (((int)threadIdx.x) * 2)) = *(float2*)(W + ((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) >> 2) * 768)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)));
    *(float2*)(W_shared + ((((int)threadIdx.x) * 2) + 192)) = *(float2*)(W + (((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) >> 2) * 768)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 18432));
    *(float2*)(W_shared + ((((int)threadIdx.x) * 2) + 384)) = *(float2*)(W + (((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) >> 2) * 768)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 36864));
    *(float2*)(W_shared + ((((int)threadIdx.x) * 2) + 576)) = *(float2*)(W + (((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) >> 2) * 768)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 55296));
    __syncthreads();
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[0] * W_shared[(((int)threadIdx.x) * 8)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[8] * W_shared[(((int)threadIdx.x) * 8)]));
    T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[16] * W_shared[(((int)threadIdx.x) * 8)]));
    T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[24] * W_shared[(((int)threadIdx.x) * 8)]));
    T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[32] * W_shared[(((int)threadIdx.x) * 8)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[1] * W_shared[((((int)threadIdx.x) * 8) + 1)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[9] * W_shared[((((int)threadIdx.x) * 8) + 1)]));
    T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[17] * W_shared[((((int)threadIdx.x) * 8) + 1)]));
    T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[25] * W_shared[((((int)threadIdx.x) * 8) + 1)]));
    T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[33] * W_shared[((((int)threadIdx.x) * 8) + 1)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[2] * W_shared[((((int)threadIdx.x) * 8) + 2)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[10] * W_shared[((((int)threadIdx.x) * 8) + 2)]));
    T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[18] * W_shared[((((int)threadIdx.x) * 8) + 2)]));
    T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[26] * W_shared[((((int)threadIdx.x) * 8) + 2)]));
    T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[34] * W_shared[((((int)threadIdx.x) * 8) + 2)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[3] * W_shared[((((int)threadIdx.x) * 8) + 3)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[11] * W_shared[((((int)threadIdx.x) * 8) + 3)]));
    T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[19] * W_shared[((((int)threadIdx.x) * 8) + 3)]));
    T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[27] * W_shared[((((int)threadIdx.x) * 8) + 3)]));
    T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[35] * W_shared[((((int)threadIdx.x) * 8) + 3)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[4] * W_shared[((((int)threadIdx.x) * 8) + 4)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[12] * W_shared[((((int)threadIdx.x) * 8) + 4)]));
    T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[20] * W_shared[((((int)threadIdx.x) * 8) + 4)]));
    T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[28] * W_shared[((((int)threadIdx.x) * 8) + 4)]));
    T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[36] * W_shared[((((int)threadIdx.x) * 8) + 4)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[5] * W_shared[((((int)threadIdx.x) * 8) + 5)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[13] * W_shared[((((int)threadIdx.x) * 8) + 5)]));
    T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[21] * W_shared[((((int)threadIdx.x) * 8) + 5)]));
    T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[29] * W_shared[((((int)threadIdx.x) * 8) + 5)]));
    T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[37] * W_shared[((((int)threadIdx.x) * 8) + 5)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[6] * W_shared[((((int)threadIdx.x) * 8) + 6)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[14] * W_shared[((((int)threadIdx.x) * 8) + 6)]));
    T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[22] * W_shared[((((int)threadIdx.x) * 8) + 6)]));
    T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[30] * W_shared[((((int)threadIdx.x) * 8) + 6)]));
    T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[38] * W_shared[((((int)threadIdx.x) * 8) + 6)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[7] * W_shared[((((int)threadIdx.x) * 8) + 7)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[15] * W_shared[((((int)threadIdx.x) * 8) + 7)]));
    T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[23] * W_shared[((((int)threadIdx.x) * 8) + 7)]));
    T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[31] * W_shared[((((int)threadIdx.x) * 8) + 7)]));
    T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[39] * W_shared[((((int)threadIdx.x) * 8) + 7)]));
  }
  for (int i_inner = 0; i_inner < 5; ++i_inner) {
    T_batch_matmul_NT[(((i_inner * 3072) + (((int)blockIdx.x) * 96)) + ((int)threadIdx.x))] = T_batch_matmul_NT_local[i_inner];
  }
}


