
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(120) candidate0(float* __restrict__ I, float* __restrict__ W, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[2];
  __shared__ float I_shared[120];
  __shared__ float W_shared[1152];
  T_batch_matmul_NT_local[0] = 0.000000e+00f;
  T_batch_matmul_NT_local[1] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 30) {
      *(float4*)(I_shared + (((int)threadIdx.x) * 4)) = *(float4*)(I + ((((((int)threadIdx.x) / 6) * 768) + (k_outer_outer * 24)) + ((((int)threadIdx.x) % 6) * 4)));
    }
    W_shared[((int)threadIdx.x)] = W[((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24))];
    W_shared[(((int)threadIdx.x) + 120)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 3840)];
    W_shared[(((int)threadIdx.x) + 240)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 7680)];
    W_shared[(((int)threadIdx.x) + 360)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 11520)];
    W_shared[(((int)threadIdx.x) + 480)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 15360)];
    W_shared[(((int)threadIdx.x) + 600)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 19200)];
    W_shared[(((int)threadIdx.x) + 720)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 23040)];
    W_shared[(((int)threadIdx.x) + 840)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 26880)];
    W_shared[(((int)threadIdx.x) + 960)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 30720)];
    if (((int)threadIdx.x) < 72) {
      W_shared[(((int)threadIdx.x) + 1080)] = W[(((((((int)blockIdx.x) * 36864) + ((((int)threadIdx.x) / 24) * 768)) + (k_outer_outer * 24)) + (((int)threadIdx.x) % 24)) + 34560)];
    }
    __syncthreads();
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[((((int)threadIdx.x) / 24) * 24)] * W_shared[((((int)threadIdx.x) % 24) * 24)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[((((int)threadIdx.x) / 24) * 24)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 576)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 1)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 1)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 1)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 577)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 2)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 2)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 2)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 578)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 3)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 3)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 3)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 579)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 4)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 4)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 4)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 580)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 5)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 5)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 5)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 581)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 6)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 6)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 6)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 582)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 7)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 7)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 7)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 583)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 8)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 8)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 8)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 584)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 9)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 9)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 9)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 585)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 10)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 10)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 10)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 586)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 11)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 11)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 11)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 587)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 12)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 12)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 12)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 588)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 13)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 13)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 13)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 589)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 14)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 14)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 14)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 590)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 15)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 15)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 15)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 591)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 16)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 16)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 16)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 592)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 17)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 17)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 17)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 593)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 18)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 18)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 18)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 594)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 19)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 19)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 19)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 595)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 20)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 20)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 20)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 596)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 21)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 21)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 21)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 597)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 22)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 22)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 22)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 598)]));
    T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 23)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 23)]));
    T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((((int)threadIdx.x) / 24) * 24) + 23)] * W_shared[(((((int)threadIdx.x) % 24) * 24) + 599)]));
  }
  T_batch_matmul_NT[((((((int)threadIdx.x) / 24) * 3072) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) % 24))] = T_batch_matmul_NT_local[0];
  T_batch_matmul_NT[(((((((int)threadIdx.x) / 24) * 3072) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) % 24)) + 24)] = T_batch_matmul_NT_local[1];
}


