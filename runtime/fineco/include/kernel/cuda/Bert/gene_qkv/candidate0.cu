
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) candidate0(float* __restrict__ I, float* __restrict__ W, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[5];
  __shared__ float I_shared[640];
  __shared__ float W_shared[4096];
  T_batch_matmul_NT_local[0] = 0.000000e+00f;
  T_batch_matmul_NT_local[1] = 0.000000e+00f;
  T_batch_matmul_NT_local[2] = 0.000000e+00f;
  T_batch_matmul_NT_local[3] = 0.000000e+00f;
  T_batch_matmul_NT_local[4] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 6; ++k_outer_outer) {
    __syncthreads();
    *(float4*)(I_shared + (((int)threadIdx.x) * 4)) = *(float4*)(I + ((k_outer_outer * 128) + (((int)threadIdx.x) * 4)));
    *(float4*)(I_shared + ((((int)threadIdx.x) * 4) + 128)) = *(float4*)(I + (((k_outer_outer * 128) + (((int)threadIdx.x) * 4)) + 768));
    *(float4*)(I_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(I + (((k_outer_outer * 128) + (((int)threadIdx.x) * 4)) + 1536));
    *(float4*)(I_shared + ((((int)threadIdx.x) * 4) + 384)) = *(float4*)(I + (((k_outer_outer * 128) + (((int)threadIdx.x) * 4)) + 2304));
    *(float4*)(I_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(I + (((k_outer_outer * 128) + (((int)threadIdx.x) * 4)) + 3072));
    *(float4*)(W_shared + (((int)threadIdx.x) * 4)) = *(float4*)(W + (((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 128)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 768));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 1536));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 384)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 2304));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 3072));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 640)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 3840));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 4608));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 896)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 5376));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 6144));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 1152)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 6912));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 1280)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 7680));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 1408)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 8448));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 9216));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 1664)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 9984));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 1792)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 10752));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 1920)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 11520));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 12288));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 2176)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 13056));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 2304)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 13824));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 2432)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 14592));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 15360));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 2688)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 16128));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 2816)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 16896));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 2944)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 17664));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 18432));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 3200)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 19200));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 3328)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 19968));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 3456)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 20736));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 21504));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 3712)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 22272));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 3840)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 23040));
    *(float4*)(W_shared + ((((int)threadIdx.x) * 4) + 3968)) = *(float4*)(W + ((((((int)blockIdx.x) * 24576) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 23808));
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 64; ++k_inner) {
        T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[((k_outer_inner * 64) + k_inner)] * W_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 64)) + k_inner)]));
        T_batch_matmul_NT_local[1] = (T_batch_matmul_NT_local[1] + (I_shared[(((k_outer_inner * 64) + k_inner) + 128)] * W_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 64)) + k_inner)]));
        T_batch_matmul_NT_local[2] = (T_batch_matmul_NT_local[2] + (I_shared[(((k_outer_inner * 64) + k_inner) + 256)] * W_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 64)) + k_inner)]));
        T_batch_matmul_NT_local[3] = (T_batch_matmul_NT_local[3] + (I_shared[(((k_outer_inner * 64) + k_inner) + 384)] * W_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 64)) + k_inner)]));
        T_batch_matmul_NT_local[4] = (T_batch_matmul_NT_local[4] + (I_shared[(((k_outer_inner * 64) + k_inner) + 512)] * W_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 64)) + k_inner)]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 5; ++i_inner) {
    T_batch_matmul_NT[(((i_inner * 768) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x))] = T_batch_matmul_NT_local[i_inner];
  }
}


