
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate3(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[64];
  __shared__ float pad_temp_shared[4096];
  __shared__ float kernel_shared[1024];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[16] = 0.000000e+00f;
  conv2d_nchw[32] = 0.000000e+00f;
  conv2d_nchw[48] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[17] = 0.000000e+00f;
  conv2d_nchw[33] = 0.000000e+00f;
  conv2d_nchw[49] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[18] = 0.000000e+00f;
  conv2d_nchw[34] = 0.000000e+00f;
  conv2d_nchw[50] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[19] = 0.000000e+00f;
  conv2d_nchw[35] = 0.000000e+00f;
  conv2d_nchw[51] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[20] = 0.000000e+00f;
  conv2d_nchw[36] = 0.000000e+00f;
  conv2d_nchw[52] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[21] = 0.000000e+00f;
  conv2d_nchw[37] = 0.000000e+00f;
  conv2d_nchw[53] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[22] = 0.000000e+00f;
  conv2d_nchw[38] = 0.000000e+00f;
  conv2d_nchw[54] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[23] = 0.000000e+00f;
  conv2d_nchw[39] = 0.000000e+00f;
  conv2d_nchw[55] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[24] = 0.000000e+00f;
  conv2d_nchw[40] = 0.000000e+00f;
  conv2d_nchw[56] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[25] = 0.000000e+00f;
  conv2d_nchw[41] = 0.000000e+00f;
  conv2d_nchw[57] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[26] = 0.000000e+00f;
  conv2d_nchw[42] = 0.000000e+00f;
  conv2d_nchw[58] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[27] = 0.000000e+00f;
  conv2d_nchw[43] = 0.000000e+00f;
  conv2d_nchw[59] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[28] = 0.000000e+00f;
  conv2d_nchw[44] = 0.000000e+00f;
  conv2d_nchw[60] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[29] = 0.000000e+00f;
  conv2d_nchw[45] = 0.000000e+00f;
  conv2d_nchw[61] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[30] = 0.000000e+00f;
  conv2d_nchw[46] = 0.000000e+00f;
  conv2d_nchw[62] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  conv2d_nchw[31] = 0.000000e+00f;
  conv2d_nchw[47] = 0.000000e+00f;
  conv2d_nchw[63] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[(((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15))];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 512)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 768)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 37632)];
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 62720)];
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 75264)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 87808)];
    pad_temp_shared[(((int)threadIdx.x) + 2048)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 100352)];
    pad_temp_shared[(((int)threadIdx.x) + 2304)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 112896)];
    pad_temp_shared[(((int)threadIdx.x) + 2560)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 125440)];
    pad_temp_shared[(((int)threadIdx.x) + 2816)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 137984)];
    pad_temp_shared[(((int)threadIdx.x) + 3072)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 150528)];
    pad_temp_shared[(((int)threadIdx.x) + 3328)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 163072)];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 175616)];
    pad_temp_shared[(((int)threadIdx.x) + 3840)] = Input[((((((rc_outer_outer * 200704) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 188160)];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 4) * 32) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) >> 4) * 32) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 512)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) >> 4) * 32) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 1024)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) >> 4) * 32) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 1536)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((rc_outer_inner * 256) + (((int)threadIdx.x) & 15))] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[16] = (conv2d_nchw[16] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 128)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[32] = (conv2d_nchw[32] + (pad_temp_shared[((rc_outer_inner * 256) + (((int)threadIdx.x) & 15))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[48] = (conv2d_nchw[48] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[17] = (conv2d_nchw[17] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 144)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[33] = (conv2d_nchw[33] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[49] = (conv2d_nchw[49] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 144)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[18] = (conv2d_nchw[18] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 160)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[34] = (conv2d_nchw[34] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[50] = (conv2d_nchw[50] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 160)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 48)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[19] = (conv2d_nchw[19] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 176)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[35] = (conv2d_nchw[35] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[51] = (conv2d_nchw[51] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 176)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 64)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[20] = (conv2d_nchw[20] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 192)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[36] = (conv2d_nchw[36] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[52] = (conv2d_nchw[52] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 192)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 80)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[21] = (conv2d_nchw[21] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 208)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[37] = (conv2d_nchw[37] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[53] = (conv2d_nchw[53] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 208)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 96)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[22] = (conv2d_nchw[22] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 224)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[38] = (conv2d_nchw[38] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[54] = (conv2d_nchw[54] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 224)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 112)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[23] = (conv2d_nchw[23] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 240)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner)]));
      conv2d_nchw[39] = (conv2d_nchw[39] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[55] = (conv2d_nchw[55] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 240)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 512)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((rc_outer_inner * 256) + (((int)threadIdx.x) & 15))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[24] = (conv2d_nchw[24] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[40] = (conv2d_nchw[40] + (pad_temp_shared[((rc_outer_inner * 256) + (((int)threadIdx.x) & 15))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[56] = (conv2d_nchw[56] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[25] = (conv2d_nchw[25] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 144)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[41] = (conv2d_nchw[41] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[57] = (conv2d_nchw[57] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 144)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[26] = (conv2d_nchw[26] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 160)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[42] = (conv2d_nchw[42] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[58] = (conv2d_nchw[58] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 160)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[27] = (conv2d_nchw[27] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 176)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[43] = (conv2d_nchw[43] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[59] = (conv2d_nchw[59] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 176)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[28] = (conv2d_nchw[28] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 192)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[44] = (conv2d_nchw[44] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[60] = (conv2d_nchw[60] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 192)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[29] = (conv2d_nchw[29] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 208)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[45] = (conv2d_nchw[45] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[61] = (conv2d_nchw[61] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 208)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[30] = (conv2d_nchw[30] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 224)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[46] = (conv2d_nchw[46] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[62] = (conv2d_nchw[62] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 224)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[31] = (conv2d_nchw[31] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 240)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 16)]));
      conv2d_nchw[47] = (conv2d_nchw[47] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
      conv2d_nchw[63] = (conv2d_nchw[63] + (pad_temp_shared[(((rc_outer_inner * 256) + (((int)threadIdx.x) & 15)) + 240)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 32) + rc_outer_inner) + 528)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 8; ++i2_inner) {
      compute[(((((((((int)threadIdx.x) >> 4) * 25088) + (i1_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (i2_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15))] = max(conv2d_nchw[((i1_inner * 8) + i2_inner)], 0.000000e+00f);
      compute[((((((((((int)threadIdx.x) >> 4) * 25088) + (i1_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (i2_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 896)] = max(conv2d_nchw[(((i1_inner * 8) + i2_inner) + 16)], 0.000000e+00f);
      compute[((((((((((int)threadIdx.x) >> 4) * 25088) + (i1_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (i2_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 401408)] = max(conv2d_nchw[(((i1_inner * 8) + i2_inner) + 32)], 0.000000e+00f);
      compute[((((((((((int)threadIdx.x) >> 4) * 25088) + (i1_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (i2_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15)) + 402304)] = max(conv2d_nchw[(((i1_inner * 8) + i2_inner) + 48)], 0.000000e+00f);
    }
  }
}


