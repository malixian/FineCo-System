
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate2(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[64];
  __shared__ float PaddedInput_shared[8960];
  __shared__ float kernel_shared[48];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[16] = 0.000000e+00f;
  DepthwiseConv2d[32] = 0.000000e+00f;
  DepthwiseConv2d[48] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[17] = 0.000000e+00f;
  DepthwiseConv2d[33] = 0.000000e+00f;
  DepthwiseConv2d[49] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[18] = 0.000000e+00f;
  DepthwiseConv2d[34] = 0.000000e+00f;
  DepthwiseConv2d[50] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  DepthwiseConv2d[19] = 0.000000e+00f;
  DepthwiseConv2d[35] = 0.000000e+00f;
  DepthwiseConv2d[51] = 0.000000e+00f;
  DepthwiseConv2d[4] = 0.000000e+00f;
  DepthwiseConv2d[20] = 0.000000e+00f;
  DepthwiseConv2d[36] = 0.000000e+00f;
  DepthwiseConv2d[52] = 0.000000e+00f;
  DepthwiseConv2d[5] = 0.000000e+00f;
  DepthwiseConv2d[21] = 0.000000e+00f;
  DepthwiseConv2d[37] = 0.000000e+00f;
  DepthwiseConv2d[53] = 0.000000e+00f;
  DepthwiseConv2d[6] = 0.000000e+00f;
  DepthwiseConv2d[22] = 0.000000e+00f;
  DepthwiseConv2d[38] = 0.000000e+00f;
  DepthwiseConv2d[54] = 0.000000e+00f;
  DepthwiseConv2d[7] = 0.000000e+00f;
  DepthwiseConv2d[23] = 0.000000e+00f;
  DepthwiseConv2d[39] = 0.000000e+00f;
  DepthwiseConv2d[55] = 0.000000e+00f;
  DepthwiseConv2d[8] = 0.000000e+00f;
  DepthwiseConv2d[24] = 0.000000e+00f;
  DepthwiseConv2d[40] = 0.000000e+00f;
  DepthwiseConv2d[56] = 0.000000e+00f;
  DepthwiseConv2d[9] = 0.000000e+00f;
  DepthwiseConv2d[25] = 0.000000e+00f;
  DepthwiseConv2d[41] = 0.000000e+00f;
  DepthwiseConv2d[57] = 0.000000e+00f;
  DepthwiseConv2d[10] = 0.000000e+00f;
  DepthwiseConv2d[26] = 0.000000e+00f;
  DepthwiseConv2d[42] = 0.000000e+00f;
  DepthwiseConv2d[58] = 0.000000e+00f;
  DepthwiseConv2d[11] = 0.000000e+00f;
  DepthwiseConv2d[27] = 0.000000e+00f;
  DepthwiseConv2d[43] = 0.000000e+00f;
  DepthwiseConv2d[59] = 0.000000e+00f;
  DepthwiseConv2d[12] = 0.000000e+00f;
  DepthwiseConv2d[28] = 0.000000e+00f;
  DepthwiseConv2d[44] = 0.000000e+00f;
  DepthwiseConv2d[60] = 0.000000e+00f;
  DepthwiseConv2d[13] = 0.000000e+00f;
  DepthwiseConv2d[29] = 0.000000e+00f;
  DepthwiseConv2d[45] = 0.000000e+00f;
  DepthwiseConv2d[61] = 0.000000e+00f;
  DepthwiseConv2d[14] = 0.000000e+00f;
  DepthwiseConv2d[30] = 0.000000e+00f;
  DepthwiseConv2d[46] = 0.000000e+00f;
  DepthwiseConv2d[62] = 0.000000e+00f;
  DepthwiseConv2d[15] = 0.000000e+00f;
  DepthwiseConv2d[31] = 0.000000e+00f;
  DepthwiseConv2d[47] = 0.000000e+00f;
  DepthwiseConv2d[63] = 0.000000e+00f;
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 80; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x))] = (((((1 <= ((((((int)blockIdx.x) % 7) * 8) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 5) * 2)) + (((int)threadIdx.x) / 56))) && (((((((int)blockIdx.x) % 7) * 8) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 5) * 2)) + (((int)threadIdx.x) / 56)) < 57)) && (1 <= (dj_outer_outer + (((int)threadIdx.x) % 56)))) && ((dj_outer_outer + (((int)threadIdx.x) % 56)) < 57)) ? Input[((((((((((int)blockIdx.x) / 7) * 50176) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 5) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 5) * 112)) + ((int)threadIdx.x)) + dj_outer_outer) - 57)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 48) {
      kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 144) + (((int)threadIdx.x) * 3)) + dj_outer_outer)];
    }
    __syncthreads();
    for (int c_outer_inner = 0; c_outer_inner < 4; ++c_outer_inner) {
      DepthwiseConv2d[(c_outer_inner * 4)] = (DepthwiseConv2d[(c_outer_inner * 4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56))] * kernel_shared[(((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3))]));
      DepthwiseConv2d[((c_outer_inner * 4) + 16)] = (DepthwiseConv2d[((c_outer_inner * 4) + 16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3))]));
      DepthwiseConv2d[((c_outer_inner * 4) + 32)] = (DepthwiseConv2d[((c_outer_inner * 4) + 32)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4480)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 24)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 48)] = (DepthwiseConv2d[((c_outer_inner * 4) + 48)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4704)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 24)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 1)] = (DepthwiseConv2d[((c_outer_inner * 4) + 1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 56)] * kernel_shared[(((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3))]));
      DepthwiseConv2d[((c_outer_inner * 4) + 17)] = (DepthwiseConv2d[((c_outer_inner * 4) + 17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 280)] * kernel_shared[(((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3))]));
      DepthwiseConv2d[((c_outer_inner * 4) + 33)] = (DepthwiseConv2d[((c_outer_inner * 4) + 33)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4536)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 24)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 49)] = (DepthwiseConv2d[((c_outer_inner * 4) + 49)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4760)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 24)]));
      DepthwiseConv2d[(c_outer_inner * 4)] = (DepthwiseConv2d[(c_outer_inner * 4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 1)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 16)] = (DepthwiseConv2d[((c_outer_inner * 4) + 16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 280)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 1)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 32)] = (DepthwiseConv2d[((c_outer_inner * 4) + 32)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4536)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 25)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 48)] = (DepthwiseConv2d[((c_outer_inner * 4) + 48)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4760)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 25)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 1)] = (DepthwiseConv2d[((c_outer_inner * 4) + 1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 1)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 17)] = (DepthwiseConv2d[((c_outer_inner * 4) + 17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 336)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 1)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 33)] = (DepthwiseConv2d[((c_outer_inner * 4) + 33)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4592)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 25)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 49)] = (DepthwiseConv2d[((c_outer_inner * 4) + 49)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4816)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 25)]));
      DepthwiseConv2d[(c_outer_inner * 4)] = (DepthwiseConv2d[(c_outer_inner * 4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 2)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 16)] = (DepthwiseConv2d[((c_outer_inner * 4) + 16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 336)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 2)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 32)] = (DepthwiseConv2d[((c_outer_inner * 4) + 32)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4592)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 26)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 48)] = (DepthwiseConv2d[((c_outer_inner * 4) + 48)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4816)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 26)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 1)] = (DepthwiseConv2d[((c_outer_inner * 4) + 1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 2)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 17)] = (DepthwiseConv2d[((c_outer_inner * 4) + 17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 392)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 2)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 33)] = (DepthwiseConv2d[((c_outer_inner * 4) + 33)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4648)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 26)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 49)] = (DepthwiseConv2d[((c_outer_inner * 4) + 49)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4872)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 26)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 2)] = (DepthwiseConv2d[((c_outer_inner * 4) + 2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3))]));
      DepthwiseConv2d[((c_outer_inner * 4) + 18)] = (DepthwiseConv2d[((c_outer_inner * 4) + 18)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3))]));
      DepthwiseConv2d[((c_outer_inner * 4) + 34)] = (DepthwiseConv2d[((c_outer_inner * 4) + 34)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4592)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 24)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 50)] = (DepthwiseConv2d[((c_outer_inner * 4) + 50)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4816)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 24)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 3)] = (DepthwiseConv2d[((c_outer_inner * 4) + 3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 168)] * kernel_shared[(((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3))]));
      DepthwiseConv2d[((c_outer_inner * 4) + 19)] = (DepthwiseConv2d[((c_outer_inner * 4) + 19)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 392)] * kernel_shared[(((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3))]));
      DepthwiseConv2d[((c_outer_inner * 4) + 35)] = (DepthwiseConv2d[((c_outer_inner * 4) + 35)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4648)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 24)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 51)] = (DepthwiseConv2d[((c_outer_inner * 4) + 51)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4872)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 24)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 2)] = (DepthwiseConv2d[((c_outer_inner * 4) + 2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 1)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 18)] = (DepthwiseConv2d[((c_outer_inner * 4) + 18)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 392)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 1)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 34)] = (DepthwiseConv2d[((c_outer_inner * 4) + 34)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4648)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 25)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 50)] = (DepthwiseConv2d[((c_outer_inner * 4) + 50)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4872)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 25)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 3)] = (DepthwiseConv2d[((c_outer_inner * 4) + 3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 224)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 1)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 19)] = (DepthwiseConv2d[((c_outer_inner * 4) + 19)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 448)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 1)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 35)] = (DepthwiseConv2d[((c_outer_inner * 4) + 35)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4704)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 25)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 51)] = (DepthwiseConv2d[((c_outer_inner * 4) + 51)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4928)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 25)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 2)] = (DepthwiseConv2d[((c_outer_inner * 4) + 2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 224)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 2)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 18)] = (DepthwiseConv2d[((c_outer_inner * 4) + 18)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 448)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 2)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 34)] = (DepthwiseConv2d[((c_outer_inner * 4) + 34)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4704)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 26)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 50)] = (DepthwiseConv2d[((c_outer_inner * 4) + 50)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4928)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 26)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 3)] = (DepthwiseConv2d[((c_outer_inner * 4) + 3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 280)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 2)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 19)] = (DepthwiseConv2d[((c_outer_inner * 4) + 19)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 504)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 2)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 35)] = (DepthwiseConv2d[((c_outer_inner * 4) + 35)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4760)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 26)]));
      DepthwiseConv2d[((c_outer_inner * 4) + 51)] = (DepthwiseConv2d[((c_outer_inner * 4) + 51)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 2240) + (c_outer_inner * 560)) + (((int)threadIdx.x) % 56)) + 4984)] * kernel_shared[((((((int)threadIdx.x) / 56) * 12) + (c_outer_inner * 3)) + 26)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
      compute[(((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 56) * 12544)) + (i1_inner * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (i2_inner * 56)) + (((int)threadIdx.x) % 56))] = max(DepthwiseConv2d[((i1_inner * 4) + i2_inner)], 0.000000e+00f);
      compute[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 56) * 12544)) + (i1_inner * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (i2_inner * 56)) + (((int)threadIdx.x) % 56)) + 224)] = max(DepthwiseConv2d[(((i1_inner * 4) + i2_inner) + 16)], 0.000000e+00f);
      compute[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 56) * 12544)) + (i1_inner * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (i2_inner * 56)) + (((int)threadIdx.x) % 56)) + 25088)] = max(DepthwiseConv2d[(((i1_inner * 4) + i2_inner) + 32)], 0.000000e+00f);
      compute[((((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 56) * 12544)) + (i1_inner * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (i2_inner * 56)) + (((int)threadIdx.x) % 56)) + 25312)] = max(DepthwiseConv2d[(((i1_inner * 4) + i2_inner) + 48)], 0.000000e+00f);
    }
  }
}


