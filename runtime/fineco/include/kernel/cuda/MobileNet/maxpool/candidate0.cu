
#include <hip/hip_runtime.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))
#endif


#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) candidate0(float* __restrict__ data, float* __restrict__ tensor) {
  float normal_reduce_temp0[1];
  float red_buf0[1];
  __shared__ float tensor1[1];
  normal_reduce_temp0[0] = 0.000000e+00f;
  normal_reduce_temp0[0] = (normal_reduce_temp0[0] + data[((((int)blockIdx.x) * 49) + ((int)threadIdx.x))]);
  if (((int)threadIdx.x) < 17) {
    normal_reduce_temp0[0] = (normal_reduce_temp0[0] + data[(((((int)blockIdx.x) * 49) + ((int)threadIdx.x)) + 32)]);
  }
  uint mask[1];
  float t0[1];
  red_buf0[0] = normal_reduce_temp0[0];
  mask[0] = __activemask();
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], 0, 32);
  tensor1[0] = red_buf0[0];
  __syncthreads();
  if (((int)threadIdx.x) < 1) {
    tensor[(((int)blockIdx.x) + ((int)threadIdx.x))] = (tensor1[((int)threadIdx.x)] * 2.040816e-02f);
  }
}


