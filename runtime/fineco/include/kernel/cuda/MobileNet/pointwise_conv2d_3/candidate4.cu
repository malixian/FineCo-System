
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate4(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[28];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[4096];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[16] = 0.000000e+00f;
  conv2d_nchw[20] = 0.000000e+00f;
  conv2d_nchw[24] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[17] = 0.000000e+00f;
  conv2d_nchw[21] = 0.000000e+00f;
  conv2d_nchw[25] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[18] = 0.000000e+00f;
  conv2d_nchw[22] = 0.000000e+00f;
  conv2d_nchw[26] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  conv2d_nchw[19] = 0.000000e+00f;
  conv2d_nchw[23] = 0.000000e+00f;
  conv2d_nchw[27] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + (((((int)threadIdx.x) % 98) / 14) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 224) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 2) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 448) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 672) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 6) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 896) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 1120) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 3) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 1344) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = Input[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + (((((int)threadIdx.x) % 98) / 14) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 1792) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 2) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2016) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2240) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 6) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2464) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2688) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 3) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2912) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = Input[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + (((((int)threadIdx.x) % 98) / 14) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 100352)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 3360) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 2) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 3584) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 3808)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 3808) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 6) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 4032)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 4032) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 4256)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 4256) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 3) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 4480)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 4480) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = Input[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + (((((int)threadIdx.x) % 98) / 14) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 150528)];
    pad_temp_shared[(((int)threadIdx.x) + 4928)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 4928) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 2) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 5152)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 5152) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 5376)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 5376) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 6) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 5600)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 5600) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 5824)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 5824) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 3) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 6048)] = Input[((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 6048) / 98) * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 224) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 896)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 672) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 1120) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 2688)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 1568) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 2016) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 4480)];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 2464) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 5376)];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 2912) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 6272)];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 3360) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) >> 5) * 8192) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (((((int)threadIdx.x) + 3808) >> 6) * 128)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[(((((((int)blockIdx.x) >> 5) * 8192) + (rc_outer_outer * 64)) + ((int)threadIdx.x)) + 8064)];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 64; ++rc_inner) {
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((rc_inner * 98) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + rc_inner)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + rc_inner)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + rc_inner)]));
      conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + rc_inner)]));
      conv2d_nchw[16] = (conv2d_nchw[16] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + rc_inner)]));
      conv2d_nchw[20] = (conv2d_nchw[20] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + rc_inner)]));
      conv2d_nchw[24] = (conv2d_nchw[24] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + rc_inner)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((rc_inner * 98) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 64)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 64)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 64)]));
      conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 64)]));
      conv2d_nchw[17] = (conv2d_nchw[17] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 64)]));
      conv2d_nchw[21] = (conv2d_nchw[21] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 64)]));
      conv2d_nchw[25] = (conv2d_nchw[25] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 64)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((rc_inner * 98) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 128)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 128)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 128)]));
      conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 128)]));
      conv2d_nchw[18] = (conv2d_nchw[18] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 128)]));
      conv2d_nchw[22] = (conv2d_nchw[22] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 128)]));
      conv2d_nchw[26] = (conv2d_nchw[26] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 128)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((rc_inner * 98) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 192)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 192)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 192)]));
      conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 192)]));
      conv2d_nchw[19] = (conv2d_nchw[19] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 192)]));
      conv2d_nchw[23] = (conv2d_nchw[23] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 192)]));
      conv2d_nchw[27] = (conv2d_nchw[27] + (pad_temp_shared[(((rc_inner * 98) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + rc_inner) + 192)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    compute[(((((((((int)blockIdx.x) >> 5) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))] = max(conv2d_nchw[i1_inner], 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) >> 5) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 56)] = max(conv2d_nchw[(i1_inner + 4)], 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) >> 5) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 112)] = max(conv2d_nchw[(i1_inner + 8)], 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) >> 5) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 168)] = max(conv2d_nchw[(i1_inner + 12)], 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) >> 5) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 224)] = max(conv2d_nchw[(i1_inner + 16)], 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) >> 5) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 280)] = max(conv2d_nchw[(i1_inner + 20)], 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) >> 5) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) & 31) >> 2) * 392)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14)) + 336)] = max(conv2d_nchw[(i1_inner + 24)], 0.000000e+00f);
  }
}


