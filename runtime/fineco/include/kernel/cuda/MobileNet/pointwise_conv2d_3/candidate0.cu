
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(56) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[16];
  __shared__ float pad_temp_shared[1792];
  __shared__ float kernel_shared[512];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 4; ++ff_inner_init) {
      conv2d_nchw[((ff_outer_inner_init * 4) + ff_inner_init)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 4) + ff_inner_init) + 8)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x))] = Input[((((((rc_outer_outer * 100352) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 10; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) >> 3)) < 64) {
        kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x))] = kernel[(((((((int)blockIdx.x) / 56) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) >> 3)) >> 2) * 128)) + (rc_outer_outer * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) & 31))];
      }
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 32; ++rc_inner) {
        for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
          conv2d_nchw[((ff_outer_inner * 4) + ff_inner)] = (conv2d_nchw[((ff_outer_inner * 4) + ff_inner)] + (pad_temp_shared[((rc_inner * 56) + ((int)threadIdx.x))] * kernel_shared[(((ff_outer_inner * 128) + (ff_inner * 32)) + rc_inner)]));
          conv2d_nchw[(((ff_outer_inner * 4) + ff_inner) + 8)] = (conv2d_nchw[(((ff_outer_inner * 4) + ff_inner) + 8)] + (pad_temp_shared[((rc_inner * 56) + ((int)threadIdx.x))] * kernel_shared[((((ff_outer_inner * 128) + (ff_inner * 32)) + rc_inner) + 256)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    compute[(((((((((int)blockIdx.x) / 56) * 50176) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))] = max(conv2d_nchw[i1_inner], 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) / 56) * 50176) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((int)threadIdx.x) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 25088)] = max(conv2d_nchw[(i1_inner + 8)], 0.000000e+00f);
  }
}


