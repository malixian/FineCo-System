
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate3(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[32];
  __shared__ float pad_temp_shared[4096];
  __shared__ float kernel_shared[8192];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
      for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
        conv2d_nchw[(((ff_outer_inner_init * 4) + (yy_inner_init * 2)) + xx_outer_inner_init)] = 0.000000e+00f;
        conv2d_nchw[((((ff_outer_inner_init * 4) + (yy_inner_init * 2)) + xx_outer_inner_init) + 8)] = 0.000000e+00f;
        conv2d_nchw[((((ff_outer_inner_init * 4) + (yy_inner_init * 2)) + xx_outer_inner_init) + 16)] = 0.000000e+00f;
        conv2d_nchw[((((ff_outer_inner_init * 4) + (yy_inner_init * 2)) + xx_outer_inner_init) + 24)] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x))] = Input[(((((((rc_outer_outer * 200704) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12544)) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 256) + ((int)threadIdx.x))] = kernel[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 512) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 64; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            conv2d_nchw[(((ff_outer_inner * 4) + (yy_inner * 2)) + xx_outer_inner)] = (conv2d_nchw[(((ff_outer_inner * 4) + (yy_inner * 2)) + xx_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 2) * 16)) + (yy_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + xx_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 128) + (ff_outer_inner * 64)) + rc_outer_inner)]));
            conv2d_nchw[((((ff_outer_inner * 4) + (yy_inner * 2)) + xx_outer_inner) + 8)] = (conv2d_nchw[((((ff_outer_inner * 4) + (yy_inner * 2)) + xx_outer_inner) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 2) * 16)) + (yy_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + xx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 128) + (ff_outer_inner * 64)) + rc_outer_inner) + 2048)]));
            conv2d_nchw[((((ff_outer_inner * 4) + (yy_inner * 2)) + xx_outer_inner) + 16)] = (conv2d_nchw[((((ff_outer_inner * 4) + (yy_inner * 2)) + xx_outer_inner) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 2) * 16)) + (yy_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + xx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 128) + (ff_outer_inner * 64)) + rc_outer_inner) + 4096)]));
            conv2d_nchw[((((ff_outer_inner * 4) + (yy_inner * 2)) + xx_outer_inner) + 24)] = (conv2d_nchw[((((ff_outer_inner * 4) + (yy_inner * 2)) + xx_outer_inner) + 24)] + (pad_temp_shared[(((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 2) * 16)) + (yy_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + xx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 128) + (ff_outer_inner * 64)) + rc_outer_inner) + 6144)]));
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
        compute[(((((((((((int)threadIdx.x) >> 4) * 6272) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 2) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + i3_inner)] = max(conv2d_nchw[(((i1_inner * 4) + (i2_inner * 2)) + i3_inner)], 0.000000e+00f);
        compute[((((((((((((int)threadIdx.x) >> 4) * 6272) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 2) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + i3_inner) + 100352)] = max(conv2d_nchw[((((i1_inner * 4) + (i2_inner * 2)) + i3_inner) + 8)], 0.000000e+00f);
        compute[((((((((((((int)threadIdx.x) >> 4) * 6272) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 2) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + i3_inner) + 200704)] = max(conv2d_nchw[((((i1_inner * 4) + (i2_inner * 2)) + i3_inner) + 16)], 0.000000e+00f);
        compute[((((((((((((int)threadIdx.x) >> 4) * 6272) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 2) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + i3_inner) + 301056)] = max(conv2d_nchw[((((i1_inner * 4) + (i2_inner * 2)) + i3_inner) + 24)], 0.000000e+00f);
      }
    }
  }
}


