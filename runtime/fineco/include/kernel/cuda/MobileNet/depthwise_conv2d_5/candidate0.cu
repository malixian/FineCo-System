
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(98) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[8];
  __shared__ float PaddedInput_shared[1152];
  __shared__ float kernel_shared[72];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  DepthwiseConv2d[4] = 0.000000e+00f;
  DepthwiseConv2d[6] = 0.000000e+00f;
  DepthwiseConv2d[5] = 0.000000e+00f;
  DepthwiseConv2d[7] = 0.000000e+00f;
  if (((int)threadIdx.x) < 64) {
    PaddedInput_shared[(((int)threadIdx.x) * 18)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) && (1 <= (((int)blockIdx.x) & 3))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 1)] = ((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 28)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 2)] = ((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 27)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 3)] = ((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 26)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 4)] = ((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 25)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 5)] = ((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 24)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 6)] = ((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 23)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 7)] = ((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 22)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 8)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2))) && ((((int)blockIdx.x) & 3) < 3)) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 21)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 9)] = (((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) && (1 <= (((int)blockIdx.x) & 3))) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) - 1)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 10)] = ((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) ? Input[((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 11)] = ((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) + 1)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 12)] = ((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) + 2)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 13)] = ((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) + 3)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 14)] = ((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) + 4)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 15)] = ((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) + 5)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 16)] = ((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) + 6)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 18) + 17)] = (((((((((int)blockIdx.x) & 7) >> 2) * 14) + ((((int)threadIdx.x) & 7) * 2)) < 28) && ((((int)blockIdx.x) & 3) < 3)) ? Input[(((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) & 7) * 56)) + ((((int)blockIdx.x) & 3) * 7)) + 7)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 72) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) >> 3) * 72) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int dj_outer_inner = 0; dj_outer_inner < 3; ++dj_outer_inner) {
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7))] * kernel_shared[(((((int)threadIdx.x) / 49) * 36) + dj_outer_inner)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 144)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 9)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 9)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 3)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 153)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 12)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 6)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 162)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 15)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 9)] * kernel_shared[(((((int)threadIdx.x) / 49) * 36) + dj_outer_inner)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 153)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 9)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 3)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 162)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 12)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 27)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 6)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 171)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 15)]));
    DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 288)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 18)]));
    DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 432)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 27)]));
    DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 297)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 21)]));
    DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 441)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 30)]));
    DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 306)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 24)]));
    DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 450)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 33)]));
    DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 297)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 18)]));
    DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 441)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 27)]));
    DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 306)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 21)]));
    DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 450)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 30)]));
    DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 315)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 24)]));
    DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 576) + (((((int)threadIdx.x) % 49) / 7) * 18)) + dj_outer_inner) + (((int)threadIdx.x) % 7)) + 459)] * kernel_shared[((((((int)threadIdx.x) / 49) * 36) + dj_outer_inner) + 33)]));
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[(((((((((((int)blockIdx.x) >> 3) * 6272) + ((((int)threadIdx.x) / 49) * 3136)) + (i1_inner * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + (((((int)threadIdx.x) % 49) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))] = max(DepthwiseConv2d[((i1_inner * 2) + i2_inner)], 0.000000e+00f);
    }
  }
}


