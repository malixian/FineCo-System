
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(98) candidate4(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[32];
  __shared__ float PaddedInput_shared[3360];
  __shared__ float kernel_shared[12];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[8] = 0.000000e+00f;
  DepthwiseConv2d[16] = 0.000000e+00f;
  DepthwiseConv2d[24] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[9] = 0.000000e+00f;
  DepthwiseConv2d[17] = 0.000000e+00f;
  DepthwiseConv2d[25] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[10] = 0.000000e+00f;
  DepthwiseConv2d[18] = 0.000000e+00f;
  DepthwiseConv2d[26] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  DepthwiseConv2d[11] = 0.000000e+00f;
  DepthwiseConv2d[19] = 0.000000e+00f;
  DepthwiseConv2d[27] = 0.000000e+00f;
  DepthwiseConv2d[4] = 0.000000e+00f;
  DepthwiseConv2d[12] = 0.000000e+00f;
  DepthwiseConv2d[20] = 0.000000e+00f;
  DepthwiseConv2d[28] = 0.000000e+00f;
  DepthwiseConv2d[5] = 0.000000e+00f;
  DepthwiseConv2d[13] = 0.000000e+00f;
  DepthwiseConv2d[21] = 0.000000e+00f;
  DepthwiseConv2d[29] = 0.000000e+00f;
  DepthwiseConv2d[6] = 0.000000e+00f;
  DepthwiseConv2d[14] = 0.000000e+00f;
  DepthwiseConv2d[22] = 0.000000e+00f;
  DepthwiseConv2d[30] = 0.000000e+00f;
  DepthwiseConv2d[7] = 0.000000e+00f;
  DepthwiseConv2d[15] = 0.000000e+00f;
  DepthwiseConv2d[23] = 0.000000e+00f;
  DepthwiseConv2d[31] = 0.000000e+00f;
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((int)threadIdx.x)] = ((((28 <= ((int)threadIdx.x)) && (1 <= (dj_outer_outer + (((int)threadIdx.x) % 28)))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[((((((int)blockIdx.x) * 3136) + dj_outer_outer) + ((int)threadIdx.x)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 98)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[((((((int)blockIdx.x) * 3136) + dj_outer_outer) + ((int)threadIdx.x)) + 69)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 196)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[((((((int)blockIdx.x) * 3136) + dj_outer_outer) + ((int)threadIdx.x)) + 167)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 294)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[((((((int)blockIdx.x) * 3136) + dj_outer_outer) + ((int)threadIdx.x)) + 265)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 392)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[((((((int)blockIdx.x) * 3136) + dj_outer_outer) + ((int)threadIdx.x)) + 363)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 490)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[((((((int)blockIdx.x) * 3136) + dj_outer_outer) + ((int)threadIdx.x)) + 461)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 588)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[((((((int)blockIdx.x) * 3136) + dj_outer_outer) + ((int)threadIdx.x)) + 559)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 686)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[((((((int)blockIdx.x) * 3136) + dj_outer_outer) + ((int)threadIdx.x)) + 657)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 784)] = (((((1 <= (((((int)threadIdx.x) / 28) + 28) % 30)) && (((((int)threadIdx.x) + 784) % 840) < 812)) && (1 <= (dj_outer_outer + (((int)threadIdx.x) % 28)))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 784) / 840) * 784)) + dj_outer_outer) + ((((int)threadIdx.x) + 784) % 840)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 882)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 882) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 42)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 980)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 980) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 140)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1078)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1078) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 238)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1176)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1176) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 336)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1274)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1274) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 434)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1372)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1372) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 532)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1470)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1470) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 630)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1568)] = ((((((int)threadIdx.x) < 84) && (1 <= (dj_outer_outer + (((int)threadIdx.x) % 28)))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1568) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 728)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1666)] = (((((2 <= (((((int)threadIdx.x) / 14) + 59) % 60)) && (((((int)threadIdx.x) + 826) % 840) < 812)) && (1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1666) / 840) * 784)) + dj_outer_outer) + ((((int)threadIdx.x) + 826) % 840)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1764)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1764) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 84)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1862)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1862) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 182)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1960)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1960) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 280)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2058)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2058) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 378)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2156)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2156) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 476)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2254)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2254) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 574)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2352)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2352) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 672)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2450)] = (((((2 <= (((((int)threadIdx.x) / 14) + 55) % 60)) && (((((int)threadIdx.x) + 770) % 840) < 812)) && (1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2450) / 840) * 784)) + dj_outer_outer) + ((((int)threadIdx.x) + 770) % 840)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2548)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2548) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 28)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2646)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2646) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 126)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2744)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2744) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 224)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2842)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2842) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 322)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2940)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2940) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 420)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 3038)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3038) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 518)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 3136)] = (((1 <= (dj_outer_outer + (((int)threadIdx.x) % 28))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3136) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 616)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 3234)] = (((1 <= (dj_outer_outer + ((((int)threadIdx.x) + 14) % 28))) && ((dj_outer_outer + ((((int)threadIdx.x) + 14) % 28)) < 29)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3234) / 840) * 784)) + dj_outer_outer) + (((int)threadIdx.x) + 714)) - 29)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((int)threadIdx.x) + 3332)] = 0.000000e+00f;
    }
    if (((int)threadIdx.x) < 12) {
      kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) * 36) + (((int)threadIdx.x) * 3)) + dj_outer_outer)];
    }
    __syncthreads();
    for (int i_outer_inner = 0; i_outer_inner < 4; ++i_outer_inner) {
      DepthwiseConv2d[(i_outer_inner * 2)] = (DepthwiseConv2d[(i_outer_inner * 2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[0]));
      DepthwiseConv2d[((i_outer_inner * 2) + 8)] = (DepthwiseConv2d[((i_outer_inner * 2) + 8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 840)] * kernel_shared[3]));
      DepthwiseConv2d[((i_outer_inner * 2) + 16)] = (DepthwiseConv2d[((i_outer_inner * 2) + 16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1680)] * kernel_shared[6]));
      DepthwiseConv2d[((i_outer_inner * 2) + 24)] = (DepthwiseConv2d[((i_outer_inner * 2) + 24)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 2520)] * kernel_shared[9]));
      DepthwiseConv2d[(i_outer_inner * 2)] = (DepthwiseConv2d[(i_outer_inner * 2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 28)] * kernel_shared[1]));
      DepthwiseConv2d[((i_outer_inner * 2) + 8)] = (DepthwiseConv2d[((i_outer_inner * 2) + 8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 868)] * kernel_shared[4]));
      DepthwiseConv2d[((i_outer_inner * 2) + 16)] = (DepthwiseConv2d[((i_outer_inner * 2) + 16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1708)] * kernel_shared[7]));
      DepthwiseConv2d[((i_outer_inner * 2) + 24)] = (DepthwiseConv2d[((i_outer_inner * 2) + 24)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 2548)] * kernel_shared[10]));
      DepthwiseConv2d[(i_outer_inner * 2)] = (DepthwiseConv2d[(i_outer_inner * 2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 56)] * kernel_shared[2]));
      DepthwiseConv2d[((i_outer_inner * 2) + 8)] = (DepthwiseConv2d[((i_outer_inner * 2) + 8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 896)] * kernel_shared[5]));
      DepthwiseConv2d[((i_outer_inner * 2) + 16)] = (DepthwiseConv2d[((i_outer_inner * 2) + 16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1736)] * kernel_shared[8]));
      DepthwiseConv2d[((i_outer_inner * 2) + 24)] = (DepthwiseConv2d[((i_outer_inner * 2) + 24)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 2576)] * kernel_shared[11]));
      DepthwiseConv2d[((i_outer_inner * 2) + 1)] = (DepthwiseConv2d[((i_outer_inner * 2) + 1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[0]));
      DepthwiseConv2d[((i_outer_inner * 2) + 9)] = (DepthwiseConv2d[((i_outer_inner * 2) + 9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 841)] * kernel_shared[3]));
      DepthwiseConv2d[((i_outer_inner * 2) + 17)] = (DepthwiseConv2d[((i_outer_inner * 2) + 17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1681)] * kernel_shared[6]));
      DepthwiseConv2d[((i_outer_inner * 2) + 25)] = (DepthwiseConv2d[((i_outer_inner * 2) + 25)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 2521)] * kernel_shared[9]));
      DepthwiseConv2d[((i_outer_inner * 2) + 1)] = (DepthwiseConv2d[((i_outer_inner * 2) + 1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 29)] * kernel_shared[1]));
      DepthwiseConv2d[((i_outer_inner * 2) + 9)] = (DepthwiseConv2d[((i_outer_inner * 2) + 9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 869)] * kernel_shared[4]));
      DepthwiseConv2d[((i_outer_inner * 2) + 17)] = (DepthwiseConv2d[((i_outer_inner * 2) + 17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1709)] * kernel_shared[7]));
      DepthwiseConv2d[((i_outer_inner * 2) + 25)] = (DepthwiseConv2d[((i_outer_inner * 2) + 25)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 2549)] * kernel_shared[10]));
      DepthwiseConv2d[((i_outer_inner * 2) + 1)] = (DepthwiseConv2d[((i_outer_inner * 2) + 1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 57)] * kernel_shared[2]));
      DepthwiseConv2d[((i_outer_inner * 2) + 9)] = (DepthwiseConv2d[((i_outer_inner * 2) + 9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 897)] * kernel_shared[5]));
      DepthwiseConv2d[((i_outer_inner * 2) + 17)] = (DepthwiseConv2d[((i_outer_inner * 2) + 17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1737)] * kernel_shared[8]));
      DepthwiseConv2d[((i_outer_inner * 2) + 25)] = (DepthwiseConv2d[((i_outer_inner * 2) + 25)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 14) * 112) + (i_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 2577)] * kernel_shared[11]));
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
      for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
        compute[((((((((int)blockIdx.x) * 3136) + (i1_inner * 784)) + ((((int)threadIdx.x) / 14) * 112)) + (i2_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner)] = max(DepthwiseConv2d[(((i1_inner * 8) + (i2_inner * 2)) + i3_inner)], 0.000000e+00f);
      }
    }
  }
}


