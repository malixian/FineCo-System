
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(896) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[8];
  __shared__ float PaddedInput_shared[7168];
  __shared__ float kernel_shared[64];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  DepthwiseConv2d[4] = 0.000000e+00f;
  DepthwiseConv2d[5] = 0.000000e+00f;
  DepthwiseConv2d[6] = 0.000000e+00f;
  DepthwiseConv2d[7] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) - 29)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6243)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12515)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18787)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25059)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31331)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37603)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43875)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9))];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) - 28)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6244)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = ((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12516)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = ((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18788)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = ((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25060)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = ((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31332)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = ((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37604)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = ((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43876)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9)) + 1)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) - 27)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6245)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12517)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18789)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25061)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31333)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37605)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = (((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28))) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43877)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9)) + 2)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((1 <= (((int)threadIdx.x) % 28)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) - 1)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((1 <= (((int)threadIdx.x) % 28)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6271)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = ((1 <= (((int)threadIdx.x) % 28)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12543)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = ((1 <= (((int)threadIdx.x) % 28)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18815)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = ((1 <= (((int)threadIdx.x) % 28)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25087)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = ((1 <= (((int)threadIdx.x) % 28)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31359)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = ((1 <= (((int)threadIdx.x) % 28)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37631)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = ((1 <= (((int)threadIdx.x) % 28)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43903)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9)) + 3)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = Input[(((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112))];
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6272)];
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12544)];
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18816)];
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25088)];
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31360)];
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37632)];
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43904)];
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9)) + 4)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = (((((int)threadIdx.x) % 28) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 1)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((((int)threadIdx.x) % 28) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6273)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((((int)threadIdx.x) % 28) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12545)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((((int)threadIdx.x) % 28) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18817)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = (((((int)threadIdx.x) % 28) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25089)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = (((((int)threadIdx.x) % 28) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31361)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = (((((int)threadIdx.x) % 28) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37633)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = (((((int)threadIdx.x) % 28) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43905)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9)) + 5)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 27)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6299)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12571)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18843)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25115)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31387)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37659)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && (1 <= (((int)threadIdx.x) % 28))) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43931)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9)) + 6)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = (((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 28)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6300)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12572)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18844)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = (((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25116)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = (((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31388)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = (((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37660)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = (((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43932)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9)) + 7)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 29)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6301)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12573)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18845)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3584)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 25117)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4480)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 31389)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5376)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 37661)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = ((((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) % 112) / 28)) < 27) && ((((int)threadIdx.x) % 28) < 27)) ? Input[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 43933)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 64) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 576) + (((int)threadIdx.x) * 9)) + 8)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112))] * kernel_shared[((((int)threadIdx.x) / 112) * 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 224)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 336)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 448)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 560)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 672)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 6)]));
  DepthwiseConv2d[7] = (DepthwiseConv2d[7] + (PaddedInput_shared[((((((int)threadIdx.x) / 112) * 896) + (((int)threadIdx.x) % 112)) + 784)] * kernel_shared[(((((int)threadIdx.x) / 112) * 8) + 7)]));
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    compute[((((((((int)blockIdx.x) / 7) * 50176) + ((((int)threadIdx.x) / 112) * 6272)) + (i1_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112))] = max(DepthwiseConv2d[i1_inner], 0.000000e+00f);
  }
}


