
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[56];
  __shared__ float pad_temp_shared[1568];
  __shared__ float kernel_shared[2048];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[28] = 0.000000e+00f;
  conv2d_nchw[42] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  conv2d_nchw[29] = 0.000000e+00f;
  conv2d_nchw[43] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[16] = 0.000000e+00f;
  conv2d_nchw[30] = 0.000000e+00f;
  conv2d_nchw[44] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[17] = 0.000000e+00f;
  conv2d_nchw[31] = 0.000000e+00f;
  conv2d_nchw[45] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[18] = 0.000000e+00f;
  conv2d_nchw[32] = 0.000000e+00f;
  conv2d_nchw[46] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[19] = 0.000000e+00f;
  conv2d_nchw[33] = 0.000000e+00f;
  conv2d_nchw[47] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[20] = 0.000000e+00f;
  conv2d_nchw[34] = 0.000000e+00f;
  conv2d_nchw[48] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[21] = 0.000000e+00f;
  conv2d_nchw[35] = 0.000000e+00f;
  conv2d_nchw[49] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[22] = 0.000000e+00f;
  conv2d_nchw[36] = 0.000000e+00f;
  conv2d_nchw[50] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[23] = 0.000000e+00f;
  conv2d_nchw[37] = 0.000000e+00f;
  conv2d_nchw[51] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[24] = 0.000000e+00f;
  conv2d_nchw[38] = 0.000000e+00f;
  conv2d_nchw[52] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[25] = 0.000000e+00f;
  conv2d_nchw[39] = 0.000000e+00f;
  conv2d_nchw[53] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[26] = 0.000000e+00f;
  conv2d_nchw[40] = 0.000000e+00f;
  conv2d_nchw[54] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[27] = 0.000000e+00f;
  conv2d_nchw[41] = 0.000000e+00f;
  conv2d_nchw[55] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + (((((int)threadIdx.x) % 98) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = Input[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 224) / 98) * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + ((((((int)threadIdx.x) / 14) + 2) % 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = Input[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 448) / 98) * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + ((((((int)threadIdx.x) / 14) + 4) % 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = Input[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 672) / 98) * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + ((((((int)threadIdx.x) / 14) + 6) % 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = Input[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 896) / 98) * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + ((((((int)threadIdx.x) / 14) + 1) % 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = Input[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1120) / 98) * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + ((((((int)threadIdx.x) / 14) + 3) % 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = Input[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1344) / 98) * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + ((((((int)threadIdx.x) / 14) + 5) % 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 17920)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 21504)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 25088)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 28672)];
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) >> 3) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 32256)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2))]));
      conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[28] = (conv2d_nchw[28] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1024)]));
      conv2d_nchw[42] = (conv2d_nchw[42] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1536)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2))]));
      conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[29] = (conv2d_nchw[29] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1024)]));
      conv2d_nchw[43] = (conv2d_nchw[43] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1536)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[(((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2))]));
      conv2d_nchw[16] = (conv2d_nchw[16] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[30] = (conv2d_nchw[30] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1024)]));
      conv2d_nchw[44] = (conv2d_nchw[44] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1536)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[(((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2))]));
      conv2d_nchw[17] = (conv2d_nchw[17] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[31] = (conv2d_nchw[31] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1024)]));
      conv2d_nchw[45] = (conv2d_nchw[45] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1536)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[(((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2))]));
      conv2d_nchw[18] = (conv2d_nchw[18] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[32] = (conv2d_nchw[32] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1024)]));
      conv2d_nchw[46] = (conv2d_nchw[46] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1536)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[(((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2))]));
      conv2d_nchw[19] = (conv2d_nchw[19] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[33] = (conv2d_nchw[33] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1024)]));
      conv2d_nchw[47] = (conv2d_nchw[47] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1536)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[(((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2))]));
      conv2d_nchw[20] = (conv2d_nchw[20] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[34] = (conv2d_nchw[34] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1024)]));
      conv2d_nchw[48] = (conv2d_nchw[48] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1536)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[28] = (conv2d_nchw[28] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1025)]));
      conv2d_nchw[42] = (conv2d_nchw[42] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1537)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[29] = (conv2d_nchw[29] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1025)]));
      conv2d_nchw[43] = (conv2d_nchw[43] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1537)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[16] = (conv2d_nchw[16] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[30] = (conv2d_nchw[30] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1025)]));
      conv2d_nchw[44] = (conv2d_nchw[44] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1537)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[17] = (conv2d_nchw[17] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[31] = (conv2d_nchw[31] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1025)]));
      conv2d_nchw[45] = (conv2d_nchw[45] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1537)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[18] = (conv2d_nchw[18] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[32] = (conv2d_nchw[32] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1025)]));
      conv2d_nchw[46] = (conv2d_nchw[46] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1537)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[19] = (conv2d_nchw[19] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[33] = (conv2d_nchw[33] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1025)]));
      conv2d_nchw[47] = (conv2d_nchw[47] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1537)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[20] = (conv2d_nchw[20] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[34] = (conv2d_nchw[34] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1025)]));
      conv2d_nchw[48] = (conv2d_nchw[48] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1537)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 16)]));
      conv2d_nchw[21] = (conv2d_nchw[21] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 528)]));
      conv2d_nchw[35] = (conv2d_nchw[35] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1040)]));
      conv2d_nchw[49] = (conv2d_nchw[49] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1552)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 16)]));
      conv2d_nchw[22] = (conv2d_nchw[22] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 528)]));
      conv2d_nchw[36] = (conv2d_nchw[36] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1040)]));
      conv2d_nchw[50] = (conv2d_nchw[50] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1552)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 16)]));
      conv2d_nchw[23] = (conv2d_nchw[23] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 528)]));
      conv2d_nchw[37] = (conv2d_nchw[37] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1040)]));
      conv2d_nchw[51] = (conv2d_nchw[51] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1552)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 16)]));
      conv2d_nchw[24] = (conv2d_nchw[24] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 528)]));
      conv2d_nchw[38] = (conv2d_nchw[38] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1040)]));
      conv2d_nchw[52] = (conv2d_nchw[52] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1552)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 16)]));
      conv2d_nchw[25] = (conv2d_nchw[25] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 528)]));
      conv2d_nchw[39] = (conv2d_nchw[39] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1040)]));
      conv2d_nchw[53] = (conv2d_nchw[53] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1552)]));
      conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 16)]));
      conv2d_nchw[26] = (conv2d_nchw[26] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 528)]));
      conv2d_nchw[40] = (conv2d_nchw[40] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1040)]));
      conv2d_nchw[54] = (conv2d_nchw[54] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1552)]));
      conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 16)]));
      conv2d_nchw[27] = (conv2d_nchw[27] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 528)]));
      conv2d_nchw[41] = (conv2d_nchw[41] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1040)]));
      conv2d_nchw[55] = (conv2d_nchw[55] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1552)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 17)]));
      conv2d_nchw[21] = (conv2d_nchw[21] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 529)]));
      conv2d_nchw[35] = (conv2d_nchw[35] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1041)]));
      conv2d_nchw[49] = (conv2d_nchw[49] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1553)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 17)]));
      conv2d_nchw[22] = (conv2d_nchw[22] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 529)]));
      conv2d_nchw[36] = (conv2d_nchw[36] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1041)]));
      conv2d_nchw[50] = (conv2d_nchw[50] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1553)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 17)]));
      conv2d_nchw[23] = (conv2d_nchw[23] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 529)]));
      conv2d_nchw[37] = (conv2d_nchw[37] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1041)]));
      conv2d_nchw[51] = (conv2d_nchw[51] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1553)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 17)]));
      conv2d_nchw[24] = (conv2d_nchw[24] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 529)]));
      conv2d_nchw[38] = (conv2d_nchw[38] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1041)]));
      conv2d_nchw[52] = (conv2d_nchw[52] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1553)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 17)]));
      conv2d_nchw[25] = (conv2d_nchw[25] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 529)]));
      conv2d_nchw[39] = (conv2d_nchw[39] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1041)]));
      conv2d_nchw[53] = (conv2d_nchw[53] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1553)]));
      conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 17)]));
      conv2d_nchw[26] = (conv2d_nchw[26] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 529)]));
      conv2d_nchw[40] = (conv2d_nchw[40] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1041)]));
      conv2d_nchw[54] = (conv2d_nchw[54] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1553)]));
      conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 17)]));
      conv2d_nchw[27] = (conv2d_nchw[27] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 529)]));
      conv2d_nchw[41] = (conv2d_nchw[41] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1041)]));
      conv2d_nchw[55] = (conv2d_nchw[55] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 32) + (rc_outer_inner * 2)) + 1553)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 7; ++i2_inner) {
      compute[((((((((((int)blockIdx.x) >> 3) * 100352) + ((((int)threadIdx.x) / 14) * 1568)) + (i1_inner * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))] = max(conv2d_nchw[((i1_inner * 7) + i2_inner)], 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) >> 3) * 100352) + ((((int)threadIdx.x) / 14) * 1568)) + (i1_inner * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 25088)] = max(conv2d_nchw[(((i1_inner * 7) + i2_inner) + 14)], 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) >> 3) * 100352) + ((((int)threadIdx.x) / 14) * 1568)) + (i1_inner * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 50176)] = max(conv2d_nchw[(((i1_inner * 7) + i2_inner) + 28)], 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) >> 3) * 100352) + ((((int)threadIdx.x) / 14) * 1568)) + (i1_inner * 784)) + (((((int)blockIdx.x) & 7) >> 1) * 196)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 75264)] = max(conv2d_nchw[(((i1_inner * 7) + i2_inner) + 42)], 0.000000e+00f);
    }
  }
}


