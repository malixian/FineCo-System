
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate2(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[8];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[4096];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((rc_outer_outer * 6272) + ((int)threadIdx.x))];
    pad_temp_shared[(((int)threadIdx.x) + 196)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 196)];
    pad_temp_shared[(((int)threadIdx.x) + 392)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 392)];
    pad_temp_shared[(((int)threadIdx.x) + 588)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 588)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 980)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 980)];
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1176)];
    pad_temp_shared[(((int)threadIdx.x) + 1372)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1372)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 1764)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1764)];
    pad_temp_shared[(((int)threadIdx.x) + 1960)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1960)];
    pad_temp_shared[(((int)threadIdx.x) + 2156)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2156)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 2548)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2548)];
    pad_temp_shared[(((int)threadIdx.x) + 2744)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2744)];
    pad_temp_shared[(((int)threadIdx.x) + 2940)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2940)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 3332)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3332)];
    pad_temp_shared[(((int)threadIdx.x) + 3528)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3528)];
    pad_temp_shared[(((int)threadIdx.x) + 3724)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3724)];
    pad_temp_shared[(((int)threadIdx.x) + 3920)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3920)];
    pad_temp_shared[(((int)threadIdx.x) + 4116)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4116)];
    pad_temp_shared[(((int)threadIdx.x) + 4312)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4312)];
    pad_temp_shared[(((int)threadIdx.x) + 4508)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4508)];
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 4900)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4900)];
    pad_temp_shared[(((int)threadIdx.x) + 5096)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5096)];
    pad_temp_shared[(((int)threadIdx.x) + 5292)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5292)];
    pad_temp_shared[(((int)threadIdx.x) + 5488)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5488)];
    pad_temp_shared[(((int)threadIdx.x) + 5684)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5684)];
    pad_temp_shared[(((int)threadIdx.x) + 5880)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5880)];
    pad_temp_shared[(((int)threadIdx.x) + 6076)] = Input[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 6076)];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) * 32768) + ((((int)threadIdx.x) >> 7) * 1024)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127))];
    kernel_shared[(((int)threadIdx.x) + 196)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 196) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 68) & 127))];
    kernel_shared[(((int)threadIdx.x) + 392)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 392) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 8) & 127))];
    kernel_shared[(((int)threadIdx.x) + 588)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 588) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 76) & 127))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 784) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 16) & 127))];
    kernel_shared[(((int)threadIdx.x) + 980)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 980) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 84) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1176)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1176) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 24) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1372)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1372) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 92) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1568) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1764)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1764) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 100) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1960)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1960) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 40) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2156)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2156) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 108) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2352)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2352) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2548)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2548) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 116) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2744)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2744) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 56) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2940)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2940) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 124) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3136) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3332)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3332) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 4) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3528)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3528) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 72) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3724)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3724) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 12) & 127))];
    if (((int)threadIdx.x) < 176) {
      kernel_shared[(((int)threadIdx.x) + 3920)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3920) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 32; ++rc_inner) {
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[((((((int)threadIdx.x) / 49) * 1024) + (rc_outer_inner * 32)) + rc_inner)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 1024) + (rc_outer_inner * 32)) + rc_inner) + 128)]));
        conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 1024) + (rc_outer_inner * 32)) + rc_inner) + 256)]));
        conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 1024) + (rc_outer_inner * 32)) + rc_inner) + 384)]));
        conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 1024) + (rc_outer_inner * 32)) + rc_inner) + 512)]));
        conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 1024) + (rc_outer_inner * 32)) + rc_inner) + 640)]));
        conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 1024) + (rc_outer_inner * 32)) + rc_inner) + 768)]));
        conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 1024) + (rc_outer_inner * 32)) + rc_inner) + 896)]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    compute[((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 49) * 392)) + (i1_inner * 49)) + (((int)threadIdx.x) % 49))] = max(conv2d_nchw[i1_inner], 0.000000e+00f);
  }
}


