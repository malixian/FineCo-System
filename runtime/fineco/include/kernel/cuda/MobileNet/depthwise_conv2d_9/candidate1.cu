
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[7];
  __shared__ float PaddedInput_shared[4032];
  __shared__ float kernel_shared[192];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 7; ++i_outer_inner_init) {
    DepthwiseConv2d[i_outer_inner_init] = 0.000000e+00f;
  }
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + ((int)threadIdx.x))] = (((((1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) / 7)) % 9)) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) / 7)) % 9) < 8)) && (1 <= (dj_outer_outer + (((int)threadIdx.x) % 7)))) && ((dj_outer_outer + (((int)threadIdx.x) % 7)) < 8)) ? Input[((((((((int)blockIdx.x) * 3136) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) / 7)) / 9) * 49)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) / 7)) % 9) * 7)) + dj_outer_outer) + (((int)threadIdx.x) % 7)) - 8)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 192) {
      kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) * 576) + (((int)threadIdx.x) * 3)) + dj_outer_outer)];
    }
    __syncthreads();
    for (int i_outer_inner = 0; i_outer_inner < 7; ++i_outer_inner) {
      for (int di_inner = 0; di_inner < 3; ++di_inner) {
        DepthwiseConv2d[i_outer_inner] = (DepthwiseConv2d[i_outer_inner] + (PaddedInput_shared[(((((((int)threadIdx.x) / 7) * 63) + (i_outer_inner * 7)) + (di_inner * 7)) + (((int)threadIdx.x) % 7))] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + di_inner)]));
      }
    }
  }
  for (int i2_inner = 0; i2_inner < 7; ++i2_inner) {
    compute[((((((int)blockIdx.x) * 3136) + ((((int)threadIdx.x) / 7) * 49)) + (i2_inner * 7)) + (((int)threadIdx.x) % 7))] = max(DepthwiseConv2d[i2_inner], 0.000000e+00f);
  }
}


