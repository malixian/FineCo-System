
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(392) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[2];
  __shared__ float PaddedInput_shared[1296];
  __shared__ float kernel_shared[144];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  PaddedInput_shared[(((int)threadIdx.x) * 3)] = ((((3 <= (((int)threadIdx.x) % 27)) && ((((int)threadIdx.x) % 27) < 24)) && (1 <= (((int)threadIdx.x) % 3))) ? Input[(((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 27) * 49)) + (((((int)threadIdx.x) % 27) / 3) * 7)) + ((((int)threadIdx.x) % 3) * 3)) - 8)] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) * 3) + 1)] = (((3 <= (((int)threadIdx.x) % 27)) && ((((int)threadIdx.x) % 27) < 24)) ? Input[(((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 27) * 49)) + (((((int)threadIdx.x) % 27) / 3) * 7)) + ((((int)threadIdx.x) % 3) * 3)) - 7)] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) * 3) + 2)] = ((((3 <= (((int)threadIdx.x) % 27)) && ((((int)threadIdx.x) % 27) < 24)) && ((((int)threadIdx.x) % 3) < 2)) ? Input[(((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 27) * 49)) + (((((int)threadIdx.x) % 27) / 3) * 7)) + ((((int)threadIdx.x) % 3) * 3)) - 6)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 40) {
    PaddedInput_shared[((((int)threadIdx.x) * 3) + 1176)] = (((((3 <= ((((int)threadIdx.x) + 14) % 27)) && (((((int)threadIdx.x) + 14) % 27) < 24)) && (1 <= (((((int)threadIdx.x) * 3) + 6) % 9))) && ((((((int)threadIdx.x) * 3) + 6) % 9) < 8)) ? Input[(((((((int)blockIdx.x) * 784) + (((((int)threadIdx.x) + 392) / 27) * 49)) + ((((((int)threadIdx.x) + 14) % 27) / 3) * 7)) + (((((int)threadIdx.x) * 3) + 6) % 9)) - 8)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 3) + 1177)] = (((((3 <= ((((int)threadIdx.x) + 14) % 27)) && (((((int)threadIdx.x) + 14) % 27) < 24)) && (1 <= (((((int)threadIdx.x) * 3) + 7) % 9))) && ((((((int)threadIdx.x) * 3) + 7) % 9) < 8)) ? Input[(((((((int)blockIdx.x) * 784) + (((((int)threadIdx.x) + 392) / 27) * 49)) + ((((((int)threadIdx.x) + 14) % 27) / 3) * 7)) + (((((int)threadIdx.x) * 3) + 7) % 9)) - 8)] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) * 3) + 1178)] = (((((3 <= ((((int)threadIdx.x) + 14) % 27)) && (((((int)threadIdx.x) + 14) % 27) < 24)) && (1 <= (((((int)threadIdx.x) * 3) + 8) % 9))) && ((((((int)threadIdx.x) * 3) + 8) % 9) < 8)) ? Input[(((((((int)blockIdx.x) * 784) + (((((int)threadIdx.x) + 392) / 27) * 49)) + ((((((int)threadIdx.x) + 14) % 27) / 3) * 7)) + (((((int)threadIdx.x) * 3) + 8) % 9)) - 8)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 144) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((int)blockIdx.x) * 144) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int di_inner = 0; di_inner < 3; ++di_inner) {
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 49) * 81) + (((((int)threadIdx.x) % 49) / 7) * 9)) + (di_inner * 9)) + (((int)threadIdx.x) % 7))] * kernel_shared[(((((int)threadIdx.x) / 49) * 9) + (di_inner * 3))]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 81) + (((((int)threadIdx.x) % 49) / 7) * 9)) + (di_inner * 9)) + (((int)threadIdx.x) % 7)) + 648)] * kernel_shared[((((((int)threadIdx.x) / 49) * 9) + (di_inner * 3)) + 72)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 81) + (((((int)threadIdx.x) % 49) / 7) * 9)) + (di_inner * 9)) + (((int)threadIdx.x) % 7)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 49) * 9) + (di_inner * 3)) + 1)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 81) + (((((int)threadIdx.x) % 49) / 7) * 9)) + (di_inner * 9)) + (((int)threadIdx.x) % 7)) + 649)] * kernel_shared[((((((int)threadIdx.x) / 49) * 9) + (di_inner * 3)) + 73)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 81) + (((((int)threadIdx.x) % 49) / 7) * 9)) + (di_inner * 9)) + (((int)threadIdx.x) % 7)) + 2)] * kernel_shared[((((((int)threadIdx.x) / 49) * 9) + (di_inner * 3)) + 2)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 81) + (((((int)threadIdx.x) % 49) / 7) * 9)) + (di_inner * 9)) + (((int)threadIdx.x) % 7)) + 650)] * kernel_shared[((((((int)threadIdx.x) / 49) * 9) + (di_inner * 3)) + 74)]));
  }
  compute[((((int)blockIdx.x) * 784) + ((int)threadIdx.x))] = max(DepthwiseConv2d[0], 0.000000e+00f);
  compute[(((((int)blockIdx.x) * 784) + ((int)threadIdx.x)) + 392)] = max(DepthwiseConv2d[1], 0.000000e+00f);
}


