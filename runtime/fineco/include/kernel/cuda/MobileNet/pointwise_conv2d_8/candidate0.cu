
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(49) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[4];
  __shared__ float pad_temp_shared[3136];
  __shared__ float kernel_shared[256];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    *(float4*)(pad_temp_shared + (((int)threadIdx.x) * 4)) = *(float4*)(Input + ((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 196)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 196));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 392)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 392));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 588)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 588));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 784)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 784));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 980)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 980));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 1176)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 1176));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 1372)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 1372));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 1568)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 1568));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 1764)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 1764));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 1960)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 1960));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 2156)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 2156));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 2352)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 2352));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 2548)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 2548));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 2744)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 2744));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 2940)) = *(float4*)(Input + (((rc_outer_outer * 3136) + (((int)threadIdx.x) * 4)) + 2940));
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) * 2048) + (rc_outer_outer * 64)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 49)] = kernel[((((((int)blockIdx.x) * 2048) + (((((int)threadIdx.x) + 49) >> 6) * 512)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 49) & 63))];
    kernel_shared[(((int)threadIdx.x) + 98)] = kernel[((((((int)blockIdx.x) * 2048) + (((((int)threadIdx.x) + 98) >> 6) * 512)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 34) & 63))];
    kernel_shared[(((int)threadIdx.x) + 147)] = kernel[((((((int)blockIdx.x) * 2048) + (((((int)threadIdx.x) + 147) >> 6) * 512)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 19) & 63))];
    kernel_shared[(((int)threadIdx.x) + 196)] = kernel[((((((int)blockIdx.x) * 2048) + (((((int)threadIdx.x) + 196) >> 6) * 512)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 4))];
    if (((int)threadIdx.x) < 11) {
      kernel_shared[(((int)threadIdx.x) + 245)] = kernel[((((((int)blockIdx.x) * 2048) + (((((int)threadIdx.x) + 245) >> 6) * 512)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 53))];
    }
    __syncthreads();
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[0]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[64]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[128]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((int)threadIdx.x)] * kernel_shared[192]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 49)] * kernel_shared[1]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 49)] * kernel_shared[65]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 49)] * kernel_shared[129]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 49)] * kernel_shared[193]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 98)] * kernel_shared[2]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 98)] * kernel_shared[66]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 98)] * kernel_shared[130]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 98)] * kernel_shared[194]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 147)] * kernel_shared[3]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 147)] * kernel_shared[67]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 147)] * kernel_shared[131]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 147)] * kernel_shared[195]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 196)] * kernel_shared[4]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 196)] * kernel_shared[68]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 196)] * kernel_shared[132]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 196)] * kernel_shared[196]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 245)] * kernel_shared[5]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 245)] * kernel_shared[69]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 245)] * kernel_shared[133]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 245)] * kernel_shared[197]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 294)] * kernel_shared[6]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 294)] * kernel_shared[70]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 294)] * kernel_shared[134]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 294)] * kernel_shared[198]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 343)] * kernel_shared[7]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 343)] * kernel_shared[71]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 343)] * kernel_shared[135]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 343)] * kernel_shared[199]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 392)] * kernel_shared[8]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 392)] * kernel_shared[72]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 392)] * kernel_shared[136]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 392)] * kernel_shared[200]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 441)] * kernel_shared[9]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 441)] * kernel_shared[73]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 441)] * kernel_shared[137]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 441)] * kernel_shared[201]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 490)] * kernel_shared[10]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 490)] * kernel_shared[74]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 490)] * kernel_shared[138]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 490)] * kernel_shared[202]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 539)] * kernel_shared[11]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 539)] * kernel_shared[75]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 539)] * kernel_shared[139]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 539)] * kernel_shared[203]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 588)] * kernel_shared[12]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 588)] * kernel_shared[76]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 588)] * kernel_shared[140]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 588)] * kernel_shared[204]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 637)] * kernel_shared[13]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 637)] * kernel_shared[77]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 637)] * kernel_shared[141]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 637)] * kernel_shared[205]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 686)] * kernel_shared[14]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 686)] * kernel_shared[78]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 686)] * kernel_shared[142]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 686)] * kernel_shared[206]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 735)] * kernel_shared[15]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 735)] * kernel_shared[79]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 735)] * kernel_shared[143]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 735)] * kernel_shared[207]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 784)] * kernel_shared[16]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 784)] * kernel_shared[80]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 784)] * kernel_shared[144]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 784)] * kernel_shared[208]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 833)] * kernel_shared[17]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 833)] * kernel_shared[81]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 833)] * kernel_shared[145]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 833)] * kernel_shared[209]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 882)] * kernel_shared[18]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 882)] * kernel_shared[82]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 882)] * kernel_shared[146]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 882)] * kernel_shared[210]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 931)] * kernel_shared[19]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 931)] * kernel_shared[83]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 931)] * kernel_shared[147]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 931)] * kernel_shared[211]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 980)] * kernel_shared[20]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 980)] * kernel_shared[84]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 980)] * kernel_shared[148]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 980)] * kernel_shared[212]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1029)] * kernel_shared[21]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1029)] * kernel_shared[85]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1029)] * kernel_shared[149]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1029)] * kernel_shared[213]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1078)] * kernel_shared[22]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1078)] * kernel_shared[86]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1078)] * kernel_shared[150]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1078)] * kernel_shared[214]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1127)] * kernel_shared[23]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1127)] * kernel_shared[87]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1127)] * kernel_shared[151]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1127)] * kernel_shared[215]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1176)] * kernel_shared[24]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1176)] * kernel_shared[88]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1176)] * kernel_shared[152]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1176)] * kernel_shared[216]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1225)] * kernel_shared[25]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1225)] * kernel_shared[89]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1225)] * kernel_shared[153]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1225)] * kernel_shared[217]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1274)] * kernel_shared[26]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1274)] * kernel_shared[90]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1274)] * kernel_shared[154]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1274)] * kernel_shared[218]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1323)] * kernel_shared[27]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1323)] * kernel_shared[91]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1323)] * kernel_shared[155]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1323)] * kernel_shared[219]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1372)] * kernel_shared[28]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1372)] * kernel_shared[92]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1372)] * kernel_shared[156]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1372)] * kernel_shared[220]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1421)] * kernel_shared[29]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1421)] * kernel_shared[93]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1421)] * kernel_shared[157]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1421)] * kernel_shared[221]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1470)] * kernel_shared[30]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1470)] * kernel_shared[94]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1470)] * kernel_shared[158]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1470)] * kernel_shared[222]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1519)] * kernel_shared[31]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1519)] * kernel_shared[95]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1519)] * kernel_shared[159]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1519)] * kernel_shared[223]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1568)] * kernel_shared[32]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1568)] * kernel_shared[96]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1568)] * kernel_shared[160]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1568)] * kernel_shared[224]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1617)] * kernel_shared[33]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1617)] * kernel_shared[97]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1617)] * kernel_shared[161]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1617)] * kernel_shared[225]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1666)] * kernel_shared[34]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1666)] * kernel_shared[98]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1666)] * kernel_shared[162]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1666)] * kernel_shared[226]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1715)] * kernel_shared[35]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1715)] * kernel_shared[99]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1715)] * kernel_shared[163]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1715)] * kernel_shared[227]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1764)] * kernel_shared[36]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1764)] * kernel_shared[100]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1764)] * kernel_shared[164]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1764)] * kernel_shared[228]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1813)] * kernel_shared[37]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1813)] * kernel_shared[101]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1813)] * kernel_shared[165]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1813)] * kernel_shared[229]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1862)] * kernel_shared[38]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1862)] * kernel_shared[102]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1862)] * kernel_shared[166]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1862)] * kernel_shared[230]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1911)] * kernel_shared[39]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1911)] * kernel_shared[103]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1911)] * kernel_shared[167]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1911)] * kernel_shared[231]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 1960)] * kernel_shared[40]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 1960)] * kernel_shared[104]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 1960)] * kernel_shared[168]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 1960)] * kernel_shared[232]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2009)] * kernel_shared[41]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2009)] * kernel_shared[105]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2009)] * kernel_shared[169]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2009)] * kernel_shared[233]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2058)] * kernel_shared[42]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2058)] * kernel_shared[106]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2058)] * kernel_shared[170]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2058)] * kernel_shared[234]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2107)] * kernel_shared[43]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2107)] * kernel_shared[107]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2107)] * kernel_shared[171]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2107)] * kernel_shared[235]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2156)] * kernel_shared[44]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2156)] * kernel_shared[108]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2156)] * kernel_shared[172]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2156)] * kernel_shared[236]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2205)] * kernel_shared[45]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2205)] * kernel_shared[109]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2205)] * kernel_shared[173]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2205)] * kernel_shared[237]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2254)] * kernel_shared[46]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2254)] * kernel_shared[110]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2254)] * kernel_shared[174]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2254)] * kernel_shared[238]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2303)] * kernel_shared[47]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2303)] * kernel_shared[111]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2303)] * kernel_shared[175]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2303)] * kernel_shared[239]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2352)] * kernel_shared[48]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2352)] * kernel_shared[112]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2352)] * kernel_shared[176]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2352)] * kernel_shared[240]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2401)] * kernel_shared[49]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2401)] * kernel_shared[113]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2401)] * kernel_shared[177]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2401)] * kernel_shared[241]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2450)] * kernel_shared[50]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2450)] * kernel_shared[114]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2450)] * kernel_shared[178]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2450)] * kernel_shared[242]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2499)] * kernel_shared[51]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2499)] * kernel_shared[115]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2499)] * kernel_shared[179]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2499)] * kernel_shared[243]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2548)] * kernel_shared[52]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2548)] * kernel_shared[116]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2548)] * kernel_shared[180]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2548)] * kernel_shared[244]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2597)] * kernel_shared[53]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2597)] * kernel_shared[117]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2597)] * kernel_shared[181]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2597)] * kernel_shared[245]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2646)] * kernel_shared[54]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2646)] * kernel_shared[118]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2646)] * kernel_shared[182]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2646)] * kernel_shared[246]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2695)] * kernel_shared[55]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2695)] * kernel_shared[119]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2695)] * kernel_shared[183]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2695)] * kernel_shared[247]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2744)] * kernel_shared[56]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2744)] * kernel_shared[120]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2744)] * kernel_shared[184]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2744)] * kernel_shared[248]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2793)] * kernel_shared[57]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2793)] * kernel_shared[121]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2793)] * kernel_shared[185]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2793)] * kernel_shared[249]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2842)] * kernel_shared[58]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2842)] * kernel_shared[122]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2842)] * kernel_shared[186]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2842)] * kernel_shared[250]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2891)] * kernel_shared[59]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2891)] * kernel_shared[123]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2891)] * kernel_shared[187]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2891)] * kernel_shared[251]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2940)] * kernel_shared[60]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2940)] * kernel_shared[124]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2940)] * kernel_shared[188]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2940)] * kernel_shared[252]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 2989)] * kernel_shared[61]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 2989)] * kernel_shared[125]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 2989)] * kernel_shared[189]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 2989)] * kernel_shared[253]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 3038)] * kernel_shared[62]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 3038)] * kernel_shared[126]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 3038)] * kernel_shared[190]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 3038)] * kernel_shared[254]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) + 3087)] * kernel_shared[63]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) + 3087)] * kernel_shared[127]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) + 3087)] * kernel_shared[191]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) + 3087)] * kernel_shared[255]));
  }
  compute[((((int)blockIdx.x) * 196) + ((int)threadIdx.x))] = max(conv2d_nchw[0], 0.000000e+00f);
  compute[(((((int)blockIdx.x) * 196) + ((int)threadIdx.x)) + 49)] = max(conv2d_nchw[1], 0.000000e+00f);
  compute[(((((int)blockIdx.x) * 196) + ((int)threadIdx.x)) + 98)] = max(conv2d_nchw[2], 0.000000e+00f);
  compute[(((((int)blockIdx.x) * 196) + ((int)threadIdx.x)) + 147)] = max(conv2d_nchw[3], 0.000000e+00f);
}


