
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[4];
  __shared__ float pad_temp_shared[896];
  __shared__ float kernel_shared[512];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    *(float4*)(pad_temp_shared + (((int)threadIdx.x) * 4)) = *(float4*)(Input + ((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 7) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) % 7) * 4)));
    *(float4*)(pad_temp_shared + ((((int)threadIdx.x) * 4) + 448)) = *(float4*)(Input + (((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 7) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) % 7) * 4)) + 3136));
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 8192) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[(((((((int)blockIdx.x) / 7) * 8192) + (((((int)threadIdx.x) + 112) >> 5) * 512)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 16) & 31))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)blockIdx.x) / 7) * 8192) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[(((((((int)blockIdx.x) / 7) * 8192) + (((((int)threadIdx.x) + 336) >> 5) * 512)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 16) & 31))];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) / 7) * 8192) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    }
    __syncthreads();
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((int)threadIdx.x) % 28)] * kernel_shared[((((int)threadIdx.x) / 28) * 128)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((int)threadIdx.x) % 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 32)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((int)threadIdx.x) % 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 64)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((int)threadIdx.x) % 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 96)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 1)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 33)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 65)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 97)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 56)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 2)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 56)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 34)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 56)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 66)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 56)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 98)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 84)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 3)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 84)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 35)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 84)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 67)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 84)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 99)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 112)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 4)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 112)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 36)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 112)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 68)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 112)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 100)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 140)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 5)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 140)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 37)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 140)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 69)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 140)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 101)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 168)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 6)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 168)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 38)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 168)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 70)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 168)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 102)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 196)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 7)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 196)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 39)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 196)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 71)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 196)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 103)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 224)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 8)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 224)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 40)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 224)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 72)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 224)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 104)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 252)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 9)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 252)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 41)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 252)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 73)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 252)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 105)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 280)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 10)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 280)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 42)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 280)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 74)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 280)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 106)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 308)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 11)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 308)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 43)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 308)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 75)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 308)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 107)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 336)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 12)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 336)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 44)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 336)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 76)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 336)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 108)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 364)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 13)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 364)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 45)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 364)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 77)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 364)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 109)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 392)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 14)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 392)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 46)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 392)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 78)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 392)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 110)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 420)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 15)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 420)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 47)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 420)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 79)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 420)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 111)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 448)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 16)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 448)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 48)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 448)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 80)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 448)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 112)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 476)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 17)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 476)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 49)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 476)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 81)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 476)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 113)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 504)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 18)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 504)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 50)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 504)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 82)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 504)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 114)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 532)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 19)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 532)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 51)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 532)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 83)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 532)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 115)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 560)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 20)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 560)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 52)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 560)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 84)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 560)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 116)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 588)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 21)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 588)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 53)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 588)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 85)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 588)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 117)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 616)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 22)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 616)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 54)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 616)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 86)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 616)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 118)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 644)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 23)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 644)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 55)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 644)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 87)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 644)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 119)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 672)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 24)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 672)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 56)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 672)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 88)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 672)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 120)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 700)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 25)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 700)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 57)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 700)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 89)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 700)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 121)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 728)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 26)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 728)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 58)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 728)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 90)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 728)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 122)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 756)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 27)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 756)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 59)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 756)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 91)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 756)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 123)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 784)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 28)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 784)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 60)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 784)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 92)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 784)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 124)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 812)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 29)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 812)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 61)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 812)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 93)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 812)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 125)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 840)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 30)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 840)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 62)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 840)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 94)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 840)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 126)]));
    conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 868)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 31)]));
    conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 868)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 63)]));
    conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 868)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 95)]));
    conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((int)threadIdx.x) % 28) + 868)] * kernel_shared[(((((int)threadIdx.x) / 28) * 128) + 127)]));
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    compute[((((((((int)blockIdx.x) / 7) * 3136) + ((((int)threadIdx.x) / 28) * 784)) + (i1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28))] = max(conv2d_nchw[i1_inner], 0.000000e+00f);
  }
}


