
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate3(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[14];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[2048];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 448)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 896)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 1344)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 1792)];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 2240)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 2688)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 3584)];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 4032)];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 4480)];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 4928)];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 5376)];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 5824)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 6720)];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 7168)];
    pad_temp_shared[(((int)threadIdx.x) + 3808)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 7616)];
    pad_temp_shared[(((int)threadIdx.x) + 4032)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 8064)];
    pad_temp_shared[(((int)threadIdx.x) + 4256)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 8512)];
    pad_temp_shared[(((int)threadIdx.x) + 4480)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 8960)];
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 4928)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 9856)];
    pad_temp_shared[(((int)threadIdx.x) + 5152)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 10304)];
    pad_temp_shared[(((int)threadIdx.x) + 5376)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 10752)];
    pad_temp_shared[(((int)threadIdx.x) + 5600)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 11200)];
    pad_temp_shared[(((int)threadIdx.x) + 5824)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 11648)];
    pad_temp_shared[(((int)threadIdx.x) + 6048)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 12096)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + ((((int)threadIdx.x) >> 6) * 512)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 224) >> 6) * 512)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) >> 1) * 16384) + ((((int)threadIdx.x) >> 6) * 512)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 672) >> 6) * 512)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) >> 1) * 16384) + ((((int)threadIdx.x) >> 6) * 512)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 1120) >> 6) * 512)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) >> 1) * 16384) + ((((int)threadIdx.x) >> 6) * 512)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 1568) >> 6) * 512)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) >> 1) * 16384) + ((((int)threadIdx.x) >> 6) * 512)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 14336)];
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 32)) + 15872)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        conv2d_nchw[(ff_outer_inner * 7)] = (conv2d_nchw[(ff_outer_inner * 7)] + (pad_temp_shared[(((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8))]));
        conv2d_nchw[(ff_outer_inner * 7)] = (conv2d_nchw[(ff_outer_inner * 7)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 98)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 1)]));
        conv2d_nchw[(ff_outer_inner * 7)] = (conv2d_nchw[(ff_outer_inner * 7)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 196)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 2)]));
        conv2d_nchw[(ff_outer_inner * 7)] = (conv2d_nchw[(ff_outer_inner * 7)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 294)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 3)]));
        conv2d_nchw[(ff_outer_inner * 7)] = (conv2d_nchw[(ff_outer_inner * 7)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 392)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 4)]));
        conv2d_nchw[(ff_outer_inner * 7)] = (conv2d_nchw[(ff_outer_inner * 7)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 490)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 5)]));
        conv2d_nchw[(ff_outer_inner * 7)] = (conv2d_nchw[(ff_outer_inner * 7)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 588)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 6)]));
        conv2d_nchw[(ff_outer_inner * 7)] = (conv2d_nchw[(ff_outer_inner * 7)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 686)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 7)]));
        conv2d_nchw[((ff_outer_inner * 7) + 1)] = (conv2d_nchw[((ff_outer_inner * 7) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 7)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8))]));
        conv2d_nchw[((ff_outer_inner * 7) + 1)] = (conv2d_nchw[((ff_outer_inner * 7) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 105)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 1)]));
        conv2d_nchw[((ff_outer_inner * 7) + 1)] = (conv2d_nchw[((ff_outer_inner * 7) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 203)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 2)]));
        conv2d_nchw[((ff_outer_inner * 7) + 1)] = (conv2d_nchw[((ff_outer_inner * 7) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 301)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 3)]));
        conv2d_nchw[((ff_outer_inner * 7) + 1)] = (conv2d_nchw[((ff_outer_inner * 7) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 399)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 4)]));
        conv2d_nchw[((ff_outer_inner * 7) + 1)] = (conv2d_nchw[((ff_outer_inner * 7) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 497)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 5)]));
        conv2d_nchw[((ff_outer_inner * 7) + 1)] = (conv2d_nchw[((ff_outer_inner * 7) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 595)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 6)]));
        conv2d_nchw[((ff_outer_inner * 7) + 1)] = (conv2d_nchw[((ff_outer_inner * 7) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 693)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 7)]));
        conv2d_nchw[((ff_outer_inner * 7) + 2)] = (conv2d_nchw[((ff_outer_inner * 7) + 2)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8))]));
        conv2d_nchw[((ff_outer_inner * 7) + 2)] = (conv2d_nchw[((ff_outer_inner * 7) + 2)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 112)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 1)]));
        conv2d_nchw[((ff_outer_inner * 7) + 2)] = (conv2d_nchw[((ff_outer_inner * 7) + 2)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 210)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 2)]));
        conv2d_nchw[((ff_outer_inner * 7) + 2)] = (conv2d_nchw[((ff_outer_inner * 7) + 2)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 308)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 3)]));
        conv2d_nchw[((ff_outer_inner * 7) + 2)] = (conv2d_nchw[((ff_outer_inner * 7) + 2)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 406)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 4)]));
        conv2d_nchw[((ff_outer_inner * 7) + 2)] = (conv2d_nchw[((ff_outer_inner * 7) + 2)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 504)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 5)]));
        conv2d_nchw[((ff_outer_inner * 7) + 2)] = (conv2d_nchw[((ff_outer_inner * 7) + 2)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 602)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 6)]));
        conv2d_nchw[((ff_outer_inner * 7) + 2)] = (conv2d_nchw[((ff_outer_inner * 7) + 2)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 700)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 7)]));
        conv2d_nchw[((ff_outer_inner * 7) + 3)] = (conv2d_nchw[((ff_outer_inner * 7) + 3)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 21)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8))]));
        conv2d_nchw[((ff_outer_inner * 7) + 3)] = (conv2d_nchw[((ff_outer_inner * 7) + 3)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 119)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 1)]));
        conv2d_nchw[((ff_outer_inner * 7) + 3)] = (conv2d_nchw[((ff_outer_inner * 7) + 3)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 217)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 2)]));
        conv2d_nchw[((ff_outer_inner * 7) + 3)] = (conv2d_nchw[((ff_outer_inner * 7) + 3)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 315)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 3)]));
        conv2d_nchw[((ff_outer_inner * 7) + 3)] = (conv2d_nchw[((ff_outer_inner * 7) + 3)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 413)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 4)]));
        conv2d_nchw[((ff_outer_inner * 7) + 3)] = (conv2d_nchw[((ff_outer_inner * 7) + 3)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 511)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 5)]));
        conv2d_nchw[((ff_outer_inner * 7) + 3)] = (conv2d_nchw[((ff_outer_inner * 7) + 3)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 609)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 6)]));
        conv2d_nchw[((ff_outer_inner * 7) + 3)] = (conv2d_nchw[((ff_outer_inner * 7) + 3)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 707)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 7)]));
        conv2d_nchw[((ff_outer_inner * 7) + 4)] = (conv2d_nchw[((ff_outer_inner * 7) + 4)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8))]));
        conv2d_nchw[((ff_outer_inner * 7) + 4)] = (conv2d_nchw[((ff_outer_inner * 7) + 4)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 126)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 1)]));
        conv2d_nchw[((ff_outer_inner * 7) + 4)] = (conv2d_nchw[((ff_outer_inner * 7) + 4)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 224)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 2)]));
        conv2d_nchw[((ff_outer_inner * 7) + 4)] = (conv2d_nchw[((ff_outer_inner * 7) + 4)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 322)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 3)]));
        conv2d_nchw[((ff_outer_inner * 7) + 4)] = (conv2d_nchw[((ff_outer_inner * 7) + 4)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 420)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 4)]));
        conv2d_nchw[((ff_outer_inner * 7) + 4)] = (conv2d_nchw[((ff_outer_inner * 7) + 4)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 518)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 5)]));
        conv2d_nchw[((ff_outer_inner * 7) + 4)] = (conv2d_nchw[((ff_outer_inner * 7) + 4)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 616)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 6)]));
        conv2d_nchw[((ff_outer_inner * 7) + 4)] = (conv2d_nchw[((ff_outer_inner * 7) + 4)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 714)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 7)]));
        conv2d_nchw[((ff_outer_inner * 7) + 5)] = (conv2d_nchw[((ff_outer_inner * 7) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 35)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8))]));
        conv2d_nchw[((ff_outer_inner * 7) + 5)] = (conv2d_nchw[((ff_outer_inner * 7) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 133)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 1)]));
        conv2d_nchw[((ff_outer_inner * 7) + 5)] = (conv2d_nchw[((ff_outer_inner * 7) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 231)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 2)]));
        conv2d_nchw[((ff_outer_inner * 7) + 5)] = (conv2d_nchw[((ff_outer_inner * 7) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 329)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 3)]));
        conv2d_nchw[((ff_outer_inner * 7) + 5)] = (conv2d_nchw[((ff_outer_inner * 7) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 427)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 4)]));
        conv2d_nchw[((ff_outer_inner * 7) + 5)] = (conv2d_nchw[((ff_outer_inner * 7) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 525)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 5)]));
        conv2d_nchw[((ff_outer_inner * 7) + 5)] = (conv2d_nchw[((ff_outer_inner * 7) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 623)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 6)]));
        conv2d_nchw[((ff_outer_inner * 7) + 5)] = (conv2d_nchw[((ff_outer_inner * 7) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 721)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 7)]));
        conv2d_nchw[((ff_outer_inner * 7) + 6)] = (conv2d_nchw[((ff_outer_inner * 7) + 6)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8))]));
        conv2d_nchw[((ff_outer_inner * 7) + 6)] = (conv2d_nchw[((ff_outer_inner * 7) + 6)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 140)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 1)]));
        conv2d_nchw[((ff_outer_inner * 7) + 6)] = (conv2d_nchw[((ff_outer_inner * 7) + 6)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 238)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 2)]));
        conv2d_nchw[((ff_outer_inner * 7) + 6)] = (conv2d_nchw[((ff_outer_inner * 7) + 6)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 336)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 3)]));
        conv2d_nchw[((ff_outer_inner * 7) + 6)] = (conv2d_nchw[((ff_outer_inner * 7) + 6)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 434)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 4)]));
        conv2d_nchw[((ff_outer_inner * 7) + 6)] = (conv2d_nchw[((ff_outer_inner * 7) + 6)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 532)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 5)]));
        conv2d_nchw[((ff_outer_inner * 7) + 6)] = (conv2d_nchw[((ff_outer_inner * 7) + 6)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 630)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 6)]));
        conv2d_nchw[((ff_outer_inner * 7) + 6)] = (conv2d_nchw[((ff_outer_inner * 7) + 6)] + (pad_temp_shared[((((rc_outer_inner * 784) + (((((int)threadIdx.x) % 14) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 728)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 8)) + 7)]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 7; ++i2_inner) {
      compute[((((((((((int)blockIdx.x) >> 1) * 6272) + ((((int)threadIdx.x) / 14) * 392)) + (i1_inner * 196)) + (((((int)threadIdx.x) % 14) / 7) * 98)) + (i2_inner * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7))] = max(conv2d_nchw[((i1_inner * 7) + i2_inner)], 0.000000e+00f);
    }
  }
}


