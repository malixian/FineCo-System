
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate2(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[16];
  __shared__ float pad_temp_shared[448];
  __shared__ float kernel_shared[2048];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((rc_outer_outer * 3136) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = Input[(((((rc_outer_outer * 3136) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 1568)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 17920)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 21504)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 25088)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 28672)];
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 32256)];
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          conv2d_nchw[((ff_outer_inner * 2) + yy_outer_inner)] = (conv2d_nchw[((ff_outer_inner * 2) + yy_outer_inner)] + (pad_temp_shared[(((rc_inner * 28) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + (ff_outer_inner * 16)) + rc_inner)]));
          conv2d_nchw[(((ff_outer_inner * 2) + yy_outer_inner) + 8)] = (conv2d_nchw[(((ff_outer_inner * 2) + yy_outer_inner) + 8)] + (pad_temp_shared[(((rc_inner * 28) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 64) + (ff_outer_inner * 16)) + rc_inner) + 1024)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[(((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 14) * 784)) + (i1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (i2_inner * 14)) + (((int)threadIdx.x) % 14))] = max(conv2d_nchw[((i1_inner * 2) + i2_inner)], 0.000000e+00f);
      compute[((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 14) * 784)) + (i1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (i2_inner * 14)) + (((int)threadIdx.x) % 14)) + 12544)] = max(conv2d_nchw[(((i1_inner * 2) + i2_inner) + 8)], 0.000000e+00f);
    }
  }
}


