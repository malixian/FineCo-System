
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(392) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[16];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[4096];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98))];
    pad_temp_shared[(((int)threadIdx.x) + 392)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 1960)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 3920)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 2744)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 5488)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 3528)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 7056)];
    pad_temp_shared[(((int)threadIdx.x) + 3920)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 7840)];
    pad_temp_shared[(((int)threadIdx.x) + 4312)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 8624)];
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 5096)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 10192)];
    pad_temp_shared[(((int)threadIdx.x) + 5488)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 10976)];
    pad_temp_shared[(((int)threadIdx.x) + 5880)] = Input[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 98) * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98)) + 11760)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 392)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 392) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 8) & 63))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 784) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 16) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1176)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 1176) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 24) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 1568) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1960)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 1960) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63))];
    kernel_shared[(((int)threadIdx.x) + 2352)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 2352) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63))];
    kernel_shared[(((int)threadIdx.x) + 2744)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 2744) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63))];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((((int)blockIdx.x) >> 1) * 16384) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 12544)];
    kernel_shared[(((int)threadIdx.x) + 3528)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 3528) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 8) & 63))];
    if (((int)threadIdx.x) < 176) {
      kernel_shared[(((int)threadIdx.x) + 3920)] = kernel[(((((((int)blockIdx.x) >> 1) * 16384) + (((((int)threadIdx.x) + 3920) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 16) & 63))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          conv2d_nchw[(ff_outer_inner * 8)] = (conv2d_nchw[(ff_outer_inner * 8)] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 98)) + ((((int)threadIdx.x) % 49) * 2))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_outer_inner * 256)) + (rc_outer_inner * 16)) + rc_inner)]));
          conv2d_nchw[((ff_outer_inner * 8) + 1)] = (conv2d_nchw[((ff_outer_inner * 8) + 1)] + (pad_temp_shared[((((rc_outer_inner * 1568) + (rc_inner * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 49) * 512) + (ff_outer_inner * 256)) + (rc_outer_inner * 16)) + rc_inner)]));
          conv2d_nchw[((ff_outer_inner * 8) + 2)] = (conv2d_nchw[((ff_outer_inner * 8) + 2)] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 98)) + ((((int)threadIdx.x) % 49) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 49) * 512) + (ff_outer_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 64)]));
          conv2d_nchw[((ff_outer_inner * 8) + 3)] = (conv2d_nchw[((ff_outer_inner * 8) + 3)] + (pad_temp_shared[((((rc_outer_inner * 1568) + (rc_inner * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 49) * 512) + (ff_outer_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 64)]));
          conv2d_nchw[((ff_outer_inner * 8) + 4)] = (conv2d_nchw[((ff_outer_inner * 8) + 4)] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 98)) + ((((int)threadIdx.x) % 49) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 49) * 512) + (ff_outer_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 128)]));
          conv2d_nchw[((ff_outer_inner * 8) + 5)] = (conv2d_nchw[((ff_outer_inner * 8) + 5)] + (pad_temp_shared[((((rc_outer_inner * 1568) + (rc_inner * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 49) * 512) + (ff_outer_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 128)]));
          conv2d_nchw[((ff_outer_inner * 8) + 6)] = (conv2d_nchw[((ff_outer_inner * 8) + 6)] + (pad_temp_shared[(((rc_outer_inner * 1568) + (rc_inner * 98)) + ((((int)threadIdx.x) % 49) * 2))] * kernel_shared[((((((((int)threadIdx.x) / 49) * 512) + (ff_outer_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 192)]));
          conv2d_nchw[((ff_outer_inner * 8) + 7)] = (conv2d_nchw[((ff_outer_inner * 8) + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + (rc_inner * 98)) + ((((int)threadIdx.x) % 49) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 49) * 512) + (ff_outer_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 192)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
      compute[(((((((((int)blockIdx.x) >> 1) * 12544) + ((((int)threadIdx.x) / 49) * 1568)) + (i1_inner * 196)) + ((((int)blockIdx.x) & 1) * 98)) + ((((int)threadIdx.x) % 49) * 2)) + i3_inner)] = max(conv2d_nchw[((i1_inner * 2) + i3_inner)], 0.000000e+00f);
    }
  }
}


