
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate3(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[16];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[2048];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
      conv2d_nchw[((ff_outer_inner_init * 2) + yy_inner_init)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 2) + yy_inner_init) + 8)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 196) + ((int)threadIdx.x))] = Input[(((((rc_outer_outer * 12544) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 392)) + ((((int)threadIdx.x) / 7) * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 49) + (((int)threadIdx.x) >> 2)) < 512) {
        kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 196) + ((int)threadIdx.x))] = kernel[(((((((int)blockIdx.x) >> 1) * 8192) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 49) + (((int)threadIdx.x) >> 2)) >> 4) * 256)) + (rc_outer_outer * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 196) + ((int)threadIdx.x)) & 63))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            conv2d_nchw[((ff_outer_inner * 2) + yy_inner)] = (conv2d_nchw[((ff_outer_inner * 2) + yy_inner)] + (pad_temp_shared[(((((rc_outer_inner * 1568) + (rc_inner * 98)) + (((((int)threadIdx.x) % 49) / 7) * 14)) + (yy_inner * 7)) + (((int)threadIdx.x) % 7))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 256) + (ff_outer_inner * 64)) + (rc_outer_inner * 16)) + rc_inner)]));
            conv2d_nchw[(((ff_outer_inner * 2) + yy_inner) + 8)] = (conv2d_nchw[(((ff_outer_inner * 2) + yy_inner) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 1568) + (rc_inner * 98)) + (((((int)threadIdx.x) % 49) / 7) * 14)) + (yy_inner * 7)) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((((int)threadIdx.x) / 49) * 256) + (ff_outer_inner * 64)) + (rc_outer_inner * 16)) + rc_inner) + 1024)]));
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[((((((((((int)blockIdx.x) >> 1) * 6272) + ((((int)threadIdx.x) / 49) * 784)) + (i1_inner * 196)) + (((((int)threadIdx.x) % 49) / 7) * 28)) + (i2_inner * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7))] = max(conv2d_nchw[((i1_inner * 2) + i2_inner)], 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) >> 1) * 6272) + ((((int)threadIdx.x) / 49) * 784)) + (i1_inner * 196)) + (((((int)threadIdx.x) % 49) / 7) * 28)) + (i2_inner * 14)) + ((((int)blockIdx.x) & 1) * 7)) + (((int)threadIdx.x) % 7)) + 3136)] = max(conv2d_nchw[(((i1_inner * 2) + i2_inner) + 8)], 0.000000e+00f);
    }
  }
}


