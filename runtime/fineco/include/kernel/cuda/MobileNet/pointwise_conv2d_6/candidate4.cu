
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate4(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[16];
  __shared__ float pad_temp_shared[3584];
  __shared__ float kernel_shared[8192];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 112)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 336)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 560)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 3920)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 5488)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 1008)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 7056)];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 7840)];
    pad_temp_shared[(((int)threadIdx.x) + 1232)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 8624)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 1456)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 10192)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 10976)];
    pad_temp_shared[(((int)threadIdx.x) + 1680)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 11760)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 1904)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 13328)];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 14112)];
    pad_temp_shared[(((int)threadIdx.x) + 2128)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 14896)];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 15680)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 16464)];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 17248)];
    pad_temp_shared[(((int)threadIdx.x) + 2576)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 18032)];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 2800)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 19600)];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 20384)];
    pad_temp_shared[(((int)threadIdx.x) + 3024)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 21168)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 21952)];
    pad_temp_shared[(((int)threadIdx.x) + 3248)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 22736)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 23520)];
    pad_temp_shared[(((int)threadIdx.x) + 3472)] = Input[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 24304)];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 112) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 224) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 336) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 448) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 560)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 560) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 672) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 784) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 1008) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 1120) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1232)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 1232) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 1344) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1456)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 1456) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 1568) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1680)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 1680) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1904)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 1904) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 2016) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2128)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 2128) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 2240) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2352)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 2352) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 2464) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2576)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 2576) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 5376)];
    kernel_shared[(((int)threadIdx.x) + 2800)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 2800) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 2912) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3024)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 3024) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 3136) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3248)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 3248) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 3360) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3472)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 3472) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 3696)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 3696) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 3808) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3920)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 3920) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 4032) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4144)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 4144) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4256)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 4256) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4368)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 4368) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 8960)];
    kernel_shared[(((int)threadIdx.x) + 4592)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 4592) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4704)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 4704) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4816)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 4816) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4928)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 4928) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5040)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 5040) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5152)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 5152) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5264)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 5264) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 5488)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 5488) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5600)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 5600) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5712)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 5712) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5824)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 5824) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5936)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 5936) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6048)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 6048) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6160)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 6160) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 12544)];
    kernel_shared[(((int)threadIdx.x) + 6384)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 6384) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6496)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 6496) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6608)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 6608) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6720)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 6720) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6832)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 6832) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6944)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 6944) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7056)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 7056) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 7280)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 7280) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7392)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 7392) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7504)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 7504) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7616)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 7616) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7728)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 7728) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7840)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 7840) >> 7) * 256)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7952)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 7952) >> 7) * 256)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 16128)];
    if (((int)threadIdx.x) < 16) {
      kernel_shared[(((int)threadIdx.x) + 8176)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 112)) + 16128)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 64; ++rc_outer_inner) {
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((rc_outer_inner * 56) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2))]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2))]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((rc_outer_inner * 56) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 128)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 128)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((rc_outer_inner * 56) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 256)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 256)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((rc_outer_inner * 56) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 384)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 384)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 129)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 129)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 257)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 257)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 385)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 385)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((rc_outer_inner * 56) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 512)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((rc_outer_inner * 56) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 640)]));
      conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 640)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((rc_outer_inner * 56) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 768)]));
      conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 768)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((rc_outer_inner * 56) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 896)]));
      conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 896)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 513)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 641)]));
      conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 641)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 769)]));
      conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 769)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 897)]));
      conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((rc_outer_inner * 56) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 1024) + (rc_outer_inner * 2)) + 897)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    compute[((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 1568)) + (i1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14))] = max(conv2d_nchw[i1_inner], 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 1568)) + (i1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 14)) + 14)] = max(conv2d_nchw[(i1_inner + 8)], 0.000000e+00f);
  }
}


