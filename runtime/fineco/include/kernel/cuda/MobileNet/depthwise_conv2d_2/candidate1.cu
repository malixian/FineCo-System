
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(896) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[4];
  __shared__ float PaddedInput_shared[10848];
  __shared__ float kernel_shared[96];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((int)threadIdx.x)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + ((((int)threadIdx.x) % 339) / 113)) + di_outer_outer)) && (1 <= (((int)threadIdx.x) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + ((((int)threadIdx.x) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + (((((int)threadIdx.x) % 339) / 113) * 112)) + (di_outer_outer * 112)) + (((int)threadIdx.x) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 218) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 105) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 896) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 218) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 105) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 97) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 97) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 1792) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 97) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 97) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 315) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 89) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 2688) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 315) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 89) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 3584)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 194) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 81) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 3584) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 194) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 81) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 4480)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 73) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 73) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 4480) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 73) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 73) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 5376)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 291) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 65) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 5376) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 291) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 65) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 6272)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 170) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 57) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 6272) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 170) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 57) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 7168)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 49) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 49) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 7168) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 49) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 49) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 8064)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 267) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 41) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 8064) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 267) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 41) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 8960)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 146) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 33) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 8960) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 146) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 33) % 113)) - 113)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 9856)] = (((1 <= ((((((int)blockIdx.x) % 28) * 4) + (((((int)threadIdx.x) + 25) % 339) / 113)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 25) % 113))) ? Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 9856) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 25) % 339) / 113) * 112)) + (di_outer_outer * 112)) + ((((int)threadIdx.x) + 25) % 113)) - 113)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 96) {
      PaddedInput_shared[(((int)threadIdx.x) + 10752)] = Input[((((((((((int)blockIdx.x) / 28) * 401408) + (((((int)threadIdx.x) + 10752) / 339) * 12544)) + ((((int)blockIdx.x) % 28) * 448)) + ((((((int)threadIdx.x) + 243) % 339) / 113) * 112)) + (di_outer_outer * 112)) + (((int)threadIdx.x) + 17)) - 113)];
    }
    if (((int)threadIdx.x) < 96) {
      kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 28) * 288) + ((((int)threadIdx.x) / 3) * 9)) + (di_outer_outer * 3)) + (((int)threadIdx.x) % 3))];
    }
    __syncthreads();
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2))] * kernel_shared[((((int)threadIdx.x) / 56) * 3)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 56)] * kernel_shared[((((int)threadIdx.x) / 56) * 3)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 5424)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 48)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 5480)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 48)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 1)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 57)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 1)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 5425)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 49)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 5481)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 49)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 2)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 58)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 2)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 5426)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 50)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 339) + (((((int)threadIdx.x) % 56) / 28) * 226)) + ((((int)threadIdx.x) % 28) * 2)) + 5482)] * kernel_shared[(((((int)threadIdx.x) / 56) * 3) + 50)]));
  }
  compute[((((((((int)blockIdx.x) / 28) * 100352) + ((((int)threadIdx.x) / 56) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) % 56) / 28) * 56)) + (((int)threadIdx.x) % 28))] = max(DepthwiseConv2d[0], 0.000000e+00f);
  compute[(((((((((int)blockIdx.x) / 28) * 100352) + ((((int)threadIdx.x) / 56) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) % 56) / 28) * 56)) + (((int)threadIdx.x) % 28)) + 28)] = max(DepthwiseConv2d[1], 0.000000e+00f);
  compute[(((((((((int)blockIdx.x) / 28) * 100352) + ((((int)threadIdx.x) / 56) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) % 56) / 28) * 56)) + (((int)threadIdx.x) % 28)) + 50176)] = max(DepthwiseConv2d[2], 0.000000e+00f);
  compute[(((((((((int)blockIdx.x) / 28) * 100352) + ((((int)threadIdx.x) / 56) * 3136)) + ((((int)blockIdx.x) % 28) * 112)) + (((((int)threadIdx.x) % 56) / 28) * 56)) + (((int)threadIdx.x) % 28)) + 50204)] = max(DepthwiseConv2d[3], 0.000000e+00f);
}


