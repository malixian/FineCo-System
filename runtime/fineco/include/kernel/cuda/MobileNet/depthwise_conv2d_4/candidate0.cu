
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[1];
  __shared__ float PaddedInput_shared[2784];
  __shared__ float kernel_shared[288];
  DepthwiseConv2d[0] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 2) + ((((int)threadIdx.x) % 87) / 29))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + (((int)threadIdx.x) % 29)))) ? Input[((((((((((int)blockIdx.x) / 56) * 100352) + ((((int)threadIdx.x) / 87) * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + (((((int)threadIdx.x) % 87) / 29) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 29)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = (((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 2) + (((((int)threadIdx.x) + 13) % 87) / 29))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 13) % 29)))) ? Input[((((((((((int)blockIdx.x) / 56) * 100352) + (((((int)threadIdx.x) + 448) / 87) * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((((int)threadIdx.x) + 13) % 87) / 29) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 13) % 29)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 2) + (((((int)threadIdx.x) + 26) % 87) / 29))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 26) % 29)))) ? Input[((((((((((int)blockIdx.x) / 56) * 100352) + (((((int)threadIdx.x) + 896) / 87) * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((((int)threadIdx.x) + 26) % 87) / 29) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 26) % 29)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1344)] = (((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 2) + (((((int)threadIdx.x) + 39) % 87) / 29))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 10) % 29)))) ? Input[((((((((((int)blockIdx.x) / 56) * 100352) + (((((int)threadIdx.x) + 1344) / 87) * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((((int)threadIdx.x) + 39) % 87) / 29) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 10) % 29)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 2) + (((((int)threadIdx.x) + 52) % 87) / 29))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 23) % 29)))) ? Input[((((((((((int)blockIdx.x) / 56) * 100352) + (((((int)threadIdx.x) + 1792) / 87) * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((((int)threadIdx.x) + 52) % 87) / 29) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 23) % 29)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2240)] = (((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 2) + (((((int)threadIdx.x) + 65) % 87) / 29))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 7) % 29)))) ? Input[((((((((((int)blockIdx.x) / 56) * 100352) + (((((int)threadIdx.x) + 2240) / 87) * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((((int)threadIdx.x) + 65) % 87) / 29) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 7) % 29)) - 57)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 96) {
    PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 2) + (((((int)threadIdx.x) + 78) % 87) / 29))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 20) % 29)))) ? Input[((((((((((int)blockIdx.x) / 56) * 100352) + (((((int)threadIdx.x) + 2688) / 87) * 3136)) + (((((int)blockIdx.x) % 56) >> 1) * 112)) + ((((((int)threadIdx.x) + 78) % 87) / 29) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 20) % 29)) - 57)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 288) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) / 56) * 288) + ((int)threadIdx.x))];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[((((int)threadIdx.x) / 14) * 9)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2)) + 29)] * kernel_shared[(((((int)threadIdx.x) / 14) * 9) + 3)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2)) + 58)] * kernel_shared[(((((int)threadIdx.x) / 14) * 9) + 6)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 14) * 9) + 1)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2)) + 30)] * kernel_shared[(((((int)threadIdx.x) / 14) * 9) + 4)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2)) + 59)] * kernel_shared[(((((int)threadIdx.x) / 14) * 9) + 7)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 14) * 9) + 2)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2)) + 31)] * kernel_shared[(((((int)threadIdx.x) / 14) * 9) + 5)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 14) * 87) + ((((int)threadIdx.x) % 14) * 2)) + 60)] * kernel_shared[(((((int)threadIdx.x) / 14) * 9) + 8)]));
  compute[(((((((int)blockIdx.x) / 56) * 25088) + ((((int)threadIdx.x) / 14) * 784)) + ((((int)blockIdx.x) % 56) * 14)) + (((int)threadIdx.x) % 14))] = max(DepthwiseConv2d[0], 0.000000e+00f);
}


