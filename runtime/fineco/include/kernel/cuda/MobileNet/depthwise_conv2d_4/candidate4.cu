
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(392) candidate4(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[4];
  __shared__ float PaddedInput_shared[6498];
  __shared__ float kernel_shared[18];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((57 <= ((int)threadIdx.x)) && (1 <= (((int)threadIdx.x) % 57))) ? Input[((((((int)blockIdx.x) * 6272) + ((((int)threadIdx.x) / 57) * 56)) + (((int)threadIdx.x) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 392)] = ((1 <= ((((int)threadIdx.x) + 50) % 57)) ? Input[((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 392) / 57) * 56)) + ((((int)threadIdx.x) + 50) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 784)] = ((1 <= ((((int)threadIdx.x) + 43) % 57)) ? Input[((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 784) / 57) * 56)) + ((((int)threadIdx.x) + 43) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1176)] = ((1 <= ((((int)threadIdx.x) + 36) % 57)) ? Input[((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 1176) / 57) * 56)) + ((((int)threadIdx.x) + 36) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1568)] = ((1 <= ((((int)threadIdx.x) + 29) % 57)) ? Input[((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 1568) / 57) * 56)) + ((((int)threadIdx.x) + 29) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1960)] = ((1 <= ((((int)threadIdx.x) + 22) % 57)) ? Input[((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 1960) / 57) * 56)) + ((((int)threadIdx.x) + 22) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2352)] = ((1 <= ((((int)threadIdx.x) + 15) % 57)) ? Input[((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 2352) / 57) * 56)) + ((((int)threadIdx.x) + 15) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2744)] = ((1 <= ((((int)threadIdx.x) + 8) % 57)) ? Input[((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 2744) / 57) * 56)) + ((((int)threadIdx.x) + 8) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3136)] = (((57 <= ((((int)threadIdx.x) + 3136) % 3249)) && (1 <= ((((int)threadIdx.x) + 1) % 57))) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 3136) / 3249) * 3136)) + ((((((int)threadIdx.x) + 3136) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 1) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3528)] = ((1 <= ((((int)threadIdx.x) + 51) % 57)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 3528) / 3249) * 3136)) + ((((((int)threadIdx.x) + 279) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 51) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3920)] = ((1 <= ((((int)threadIdx.x) + 44) % 57)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 3920) / 3249) * 3136)) + ((((((int)threadIdx.x) + 671) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 44) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4312)] = ((1 <= ((((int)threadIdx.x) + 37) % 57)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 4312) / 3249) * 3136)) + ((((((int)threadIdx.x) + 1063) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 37) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4704)] = ((1 <= ((((int)threadIdx.x) + 30) % 57)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 4704) / 3249) * 3136)) + ((((((int)threadIdx.x) + 1455) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 30) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5096)] = ((1 <= ((((int)threadIdx.x) + 23) % 57)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 5096) / 3249) * 3136)) + ((((((int)threadIdx.x) + 1847) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 23) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5488)] = ((1 <= ((((int)threadIdx.x) + 16) % 57)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 5488) / 3249) * 3136)) + ((((((int)threadIdx.x) + 2239) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 16) % 57)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5880)] = ((1 <= ((((int)threadIdx.x) + 9) % 57)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 5880) / 3249) * 3136)) + ((((((int)threadIdx.x) + 2631) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 9) % 57)) - 57)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 226) {
    PaddedInput_shared[(((int)threadIdx.x) + 6272)] = ((1 <= ((((int)threadIdx.x) + 2) % 57)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 6272) / 3249) * 3136)) + ((((((int)threadIdx.x) + 3023) % 3249) / 57) * 56)) + ((((int)threadIdx.x) + 2) % 57)) - 57)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 18) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8))] * kernel_shared[((((int)threadIdx.x) / 196) * 9)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 1)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 2)] * kernel_shared[((((int)threadIdx.x) / 196) * 9)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 1)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 4)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 2)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 4)] * kernel_shared[((((int)threadIdx.x) / 196) * 9)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 6)] * kernel_shared[((((int)threadIdx.x) / 196) * 9)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 1)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 8)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 2)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 57)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 3)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 58)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 4)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 59)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 5)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 59)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 3)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 60)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 4)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 61)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 5)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 61)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 3)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 62)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 4)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 63)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 5)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 63)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 3)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 64)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 4)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 65)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 5)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 114)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 6)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 115)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 7)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 116)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 116)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 6)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 117)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 7)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 118)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 8)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 118)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 6)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 119)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 7)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 120)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 8)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 120)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 6)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 121)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 7)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 3249) + (((((int)threadIdx.x) % 196) / 7) * 114)) + ((((int)threadIdx.x) % 7) * 8)) + 122)] * kernel_shared[(((((int)threadIdx.x) / 196) * 9) + 8)]));
  for (int i3_inner = 0; i3_inner < 4; ++i3_inner) {
    compute[(((((int)blockIdx.x) * 1568) + (((int)threadIdx.x) * 4)) + i3_inner)] = max(DepthwiseConv2d[i3_inner], 0.000000e+00f);
  }
}


