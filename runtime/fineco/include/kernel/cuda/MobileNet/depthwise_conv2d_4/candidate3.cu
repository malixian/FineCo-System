
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate3(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[4];
  __shared__ float PaddedInput_shared[7776];
  __shared__ float kernel_shared[96];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((int)threadIdx.x)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + ((((int)threadIdx.x) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + (((int)threadIdx.x) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + ((((int)threadIdx.x) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + (((((int)threadIdx.x) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + (((int)threadIdx.x) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 448)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 205) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 16) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 448) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 205) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 16) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 167) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 5) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 896) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 167) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 5) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 129) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 21) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 1344) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 129) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 21) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 91) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 10) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 1792) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 91) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 10) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2240)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 53) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 26) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 2240) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 53) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 26) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 15) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 15) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 2688) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 15) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 15) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 3136)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 220) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 4) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 3136) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 220) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 4) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 3584)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 182) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 20) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 3584) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 182) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 20) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 4032)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 144) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 9) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 4032) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 144) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 9) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 4480)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 106) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 25) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 4480) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 106) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 25) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 4928)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 68) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 14) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 4928) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 68) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 14) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 5376)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 30) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 3) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 5376) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 30) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 3) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 5824)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 235) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 19) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 5824) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 235) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 19) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 6272)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 197) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 8) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 6272) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 197) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 8) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 6720)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 159) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 24) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 6720) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 159) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 24) % 27)) - 57)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 7168)] = (((1 <= ((((((int)blockIdx.x) % 14) >> 1) * 8) + (((((int)threadIdx.x) + 121) % 243) / 27))) && (1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 13) % 27)))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 7168) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 121) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 13) % 27)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 160) {
      PaddedInput_shared[(((int)threadIdx.x) + 7616)] = ((1 <= ((((((int)blockIdx.x) & 1) * 28) + dj_outer_outer) + ((((int)threadIdx.x) + 2) % 27))) ? Input[(((((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 7616) / 243) * 3136)) + (((((int)blockIdx.x) % 14) >> 1) * 448)) + ((((((int)threadIdx.x) + 83) % 243) / 27) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + dj_outer_outer) + ((((int)threadIdx.x) + 2) % 27)) - 57)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 96) {
      kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 14) * 288) + (((int)threadIdx.x) * 3)) + dj_outer_outer)];
    }
    __syncthreads();
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[((((int)threadIdx.x) / 28) * 3)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 108)] * kernel_shared[((((int)threadIdx.x) / 28) * 3)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 3888)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 48)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 3996)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 48)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 27)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 1)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 135)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 1)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 3915)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 49)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 4023)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 49)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 54)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 2)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 162)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 2)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 3942)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 50)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 243) + (((((int)threadIdx.x) % 28) / 14) * 54)) + ((((int)threadIdx.x) % 14) * 2)) + 4050)] * kernel_shared[(((((int)threadIdx.x) / 28) * 3) + 50)]));
  }
  compute[(((((((((int)blockIdx.x) / 14) * 25088) + ((((int)threadIdx.x) / 28) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 28) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))] = max(DepthwiseConv2d[0], 0.000000e+00f);
  compute[((((((((((int)blockIdx.x) / 14) * 25088) + ((((int)threadIdx.x) / 28) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 28) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 56)] = max(DepthwiseConv2d[1], 0.000000e+00f);
  compute[((((((((((int)blockIdx.x) / 14) * 25088) + ((((int)threadIdx.x) / 28) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 28) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 12544)] = max(DepthwiseConv2d[2], 0.000000e+00f);
  compute[((((((((((int)blockIdx.x) / 14) * 25088) + ((((int)threadIdx.x) / 28) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 28) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 12600)] = max(DepthwiseConv2d[3], 0.000000e+00f);
}

