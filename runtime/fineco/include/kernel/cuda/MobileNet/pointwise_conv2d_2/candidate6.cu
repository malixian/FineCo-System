
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate6(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[32];
  __shared__ float pad_temp_shared[1024];
  __shared__ float kernel_shared[4096];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
    conv2d_nchw[yy_outer_inner_init] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 16)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 4)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 20)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 8)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 24)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 12)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 28)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = Input[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = Input[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 384)] = Input[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 37632)];
    pad_temp_shared[(((int)threadIdx.x) + 512)] = Input[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 640)] = Input[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 62720)];
    pad_temp_shared[(((int)threadIdx.x) + 768)] = Input[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 75264)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = Input[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 5) * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (((((int)threadIdx.x) & 31) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 87808)];
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x))] = kernel[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          conv2d_nchw[yy_outer_inner] = (conv2d_nchw[yy_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 32)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_inner * 8)) + rc_inner)]));
          conv2d_nchw[(yy_outer_inner + 16)] = (conv2d_nchw[(yy_outer_inner + 16)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 32)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_inner * 8)) + rc_inner) + 2048)]));
          conv2d_nchw[(yy_outer_inner + 4)] = (conv2d_nchw[(yy_outer_inner + 4)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 32)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_inner * 8)) + rc_inner) + 32)]));
          conv2d_nchw[(yy_outer_inner + 20)] = (conv2d_nchw[(yy_outer_inner + 20)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 32)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_inner * 8)) + rc_inner) + 2080)]));
          conv2d_nchw[(yy_outer_inner + 8)] = (conv2d_nchw[(yy_outer_inner + 8)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 32)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_inner * 8)) + rc_inner) + 64)]));
          conv2d_nchw[(yy_outer_inner + 24)] = (conv2d_nchw[(yy_outer_inner + 24)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 32)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_inner * 8)) + rc_inner) + 2112)]));
          conv2d_nchw[(yy_outer_inner + 12)] = (conv2d_nchw[(yy_outer_inner + 12)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 32)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_inner * 8)) + rc_inner) + 96)]));
          conv2d_nchw[(yy_outer_inner + 28)] = (conv2d_nchw[(yy_outer_inner + 28)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 32)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (rc_outer_inner * 8)) + rc_inner) + 2144)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
      compute[(((((((((int)threadIdx.x) >> 3) * 12544) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = max(conv2d_nchw[((i1_inner * 4) + i2_inner)], 0.000000e+00f);
      compute[((((((((((int)threadIdx.x) >> 3) * 12544) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 224)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 200704)] = max(conv2d_nchw[(((i1_inner * 4) + i2_inner) + 16)], 0.000000e+00f);
    }
  }
}


