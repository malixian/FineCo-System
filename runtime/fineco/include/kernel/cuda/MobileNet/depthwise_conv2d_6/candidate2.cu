
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(896) candidate2(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[2];
  __shared__ float PaddedInput_shared[3712];
  __shared__ float kernel_shared[384];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((1 <= (((int)blockIdx.x) % 14)) && (1 <= (((int)threadIdx.x) % 29))) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + ((((int)threadIdx.x) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + (((int)threadIdx.x) % 29)) - 29)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((1 <= (((int)blockIdx.x) % 14)) && (1 <= ((((int)threadIdx.x) + 26) % 29))) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 896) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 26) % 29)) - 29)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((1 <= (((int)blockIdx.x) % 14)) && (1 <= ((((int)threadIdx.x) + 23) % 29))) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 1792) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 23) % 29)) - 29)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((1 <= (((int)blockIdx.x) % 14)) && (1 <= ((((int)threadIdx.x) + 20) % 29))) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 2688) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 20) % 29)) - 29)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 128) {
    PaddedInput_shared[(((int)threadIdx.x) + 3584)] = (((1 <= (((int)blockIdx.x) % 14)) && (1 <= ((((int)threadIdx.x) + 17) % 29))) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 3584) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 17) % 29)) - 29)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 384) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 14) * 1152) + ((((int)threadIdx.x) / 3) * 9)) + (((int)threadIdx.x) % 3))];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 3)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[((((int)threadIdx.x) / 7) * 3)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 1)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 1)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 16)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 2)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((1 <= (((int)threadIdx.x) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + ((((int)threadIdx.x) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + (((int)threadIdx.x) % 29)) - 1)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((1 <= ((((int)threadIdx.x) + 26) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 896) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 26) % 29)) - 1)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = ((1 <= ((((int)threadIdx.x) + 23) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 1792) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 23) % 29)) - 1)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = ((1 <= ((((int)threadIdx.x) + 20) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 2688) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 20) % 29)) - 1)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 128) {
    PaddedInput_shared[(((int)threadIdx.x) + 3584)] = ((1 <= ((((int)threadIdx.x) + 17) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 3584) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 17) % 29)) - 1)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 384) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 14) * 1152) + ((((int)threadIdx.x) / 3) * 9)) + (((int)threadIdx.x) % 3)) + 3)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 3)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[((((int)threadIdx.x) / 7) * 3)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 1)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 1)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 16)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 2)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((1 <= (((int)threadIdx.x) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + ((((int)threadIdx.x) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + (((int)threadIdx.x) % 29)) + 27)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((1 <= ((((int)threadIdx.x) + 26) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 896) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 26) % 29)) + 27)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = ((1 <= ((((int)threadIdx.x) + 23) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 1792) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 23) % 29)) + 27)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = ((1 <= ((((int)threadIdx.x) + 20) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 2688) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 20) % 29)) + 27)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 128) {
    PaddedInput_shared[(((int)threadIdx.x) + 3584)] = ((1 <= ((((int)threadIdx.x) + 17) % 29)) ? Input[((((((((int)blockIdx.x) / 14) * 100352) + (((((int)threadIdx.x) + 3584) / 29) * 784)) + ((((int)blockIdx.x) % 14) * 56)) + ((((int)threadIdx.x) + 17) % 29)) + 27)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 384) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 14) * 1152) + ((((int)threadIdx.x) / 3) * 9)) + (((int)threadIdx.x) % 3)) + 6)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 3)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[((((int)threadIdx.x) / 7) * 3)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 1)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 1)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 29) + ((((int)threadIdx.x) % 7) * 2)) + 16)] * kernel_shared[(((((int)threadIdx.x) / 7) * 3) + 2)]));
  compute[(((((((int)blockIdx.x) / 14) * 25088) + ((((int)threadIdx.x) / 7) * 196)) + ((((int)blockIdx.x) % 14) * 14)) + (((int)threadIdx.x) % 7))] = max(DepthwiseConv2d[0], 0.000000e+00f);
  compute[((((((((int)blockIdx.x) / 14) * 25088) + ((((int)threadIdx.x) / 7) * 196)) + ((((int)blockIdx.x) % 14) * 14)) + (((int)threadIdx.x) % 7)) + 7)] = max(DepthwiseConv2d[1], 0.000000e+00f);
}


