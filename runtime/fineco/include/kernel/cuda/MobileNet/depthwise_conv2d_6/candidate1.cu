
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(784) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[4];
  __shared__ float PaddedInput_shared[12064];
  __shared__ float kernel_shared[96];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((int)threadIdx.x)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + ((((int)threadIdx.x) % 377) / 29)) + di_outer_outer)) && (1 <= (((int)threadIdx.x) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + ((((int)threadIdx.x) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + (((((int)threadIdx.x) % 377) / 29) * 28)) + (di_outer_outer * 28)) + (((int)threadIdx.x) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 784)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 30) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 1) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 784) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 30) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 1) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 1568)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 60) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 2) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 1568) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 60) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 2) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 2352)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 90) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 3) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 2352) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 90) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 3) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 3136)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 120) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 4) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 3136) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 120) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 4) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 3920)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 150) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 5) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 3920) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 150) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 5) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 4704)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 180) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 6) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 4704) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 180) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 6) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 5488)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 210) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 7) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 5488) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 210) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 7) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 6272)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 240) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 8) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 6272) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 240) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 8) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 7056)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 270) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 9) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 7056) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 270) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 9) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 7840)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 300) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 10) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 7840) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 300) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 10) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 8624)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 330) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 11) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 8624) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 330) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 11) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 9408)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 360) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 12) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 9408) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 360) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 12) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 10192)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 13) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 13) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 10192) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 13) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 13) % 29)) - 29)] : 0.000000e+00f);
    PaddedInput_shared[(((int)threadIdx.x) + 10976)] = (((1 <= ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.x) + 43) % 377) / 29)) + di_outer_outer)) && (1 <= ((((int)threadIdx.x) + 14) % 29))) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 10976) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 43) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 14) % 29)) - 29)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 304) {
      PaddedInput_shared[(((int)threadIdx.x) + 11760)] = ((1 <= ((((int)threadIdx.x) + 15) % 29)) ? Input[((((((((((int)blockIdx.x) >> 1) * 25088) + (((((int)threadIdx.x) + 11760) / 377) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + ((((((int)threadIdx.x) + 73) % 377) / 29) * 28)) + (di_outer_outer * 28)) + ((((int)threadIdx.x) + 15) % 29)) - 29)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 96) {
      kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 1) * 288) + ((((int)threadIdx.x) / 3) * 9)) + (di_outer_outer * 3)) + (((int)threadIdx.x) % 3))];
    }
    __syncthreads();
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[((((int)threadIdx.x) / 98) * 12)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 377)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 3)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 754)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 6)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 1131)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 9)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 1)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 378)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 4)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 755)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 7)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 1132)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 10)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 2)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 379)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 5)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 756)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 8)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 1508) + (((((int)threadIdx.x) % 98) / 14) * 58)) + ((((int)threadIdx.x) % 14) * 2)) + 1133)] * kernel_shared[(((((int)threadIdx.x) / 98) * 12) + 11)]));
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    compute[((((((((int)blockIdx.x) >> 1) * 6272) + ((((int)threadIdx.x) / 98) * 784)) + (i1_inner * 196)) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) % 98))] = max(DepthwiseConv2d[i1_inner], 0.000000e+00f);
  }
}


