
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate2(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[2];
  __shared__ float PaddedInput_shared[1664];
  __shared__ float kernel_shared[128];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((1 <= (((int)blockIdx.x) % 7)) && (1 <= (((int)threadIdx.x) % 13))) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = (((1 <= (((int)blockIdx.x) % 7)) && (1 <= ((((int)threadIdx.x) + 6) % 13))) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((1 <= (((int)blockIdx.x) % 7)) && (1 <= ((((int)threadIdx.x) + 12) % 13))) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13)) - 15)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = (((1 <= (((int)blockIdx.x) % 7)) && (1 <= ((((int)threadIdx.x) + 5) % 13))) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13)) - 15)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9))];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((1 <= (((int)blockIdx.x) % 7)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) - 14)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = ((1 <= (((int)blockIdx.x) % 7)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13)) - 14)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((1 <= (((int)blockIdx.x) % 7)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13)) - 14)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = ((1 <= (((int)blockIdx.x) % 7)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13)) - 14)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9)) + 1)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((1 <= (((int)blockIdx.x) % 7)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) - 13)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = ((1 <= (((int)blockIdx.x) % 7)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13)) - 13)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((1 <= (((int)blockIdx.x) % 7)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13)) - 13)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = ((1 <= (((int)blockIdx.x) % 7)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13)) - 13)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9)) + 2)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((1 <= (((int)threadIdx.x) % 13)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) - 1)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = ((1 <= ((((int)threadIdx.x) + 6) % 13)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13)) - 1)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((1 <= ((((int)threadIdx.x) + 12) % 13)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13)) - 1)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = ((1 <= ((((int)threadIdx.x) + 5) % 13)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13)) - 1)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9)) + 3)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = Input[(((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13))];
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = Input[(((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13))];
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = Input[(((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13))];
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = Input[(((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13))];
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9)) + 4)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 1)];
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13)) + 1)];
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13)) + 1)];
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13)) + 1)];
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9)) + 5)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = ((1 <= (((int)threadIdx.x) % 13)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 13)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = ((1 <= ((((int)threadIdx.x) + 6) % 13)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13)) + 13)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = ((1 <= ((((int)threadIdx.x) + 12) % 13)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13)) + 13)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = ((1 <= ((((int)threadIdx.x) + 5) % 13)) ? Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13)) + 13)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9)) + 6)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 14)];
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13)) + 14)];
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13)) + 14)];
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13)) + 14)];
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9)) + 7)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  __syncthreads();
  PaddedInput_shared[((int)threadIdx.x)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 13)) + 15)];
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 448) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 6) % 13)) + 15)];
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 896) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 12) % 13)) + 15)];
  if (((int)threadIdx.x) < 320) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = Input[((((((((int)blockIdx.x) / 7) * 25088) + (((((int)threadIdx.x) + 1344) / 13) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) + 5) % 13)) + 15)];
  }
  if (((int)threadIdx.x) < 128) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 1152) + (((int)threadIdx.x) * 9)) + 8)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 26) + ((((int)threadIdx.x) % 7) * 2)) + 13)] * kernel_shared[(((((int)threadIdx.x) / 7) * 2) + 1)]));
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    compute[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 7) * 98)) + (i1_inner * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7))] = max(DepthwiseConv2d[i1_inner], 0.000000e+00f);
  }
}


