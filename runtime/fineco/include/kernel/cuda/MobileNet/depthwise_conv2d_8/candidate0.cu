
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(14) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[1];
  __shared__ float PaddedInput_shared[90];
  __shared__ float kernel_shared[18];
  DepthwiseConv2d[0] = 0.000000e+00f;
  PaddedInput_shared[(((int)threadIdx.x) * 2)] = (((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) * 2) / 15))) && (1 <= ((((int)threadIdx.x) * 2) % 15))) ? Input[((((((((int)blockIdx.x) / 7) * 392) + ((((int)blockIdx.x) % 7) * 28)) + (((((int)threadIdx.x) * 2) / 15) * 14)) + ((((int)threadIdx.x) * 2) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) * 2) + 1)] = (((1 <= (((((int)blockIdx.x) % 7) * 2) + (((((int)threadIdx.x) * 2) + 1) / 15))) && (1 <= (((((int)threadIdx.x) * 2) + 1) % 15))) ? Input[((((((((int)blockIdx.x) / 7) * 392) + ((((int)blockIdx.x) % 7) * 28)) + ((((((int)threadIdx.x) * 2) + 1) / 15) * 14)) + (((((int)threadIdx.x) * 2) + 1) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) * 2) + 28)] = (((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((((int)threadIdx.x) * 2) + 28) % 45) / 15))) && (1 <= (((((int)threadIdx.x) * 2) + 13) % 15))) ? Input[(((((((((int)blockIdx.x) / 7) * 392) + ((((((int)threadIdx.x) * 2) + 28) / 45) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((((int)threadIdx.x) * 2) + 28) % 45) / 15) * 14)) + (((((int)threadIdx.x) * 2) + 13) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) * 2) + 29)] = (((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((((int)threadIdx.x) * 2) + 29) % 45) / 15))) && (1 <= (((((int)threadIdx.x) * 2) + 14) % 15))) ? Input[(((((((((int)blockIdx.x) / 7) * 392) + ((((((int)threadIdx.x) * 2) + 29) / 45) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((((int)threadIdx.x) * 2) + 29) % 45) / 15) * 14)) + (((((int)threadIdx.x) * 2) + 14) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) * 2) + 56)] = (((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((((int)threadIdx.x) * 2) + 11) % 45) / 15))) && (1 <= (((((int)threadIdx.x) * 2) + 11) % 15))) ? Input[(((((((((int)blockIdx.x) / 7) * 392) + ((((((int)threadIdx.x) * 2) + 56) / 45) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((((int)threadIdx.x) * 2) + 11) % 45) / 15) * 14)) + (((((int)threadIdx.x) * 2) + 11) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[((((int)threadIdx.x) * 2) + 57)] = (((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((((int)threadIdx.x) * 2) + 12) % 45) / 15))) && (1 <= (((((int)threadIdx.x) * 2) + 12) % 15))) ? Input[(((((((((int)blockIdx.x) / 7) * 392) + ((((((int)threadIdx.x) * 2) + 57) / 45) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((((int)threadIdx.x) * 2) + 12) % 45) / 15) * 14)) + (((((int)threadIdx.x) * 2) + 12) % 15)) - 15)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 3) {
    PaddedInput_shared[((((int)threadIdx.x) * 2) + 84)] = Input[(((((((((int)blockIdx.x) / 7) * 392) + ((((((int)threadIdx.x) * 2) + 84) / 45) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((((int)threadIdx.x) * 2) + 39) % 45) / 15) * 14)) + ((((int)threadIdx.x) * 2) + 9)) - 15)];
  }
  if (((int)threadIdx.x) < 3) {
    PaddedInput_shared[((((int)threadIdx.x) * 2) + 85)] = Input[(((((((((int)blockIdx.x) / 7) * 392) + ((((((int)threadIdx.x) * 2) + 85) / 45) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((((int)threadIdx.x) * 2) + 40) % 45) / 15) * 14)) + ((((int)threadIdx.x) * 2) + 10)) - 15)];
  }
  if (((int)threadIdx.x) < 6) {
    *(float3*)(kernel_shared + (((int)threadIdx.x) * 3)) = *(float3*)(kernel + (((((int)blockIdx.x) / 7) * 18) + (((int)threadIdx.x) * 3)));
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 9)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 1)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 2)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 3)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2)) + 16)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 4)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2)) + 17)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 5)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2)) + 30)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 6)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2)) + 31)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 7)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 45) + ((((int)threadIdx.x) % 7) * 2)) + 32)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 8)]));
  compute[(((((((int)blockIdx.x) / 7) * 98) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7))] = max(DepthwiseConv2d[0], 0.000000e+00f);
}


