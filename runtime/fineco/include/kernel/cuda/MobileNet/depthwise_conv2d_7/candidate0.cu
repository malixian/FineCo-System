
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(784) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[4];
  __shared__ float PaddedInput_shared[3136];
  __shared__ float kernel_shared[16];
  for (int c_outer_inner_init = 0; c_outer_inner_init < 2; ++c_outer_inner_init) {
    DepthwiseConv2d[c_outer_inner_init] = 0.000000e+00f;
    DepthwiseConv2d[(c_outer_inner_init + 2)] = 0.000000e+00f;
  }
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        PaddedInput_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + ((int)threadIdx.x))] = (((((1 <= (((((int)threadIdx.x) % 196) / 14) + di_outer_outer)) && ((((((int)threadIdx.x) % 196) / 14) + di_outer_outer) < 15)) && (1 <= (dj_outer_outer + (((int)threadIdx.x) % 14)))) && ((dj_outer_outer + (((int)threadIdx.x) % 14)) < 15)) ? Input[((((((((int)blockIdx.x) * 3136) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784)) + (di_outer_outer * 14)) + ((int)threadIdx.x)) + dj_outer_outer) - 15)] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 16) {
        kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) * 144) + (((int)threadIdx.x) * 9)) + (di_outer_outer * 3)) + dj_outer_outer)];
      }
      __syncthreads();
      for (int c_outer_inner = 0; c_outer_inner < 2; ++c_outer_inner) {
        DepthwiseConv2d[c_outer_inner] = (DepthwiseConv2d[c_outer_inner] + (PaddedInput_shared[((((((int)threadIdx.x) / 196) * 392) + (c_outer_inner * 196)) + (((int)threadIdx.x) % 196))] * kernel_shared[(((((int)threadIdx.x) / 196) * 2) + c_outer_inner)]));
        DepthwiseConv2d[(c_outer_inner + 2)] = (DepthwiseConv2d[(c_outer_inner + 2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 196) * 392) + (c_outer_inner * 196)) + (((int)threadIdx.x) % 196)) + 1568)] * kernel_shared[((((((int)threadIdx.x) / 196) * 2) + c_outer_inner) + 8)]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    compute[((((((int)blockIdx.x) * 3136) + ((((int)threadIdx.x) / 196) * 392)) + (i1_inner * 196)) + (((int)threadIdx.x) % 196))] = max(DepthwiseConv2d[i1_inner], 0.000000e+00f);
    compute[(((((((int)blockIdx.x) * 3136) + ((((int)threadIdx.x) / 196) * 392)) + (i1_inner * 196)) + (((int)threadIdx.x) % 196)) + 1568)] = max(DepthwiseConv2d[(i1_inner + 2)], 0.000000e+00f);
  }
}


