
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate2(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[32];
  __shared__ float pad_temp_shared[1792];
  __shared__ float kernel_shared[4096];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
    conv2d_nchw[yy_outer_inner_init] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 16)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 4)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 20)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 8)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 24)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 12)] = 0.000000e+00f;
    conv2d_nchw[(yy_outer_inner_init + 28)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 56) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 56) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = Input[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 56) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 56) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = Input[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 56) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 56) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = Input[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 56) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 56) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = Input[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 56) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 56) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = Input[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 56) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 56) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 15680)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = Input[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 56) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 56) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = Input[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 56) * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (((((int)threadIdx.x) % 56) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 21952)];
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 19; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + (((int)threadIdx.x) >> 5)) < 128) {
        kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 14) * 16384) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 896)) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          conv2d_nchw[yy_outer_inner] = (conv2d_nchw[yy_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 448) + (rc_inner * 56)) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 8)) + rc_inner)]));
          conv2d_nchw[(yy_outer_inner + 16)] = (conv2d_nchw[(yy_outer_inner + 16)] + (pad_temp_shared[((((rc_outer_inner * 448) + (rc_inner * 56)) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 8)) + rc_inner) + 2048)]));
          conv2d_nchw[(yy_outer_inner + 4)] = (conv2d_nchw[(yy_outer_inner + 4)] + (pad_temp_shared[((((rc_outer_inner * 448) + (rc_inner * 56)) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 8)) + rc_inner) + 32)]));
          conv2d_nchw[(yy_outer_inner + 20)] = (conv2d_nchw[(yy_outer_inner + 20)] + (pad_temp_shared[((((rc_outer_inner * 448) + (rc_inner * 56)) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 8)) + rc_inner) + 2080)]));
          conv2d_nchw[(yy_outer_inner + 8)] = (conv2d_nchw[(yy_outer_inner + 8)] + (pad_temp_shared[((((rc_outer_inner * 448) + (rc_inner * 56)) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 8)) + rc_inner) + 64)]));
          conv2d_nchw[(yy_outer_inner + 24)] = (conv2d_nchw[(yy_outer_inner + 24)] + (pad_temp_shared[((((rc_outer_inner * 448) + (rc_inner * 56)) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 8)) + rc_inner) + 2112)]));
          conv2d_nchw[(yy_outer_inner + 12)] = (conv2d_nchw[(yy_outer_inner + 12)] + (pad_temp_shared[((((rc_outer_inner * 448) + (rc_inner * 56)) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 8)) + rc_inner) + 96)]));
          conv2d_nchw[(yy_outer_inner + 28)] = (conv2d_nchw[(yy_outer_inner + 28)] + (pad_temp_shared[((((rc_outer_inner * 448) + (rc_inner * 56)) + (yy_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 8)) + rc_inner) + 2144)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
      compute[((((((((((int)blockIdx.x) / 14) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (i1_inner * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))] = max(conv2d_nchw[((i1_inner * 4) + i2_inner)], 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) / 14) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (i1_inner * 784)) + (((((int)blockIdx.x) % 14) >> 1) * 112)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 50176)] = max(conv2d_nchw[(((i1_inner * 4) + i2_inner) + 16)], 0.000000e+00f);
    }
  }
}


