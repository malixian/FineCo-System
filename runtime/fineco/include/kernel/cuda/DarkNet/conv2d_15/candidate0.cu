
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[2];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[1024];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    conv2d_nchw[ff_outer_inner_init] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      *(float4*)(pad_temp_shared + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 4))) = *(float4*)(data + (((rc_outer_outer * 6272) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784)) + (((int)threadIdx.x) * 4)));
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 49) + (((int)threadIdx.x) >> 2)) < 128) {
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
          kernel_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 392) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s)] = kernel[((((((int)blockIdx.x) * 8192) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 49) + (((int)threadIdx.x) >> 2)) >> 4) * 1024)) + (rc_outer_outer * 128)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 392) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 127))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 64; ++rc_inner) {
          conv2d_nchw[ff_outer_inner] = (conv2d_nchw[ff_outer_inner] + (pad_temp_shared[(((rc_outer_inner * 3136) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 256) + (ff_outer_inner * 128)) + (rc_outer_inner * 64)) + rc_inner)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    compute[((((((int)blockIdx.x) * 392) + ((((int)threadIdx.x) / 49) * 98)) + (i1_inner * 49)) + (((int)threadIdx.x) % 49))] = max((conv2d_nchw[i1_inner] + bias[(((((int)blockIdx.x) * 8) + ((((int)threadIdx.x) / 49) * 2)) + i1_inner)]), 0.000000e+00f);
  }
}


