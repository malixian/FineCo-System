
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) candidate0(float* __restrict__ tensor, float* __restrict__ data) {
  tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 2; ++rv0) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 7) * 28) + (rv0 * 14)) + ((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 7) * 2)) + rv1)]);
    }
  }
}


