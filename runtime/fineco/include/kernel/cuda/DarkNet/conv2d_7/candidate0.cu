
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[16];
  __shared__ float pad_temp_shared[2048];
  __shared__ float kernel_shared[8192];
  for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
    conv2d_nchw[yy_inner_init] = 0.000000e+00f;
    conv2d_nchw[(yy_inner_init + 2)] = 0.000000e+00f;
    conv2d_nchw[(yy_inner_init + 4)] = 0.000000e+00f;
    conv2d_nchw[(yy_inner_init + 6)] = 0.000000e+00f;
    conv2d_nchw[(yy_inner_init + 8)] = 0.000000e+00f;
    conv2d_nchw[(yy_inner_init + 10)] = 0.000000e+00f;
    conv2d_nchw[(yy_inner_init + 12)] = 0.000000e+00f;
    conv2d_nchw[(yy_inner_init + 14)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 128; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x))] = data[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1568) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (((((int)threadIdx.x) & 7) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3))];
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 128; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    *(float4*)(kernel_shared + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + ((((((int)blockIdx.x) / 98) * 8192) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64)) + (((int)threadIdx.x) * 4)));
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
    for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
      for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
        conv2d_nchw[yy_inner] = (conv2d_nchw[yy_inner] + (pad_temp_shared[(((rc_outer_inner * 128) + (rc_inner * 8)) + (yy_inner * 4))] * kernel_shared[(((((int)threadIdx.x) * 256) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[(yy_inner + 2)] = (conv2d_nchw[(yy_inner + 2)] + (pad_temp_shared[((((rc_outer_inner * 128) + (rc_inner * 8)) + (yy_inner * 4)) + 1)] * kernel_shared[(((((int)threadIdx.x) * 256) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[(yy_inner + 4)] = (conv2d_nchw[(yy_inner + 4)] + (pad_temp_shared[((((rc_outer_inner * 128) + (rc_inner * 8)) + (yy_inner * 4)) + 2)] * kernel_shared[(((((int)threadIdx.x) * 256) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[(yy_inner + 6)] = (conv2d_nchw[(yy_inner + 6)] + (pad_temp_shared[((((rc_outer_inner * 128) + (rc_inner * 8)) + (yy_inner * 4)) + 3)] * kernel_shared[(((((int)threadIdx.x) * 256) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[(yy_inner + 8)] = (conv2d_nchw[(yy_inner + 8)] + (pad_temp_shared[(((rc_outer_inner * 128) + (rc_inner * 8)) + (yy_inner * 4))] * kernel_shared[((((((int)threadIdx.x) * 256) + (rc_outer_inner * 16)) + rc_inner) + 4096)]));
        conv2d_nchw[(yy_inner + 10)] = (conv2d_nchw[(yy_inner + 10)] + (pad_temp_shared[((((rc_outer_inner * 128) + (rc_inner * 8)) + (yy_inner * 4)) + 1)] * kernel_shared[((((((int)threadIdx.x) * 256) + (rc_outer_inner * 16)) + rc_inner) + 4096)]));
        conv2d_nchw[(yy_inner + 12)] = (conv2d_nchw[(yy_inner + 12)] + (pad_temp_shared[((((rc_outer_inner * 128) + (rc_inner * 8)) + (yy_inner * 4)) + 2)] * kernel_shared[((((((int)threadIdx.x) * 256) + (rc_outer_inner * 16)) + rc_inner) + 4096)]));
        conv2d_nchw[(yy_inner + 14)] = (conv2d_nchw[(yy_inner + 14)] + (pad_temp_shared[((((rc_outer_inner * 128) + (rc_inner * 8)) + (yy_inner * 4)) + 3)] * kernel_shared[((((((int)threadIdx.x) * 256) + (rc_outer_inner * 16)) + rc_inner) + 4096)]));
      }
    }
  }
  for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
    compute[((((((((int)blockIdx.x) / 98) * 25088) + (((int)threadIdx.x) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4))] = max((conv2d_nchw[i2_inner] + bias[(((((int)blockIdx.x) / 98) * 32) + ((int)threadIdx.x))]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 98) * 25088) + (((int)threadIdx.x) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 1)] = max((conv2d_nchw[(i2_inner + 2)] + bias[(((((int)blockIdx.x) / 98) * 32) + ((int)threadIdx.x))]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 98) * 25088) + (((int)threadIdx.x) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 2)] = max((conv2d_nchw[(i2_inner + 4)] + bias[(((((int)blockIdx.x) / 98) * 32) + ((int)threadIdx.x))]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 98) * 25088) + (((int)threadIdx.x) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 3)] = max((conv2d_nchw[(i2_inner + 6)] + bias[(((((int)blockIdx.x) / 98) * 32) + ((int)threadIdx.x))]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 98) * 25088) + (((int)threadIdx.x) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 12544)] = max((conv2d_nchw[(i2_inner + 8)] + bias[((((((int)blockIdx.x) / 98) * 32) + ((int)threadIdx.x)) + 16)]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 98) * 25088) + (((int)threadIdx.x) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 12545)] = max((conv2d_nchw[(i2_inner + 10)] + bias[((((((int)blockIdx.x) / 98) * 32) + ((int)threadIdx.x)) + 16)]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 98) * 25088) + (((int)threadIdx.x) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 12546)] = max((conv2d_nchw[(i2_inner + 12)] + bias[((((((int)blockIdx.x) / 98) * 32) + ((int)threadIdx.x)) + 16)]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 98) * 25088) + (((int)threadIdx.x) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 12547)] = max((conv2d_nchw[(i2_inner + 14)] + bias[((((((int)blockIdx.x) / 98) * 32) + ((int)threadIdx.x)) + 16)]), 0.000000e+00f);
  }
}


