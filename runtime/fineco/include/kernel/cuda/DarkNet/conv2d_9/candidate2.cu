
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[14];
  __shared__ float pad_temp_shared[128];
  __shared__ float kernel_shared[2304];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 7; ++xx_outer_inner_init) {
      conv2d_nchw[((ff_outer_inner_init * 7) + xx_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 128) {
      pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) & 63) >> 4))) && ((((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) & 63) >> 4)) < 15)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? data[((((((rc_outer_outer * 392) + ((((int)threadIdx.x) >> 6) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((((int)threadIdx.x) & 63) >> 4) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[(((((((int)blockIdx.x) / 7) * 294912) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) >> 1)) / 9) * 2304)) + (rc_outer_outer * 18)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x)) % 18))];
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
            for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
              conv2d_nchw[((ff_outer_inner * 7) + xx_outer_inner)] = (conv2d_nchw[((ff_outer_inner * 7) + xx_outer_inner)] + (pad_temp_shared[((((((rc_inner * 64) + (((((int)threadIdx.x) & 3) >> 1) * 16)) + (ry_inner * 16)) + ((((int)threadIdx.x) & 1) * 7)) + xx_outer_inner) + rx_inner)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 36) + (ff_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
            }
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i3_inner = 0; i3_inner < 7; ++i3_inner) {
      compute[(((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) >> 2) * 392)) + (i1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) & 3) * 7)) + i3_inner)] = max((conv2d_nchw[((i1_inner * 7) + i3_inner)] + bias[((((((int)blockIdx.x) / 7) * 128) + ((((int)threadIdx.x) >> 2) * 2)) + i1_inner)]), 0.000000e+00f);
    }
  }
}


