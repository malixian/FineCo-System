
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[64];
  __shared__ float pad_temp_shared[2304];
  __shared__ float kernel_shared[1536];
  for (int ff_inner_init = 0; ff_inner_init < 16; ++ff_inner_init) {
    conv2d_nchw[(ff_inner_init * 4)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 4) + 1)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 4) + 2)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 4) + 3)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      pad_temp_shared[((int)threadIdx.x)] = ((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((int)threadIdx.x) >> 4))) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[(((((((rc_outer_outer * 100352) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 256)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 16) % 18))) && ((((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 16) % 18)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 256) / 288) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + ((((((int)threadIdx.x) >> 4) + 16) % 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 512)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 14) % 18))) && ((((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 14) % 18)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 512) / 288) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + ((((((int)threadIdx.x) >> 4) + 14) % 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 768)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 12) % 18))) && ((((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 12) % 18)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 768) / 288) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + ((((((int)threadIdx.x) >> 4) + 12) % 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 10) % 18))) && ((((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 10) % 18)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1024) / 288) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + ((((((int)threadIdx.x) >> 4) + 10) % 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1280)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 8) % 18))) && ((((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 8) % 18)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1280) / 288) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + ((((((int)threadIdx.x) >> 4) + 8) % 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1536)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 6) % 18))) && ((((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 6) % 18)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1536) / 288) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + ((((((int)threadIdx.x) >> 4) + 6) % 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 4) % 18))) && ((((((int)blockIdx.x) / 7) * 16) + (((((int)threadIdx.x) >> 4) + 4) % 18)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1792) / 288) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + ((((((int)threadIdx.x) >> 4) + 4) % 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 2048)] = (((((((((int)blockIdx.x) / 7) * 16) + ((((int)threadIdx.x) >> 4) + 2)) < 113) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2048) / 288) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((int)threadIdx.x) >> 4) + 2) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)threadIdx.x) / 24) * 288) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)threadIdx.x) + 256) / 24) * 288) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)threadIdx.x) + 512) / 24) * 288) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 768)] = kernel[((((((((int)threadIdx.x) / 24) * 288) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + rx_outer_outer) + 9216)];
      kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((((((int)threadIdx.x) + 1024) / 24) * 288) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[((((((((int)threadIdx.x) + 1280) / 24) * 288) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + rx_outer_outer)];
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
        for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
          for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
            for (int ff_inner = 0; ff_inner < 16; ++ff_inner) {
              conv2d_nchw[(ff_inner * 4)] = (conv2d_nchw[(ff_inner * 4)] + (pad_temp_shared[(((((rc_outer_inner * 1152) + (rc_inner * 288)) + (((((int)threadIdx.x) & 63) >> 4) * 64)) + (ry_inner * 16)) + (((int)threadIdx.x) & 15))] * kernel_shared[((((((((int)threadIdx.x) >> 6) * 384) + (ff_inner * 24)) + (rc_outer_inner * 12)) + (rc_inner * 3)) + ry_inner)]));
              conv2d_nchw[((ff_inner * 4) + 1)] = (conv2d_nchw[((ff_inner * 4) + 1)] + (pad_temp_shared[((((((rc_outer_inner * 1152) + (rc_inner * 288)) + (((((int)threadIdx.x) & 63) >> 4) * 64)) + (ry_inner * 16)) + (((int)threadIdx.x) & 15)) + 16)] * kernel_shared[((((((((int)threadIdx.x) >> 6) * 384) + (ff_inner * 24)) + (rc_outer_inner * 12)) + (rc_inner * 3)) + ry_inner)]));
              conv2d_nchw[((ff_inner * 4) + 2)] = (conv2d_nchw[((ff_inner * 4) + 2)] + (pad_temp_shared[((((((rc_outer_inner * 1152) + (rc_inner * 288)) + (((((int)threadIdx.x) & 63) >> 4) * 64)) + (ry_inner * 16)) + (((int)threadIdx.x) & 15)) + 32)] * kernel_shared[((((((((int)threadIdx.x) >> 6) * 384) + (ff_inner * 24)) + (rc_outer_inner * 12)) + (rc_inner * 3)) + ry_inner)]));
              conv2d_nchw[((ff_inner * 4) + 3)] = (conv2d_nchw[((ff_inner * 4) + 3)] + (pad_temp_shared[((((((rc_outer_inner * 1152) + (rc_inner * 288)) + (((((int)threadIdx.x) & 63) >> 4) * 64)) + (ry_inner * 16)) + (((int)threadIdx.x) & 15)) + 48)] * kernel_shared[((((((((int)threadIdx.x) >> 6) * 384) + (ff_inner * 24)) + (rc_outer_inner * 12)) + (rc_inner * 3)) + ry_inner)]));
            }
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 16; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
      compute[((((((((((int)threadIdx.x) >> 6) * 200704) + (i1_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((int)threadIdx.x) & 63) >> 4) * 448)) + (i2_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 15))] = max((conv2d_nchw[((i1_inner * 4) + i2_inner)] + bias[(((((int)threadIdx.x) >> 6) * 16) + i1_inner)]), 0.000000e+00f);
    }
  }
}


