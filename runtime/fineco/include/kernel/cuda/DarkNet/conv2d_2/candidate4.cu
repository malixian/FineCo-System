
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[32];
  __shared__ float pad_temp_shared[2880];
  __shared__ float kernel_shared[9216];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[16] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[17] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[18] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[19] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[20] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[21] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[22] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[23] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[24] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[25] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[26] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[27] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[28] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[29] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[30] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  conv2d_nchw[31] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((int)threadIdx.x) % 180) / 10))) && ((((((int)blockIdx.x) / 14) * 16) + ((((int)threadIdx.x) % 180) / 10)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + (((int)threadIdx.x) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + (((int)threadIdx.x) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((int)threadIdx.x) % 180) / 10) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + (((int)threadIdx.x) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 256)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 38) % 90) / 5))) && ((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 38) % 90) / 5)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 6) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 6) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 256) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 38) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 6) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 512)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 76) % 90) / 5))) && ((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 76) % 90) / 5)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 2) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 2) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 512) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 76) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 2) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 768)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 24) % 90) / 5))) && ((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 24) % 90) / 5)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 8) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 8) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 768) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 24) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 8) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 62) % 90) / 5))) && ((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 62) % 90) / 5)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 4) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 4) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 1024) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 62) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 4) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + (((((int)threadIdx.x) / 10) + 2) % 18))) && ((((((int)blockIdx.x) / 14) * 16) + (((((int)threadIdx.x) / 10) + 2) % 18)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + (((int)threadIdx.x) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + (((int)threadIdx.x) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 1280) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + ((((((int)threadIdx.x) / 10) + 2) % 18) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + (((int)threadIdx.x) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 48) % 90) / 5))) && ((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 48) % 90) / 5)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 6) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 6) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 1536) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 48) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 6) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 86) % 90) / 5))) && ((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 86) % 90) / 5)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 2) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 2) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 1792) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 86) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 2) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2048)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 34) % 90) / 5))) && ((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 34) % 90) / 5)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 8) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 8) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2048) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 34) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 8) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2304)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 72) % 90) / 5))) && ((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 72) % 90) / 5)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 4) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 4) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2304) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 72) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 4) % 10)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2560)] = (((((1 <= (((((int)blockIdx.x) / 14) * 16) + (((((int)threadIdx.x) / 10) + 4) % 18))) && ((((((int)blockIdx.x) / 14) * 16) + (((((int)threadIdx.x) / 10) + 4) % 18)) < 113)) && (1 <= (((((int)blockIdx.x) % 14) * 8) + (((int)threadIdx.x) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + (((int)threadIdx.x) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2560) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + ((((((int)threadIdx.x) / 10) + 4) % 18) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + (((int)threadIdx.x) % 10)) - 113)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[(((int)threadIdx.x) + 2816)] = (((((((((int)blockIdx.x) / 14) * 16) + ((((((int)threadIdx.x) >> 1) + 58) % 90) / 5)) < 113) && (1 <= (((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 6) % 10)))) && ((((((int)blockIdx.x) % 14) * 8) + ((((int)threadIdx.x) + 6) % 10)) < 113)) ? data[(((((((rc_outer_outer * 200704) + (((((int)threadIdx.x) + 2816) / 180) * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((((int)threadIdx.x) >> 1) + 58) % 90) / 5) * 112)) + ((((int)blockIdx.x) % 14) * 8)) + ((((int)threadIdx.x) + 6) % 10)) - 113)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 144) * 288) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) + 256) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) + 512) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) + 768) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) + 1024) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 16) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) + 1280) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) + 1536) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) + 1792) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) + 2048) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((int)threadIdx.x) / 144) * 288) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144)) + 4608)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) + 2560) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)threadIdx.x) + 2816) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) + 3072) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)threadIdx.x) + 3328) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 16) % 144))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) + 3584) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)threadIdx.x) + 3840) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) + 4096) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)threadIdx.x) + 4352) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((int)threadIdx.x) / 144) * 288) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144)) + 9216)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[(((((((int)threadIdx.x) + 4864) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)threadIdx.x) + 5120) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[(((((((int)threadIdx.x) + 5376) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)threadIdx.x) + 5632) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 16) % 144))];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[(((((((int)threadIdx.x) + 5888) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)threadIdx.x) + 6144) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[(((((((int)threadIdx.x) + 6400) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)threadIdx.x) + 6656) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[(((((((int)threadIdx.x) / 144) * 288) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144)) + 13824)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)threadIdx.x) + 7168) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[(((((((int)threadIdx.x) + 7424) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)threadIdx.x) + 7680) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[(((((((int)threadIdx.x) + 7936) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 16) % 144))];
    kernel_shared[(((int)threadIdx.x) + 8192)] = kernel[(((((((int)threadIdx.x) + 8192) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 8448)] = kernel[(((((((int)threadIdx.x) + 8448) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 8704)] = kernel[(((((((int)threadIdx.x) + 8704) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 8960)] = kernel[(((((((int)threadIdx.x) + 8960) / 144) * 288) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
            conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
            conv2d_nchw[16] = (conv2d_nchw[16] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
            conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 10)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
            conv2d_nchw[17] = (conv2d_nchw[17] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 90)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
            conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 144)]));
            conv2d_nchw[18] = (conv2d_nchw[18] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 144)]));
            conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 10)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 144)]));
            conv2d_nchw[19] = (conv2d_nchw[19] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 90)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 144)]));
            conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 288)]));
            conv2d_nchw[20] = (conv2d_nchw[20] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 288)]));
            conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 10)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 288)]));
            conv2d_nchw[21] = (conv2d_nchw[21] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 90)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 288)]));
            conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
            conv2d_nchw[22] = (conv2d_nchw[22] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
            conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 10)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
            conv2d_nchw[23] = (conv2d_nchw[23] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 90)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
            conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 576)]));
            conv2d_nchw[24] = (conv2d_nchw[24] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 576)]));
            conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 10)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 576)]));
            conv2d_nchw[25] = (conv2d_nchw[25] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 90)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 576)]));
            conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 720)]));
            conv2d_nchw[26] = (conv2d_nchw[26] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 720)]));
            conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 10)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 720)]));
            conv2d_nchw[27] = (conv2d_nchw[27] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 90)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 720)]));
            conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 864)]));
            conv2d_nchw[28] = (conv2d_nchw[28] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 864)]));
            conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 10)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 864)]));
            conv2d_nchw[29] = (conv2d_nchw[29] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 90)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 864)]));
            conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 1008)]));
            conv2d_nchw[30] = (conv2d_nchw[30] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 1008)]));
            conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 10)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 1008)]));
            conv2d_nchw[31] = (conv2d_nchw[31] + (pad_temp_shared[(((((((rc_outer_inner * 720) + (rc_inner * 180)) + (((((int)threadIdx.x) & 31) >> 3) * 20)) + (ry_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 90)] * kernel_shared[(((((((((int)threadIdx.x) >> 5) * 1152) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 1008)]));
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[((((((((((int)threadIdx.x) >> 5) * 100352) + (i1_inner * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((int)threadIdx.x) & 31) >> 3) * 224)) + (i2_inner * 112)) + ((((int)blockIdx.x) % 14) * 8)) + (((int)threadIdx.x) & 7))] = max((conv2d_nchw[((i1_inner * 2) + i2_inner)] + bias[(((((int)threadIdx.x) >> 5) * 8) + i1_inner)]), 0.000000e+00f);
      compute[(((((((((((int)threadIdx.x) >> 5) * 100352) + (i1_inner * 12544)) + ((((int)blockIdx.x) / 14) * 1792)) + (((((int)threadIdx.x) & 31) >> 3) * 224)) + (i2_inner * 112)) + ((((int)blockIdx.x) % 14) * 8)) + (((int)threadIdx.x) & 7)) + 896)] = max((conv2d_nchw[(((i1_inner * 2) + i2_inner) + 16)] + bias[(((((int)threadIdx.x) >> 5) * 8) + i1_inner)]), 0.000000e+00f);
    }
  }
}


