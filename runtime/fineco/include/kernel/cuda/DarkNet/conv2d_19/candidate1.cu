
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[10];
  __shared__ float pad_temp_shared[3136];
  __shared__ float kernel_shared[2560];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((rc_outer_outer * 3136) + ((int)threadIdx.x))];
    pad_temp_shared[(((int)threadIdx.x) + 196)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 196)];
    pad_temp_shared[(((int)threadIdx.x) + 392)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 392)];
    pad_temp_shared[(((int)threadIdx.x) + 588)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 588)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 980)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 980)];
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 1176)];
    pad_temp_shared[(((int)threadIdx.x) + 1372)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 1372)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 1764)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 1764)];
    pad_temp_shared[(((int)threadIdx.x) + 1960)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 1960)];
    pad_temp_shared[(((int)threadIdx.x) + 2156)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 2156)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 2548)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 2548)];
    pad_temp_shared[(((int)threadIdx.x) + 2744)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 2744)];
    pad_temp_shared[(((int)threadIdx.x) + 2940)] = data[(((rc_outer_outer * 3136) + ((int)threadIdx.x)) + 2940)];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) * 40960) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 196)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 196) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 4) & 63))];
    kernel_shared[(((int)threadIdx.x) + 392)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 392) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 8) & 63))];
    kernel_shared[(((int)threadIdx.x) + 588)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 588) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 12) & 63))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 784) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 16) & 63))];
    kernel_shared[(((int)threadIdx.x) + 980)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 980) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 20) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1176)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 1176) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 24) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1372)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 1372) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 28) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 1568) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1764)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 1764) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 36) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1960)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 1960) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63))];
    kernel_shared[(((int)threadIdx.x) + 2156)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 2156) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 44) & 63))];
    kernel_shared[(((int)threadIdx.x) + 2352)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 2352) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63))];
    if (((int)threadIdx.x) < 12) {
      kernel_shared[(((int)threadIdx.x) + 2548)] = kernel[((((((int)blockIdx.x) * 40960) + (((((int)threadIdx.x) + 2548) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 52))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 64)]));
        conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 128)]));
        conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 192)]));
        conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 256)]));
        conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 320)]));
        conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 384)]));
        conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 448)]));
        conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 512)]));
        conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 49)) + (((int)threadIdx.x) % 49))] * kernel_shared[(((((((int)threadIdx.x) / 49) * 640) + (rc_outer_inner * 16)) + rc_inner) + 576)]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 10; ++i1_inner) {
    compute[((((((int)blockIdx.x) * 1960) + ((((int)threadIdx.x) / 49) * 490)) + (i1_inner * 49)) + (((int)threadIdx.x) % 49))] = max((conv2d_nchw[i1_inner] + bias[(((((int)blockIdx.x) * 40) + ((((int)threadIdx.x) / 49) * 10)) + i1_inner)]), 0.000000e+00f);
  }
}


