
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(896) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[14];
  __shared__ float pad_temp_shared[3584];
  __shared__ float kernel_shared[3072];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      pad_temp_shared[((int)threadIdx.x)] = (((((1 <= ((((((int)blockIdx.x) & 3) >> 1) * 14) + ((((int)threadIdx.x) % 224) / 14))) && (((((((int)blockIdx.x) & 3) >> 1) * 14) + ((((int)threadIdx.x) % 224) / 14)) < 29)) && (1 <= ((((((int)blockIdx.x) & 1) * 14) + rx_outer_outer) + (((int)threadIdx.x) % 14)))) && (((((((int)blockIdx.x) & 1) * 14) + rx_outer_outer) + (((int)threadIdx.x) % 14)) < 29)) ? data[((((((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 224) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 224) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + rx_outer_outer) + (((int)threadIdx.x) % 14)) - 29)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 896)] = (((((1 <= ((((((int)blockIdx.x) & 3) >> 1) * 14) + ((((int)threadIdx.x) % 224) / 14))) && (((((((int)blockIdx.x) & 3) >> 1) * 14) + ((((int)threadIdx.x) % 224) / 14)) < 29)) && (1 <= ((((((int)blockIdx.x) & 1) * 14) + rx_outer_outer) + (((int)threadIdx.x) % 14)))) && (((((((int)blockIdx.x) & 1) * 14) + rx_outer_outer) + (((int)threadIdx.x) % 14)) < 29)) ? data[((((((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 224) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 224) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + rx_outer_outer) + (((int)threadIdx.x) % 14)) + 3107)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((((1 <= ((((((int)blockIdx.x) & 3) >> 1) * 14) + ((((int)threadIdx.x) % 224) / 14))) && (((((((int)blockIdx.x) & 3) >> 1) * 14) + ((((int)threadIdx.x) % 224) / 14)) < 29)) && (1 <= ((((((int)blockIdx.x) & 1) * 14) + rx_outer_outer) + (((int)threadIdx.x) % 14)))) && (((((((int)blockIdx.x) & 1) * 14) + rx_outer_outer) + (((int)threadIdx.x) % 14)) < 29)) ? data[((((((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 224) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 224) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + rx_outer_outer) + (((int)threadIdx.x) % 14)) + 6243)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 2688)] = (((((1 <= ((((((int)blockIdx.x) & 3) >> 1) * 14) + ((((int)threadIdx.x) % 224) / 14))) && (((((((int)blockIdx.x) & 3) >> 1) * 14) + ((((int)threadIdx.x) % 224) / 14)) < 29)) && (1 <= ((((((int)blockIdx.x) & 1) * 14) + rx_outer_outer) + (((int)threadIdx.x) % 14)))) && (((((((int)blockIdx.x) & 1) * 14) + rx_outer_outer) + (((int)threadIdx.x) % 14)) < 29)) ? data[((((((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 224) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 224) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + rx_outer_outer) + (((int)threadIdx.x) % 14)) + 9379)] : 0.000000e+00f);
      kernel_shared[((int)threadIdx.x)] = kernel[((((((((int)blockIdx.x) >> 2) * 73728) + ((((int)threadIdx.x) / 48) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 896) / 48) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 32) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 1792) / 48) * 1152)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 16) % 48) * 3)) + rx_outer_outer)];
      if (((int)threadIdx.x) < 384) {
        kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((((int)blockIdx.x) >> 2) * 73728) + ((((int)threadIdx.x) / 48) * 1152)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer) + 64512)];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3))]));
        conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1536)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3))]));
        conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1536)]));
        conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3))]));
        conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1536)]));
        conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[(((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3))]));
        conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1536)]));
        conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[(((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3))]));
        conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1536)]));
        conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[(((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3))]));
        conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1536)]));
        conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[(((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3))]));
        conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1536)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1)]));
        conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1537)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1)]));
        conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1537)]));
        conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1)]));
        conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1537)]));
        conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1)]));
        conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1537)]));
        conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1)]));
        conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1537)]));
        conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1)]));
        conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1537)]));
        conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1)]));
        conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1537)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 2)]));
        conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1538)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 2)]));
        conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1538)]));
        conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 2)]));
        conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1538)]));
        conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 2)]));
        conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1538)]));
        conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 2)]));
        conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1538)]));
        conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 2)]));
        conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1538)]));
        conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 2)]));
        conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((((rc_outer_inner * 224) + (((((int)threadIdx.x) % 28) / 14) * 98)) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_outer_inner * 3)) + 1538)]));
      }
    }
  }
  for (int i2_inner = 0; i2_inner < 7; ++i2_inner) {
    compute[((((((((((int)blockIdx.x) >> 2) * 50176) + ((((int)threadIdx.x) / 28) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))] = max((conv2d_nchw[i2_inner] + bias[(((((int)blockIdx.x) >> 2) * 64) + (((int)threadIdx.x) / 28))]), 0.000000e+00f);
    compute[(((((((((((int)blockIdx.x) >> 2) * 50176) + ((((int)threadIdx.x) / 28) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (i2_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 25088)] = max((conv2d_nchw[(i2_inner + 7)] + bias[((((((int)blockIdx.x) >> 2) * 64) + (((int)threadIdx.x) / 28)) + 32)]), 0.000000e+00f);
  }
}


