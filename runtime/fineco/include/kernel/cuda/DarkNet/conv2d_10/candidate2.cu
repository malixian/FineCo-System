
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[8];
  __shared__ float pad_temp_shared[3584];
  __shared__ float kernel_shared[8192];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 7840)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 10976)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 14112)];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 15680)];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 17248)];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 20384)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 21952)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 23520)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 224) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 448) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 672) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 1120) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 1344) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 1568) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 2016) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 2240) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 2464) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 2912) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 3136) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 3360) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 3808) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 4032) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4256)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 4256) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)) + 17920)];
    kernel_shared[(((int)threadIdx.x) + 4704)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 4704) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 4928)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 4928) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5152)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 5152) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)) + 21504)];
    kernel_shared[(((int)threadIdx.x) + 5600)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 5600) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 5824)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 5824) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6048)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 6048) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)) + 25088)];
    kernel_shared[(((int)threadIdx.x) + 6496)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 6496) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6720)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 6720) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 6944)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 6944) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[((((((((int)blockIdx.x) / 7) * 32768) + ((((int)threadIdx.x) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 7392)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 7392) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7616)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 7616) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 7840)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (((((int)threadIdx.x) + 7840) >> 7) * 512)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    if (((int)threadIdx.x) < 128) {
      kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[(((((((int)blockIdx.x) / 7) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 32256)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((rc_outer_inner * 224) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8))]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8))]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((rc_outer_inner * 224) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4096)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4096)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4097)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4097)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4098)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4098)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4099)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4099)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4100)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4100)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4101)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4101)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4102)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4102)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4103)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4103)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((rc_outer_inner * 224) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 128)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 128)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((rc_outer_inner * 224) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4224)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4224)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 129)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 129)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4225)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4225)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 130)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 130)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4226)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4226)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 131)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 131)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4227)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4227)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 132)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 132)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4228)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4228)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 133)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 133)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4229)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4229)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 134)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 134)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4230)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4230)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 135)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 135)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4231)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((rc_outer_inner * 224) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 14) * 256) + (rc_outer_inner * 8)) + 4231)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    compute[(((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (i1_inner * 196)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))] = max((conv2d_nchw[i1_inner] + bias[((((((int)blockIdx.x) / 7) * 64) + ((((int)threadIdx.x) / 14) * 2)) + i1_inner)]), 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (i1_inner * 196)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 98)] = max((conv2d_nchw[(i1_inner + 2)] + bias[((((((int)blockIdx.x) / 7) * 64) + ((((int)threadIdx.x) / 14) * 2)) + i1_inner)]), 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (i1_inner * 196)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 6272)] = max((conv2d_nchw[(i1_inner + 4)] + bias[(((((((int)blockIdx.x) / 7) * 64) + ((((int)threadIdx.x) / 14) * 2)) + i1_inner) + 32)]), 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (i1_inner * 196)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 6370)] = max((conv2d_nchw[(i1_inner + 6)] + bias[(((((((int)blockIdx.x) / 7) * 64) + ((((int)threadIdx.x) / 14) * 2)) + i1_inner) + 32)]), 0.000000e+00f);
  }
}


