
#include <hip/hip_runtime.h>


#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(121) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[12];
  __shared__ float pad_temp_shared[2601];
  __shared__ float kernel_shared[1452];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 3; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((1 <= ((((((int)blockIdx.x) % 25) / 5) * 22) + (((int)threadIdx.x) / 102))) && (1 <= (((((int)blockIdx.x) % 5) * 22) + ((((int)threadIdx.x) % 51) >> 1)))) && ((((((int)blockIdx.x) % 5) * 22) + ((((int)threadIdx.x) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + ((((int)threadIdx.x) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + (((int)threadIdx.x) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 121)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 19) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 19) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 121) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 19) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 242)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 38) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 38) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 242) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 38) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 363)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 6) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 6) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 363) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 6) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 484)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 25) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 25) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 484) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 25) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 605)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 44) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 44) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 605) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 44) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 726)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 12) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 12) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 726) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 12) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 847)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 31) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 31) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 847) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 31) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 968)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 50) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 50) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 968) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 50) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1089)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 18) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 18) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 1089) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 18) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1210)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 37) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 37) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 1210) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 37) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1331)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 5) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 5) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 1331) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 5) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1452)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 24) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 24) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 1452) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 24) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1573)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 43) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 43) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 1573) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 43) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1694)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 11) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 11) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 1694) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 11) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1815)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 30) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 30) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 1815) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 30) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1936)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 49) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 49) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 1936) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 49) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2057)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 17) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 17) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 2057) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 17) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2178)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 36) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 36) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 2178) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 36) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2299)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 4) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 4) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 2299) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 4) % 51)) - 450)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2420)] = (((1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 23) % 51) >> 1))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 23) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 2420) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 23) % 51)) - 450)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 60) {
      pad_temp_shared[(((int)threadIdx.x) + 2541)] = ((((((((((int)blockIdx.x) % 25) / 5) * 22) + ((((int)threadIdx.x) + 2541) / 102)) < 113) && (1 <= (((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 42) % 51) >> 1)))) && ((((((int)blockIdx.x) % 5) * 22) + (((((int)threadIdx.x) + 42) % 51) >> 1)) < 113)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 25) / 5) * 9856)) + (((((int)threadIdx.x) + 2541) / 51) * 224)) + ((((int)blockIdx.x) % 5) * 44)) + ((((int)threadIdx.x) + 42) % 51)) - 450)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 121)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 363)];
    kernel_shared[(((int)threadIdx.x) + 242)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 726)];
    kernel_shared[(((int)threadIdx.x) + 363)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 1089)];
    kernel_shared[(((int)threadIdx.x) + 484)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 1452)];
    kernel_shared[(((int)threadIdx.x) + 605)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 1815)];
    kernel_shared[(((int)threadIdx.x) + 726)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 2178)];
    kernel_shared[(((int)threadIdx.x) + 847)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 2541)];
    kernel_shared[(((int)threadIdx.x) + 968)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 2904)];
    kernel_shared[(((int)threadIdx.x) + 1089)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 3267)];
    kernel_shared[(((int)threadIdx.x) + 1210)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 3630)];
    kernel_shared[(((int)threadIdx.x) + 1331)] = kernel[(((((((int)blockIdx.x) / 25) * 4356) + (rc_outer_outer * 121)) + ((int)threadIdx.x)) + 3993)];
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 11; ++ry_inner) {
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[(ry_inner * 11)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 484)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 968)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 121)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 605)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 1089)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 242)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 726)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 1210)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 363)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 847)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4))] * kernel_shared[((ry_inner * 11) + 1331)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 1)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 485)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 969)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 122)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 606)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 1090)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 243)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 727)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 1211)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 364)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 848)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 1)] * kernel_shared[((ry_inner * 11) + 1332)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 2)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 486)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 970)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 123)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 607)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 1091)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 244)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 728)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 1212)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 365)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 849)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 2)] * kernel_shared[((ry_inner * 11) + 1333)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 3)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 487)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 971)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 124)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 608)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 1092)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 245)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 729)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 1213)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 366)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 850)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 3)] * kernel_shared[((ry_inner * 11) + 1334)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 4)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 488)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 972)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 125)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 609)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 1093)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 246)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 730)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 1214)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 367)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 851)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 4)] * kernel_shared[((ry_inner * 11) + 1335)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 5)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 489)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 973)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 126)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 610)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 1094)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 247)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 731)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 1215)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 368)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 852)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 5)] * kernel_shared[((ry_inner * 11) + 1336)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 6)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 490)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 974)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 127)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 611)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 1095)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 248)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 732)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 1216)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 369)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 853)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 6)] * kernel_shared[((ry_inner * 11) + 1337)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 7)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 491)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 975)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 128)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 612)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 1096)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 249)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 733)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 1217)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 370)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 854)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 7)] * kernel_shared[((ry_inner * 11) + 1338)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 8)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 492)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 976)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 129)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 613)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 1097)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 250)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 734)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 1218)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 371)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 855)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 8)] * kernel_shared[((ry_inner * 11) + 1339)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 9)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 493)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 977)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 130)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 614)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 1098)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 251)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 735)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 1219)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 372)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 856)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 9)] * kernel_shared[((ry_inner * 11) + 1340)]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 10)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 494)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 978)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 131)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 615)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 1099)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 252)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 736)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 1220)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 373)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 857)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[(((((((int)threadIdx.x) / 11) * 204) + (ry_inner * 51)) + ((((int)threadIdx.x) % 11) * 4)) + 10)] * kernel_shared[((ry_inner * 11) + 1341)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    compute[(((((((((int)blockIdx.x) / 25) * 36300) + (i1_inner * 3025)) + (((((int)blockIdx.x) % 25) / 5) * 605)) + ((((int)threadIdx.x) / 11) * 55)) + ((((int)blockIdx.x) % 5) * 11)) + (((int)threadIdx.x) % 11))] = max((conv2d_nchw[i1_inner] + bias[(((((int)blockIdx.x) / 25) * 12) + i1_inner)]), 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) / 25) * 36300) + (i1_inner * 3025)) + (((((int)blockIdx.x) % 25) / 5) * 605)) + ((((int)threadIdx.x) / 11) * 55)) + ((((int)blockIdx.x) % 5) * 11)) + (((int)threadIdx.x) % 11)) + 12100)] = max((conv2d_nchw[(i1_inner + 4)] + bias[((((((int)blockIdx.x) / 25) * 12) + i1_inner) + 4)]), 0.000000e+00f);
    compute[((((((((((int)blockIdx.x) / 25) * 36300) + (i1_inner * 3025)) + (((((int)blockIdx.x) % 25) / 5) * 605)) + ((((int)threadIdx.x) / 11) * 55)) + ((((int)blockIdx.x) % 5) * 11)) + (((int)threadIdx.x) % 11)) + 24200)] = max((conv2d_nchw[(i1_inner + 8)] + bias[((((((int)blockIdx.x) / 25) * 12) + i1_inner) + 8)]), 0.000000e+00f);
  }
}


