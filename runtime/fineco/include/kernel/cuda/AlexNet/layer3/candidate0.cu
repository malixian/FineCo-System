
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(96) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[36];
  __shared__ float pad_temp_shared[312];
  __shared__ float kernel_shared[5120];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  conv2d_nchw[16] = 0.000000e+00f;
  conv2d_nchw[17] = 0.000000e+00f;
  conv2d_nchw[18] = 0.000000e+00f;
  conv2d_nchw[19] = 0.000000e+00f;
  conv2d_nchw[20] = 0.000000e+00f;
  conv2d_nchw[27] = 0.000000e+00f;
  conv2d_nchw[28] = 0.000000e+00f;
  conv2d_nchw[29] = 0.000000e+00f;
  conv2d_nchw[21] = 0.000000e+00f;
  conv2d_nchw[22] = 0.000000e+00f;
  conv2d_nchw[23] = 0.000000e+00f;
  conv2d_nchw[30] = 0.000000e+00f;
  conv2d_nchw[31] = 0.000000e+00f;
  conv2d_nchw[32] = 0.000000e+00f;
  conv2d_nchw[24] = 0.000000e+00f;
  conv2d_nchw[25] = 0.000000e+00f;
  conv2d_nchw[26] = 0.000000e+00f;
  conv2d_nchw[33] = 0.000000e+00f;
  conv2d_nchw[34] = 0.000000e+00f;
  conv2d_nchw[35] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 6; ++rc_outer_outer) {
    for (int ry_outer_outer = 0; ry_outer_outer < 5; ++ry_outer_outer) {
      __syncthreads();
      pad_temp_shared[((int)threadIdx.x)] = (((((2 <= ((((((int)blockIdx.x) / 3) * 3) + ((((int)threadIdx.x) % 39) / 13)) + ry_outer_outer)) && (((((((int)blockIdx.x) / 3) * 3) + ((((int)threadIdx.x) % 39) / 13)) + ry_outer_outer) < 29)) && (2 <= (((((int)blockIdx.x) % 3) * 9) + (((int)threadIdx.x) % 13)))) && ((((((int)blockIdx.x) % 3) * 9) + (((int)threadIdx.x) % 13)) < 29)) ? data[((((((((rc_outer_outer * 5832) + ((((int)threadIdx.x) / 39) * 729)) + ((((int)blockIdx.x) / 3) * 81)) + (((((int)threadIdx.x) % 39) / 13) * 27)) + (ry_outer_outer * 27)) + ((((int)blockIdx.x) % 3) * 9)) + (((int)threadIdx.x) % 13)) - 56)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 96)] = (((((2 <= ((((((int)blockIdx.x) / 3) * 3) + (((((int)threadIdx.x) + 18) % 39) / 13)) + ry_outer_outer)) && (((((((int)blockIdx.x) / 3) * 3) + (((((int)threadIdx.x) + 18) % 39) / 13)) + ry_outer_outer) < 29)) && (2 <= (((((int)blockIdx.x) % 3) * 9) + ((((int)threadIdx.x) + 5) % 13)))) && ((((((int)blockIdx.x) % 3) * 9) + ((((int)threadIdx.x) + 5) % 13)) < 29)) ? data[((((((((rc_outer_outer * 5832) + (((((int)threadIdx.x) + 96) / 39) * 729)) + ((((int)blockIdx.x) / 3) * 81)) + ((((((int)threadIdx.x) + 18) % 39) / 13) * 27)) + (ry_outer_outer * 27)) + ((((int)blockIdx.x) % 3) * 9)) + ((((int)threadIdx.x) + 5) % 13)) - 56)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 192)] = (((((2 <= ((((((int)blockIdx.x) / 3) * 3) + (((((int)threadIdx.x) + 36) % 39) / 13)) + ry_outer_outer)) && (((((((int)blockIdx.x) / 3) * 3) + (((((int)threadIdx.x) + 36) % 39) / 13)) + ry_outer_outer) < 29)) && (2 <= (((((int)blockIdx.x) % 3) * 9) + ((((int)threadIdx.x) + 10) % 13)))) && ((((((int)blockIdx.x) % 3) * 9) + ((((int)threadIdx.x) + 10) % 13)) < 29)) ? data[((((((((rc_outer_outer * 5832) + (((((int)threadIdx.x) + 192) / 39) * 729)) + ((((int)blockIdx.x) / 3) * 81)) + ((((((int)threadIdx.x) + 36) % 39) / 13) * 27)) + (ry_outer_outer * 27)) + ((((int)blockIdx.x) % 3) * 9)) + ((((int)threadIdx.x) + 10) % 13)) - 56)] : 0.000000e+00f);
      if (((int)threadIdx.x) < 24) {
        pad_temp_shared[(((int)threadIdx.x) + 288)] = (((((2 <= ((((((int)blockIdx.x) / 3) * 3) + (((((int)threadIdx.x) + 15) % 39) / 13)) + ry_outer_outer)) && (((((((int)blockIdx.x) / 3) * 3) + (((((int)threadIdx.x) + 15) % 39) / 13)) + ry_outer_outer) < 29)) && (2 <= (((((int)blockIdx.x) % 3) * 9) + ((((int)threadIdx.x) + 2) % 13)))) && ((((((int)blockIdx.x) % 3) * 9) + ((((int)threadIdx.x) + 2) % 13)) < 29)) ? data[((((((((rc_outer_outer * 5832) + (((((int)threadIdx.x) + 288) / 39) * 729)) + ((((int)blockIdx.x) / 3) * 81)) + ((((((int)threadIdx.x) + 15) % 39) / 13) * 27)) + (ry_outer_outer * 27)) + ((((int)blockIdx.x) % 3) * 9)) + ((((int)threadIdx.x) + 2) % 13)) - 56)] : 0.000000e+00f);
      }
      kernel_shared[((int)threadIdx.x)] = kernel[((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5))];
      kernel_shared[(((int)threadIdx.x) + 96)] = kernel[(((((((((int)threadIdx.x) + 96) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 192)] = kernel[(((((((((int)threadIdx.x) + 192) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 288)] = kernel[(((((((((int)threadIdx.x) + 288) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 384)] = kernel[(((((((((int)threadIdx.x) + 384) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 480)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 14400)];
      kernel_shared[(((int)threadIdx.x) + 576)] = kernel[(((((((((int)threadIdx.x) + 576) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((((int)threadIdx.x) + 672) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((((int)threadIdx.x) + 768) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 864)] = kernel[(((((((((int)threadIdx.x) + 864) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 960)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 28800)];
      kernel_shared[(((int)threadIdx.x) + 1056)] = kernel[(((((((((int)threadIdx.x) + 1056) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[(((((((((int)threadIdx.x) + 1152) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 1248)] = kernel[(((((((((int)threadIdx.x) + 1248) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((((int)threadIdx.x) + 1344) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 1440)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 43200)];
      kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((((int)threadIdx.x) + 1536) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 1632)] = kernel[(((((((((int)threadIdx.x) + 1632) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 1728)] = kernel[(((((((((int)threadIdx.x) + 1728) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 1824)] = kernel[(((((((((int)threadIdx.x) + 1824) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 57600)];
      kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((((int)threadIdx.x) + 2016) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 2112)] = kernel[(((((((((int)threadIdx.x) + 2112) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 2208)] = kernel[(((((((((int)threadIdx.x) + 2208) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((((int)threadIdx.x) + 2304) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 2400)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 72000)];
      kernel_shared[(((int)threadIdx.x) + 2496)] = kernel[(((((((((int)threadIdx.x) + 2496) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 2592)] = kernel[(((((((((int)threadIdx.x) + 2592) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((((int)threadIdx.x) + 2688) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 2784)] = kernel[(((((((((int)threadIdx.x) + 2784) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 2880)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 86400)];
      kernel_shared[(((int)threadIdx.x) + 2976)] = kernel[(((((((((int)threadIdx.x) + 2976) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((((int)threadIdx.x) + 3072) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 3168)] = kernel[(((((((((int)threadIdx.x) + 3168) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 3264)] = kernel[(((((((((int)threadIdx.x) + 3264) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 100800)];
      kernel_shared[(((int)threadIdx.x) + 3456)] = kernel[(((((((((int)threadIdx.x) + 3456) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 3552)] = kernel[(((((((((int)threadIdx.x) + 3552) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 3648)] = kernel[(((((((((int)threadIdx.x) + 3648) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 3744)] = kernel[(((((((((int)threadIdx.x) + 3744) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 115200)];
      kernel_shared[(((int)threadIdx.x) + 3936)] = kernel[(((((((((int)threadIdx.x) + 3936) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[(((((((((int)threadIdx.x) + 4032) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4128)] = kernel[(((((((((int)threadIdx.x) + 4128) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4224)] = kernel[(((((((((int)threadIdx.x) + 4224) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4320)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 129600)];
      kernel_shared[(((int)threadIdx.x) + 4416)] = kernel[(((((((((int)threadIdx.x) + 4416) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4512)] = kernel[(((((((((int)threadIdx.x) + 4512) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((((int)threadIdx.x) + 4608) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4704)] = kernel[(((((((((int)threadIdx.x) + 4704) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 24) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 4) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4800)] = kernel[(((((((((int)threadIdx.x) / 40) * 1200) + (rc_outer_outer * 200)) + (((((int)threadIdx.x) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + (((int)threadIdx.x) % 5)) + 144000)];
      kernel_shared[(((int)threadIdx.x) + 4896)] = kernel[(((((((((int)threadIdx.x) + 4896) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 16) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 1) % 5))];
      kernel_shared[(((int)threadIdx.x) + 4992)] = kernel[(((((((((int)threadIdx.x) + 4992) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 32) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 2) % 5))];
      if (((int)threadIdx.x) < 32) {
        kernel_shared[(((int)threadIdx.x) + 5088)] = kernel[(((((((((int)threadIdx.x) + 5088) / 40) * 1200) + (rc_outer_outer * 200)) + ((((((int)threadIdx.x) + 8) % 40) / 5) * 25)) + (ry_outer_outer * 5)) + ((((int)threadIdx.x) + 3) % 5))];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
        for (int rx_outer_inner = 0; rx_outer_inner < 5; ++rx_outer_inner) {
          for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
            for (int yy_outer_inner = 0; yy_outer_inner < 3; ++yy_outer_inner) {
              conv2d_nchw[((ff_outer_inner * 18) + (yy_outer_inner * 3))] = (conv2d_nchw[((ff_outer_inner * 18) + (yy_outer_inner * 3))] + (pad_temp_shared[((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 1)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 2)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 9)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 9)] + (pad_temp_shared[((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 40)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 10)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 40)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 11)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 11)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 40)]));
              conv2d_nchw[((ff_outer_inner * 18) + (yy_outer_inner * 3))] = (conv2d_nchw[((ff_outer_inner * 18) + (yy_outer_inner * 3))] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 39)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 5)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 1)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 40)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 5)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 2)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 41)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 5)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 9)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 9)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 39)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 45)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 10)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 40)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 45)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 11)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 11)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 41)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 45)]));
              conv2d_nchw[((ff_outer_inner * 18) + (yy_outer_inner * 3))] = (conv2d_nchw[((ff_outer_inner * 18) + (yy_outer_inner * 3))] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 78)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 10)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 1)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 79)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 10)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 2)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 80)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 10)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 9)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 9)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 78)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 50)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 10)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 79)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 50)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 11)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 11)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 80)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 50)]));
              conv2d_nchw[((ff_outer_inner * 18) + (yy_outer_inner * 3))] = (conv2d_nchw[((ff_outer_inner * 18) + (yy_outer_inner * 3))] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 117)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 15)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 1)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 118)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 15)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 2)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 119)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 15)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 9)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 9)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 117)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 55)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 10)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 118)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 55)]));
              conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 11)] = (conv2d_nchw[(((ff_outer_inner * 18) + (yy_outer_inner * 3)) + 11)] + (pad_temp_shared[(((((rc_outer_inner * 156) + (yy_outer_inner * 13)) + ((((int)threadIdx.x) % 3) * 3)) + rx_outer_inner) + 119)] * kernel_shared[((((((((int)threadIdx.x) / 3) * 160) + (ff_outer_inner * 80)) + (rc_outer_inner * 20)) + rx_outer_inner) + 55)]));
            }
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 3; ++i2_inner) {
      for (int i3_inner = 0; i3_inner < 3; ++i3_inner) {
        compute[((((((((((int)threadIdx.x) / 3) * 2916) + (i1_inner * 729)) + ((((int)blockIdx.x) / 3) * 81)) + (i2_inner * 27)) + ((((int)blockIdx.x) % 3) * 9)) + ((((int)threadIdx.x) % 3) * 3)) + i3_inner)] = max((conv2d_nchw[(((i1_inner * 9) + (i2_inner * 3)) + i3_inner)] + bias[(((((int)threadIdx.x) / 3) * 4) + i1_inner)]), 0.000000e+00f);
      }
    }
  }
}


