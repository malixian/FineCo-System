
#include <hip/hip_runtime.h>
#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(8) candidate0(float* __restrict__ tensor, float* __restrict__ data) {
  tensor[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      tensor[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = max(tensor[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], data[((((((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 2)) / 9) * 169) + (((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 1)) % 18) / 3) * 26)) + (rv0 * 13)) + ((((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 6) * 2)) + rv1)]);
    }
  }
}


