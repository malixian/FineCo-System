
#include <hip/hip_runtime.h>
#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) candidate0(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[1];
  __shared__ float data_shared[128];
  __shared__ float weight_shared[8192];
  T_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 16; ++k_outer_outer) {
    __syncthreads();
    *(float2*)(data_shared + (((int)threadIdx.x) * 2)) = *(float2*)(data + ((k_outer_outer * 128) + (((int)threadIdx.x) * 2)));
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(weight_shared + ((ax0_ax1_fused_outer_outer * 256) + (((int)threadIdx.x) * 4))) = *(float4*)(weight + (((((((int)blockIdx.x) * 131072) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)threadIdx.x) >> 5) * 2048)) + (k_outer_outer * 128)) + ((((int)threadIdx.x) & 31) * 4)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 16; ++k_outer_inner) {
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[(k_outer_inner * 8)] * weight_shared[((((int)threadIdx.x) * 128) + (k_outer_inner * 8))]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 8) + 1)] * weight_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 8)) + 1)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 8) + 2)] * weight_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 8)) + 2)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 8) + 3)] * weight_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 8)) + 3)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 8) + 4)] * weight_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 8)) + 4)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 8) + 5)] * weight_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 8)) + 5)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 8) + 6)] * weight_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 8)) + 6)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 8) + 7)] * weight_shared[(((((int)threadIdx.x) * 128) + (k_outer_inner * 8)) + 7)]));
    }
  }
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}


