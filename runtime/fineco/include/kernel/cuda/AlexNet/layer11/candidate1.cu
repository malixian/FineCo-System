
#include <hip/hip_runtime.h>
#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(4) candidate1(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[1];
  __shared__ float data_shared[1024];
  __shared__ float weight_shared[4096];
  T_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    *(float4*)(data_shared + (((int)threadIdx.x) * 4)) = *(float4*)(data + ((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 16)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 16));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 32)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 32));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 48)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 48));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 64)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 64));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 80)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 80));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 96)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 96));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 112)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 112));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 128)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 128));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 144)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 144));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 160)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 160));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 176)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 176));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 192)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 192));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 208)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 208));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 224)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 224));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 240)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 240));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 256));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 272)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 272));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 288)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 288));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 304)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 304));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 320)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 320));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 336)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 336));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 352)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 352));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 368)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 368));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 384)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 384));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 400)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 400));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 416)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 416));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 432)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 432));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 448)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 448));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 464)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 464));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 480)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 480));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 496)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 496));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 512));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 528)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 528));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 544)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 544));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 560)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 560));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 576)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 576));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 592)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 592));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 608)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 608));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 624)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 624));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 640)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 640));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 656)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 656));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 672)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 672));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 688)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 688));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 704)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 704));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 720)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 720));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 736)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 736));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 752)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 752));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 768));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 784)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 784));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 800)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 800));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 816)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 816));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 832)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 832));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 848)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 848));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 864)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 864));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 880)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 880));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 896)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 896));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 912)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 912));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 928)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 928));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 944)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 944));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 960)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 960));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 976)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 976));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 992)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 992));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 1008)) = *(float4*)(data + (((k_outer_outer * 1024) + (((int)threadIdx.x) * 4)) + 1008));
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 256; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(weight_shared + ((ax0_ax1_fused_outer_outer * 16) + (((int)threadIdx.x) * 4))) = *(float4*)(weight + (((((((int)blockIdx.x) * 8192) + ((ax0_ax1_fused_outer_outer >> 6) * 2048)) + (k_outer_outer * 1024)) + ((ax0_ax1_fused_outer_outer & 63) * 16)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 64; ++k_outer_inner) {
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[(k_outer_inner * 16)] * weight_shared[((((int)threadIdx.x) * 1024) + (k_outer_inner * 16))]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 1)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 1)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 2)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 2)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 3)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 3)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 4)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 4)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 5)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 5)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 6)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 6)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 7)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 7)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 8)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 8)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 9)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 9)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 10)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 10)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 11)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 11)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 12)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 12)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 13)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 13)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 14)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 14)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 16) + 15)] * weight_shared[(((((int)threadIdx.x) * 1024) + (k_outer_inner * 16)) + 15)]));
    }
  }
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}


