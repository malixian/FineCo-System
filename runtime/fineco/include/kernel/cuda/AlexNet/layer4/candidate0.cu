
#include <hip/hip_runtime.h>
#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) candidate0(float* __restrict__ tensor, float* __restrict__ data) {
  tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], data[((((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 169) * 729) + (((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 169) / 13) * 54)) + (rv0 * 27)) + ((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 13) * 2)) + rv1)]);
    }
  }
}


