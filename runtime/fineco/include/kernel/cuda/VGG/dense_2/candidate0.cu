
#include <hip/hip_runtime.h>


#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate0(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[1];
  __shared__ float data_shared[64];
  __shared__ float weight_shared[8192];
  T_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      data_shared[((int)threadIdx.x)] = data[((k_outer_outer * 64) + ((int)threadIdx.x))];
    }
    *(float4*)(weight_shared + (((int)threadIdx.x) * 4)) = *(float4*)(weight + ((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 32768));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 65536));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 98304));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 131072));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 163840));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 196608));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 229376));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4096)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 262144));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4608)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 294912));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5120)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 327680));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5632)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 360448));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6144)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 393216));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6656)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 425984));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7168)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 458752));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7680)) = *(float4*)(weight + (((((((int)blockIdx.x) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + (k_outer_outer * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 491520));
    __syncthreads();
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[0] * weight_shared[(((int)threadIdx.x) * 64)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[1] * weight_shared[((((int)threadIdx.x) * 64) + 1)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[2] * weight_shared[((((int)threadIdx.x) * 64) + 2)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[3] * weight_shared[((((int)threadIdx.x) * 64) + 3)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[4] * weight_shared[((((int)threadIdx.x) * 64) + 4)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[5] * weight_shared[((((int)threadIdx.x) * 64) + 5)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[6] * weight_shared[((((int)threadIdx.x) * 64) + 6)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[7] * weight_shared[((((int)threadIdx.x) * 64) + 7)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[8] * weight_shared[((((int)threadIdx.x) * 64) + 8)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[9] * weight_shared[((((int)threadIdx.x) * 64) + 9)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[10] * weight_shared[((((int)threadIdx.x) * 64) + 10)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[11] * weight_shared[((((int)threadIdx.x) * 64) + 11)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[12] * weight_shared[((((int)threadIdx.x) * 64) + 12)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[13] * weight_shared[((((int)threadIdx.x) * 64) + 13)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[14] * weight_shared[((((int)threadIdx.x) * 64) + 14)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[15] * weight_shared[((((int)threadIdx.x) * 64) + 15)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[16] * weight_shared[((((int)threadIdx.x) * 64) + 16)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[17] * weight_shared[((((int)threadIdx.x) * 64) + 17)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[18] * weight_shared[((((int)threadIdx.x) * 64) + 18)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[19] * weight_shared[((((int)threadIdx.x) * 64) + 19)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[20] * weight_shared[((((int)threadIdx.x) * 64) + 20)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[21] * weight_shared[((((int)threadIdx.x) * 64) + 21)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[22] * weight_shared[((((int)threadIdx.x) * 64) + 22)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[23] * weight_shared[((((int)threadIdx.x) * 64) + 23)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[24] * weight_shared[((((int)threadIdx.x) * 64) + 24)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[25] * weight_shared[((((int)threadIdx.x) * 64) + 25)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[26] * weight_shared[((((int)threadIdx.x) * 64) + 26)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[27] * weight_shared[((((int)threadIdx.x) * 64) + 27)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[28] * weight_shared[((((int)threadIdx.x) * 64) + 28)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[29] * weight_shared[((((int)threadIdx.x) * 64) + 29)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[30] * weight_shared[((((int)threadIdx.x) * 64) + 30)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[31] * weight_shared[((((int)threadIdx.x) * 64) + 31)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[32] * weight_shared[((((int)threadIdx.x) * 64) + 32)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[33] * weight_shared[((((int)threadIdx.x) * 64) + 33)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[34] * weight_shared[((((int)threadIdx.x) * 64) + 34)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[35] * weight_shared[((((int)threadIdx.x) * 64) + 35)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[36] * weight_shared[((((int)threadIdx.x) * 64) + 36)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[37] * weight_shared[((((int)threadIdx.x) * 64) + 37)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[38] * weight_shared[((((int)threadIdx.x) * 64) + 38)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[39] * weight_shared[((((int)threadIdx.x) * 64) + 39)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[40] * weight_shared[((((int)threadIdx.x) * 64) + 40)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[41] * weight_shared[((((int)threadIdx.x) * 64) + 41)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[42] * weight_shared[((((int)threadIdx.x) * 64) + 42)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[43] * weight_shared[((((int)threadIdx.x) * 64) + 43)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[44] * weight_shared[((((int)threadIdx.x) * 64) + 44)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[45] * weight_shared[((((int)threadIdx.x) * 64) + 45)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[46] * weight_shared[((((int)threadIdx.x) * 64) + 46)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[47] * weight_shared[((((int)threadIdx.x) * 64) + 47)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[48] * weight_shared[((((int)threadIdx.x) * 64) + 48)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[49] * weight_shared[((((int)threadIdx.x) * 64) + 49)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[50] * weight_shared[((((int)threadIdx.x) * 64) + 50)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[51] * weight_shared[((((int)threadIdx.x) * 64) + 51)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[52] * weight_shared[((((int)threadIdx.x) * 64) + 52)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[53] * weight_shared[((((int)threadIdx.x) * 64) + 53)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[54] * weight_shared[((((int)threadIdx.x) * 64) + 54)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[55] * weight_shared[((((int)threadIdx.x) * 64) + 55)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[56] * weight_shared[((((int)threadIdx.x) * 64) + 56)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[57] * weight_shared[((((int)threadIdx.x) * 64) + 57)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[58] * weight_shared[((((int)threadIdx.x) * 64) + 58)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[59] * weight_shared[((((int)threadIdx.x) * 64) + 59)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[60] * weight_shared[((((int)threadIdx.x) * 64) + 60)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[61] * weight_shared[((((int)threadIdx.x) * 64) + 61)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[62] * weight_shared[((((int)threadIdx.x) * 64) + 62)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[63] * weight_shared[((((int)threadIdx.x) * 64) + 63)]));
  }
  compute[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))]);
}


