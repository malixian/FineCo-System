
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[32];
  __shared__ float pad_temp_shared[200];
  __shared__ float kernel_shared[4608];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int yy_c_inner_init = 0; yy_c_inner_init < 2; ++yy_c_inner_init) {
      conv2d_nchw_local[((ff_c_outer_inner_init * 2) + yy_c_inner_init)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + yy_c_inner_init) + 4)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + yy_c_inner_init) + 8)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + yy_c_inner_init) + 12)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + yy_c_inner_init) + 16)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + yy_c_inner_init) + 20)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + yy_c_inner_init) + 24)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 2) + yy_c_inner_init) + 28)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 200) {
      pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) / 7) * 8) + ((((int)threadIdx.x) % 100) / 10))) && ((((((int)blockIdx.x) / 7) * 8) + ((((int)threadIdx.x) % 100) / 10)) < 57)) && (1 <= (((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) % 10)))) && ((((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) % 10)) < 57)) ? data[(((((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 100) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) % 100) / 10) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) % 10)) - 57)] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x))] = kernel[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + (((int)threadIdx.x) >> 1)) / 9) * 1152) + (rc_outer_outer * 18)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) % 18))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
          for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
            for (int yy_c_inner = 0; yy_c_inner < 2; ++yy_c_inner) {
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_inner)] + (pad_temp_shared[(((((rc_outer_inner * 100) + (yy_c_inner * 10)) + (ry_outer_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 36) + (ff_c_outer_inner * 18)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 100) + (yy_c_inner * 10)) + (ry_outer_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 20)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 36) + (ff_c_outer_inner * 18)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 8)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 100) + (yy_c_inner * 10)) + (ry_outer_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 36) + (ff_c_outer_inner * 18)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 12)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 12)] + (pad_temp_shared[((((((rc_outer_inner * 100) + (yy_c_inner * 10)) + (ry_outer_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 60)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 36) + (ff_c_outer_inner * 18)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 100) + (yy_c_inner * 10)) + (ry_outer_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((((int)threadIdx.x) >> 3) * 36) + (ff_c_outer_inner * 18)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 20)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 100) + (yy_c_inner * 10)) + (ry_outer_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 20)] * kernel_shared[(((((((((int)threadIdx.x) >> 3) * 36) + (ff_c_outer_inner * 18)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 24)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 100) + (yy_c_inner * 10)) + (ry_outer_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[(((((((((int)threadIdx.x) >> 3) * 36) + (ff_c_outer_inner * 18)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 28)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_inner) + 28)] + (pad_temp_shared[((((((rc_outer_inner * 100) + (yy_c_inner * 10)) + (ry_outer_inner * 10)) + rx_inner) + (((int)threadIdx.x) & 7)) + 60)] * kernel_shared[(((((((((int)threadIdx.x) >> 3) * 36) + (ff_c_outer_inner * 18)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[(((((((((int)threadIdx.x) >> 3) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 112)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 4)];
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 224)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 8)];
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 336)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 12)];
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 401408)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 16)];
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 401520)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 20)];
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 401632)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 24)];
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 6272) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 401744)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 28)];
    }
  }
}


