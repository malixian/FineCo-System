
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[112];
  __shared__ float pad_temp_shared[2320];
  __shared__ float kernel_shared[2304];
  for (int yy_c_inner_init = 0; yy_c_inner_init < 2; ++yy_c_inner_init) {
    for (int xx_c_inner_init = 0; xx_c_inner_init < 7; ++xx_c_inner_init) {
      conv2d_nchw_local[((yy_c_inner_init * 7) + xx_c_inner_init)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_inner_init * 7) + xx_c_inner_init) + 14)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_inner_init * 7) + xx_c_inner_init) + 28)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_inner_init * 7) + xx_c_inner_init) + 42)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_inner_init * 7) + xx_c_inner_init) + 56)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_inner_init * 7) + xx_c_inner_init) + 70)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_inner_init * 7) + xx_c_inner_init) + 84)] = 0.000000e+00f;
      conv2d_nchw_local[(((yy_c_inner_init * 7) + xx_c_inner_init) + 98)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((1 <= (((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) / 58))) && (1 <= (((int)threadIdx.x) % 58))) && ((((int)threadIdx.x) % 58) < 57)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) % 7) * 448)) + ((((int)threadIdx.x) / 58) * 56)) + (((int)threadIdx.x) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 256)] = (((1 <= ((((int)threadIdx.x) + 24) % 58)) && (((((int)threadIdx.x) + 24) % 58) < 57)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) % 7) * 448)) + (((((int)threadIdx.x) + 256) / 58) * 56)) + ((((int)threadIdx.x) + 24) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 512)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 256) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 256) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 48) % 58))) && (((((int)threadIdx.x) + 48) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 512) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 256) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 48) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 768)] = (((1 <= ((((int)threadIdx.x) + 14) % 58)) && (((((int)threadIdx.x) + 14) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 768) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 94) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 14) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 222) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 222) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 38) % 58))) && (((((int)threadIdx.x) + 38) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1024) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 222) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 38) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = (((1 <= ((((int)threadIdx.x) + 4) % 58)) && (((((int)threadIdx.x) + 4) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1280) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 60) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 4) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = (((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 188) % 290) / 29))) && ((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 188) % 290) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 28) % 58))) && (((((int)threadIdx.x) + 28) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1536) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 188) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 28) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = ((((1 <= (((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 26) % 290) / 29))) && (1 <= ((((int)threadIdx.x) + 52) % 58))) && (((((int)threadIdx.x) + 52) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1792) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 26) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 52) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2048)] = (((((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 154) % 290) / 29)) < 57) && (1 <= ((((int)threadIdx.x) + 18) % 58))) && (((((int)threadIdx.x) + 18) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 2048) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 154) % 290) / 29) * 56)) + ((((int)threadIdx.x) + 18) % 58)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 16) {
      pad_temp_shared[(((int)threadIdx.x) + 2304)] = ((((((((int)blockIdx.x) % 7) * 8) + ((((((int)threadIdx.x) >> 1) + 282) % 290) / 29)) < 57) && (((int)threadIdx.x) < 15)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 2304) / 580) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 282) % 290) / 29) * 56)) + (((int)threadIdx.x) + 42)) - 57)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + ((((int)threadIdx.x) / 36) * 2304)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 256) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 4) % 36))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 512) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 768) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 12) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 1024) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 16) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 1280) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 20) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 1536) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 24) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 1792) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 28) % 36))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 2048) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 32) % 36))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
            for (int yy_c_inner = 0; yy_c_inner < 2; ++yy_c_inner) {
              for (int xx_c_inner = 0; xx_c_inner < 7; ++xx_c_inner) {
                conv2d_nchw_local[((yy_c_inner * 7) + xx_c_inner)] = (conv2d_nchw_local[((yy_c_inner * 7) + xx_c_inner)] + (pad_temp_shared[(((((((rc_outer_inner * 1160) + (rc_inner * 580)) + (yy_c_inner * 58)) + (ry_inner * 58)) + ((((int)threadIdx.x) & 7) * 7)) + xx_c_inner) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 36) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
                conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 14)] = (conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 14)] + (pad_temp_shared[((((((((rc_outer_inner * 1160) + (rc_inner * 580)) + (yy_c_inner * 58)) + (ry_inner * 58)) + ((((int)threadIdx.x) & 7) * 7)) + xx_c_inner) + rx_outer_inner) + 116)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 36) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
                conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 28)] = (conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 28)] + (pad_temp_shared[((((((((rc_outer_inner * 1160) + (rc_inner * 580)) + (yy_c_inner * 58)) + (ry_inner * 58)) + ((((int)threadIdx.x) & 7) * 7)) + xx_c_inner) + rx_outer_inner) + 232)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 36) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
                conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 42)] = (conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 42)] + (pad_temp_shared[((((((((rc_outer_inner * 1160) + (rc_inner * 580)) + (yy_c_inner * 58)) + (ry_inner * 58)) + ((((int)threadIdx.x) & 7) * 7)) + xx_c_inner) + rx_outer_inner) + 348)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 36) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
                conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 56)] = (conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 56)] + (pad_temp_shared[(((((((rc_outer_inner * 1160) + (rc_inner * 580)) + (yy_c_inner * 58)) + (ry_inner * 58)) + ((((int)threadIdx.x) & 7) * 7)) + xx_c_inner) + rx_outer_inner)] * kernel_shared[(((((((((int)threadIdx.x) >> 3) * 36) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 1152)]));
                conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 70)] = (conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 70)] + (pad_temp_shared[((((((((rc_outer_inner * 1160) + (rc_inner * 580)) + (yy_c_inner * 58)) + (ry_inner * 58)) + ((((int)threadIdx.x) & 7) * 7)) + xx_c_inner) + rx_outer_inner) + 116)] * kernel_shared[(((((((((int)threadIdx.x) >> 3) * 36) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 1152)]));
                conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 84)] = (conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 84)] + (pad_temp_shared[((((((((rc_outer_inner * 1160) + (rc_inner * 580)) + (yy_c_inner * 58)) + (ry_inner * 58)) + ((((int)threadIdx.x) & 7) * 7)) + xx_c_inner) + rx_outer_inner) + 232)] * kernel_shared[(((((((((int)threadIdx.x) >> 3) * 36) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 1152)]));
                conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 98)] = (conv2d_nchw_local[(((yy_c_inner * 7) + xx_c_inner) + 98)] + (pad_temp_shared[((((((((rc_outer_inner * 1160) + (rc_inner * 580)) + (yy_c_inner * 58)) + (ry_inner * 58)) + ((((int)threadIdx.x) & 7) * 7)) + xx_c_inner) + rx_outer_inner) + 348)] * kernel_shared[(((((((((int)threadIdx.x) >> 3) * 36) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 1152)]));
              }
            }
          }
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
      conv2d_nchw[(((((((((int)blockIdx.x) / 7) * 200704) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (yy_inner * 56)) + ((((int)threadIdx.x) & 7) * 7)) + xx_inner)] = conv2d_nchw_local[((yy_inner * 7) + xx_inner)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 200704) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (yy_inner * 56)) + ((((int)threadIdx.x) & 7) * 7)) + xx_inner) + 112)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 14)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 200704) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (yy_inner * 56)) + ((((int)threadIdx.x) & 7) * 7)) + xx_inner) + 224)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 28)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 200704) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (yy_inner * 56)) + ((((int)threadIdx.x) & 7) * 7)) + xx_inner) + 336)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 42)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 200704) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (yy_inner * 56)) + ((((int)threadIdx.x) & 7) * 7)) + xx_inner) + 100352)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 56)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 200704) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (yy_inner * 56)) + ((((int)threadIdx.x) & 7) * 7)) + xx_inner) + 100464)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 70)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 200704) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (yy_inner * 56)) + ((((int)threadIdx.x) & 7) * 7)) + xx_inner) + 100576)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 84)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 200704) + ((((int)threadIdx.x) >> 3) * 3136)) + ((((int)blockIdx.x) % 7) * 448)) + (yy_inner * 56)) + ((((int)threadIdx.x) & 7) * 7)) + xx_inner) + 100688)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 98)];
    }
  }
}


