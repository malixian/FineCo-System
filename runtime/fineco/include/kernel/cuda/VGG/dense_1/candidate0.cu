
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) candidate0(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[2];
  __shared__ float data_shared[32];
  __shared__ float weight_shared[4096];
  T_matmul_NT[0] = 0.000000e+00f;
  T_matmul_NT[1] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 784; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      data_shared[((int)threadIdx.x)] = data[((k_outer_outer * 32) + ((int)threadIdx.x))];
    }
    *(float4*)(weight_shared + (((int)threadIdx.x) * 4)) = *(float4*)(weight + ((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 200704));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 401408));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 602112));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 802816));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1003520));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1204224));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1404928));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1605632));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2304)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1806336));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2007040));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2816)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2207744));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2408448));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3328)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2609152));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2809856));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3840)) = *(float4*)(weight + (((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 3) * 25088)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3010560));
    __syncthreads();
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[0] * weight_shared[(((int)threadIdx.x) * 32)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[0] * weight_shared[((((int)threadIdx.x) * 32) + 2048)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[1] * weight_shared[((((int)threadIdx.x) * 32) + 1)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[1] * weight_shared[((((int)threadIdx.x) * 32) + 2049)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[2] * weight_shared[((((int)threadIdx.x) * 32) + 2)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[2] * weight_shared[((((int)threadIdx.x) * 32) + 2050)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[3] * weight_shared[((((int)threadIdx.x) * 32) + 3)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[3] * weight_shared[((((int)threadIdx.x) * 32) + 2051)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[4] * weight_shared[((((int)threadIdx.x) * 32) + 4)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[4] * weight_shared[((((int)threadIdx.x) * 32) + 2052)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[5] * weight_shared[((((int)threadIdx.x) * 32) + 5)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[5] * weight_shared[((((int)threadIdx.x) * 32) + 2053)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[6] * weight_shared[((((int)threadIdx.x) * 32) + 6)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[6] * weight_shared[((((int)threadIdx.x) * 32) + 2054)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[7] * weight_shared[((((int)threadIdx.x) * 32) + 7)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[7] * weight_shared[((((int)threadIdx.x) * 32) + 2055)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[8] * weight_shared[((((int)threadIdx.x) * 32) + 8)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[8] * weight_shared[((((int)threadIdx.x) * 32) + 2056)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[9] * weight_shared[((((int)threadIdx.x) * 32) + 9)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[9] * weight_shared[((((int)threadIdx.x) * 32) + 2057)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[10] * weight_shared[((((int)threadIdx.x) * 32) + 10)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[10] * weight_shared[((((int)threadIdx.x) * 32) + 2058)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[11] * weight_shared[((((int)threadIdx.x) * 32) + 11)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[11] * weight_shared[((((int)threadIdx.x) * 32) + 2059)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[12] * weight_shared[((((int)threadIdx.x) * 32) + 12)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[12] * weight_shared[((((int)threadIdx.x) * 32) + 2060)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[13] * weight_shared[((((int)threadIdx.x) * 32) + 13)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[13] * weight_shared[((((int)threadIdx.x) * 32) + 2061)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[14] * weight_shared[((((int)threadIdx.x) * 32) + 14)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[14] * weight_shared[((((int)threadIdx.x) * 32) + 2062)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[15] * weight_shared[((((int)threadIdx.x) * 32) + 15)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[15] * weight_shared[((((int)threadIdx.x) * 32) + 2063)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[16] * weight_shared[((((int)threadIdx.x) * 32) + 16)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[16] * weight_shared[((((int)threadIdx.x) * 32) + 2064)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[17] * weight_shared[((((int)threadIdx.x) * 32) + 17)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[17] * weight_shared[((((int)threadIdx.x) * 32) + 2065)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[18] * weight_shared[((((int)threadIdx.x) * 32) + 18)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[18] * weight_shared[((((int)threadIdx.x) * 32) + 2066)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[19] * weight_shared[((((int)threadIdx.x) * 32) + 19)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[19] * weight_shared[((((int)threadIdx.x) * 32) + 2067)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[20] * weight_shared[((((int)threadIdx.x) * 32) + 20)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[20] * weight_shared[((((int)threadIdx.x) * 32) + 2068)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[21] * weight_shared[((((int)threadIdx.x) * 32) + 21)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[21] * weight_shared[((((int)threadIdx.x) * 32) + 2069)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[22] * weight_shared[((((int)threadIdx.x) * 32) + 22)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[22] * weight_shared[((((int)threadIdx.x) * 32) + 2070)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[23] * weight_shared[((((int)threadIdx.x) * 32) + 23)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[23] * weight_shared[((((int)threadIdx.x) * 32) + 2071)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[24] * weight_shared[((((int)threadIdx.x) * 32) + 24)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[24] * weight_shared[((((int)threadIdx.x) * 32) + 2072)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[25] * weight_shared[((((int)threadIdx.x) * 32) + 25)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[25] * weight_shared[((((int)threadIdx.x) * 32) + 2073)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[26] * weight_shared[((((int)threadIdx.x) * 32) + 26)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[26] * weight_shared[((((int)threadIdx.x) * 32) + 2074)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[27] * weight_shared[((((int)threadIdx.x) * 32) + 27)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[27] * weight_shared[((((int)threadIdx.x) * 32) + 2075)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[28] * weight_shared[((((int)threadIdx.x) * 32) + 28)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[28] * weight_shared[((((int)threadIdx.x) * 32) + 2076)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[29] * weight_shared[((((int)threadIdx.x) * 32) + 29)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[29] * weight_shared[((((int)threadIdx.x) * 32) + 2077)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[30] * weight_shared[((((int)threadIdx.x) * 32) + 30)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[30] * weight_shared[((((int)threadIdx.x) * 32) + 2078)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[31] * weight_shared[((((int)threadIdx.x) * 32) + 31)]));
    T_matmul_NT[1] = (T_matmul_NT[1] + (data_shared[31] * weight_shared[((((int)threadIdx.x) * 32) + 2079)]));
  }
  compute[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))]);
  compute[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 64)] = (T_matmul_NT[1] + bias[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 64)]);
}


