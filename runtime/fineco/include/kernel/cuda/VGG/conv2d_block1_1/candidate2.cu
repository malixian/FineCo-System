
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[512];
  __shared__ float pad_temp_shared[5916];
  __shared__ float kernel_shared[1728];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 32; ++ff_c_outer_inner_init) {
    for (int yy_c_inner_init = 0; yy_c_inner_init < 8; ++yy_c_inner_init) {
      for (int xx_c_inner_init = 0; xx_c_inner_init < 2; ++xx_c_inner_init) {
        conv2d_nchw_local[(((ff_c_outer_inner_init * 16) + (yy_c_inner_init * 2)) + xx_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 27; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + (((int)threadIdx.x) >> 2)) < 1479) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x))] = (((((1 <= (((((int)blockIdx.x) / 7) * 56) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + (((int)threadIdx.x) >> 1)) % 986) / 17))) && ((((((int)blockIdx.x) / 7) * 56) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + (((int)threadIdx.x) >> 1)) % 986) / 17)) < 225)) && (1 <= (((((int)blockIdx.x) % 7) * 32) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) % 34)))) && ((((((int)blockIdx.x) % 7) * 32) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) % 34)) < 225)) ? data[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + (((int)threadIdx.x) >> 2)) / 493) * 50176) + ((((int)blockIdx.x) / 7) * 12544)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + (((int)threadIdx.x) >> 1)) % 986) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x)) % 34)) - 225)] : 0.000000e+00f);
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) >> 5)) < 54) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 224) + ((int)threadIdx.x))] = kernel[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 224) + ((int)threadIdx.x))];
    }
  }
  __syncthreads();
  for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
    for (int ff_c_outer_inner = 0; ff_c_outer_inner < 32; ++ff_c_outer_inner) {
      for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          for (int yy_c_inner = 0; yy_c_inner < 8; ++yy_c_inner) {
            for (int xx_c_inner = 0; xx_c_inner < 2; ++xx_c_inner) {
              conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_inner * 2)) + xx_c_inner)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_inner * 2)) + xx_c_inner)] + (pad_temp_shared[(((((((rc_inner * 1972) + (((((int)threadIdx.x) % 112) >> 4) * 272)) + (yy_c_inner * 34)) + (ry_inner * 34)) + ((((int)threadIdx.x) & 15) * 2)) + xx_c_inner) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) / 112) * 864) + (ff_c_outer_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 32; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 8; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        conv2d_nchw[(((((((((((int)threadIdx.x) / 112) * 1605632) + (ff_inner * 50176)) + ((((int)blockIdx.x) / 7) * 12544)) + (((((int)threadIdx.x) % 112) >> 4) * 1792)) + (yy_inner * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 16) + (yy_inner * 2)) + xx_inner)];
      }
    }
  }
}


