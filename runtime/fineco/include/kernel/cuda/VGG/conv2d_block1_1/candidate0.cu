
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(28) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[64];
  __shared__ float pad_temp_shared[456];
  __shared__ float kernel_shared[72];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[56] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[57] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[58] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[59] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[60] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[61] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[62] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[63] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 3; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x) * 4)] = (((2 <= (((int)blockIdx.x) % 224)) && (1 <= (((((int)blockIdx.x) & 1) * 112) + (((int)threadIdx.x) * 4)))) ? data[(((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 1)] = ((2 <= (((int)blockIdx.x) % 224)) ? data[(((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) - 224)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 2)] = ((2 <= (((int)blockIdx.x) % 224)) ? data[(((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) - 223)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 3)] = ((2 <= (((int)blockIdx.x) % 224)) ? data[(((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) - 222)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 112)] = ((((1 <= ((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 56) / 57))) && (1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 112) % 114)))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 112) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 56) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 112) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 113)] = ((((1 <= ((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 56) / 57))) && (1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 113) % 114)))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 113) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 56) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 113) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 114)] = ((1 <= (((((int)blockIdx.x) & 1) * 112) + (((int)threadIdx.x) * 4))) ? data[(((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) - 1)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 115)] = data[(((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((int)blockIdx.x) & 1) * 112)) + ((((int)threadIdx.x) * 4) + 1)) - 1)];
    pad_temp_shared[((((int)threadIdx.x) * 4) + 224)] = (((1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 110) % 114))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 110) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 112) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 110) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 225)] = (((1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 111) % 114))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 111) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 112) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 111) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 226)] = (((1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 112) % 114))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 112) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 113) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 112) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 227)] = (((1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 113) % 114))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 113) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 113) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 113) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 336)] = ((((((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 168) / 57)) < 225) && (1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 108) % 114)))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 108) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 168) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 108) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 337)] = ((((((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 168) / 57)) < 225) && (1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 109) % 114)))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 109) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 168) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 109) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 338)] = ((((((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 169) / 57)) < 225) && (1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 110) % 114)))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 110) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 169) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 110) % 114)) - 225)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 4) + 339)] = ((((((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 169) / 57)) < 225) && (1 <= (((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 111) % 114)))) && ((((((int)blockIdx.x) & 1) * 112) + (((((int)threadIdx.x) * 4) + 111) % 114)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 169) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((int)threadIdx.x) * 4) + 111) % 114)) - 225)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 2) {
      pad_temp_shared[((((int)threadIdx.x) * 4) + 448)] = ((((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 224) / 57)) < 225) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 224) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + ((((int)threadIdx.x) * 4) + 106)) - 225)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 2) {
      pad_temp_shared[((((int)threadIdx.x) * 4) + 449)] = ((((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 224) / 57)) < 225) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 224) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + ((((int)threadIdx.x) * 4) + 107)) - 225)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 2) {
      pad_temp_shared[((((int)threadIdx.x) * 4) + 450)] = ((((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 225) / 57)) < 225) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 225) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + ((((int)threadIdx.x) * 4) + 108)) - 225)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 2) {
      pad_temp_shared[((((int)threadIdx.x) * 4) + 451)] = (((((((((int)blockIdx.x) % 224) >> 1) * 2) + (((((int)threadIdx.x) * 2) + 225) / 57)) < 225) && ((((((int)blockIdx.x) & 1) * 112) + ((((int)threadIdx.x) * 4) + 109)) < 225)) ? data[((((((rc_outer_outer * 50176) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + ((((((int)threadIdx.x) * 2) + 225) / 57) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + ((((int)threadIdx.x) * 4) + 109)) - 225)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 24) {
      *(float3*)(kernel_shared + (((int)threadIdx.x) * 3)) = *(float3*)(kernel + (((((((int)blockIdx.x) / 224) * 216) + ((((int)threadIdx.x) / 3) * 27)) + (rc_outer_outer * 9)) + ((((int)threadIdx.x) % 3) * 3)));
    }
    __syncthreads();
    for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
        conv2d_nchw_local[(ff_c_outer_inner * 8)] = (conv2d_nchw_local[(ff_c_outer_inner * 8)] + (pad_temp_shared[((((int)threadIdx.x) * 4) + rx_outer_inner)] * kernel_shared[((ff_c_outer_inner * 9) + rx_outer_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 32)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 32)] + (pad_temp_shared[((((int)threadIdx.x) * 4) + rx_outer_inner)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 36)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 1)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 1)] * kernel_shared[((ff_c_outer_inner * 9) + rx_outer_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 33)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 33)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 1)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 36)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 2)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 2)] * kernel_shared[((ff_c_outer_inner * 9) + rx_outer_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 34)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 34)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 2)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 36)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 3)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 3)] * kernel_shared[((ff_c_outer_inner * 9) + rx_outer_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 35)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 35)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 3)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 36)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 4)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 4)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 114)] * kernel_shared[((ff_c_outer_inner * 9) + rx_outer_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 36)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 36)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 114)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 36)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 5)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 5)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 115)] * kernel_shared[((ff_c_outer_inner * 9) + rx_outer_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 37)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 37)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 115)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 36)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 6)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 6)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 116)] * kernel_shared[((ff_c_outer_inner * 9) + rx_outer_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 38)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 38)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 116)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 36)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 7)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 7)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 117)] * kernel_shared[((ff_c_outer_inner * 9) + rx_outer_inner)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 39)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 39)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 117)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 36)]));
        conv2d_nchw_local[(ff_c_outer_inner * 8)] = (conv2d_nchw_local[(ff_c_outer_inner * 8)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 114)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 32)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 32)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 114)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 39)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 1)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 115)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 33)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 33)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 115)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 39)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 2)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 116)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 34)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 34)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 116)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 39)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 3)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 117)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 35)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 35)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 117)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 39)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 4)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 4)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 228)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 36)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 36)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 228)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 39)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 5)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 5)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 229)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 37)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 37)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 229)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 39)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 6)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 6)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 230)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 38)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 38)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 230)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 39)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 7)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 7)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 231)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 39)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 39)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 231)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 39)]));
        conv2d_nchw_local[(ff_c_outer_inner * 8)] = (conv2d_nchw_local[(ff_c_outer_inner * 8)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 228)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 32)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 32)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 228)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 42)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 1)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 229)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 33)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 33)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 229)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 42)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 2)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 230)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 34)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 34)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 230)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 42)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 3)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 231)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 35)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 35)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 231)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 42)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 4)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 4)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 342)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 36)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 36)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 342)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 42)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 5)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 5)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 343)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 37)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 37)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 343)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 42)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 6)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 6)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 344)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 38)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 38)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 344)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 42)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 7)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 7)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 345)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[((ff_c_outer_inner * 8) + 39)] = (conv2d_nchw_local[((ff_c_outer_inner * 8) + 39)] + (pad_temp_shared[(((((int)threadIdx.x) * 4) + rx_outer_inner) + 345)] * kernel_shared[(((ff_c_outer_inner * 9) + rx_outer_inner) + 42)]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 4; ++xx_inner) {
        conv2d_nchw[((((((((((int)blockIdx.x) / 224) * 401408) + (ff_inner * 50176)) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + (yy_inner * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 8) + (yy_inner * 4)) + xx_inner)];
        conv2d_nchw[(((((((((((int)blockIdx.x) / 224) * 401408) + (ff_inner * 50176)) + (((((int)blockIdx.x) % 224) >> 1) * 448)) + (yy_inner * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) + xx_inner) + 200704)] = conv2d_nchw_local[((((ff_inner * 8) + (yy_inner * 4)) + xx_inner) + 32)];
      }
    }
  }
}


