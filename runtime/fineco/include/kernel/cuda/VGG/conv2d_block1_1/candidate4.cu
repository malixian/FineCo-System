
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[64];
  __shared__ float pad_temp_shared[3264];
  __shared__ float kernel_shared[288];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[56] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[57] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[58] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[59] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[60] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[61] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[62] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[63] = 0.000000e+00f;
  for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((1 <= ((((((int)blockIdx.x) % 49) / 7) * 32) + (((int)threadIdx.x) >> 5))) && (1 <= ((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)))) && (((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)) < 225)) ? data[((((((((((int)blockIdx.x) % 49) / 7) * 7168) + ((((int)threadIdx.x) >> 5) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + rx_outer_outer) + (((int)threadIdx.x) & 31)) - 225)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 512)] = (((1 <= ((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31))) && (((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)) < 225)) ? data[((((((((((int)blockIdx.x) % 49) / 7) * 7168) + ((((int)threadIdx.x) >> 5) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + rx_outer_outer) + (((int)threadIdx.x) & 31)) + 3359)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((((1 <= ((((((int)blockIdx.x) % 49) / 7) * 32) + (((((int)threadIdx.x) >> 5) + 32) % 34))) && (((((((int)blockIdx.x) % 49) / 7) * 32) + (((((int)threadIdx.x) >> 5) + 32) % 34)) < 225)) && (1 <= ((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)))) && (((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)) < 225)) ? data[(((((((((((int)threadIdx.x) + 1024) / 1088) * 50176) + (((((int)blockIdx.x) % 49) / 7) * 7168)) + ((((((int)threadIdx.x) >> 5) + 32) % 34) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + rx_outer_outer) + (((int)threadIdx.x) & 31)) - 225)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = (((1 <= ((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31))) && (((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)) < 225)) ? data[(((((((((((int)threadIdx.x) + 1536) / 1088) * 50176) + (((((int)blockIdx.x) % 49) / 7) * 7168)) + (((((int)threadIdx.x) >> 5) + 14) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + rx_outer_outer) + (((int)threadIdx.x) & 31)) - 225)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2048)] = (((((1 <= ((((((int)blockIdx.x) % 49) / 7) * 32) + (((((int)threadIdx.x) >> 5) + 30) % 34))) && (((((((int)blockIdx.x) % 49) / 7) * 32) + (((((int)threadIdx.x) >> 5) + 30) % 34)) < 225)) && (1 <= ((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)))) && (((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)) < 225)) ? data[(((((((((((int)threadIdx.x) + 2048) / 1088) * 50176) + (((((int)blockIdx.x) % 49) / 7) * 7168)) + ((((((int)threadIdx.x) >> 5) + 30) % 34) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + rx_outer_outer) + (((int)threadIdx.x) & 31)) - 225)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2560)] = (((1 <= ((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31))) && (((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)) < 225)) ? data[(((((((((((int)threadIdx.x) + 2560) / 1088) * 50176) + (((((int)blockIdx.x) % 49) / 7) * 7168)) + (((((int)threadIdx.x) >> 5) + 12) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + rx_outer_outer) + (((int)threadIdx.x) & 31)) - 225)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 192) {
      pad_temp_shared[(((int)threadIdx.x) + 3072)] = ((((((((((int)blockIdx.x) % 49) / 7) * 32) + ((((int)threadIdx.x) >> 5) + 28)) < 225) && (1 <= ((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)))) && (((((((int)blockIdx.x) % 7) * 32) + rx_outer_outer) + (((int)threadIdx.x) & 31)) < 225)) ? data[(((((((((((int)threadIdx.x) + 3072) / 1088) * 50176) + (((((int)blockIdx.x) % 49) / 7) * 7168)) + (((((int)threadIdx.x) >> 5) + 28) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + rx_outer_outer) + (((int)threadIdx.x) & 31)) - 225)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 288) {
      kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 49) * 864) + (((int)threadIdx.x) * 3)) + rx_outer_outer)];
    }
    __syncthreads();
    for (int yy_c_outer_inner = 0; yy_c_outer_inner < 4; ++yy_c_outer_inner) {
      for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
        conv2d_nchw_local[(yy_c_outer_inner * 4)] = (conv2d_nchw_local[(yy_c_outer_inner * 4)] + (pad_temp_shared[((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31))] * kernel_shared[(((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3))]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 32)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 32)] + (pad_temp_shared[((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31))] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 144)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 1)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3))]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 33)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 33)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 144)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 2)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[(((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3))]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 34)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 34)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 144)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 3)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[(((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3))]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 35)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 35)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 144)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 16)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 16)] + (pad_temp_shared[((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31))] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 9)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 48)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 48)] + (pad_temp_shared[((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31))] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 153)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 17)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 17)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 9)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 49)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 49)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 153)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 18)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 18)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 9)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 50)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 50)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 153)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 19)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 19)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 9)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 51)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 51)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 153)]));
        conv2d_nchw_local[(yy_c_outer_inner * 4)] = (conv2d_nchw_local[(yy_c_outer_inner * 4)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 1)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 32)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 32)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 145)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 1)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 1)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 33)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 33)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 145)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 2)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 1)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 34)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 34)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 145)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 3)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 1)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 35)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 35)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 145)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 16)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 16)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 10)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 48)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 48)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 154)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 17)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 17)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 10)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 49)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 49)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 154)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 18)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 18)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 10)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 50)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 50)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 154)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 19)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 19)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 10)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 51)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 51)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 154)]));
        conv2d_nchw_local[(yy_c_outer_inner * 4)] = (conv2d_nchw_local[(yy_c_outer_inner * 4)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 2)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 32)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 32)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 146)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 1)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 1)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 2)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 33)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 33)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 146)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 2)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 2)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 2)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 34)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 34)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 146)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 3)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 3)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 160)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 2)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 35)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 35)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 160)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 146)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 16)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 16)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 11)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 48)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 48)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 155)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 17)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 17)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 11)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 49)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 49)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 155)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 18)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 18)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 11)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 50)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 50)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 128)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 155)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 19)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 19)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 160)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 11)]));
        conv2d_nchw_local[((yy_c_outer_inner * 4) + 51)] = (conv2d_nchw_local[((yy_c_outer_inner * 4) + 51)] + (pad_temp_shared[(((((rc_inner * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (yy_c_outer_inner * 128)) + (((int)threadIdx.x) & 31)) + 160)] * kernel_shared[((((((int)threadIdx.x) >> 6) * 18) + (rc_inner * 3)) + 155)]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 16; ++yy_inner) {
      conv2d_nchw[(((((((((((int)blockIdx.x) / 49) * 1605632) + ((((int)threadIdx.x) >> 6) * 100352)) + (ff_inner * 50176)) + (((((int)blockIdx.x) % 49) / 7) * 7168)) + (((((int)threadIdx.x) & 63) >> 5) * 3584)) + (yy_inner * 224)) + ((((int)blockIdx.x) % 7) * 32)) + (((int)threadIdx.x) & 31))] = conv2d_nchw_local[((ff_inner * 16) + yy_inner)];
      conv2d_nchw[((((((((((((int)blockIdx.x) / 49) * 1605632) + ((((int)threadIdx.x) >> 6) * 100352)) + (ff_inner * 50176)) + (((((int)blockIdx.x) % 49) / 7) * 7168)) + (((((int)threadIdx.x) & 63) >> 5) * 3584)) + (yy_inner * 224)) + ((((int)blockIdx.x) % 7) * 32)) + (((int)threadIdx.x) & 31)) + 802816)] = conv2d_nchw_local[(((ff_inner * 16) + yy_inner) + 32)];
    }
  }
}


