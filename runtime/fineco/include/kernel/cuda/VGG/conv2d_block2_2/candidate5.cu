
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate5(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[56];
  __shared__ float pad_temp_shared[1024];
  __shared__ float kernel_shared[1536];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) / 7) * 14) + ((((int)threadIdx.x) & 255) >> 4))) && ((((((int)blockIdx.x) / 7) * 14) + ((((int)threadIdx.x) & 255) >> 4)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 50176) + ((((int)threadIdx.x) >> 8) * 12544)) + ((((int)blockIdx.x) / 7) * 1568)) + (((((int)threadIdx.x) & 255) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 512)] = (((((1 <= (((((int)blockIdx.x) / 7) * 14) + ((((int)threadIdx.x) & 255) >> 4))) && ((((((int)blockIdx.x) / 7) * 14) + ((((int)threadIdx.x) & 255) >> 4)) < 113)) && (1 <= ((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)))) && (((((((int)blockIdx.x) % 7) * 16) + rx_outer_outer) + (((int)threadIdx.x) & 15)) < 113)) ? data[((((((((rc_outer_outer * 50176) + ((((int)threadIdx.x) >> 8) * 12544)) + ((((int)blockIdx.x) / 7) * 1568)) + (((((int)threadIdx.x) & 255) >> 4) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + rx_outer_outer) + (((int)threadIdx.x) & 15)) + 24975)] : 0.000000e+00f);
      kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)threadIdx.x) / 12) * 1152) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) % 12) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)threadIdx.x) + 512) / 12) * 1152) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) + 8) % 12) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((((((int)threadIdx.x) + 1024) / 12) * 1152) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) + 4) % 12) * 3)) + rx_outer_outer)];
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 56)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 56)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 88)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 88)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner)]));
          conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 768)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 56)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 56)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 88)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 80)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 88)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 12)]));
          conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
          conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[(((((rc_outer_inner * 256) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (ry_inner * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 24) + (rc_outer_inner * 3)) + ry_inner) + 780)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      conv2d_nchw[((((((((((int)threadIdx.x) >> 4) * 25088) + (ff_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1568)) + (((((int)threadIdx.x) & 15) >> 3) * 784)) + (yy_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 7))] = conv2d_nchw_local[((ff_inner * 7) + yy_inner)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 4) * 25088) + (ff_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1568)) + (((((int)threadIdx.x) & 15) >> 3) * 784)) + (yy_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 7)) + 8)] = conv2d_nchw_local[(((ff_inner * 7) + yy_inner) + 14)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 4) * 25088) + (ff_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1568)) + (((((int)threadIdx.x) & 15) >> 3) * 784)) + (yy_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 7)) + 802816)] = conv2d_nchw_local[(((ff_inner * 7) + yy_inner) + 28)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 4) * 25088) + (ff_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1568)) + (((((int)threadIdx.x) & 15) >> 3) * 784)) + (yy_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) & 7)) + 802824)] = conv2d_nchw_local[(((ff_inner * 7) + yy_inner) + 42)];
    }
  }
}


