
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate6(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[98];
  __shared__ float pad_temp_shared[6720];
  __shared__ float kernel_shared[768];
  for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 7; ++yy_c_outer_inner_init) {
    conv2d_nchw_local[(yy_c_outer_inner_init * 7)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 49)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 1)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 50)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 2)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 51)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 3)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 52)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 4)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 53)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 5)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 54)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 6)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 55)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      pad_temp_shared[((int)threadIdx.x)] = ((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + (((int)threadIdx.x) / 28))) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + (((int)threadIdx.x) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + (((int)threadIdx.x) % 28)) < 113)) ? data[(((((((rc_outer_outer * 100352) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + ((((int)threadIdx.x) / 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + (((int)threadIdx.x) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 256)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)) < 113)) ? data[(((((((rc_outer_outer * 100352) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((int)threadIdx.x) + 256) / 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 512)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)) < 113)) ? data[(((((((rc_outer_outer * 100352) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((int)threadIdx.x) + 512) / 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 768)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 192) % 210) / 7))) && (((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 192) % 210) / 7)) < 113)) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 768) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 192) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1024) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 46) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1280)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1280) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 110) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1536)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 174) % 210) / 7))) && (((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 174) % 210) / 7)) < 113)) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1536) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 174) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + (((int)threadIdx.x) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + (((int)threadIdx.x) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1792) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 4) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + (((int)threadIdx.x) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 2048)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2048) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 92) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 2304)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 156) % 210) / 7))) && (((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 156) % 210) / 7)) < 113)) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2304) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 156) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 2560)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2560) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 10) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 2816)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2816) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 74) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 3072)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3072) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 138) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 3328)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 202) % 210) / 7))) && (((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 202) % 210) / 7)) < 113)) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3328) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 202) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 3584)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + (((int)threadIdx.x) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + (((int)threadIdx.x) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3584) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 8) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + (((int)threadIdx.x) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 3840)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3840) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 120) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 4096)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 184) % 210) / 7))) && (((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 184) % 210) / 7)) < 113)) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 4096) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 184) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 4352)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 4352) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 38) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 4608)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 4608) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 102) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 4864)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 166) % 210) / 7))) && (((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 166) % 210) / 7)) < 113)) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 4864) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 166) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 5120)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 5120) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 20) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 24) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 5376)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + (((int)threadIdx.x) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + (((int)threadIdx.x) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 5376) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 12) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + (((int)threadIdx.x) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 5632)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 148) % 210) / 7))) && (((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 148) % 210) / 7)) < 113)) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 5632) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 148) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 4) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 5888)] = ((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 2) % 210) / 7))) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 5888) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 2) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 8) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 6144)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 6144) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 66) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 12) % 28)) - 113)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 6400)] = (((1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 6400) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 130) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 16) % 28)) - 113)] : 0.000000e+00f);
      if (((int)threadIdx.x) < 64) {
        pad_temp_shared[(((int)threadIdx.x) + 6656)] = ((((((((((int)blockIdx.x) & 15) >> 2) * 28) + ((((((int)threadIdx.x) >> 2) + 194) % 210) / 7)) < 113) && (1 <= ((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)))) && (((((((int)blockIdx.x) & 3) * 28) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)) < 113)) ? data[((((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 6656) / 840) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((((int)threadIdx.x) >> 2) + 194) % 210) / 7) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + rx_outer_outer) + ((((int)threadIdx.x) + 20) % 28)) - 113)] : 0.000000e+00f);
      }
      kernel_shared[((int)threadIdx.x)] = kernel[((((((((int)blockIdx.x) >> 4) * 36864) + ((((int)threadIdx.x) / 24) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) >> 4) * 36864) + (((((int)threadIdx.x) + 256) / 24) * 1152)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)blockIdx.x) >> 4) * 36864) + (((((int)threadIdx.x) + 512) / 24) * 1152)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + rx_outer_outer)];
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
        for (int yy_c_outer_inner = 0; yy_c_outer_inner < 7; ++yy_c_outer_inner) {
          conv2d_nchw_local[(yy_c_outer_inner * 7)] = (conv2d_nchw_local[(yy_c_outer_inner * 7)] + (pad_temp_shared[((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7))] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 49)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 14)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[(yy_c_outer_inner * 7)] = (conv2d_nchw_local[(yy_c_outer_inner * 7)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 28)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 49)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 42)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[(yy_c_outer_inner * 7)] = (conv2d_nchw_local[(yy_c_outer_inner * 7)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 56)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 49)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 70)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 1)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 50)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 50)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 15)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 1)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 29)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 50)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 50)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 43)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 1)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 57)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 50)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 50)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 71)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 2)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 51)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 51)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 2)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 30)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 51)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 51)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 44)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 2)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 58)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 51)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 51)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 72)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 3)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 3)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 3)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 52)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 52)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 17)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 3)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 3)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 31)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 52)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 52)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 45)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 3)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 3)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 59)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 52)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 52)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 73)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 4)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 4)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 53)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 53)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 18)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 4)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 53)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 53)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 46)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 4)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 60)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 53)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 53)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 74)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 5)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 5)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 5)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 54)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 54)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 19)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 5)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 5)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 33)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 54)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 54)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 47)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 5)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 5)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 61)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 54)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 54)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 75)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 6)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 6)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 55)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 55)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3))]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 6)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 6)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 34)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 55)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 55)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 6)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 6)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 62)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[((yy_c_outer_inner * 7) + 55)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + 55)] + (pad_temp_shared[(((((rc_outer_inner * 840) + (((((int)threadIdx.x) & 7) >> 1) * 196)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) & 1) * 7)) + 76)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 24) + (rc_outer_inner * 3)) + 2)]));
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 4) * 401408) + ((((int)threadIdx.x) >> 3) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((int)threadIdx.x) & 7) >> 1) * 784)) + (yy_inner * 112)) + ((((int)blockIdx.x) & 3) * 28)) + ((((int)threadIdx.x) & 1) * 7)) + xx_inner)] = conv2d_nchw_local[((yy_inner * 7) + xx_inner)];
      conv2d_nchw[((((((((((((int)blockIdx.x) >> 4) * 401408) + ((((int)threadIdx.x) >> 3) * 12544)) + (((((int)blockIdx.x) & 15) >> 2) * 3136)) + (((((int)threadIdx.x) & 7) >> 1) * 784)) + (yy_inner * 112)) + ((((int)blockIdx.x) & 3) * 28)) + ((((int)threadIdx.x) & 1) * 7)) + xx_inner) + 14)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 49)];
    }
  }
}

