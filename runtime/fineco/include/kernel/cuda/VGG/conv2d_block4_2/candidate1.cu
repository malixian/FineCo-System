
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[56];
  __shared__ float pad_temp_shared[7168];
  __shared__ float kernel_shared[3072];
  for (int yy_c_inner_init = 0; yy_c_inner_init < 7; ++yy_c_inner_init) {
    conv2d_nchw_local[(yy_c_inner_init * 2)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_inner_init * 2) + 14)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_inner_init * 2) + 28)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_inner_init * 2) + 42)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_inner_init * 2) + 1)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_inner_init * 2) + 15)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_inner_init * 2) + 29)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_inner_init * 2) + 43)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) - 29)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 448)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 755)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 896)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 1539)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1344)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 2323)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 3107)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 2240)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 3891)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 2688)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 4675)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 3136)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 5459)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 3584)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 6243)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 4032)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 7027)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 4480)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 7811)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 4928)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 8595)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 5376)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 9379)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 5824)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 10163)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 6272)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 10947)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 6720)] = (((((1 <= (((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28))) && ((((((int)blockIdx.x) & 1) * 14) + (((int)threadIdx.x) / 28)) < 29)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 28)))) && ((rx_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 1) * 392)) + rx_outer_outer) + ((int)threadIdx.x)) + 11731)] : 0.000000e+00f);
      kernel_shared[((int)threadIdx.x)] = kernel[((((((((int)blockIdx.x) >> 1) * 294912) + ((((int)threadIdx.x) / 48) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) >> 1) * 294912) + (((((int)threadIdx.x) + 448) / 48) * 4608)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 16) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) >> 1) * 294912) + (((((int)threadIdx.x) + 896) / 48) * 4608)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 32) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((((int)blockIdx.x) >> 1) * 294912) + ((((int)threadIdx.x) / 48) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer) + 129024)];
      kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) >> 1) * 294912) + (((((int)threadIdx.x) + 1792) / 48) * 4608)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 16) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)blockIdx.x) >> 1) * 294912) + (((((int)threadIdx.x) + 2240) / 48) * 4608)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 32) % 48) * 3)) + rx_outer_outer)];
      if (((int)threadIdx.x) < 384) {
        kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((((int)blockIdx.x) >> 1) * 294912) + ((((int)threadIdx.x) / 48) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer) + 258048)];
      }
      __syncthreads();
      for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          for (int yy_c_inner = 0; yy_c_inner < 7; ++yy_c_inner) {
            conv2d_nchw_local[(yy_c_inner * 2)] = (conv2d_nchw_local[(yy_c_inner * 2)] + (pad_temp_shared[(((((rc_inner * 448) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_c_inner * 28)) + (ry_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[((yy_c_inner * 2) + 14)] = (conv2d_nchw_local[((yy_c_inner * 2) + 14)] + (pad_temp_shared[(((((rc_inner * 448) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_c_inner * 28)) + (ry_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[(((((((int)threadIdx.x) / 28) * 48) + (rc_inner * 3)) + ry_inner) + 768)]));
            conv2d_nchw_local[((yy_c_inner * 2) + 28)] = (conv2d_nchw_local[((yy_c_inner * 2) + 28)] + (pad_temp_shared[(((((rc_inner * 448) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_c_inner * 28)) + (ry_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[(((((((int)threadIdx.x) / 28) * 48) + (rc_inner * 3)) + ry_inner) + 1536)]));
            conv2d_nchw_local[((yy_c_inner * 2) + 42)] = (conv2d_nchw_local[((yy_c_inner * 2) + 42)] + (pad_temp_shared[(((((rc_inner * 448) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_c_inner * 28)) + (ry_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[(((((((int)threadIdx.x) / 28) * 48) + (rc_inner * 3)) + ry_inner) + 2304)]));
            conv2d_nchw_local[((yy_c_inner * 2) + 1)] = (conv2d_nchw_local[((yy_c_inner * 2) + 1)] + (pad_temp_shared[((((((rc_inner * 448) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_c_inner * 28)) + (ry_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 28) * 48) + (rc_inner * 3)) + ry_inner)]));
            conv2d_nchw_local[((yy_c_inner * 2) + 15)] = (conv2d_nchw_local[((yy_c_inner * 2) + 15)] + (pad_temp_shared[((((((rc_inner * 448) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_c_inner * 28)) + (ry_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 48) + (rc_inner * 3)) + ry_inner) + 768)]));
            conv2d_nchw_local[((yy_c_inner * 2) + 29)] = (conv2d_nchw_local[((yy_c_inner * 2) + 29)] + (pad_temp_shared[((((((rc_inner * 448) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_c_inner * 28)) + (ry_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 48) + (rc_inner * 3)) + ry_inner) + 1536)]));
            conv2d_nchw_local[((yy_c_inner * 2) + 43)] = (conv2d_nchw_local[((yy_c_inner * 2) + 43)] + (pad_temp_shared[((((((rc_inner * 448) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_c_inner * 28)) + (ry_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 28) * 48) + (rc_inner * 3)) + ry_inner) + 2304)]));
          }
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      conv2d_nchw[((((((((((int)blockIdx.x) >> 1) * 50176) + ((((int)threadIdx.x) / 28) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + xx_inner)] = conv2d_nchw_local[((yy_inner * 2) + xx_inner)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 1) * 50176) + ((((int)threadIdx.x) / 28) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + xx_inner) + 12544)] = conv2d_nchw_local[(((yy_inner * 2) + xx_inner) + 14)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 1) * 50176) + ((((int)threadIdx.x) / 28) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + xx_inner) + 25088)] = conv2d_nchw_local[(((yy_inner * 2) + xx_inner) + 28)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 1) * 50176) + ((((int)threadIdx.x) / 28) * 784)) + ((((int)blockIdx.x) & 1) * 392)) + (((((int)threadIdx.x) % 28) / 14) * 196)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + xx_inner) + 37632)] = conv2d_nchw_local[(((yy_inner * 2) + xx_inner) + 42)];
    }
  }
}


