
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[28];
  __shared__ float pad_temp_shared[7168];
  __shared__ float kernel_shared[1536];
  for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 2; ++yy_c_outer_inner_init) {
    conv2d_nchw_local[(yy_c_outer_inner_init * 7)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 14)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 1)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 15)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 2)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 16)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 3)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 17)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 4)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 18)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 5)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 19)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 6)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 7) + 20)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x))] = (((((1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 1) * 8) + (((int)threadIdx.x) / 14))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 1) * 8) + (((int)threadIdx.x) / 14)) < 15)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 14)))) && ((rx_outer_outer + (((int)threadIdx.x) % 14)) < 15)) ? data[((((((rc_outer_outer * 6272) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 1) * 196)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 1) * 112)) + ((int)threadIdx.x)) + rx_outer_outer) - 15)] : 0.000000e+00f);
      }
      kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) / 96) * 4608)) + (rc_outer_outer * 288)) + ((((int)threadIdx.x) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 112)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 112) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 16) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 224) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 32) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 336)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 336) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 48) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 448) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 64) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 560)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 560) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 80) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) / 96) * 4608)) + (rc_outer_outer * 288)) + ((((int)threadIdx.x) % 96) * 3)) + rx_outer_outer) + 32256)];
      kernel_shared[(((int)threadIdx.x) + 784)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 784) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 16) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 896) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 32) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1008) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 48) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1120) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 64) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1232)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1232) / 96) * 4608)) + (rc_outer_outer * 288)) + (((((int)threadIdx.x) + 80) % 96) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) / 96) * 4608)) + (rc_outer_outer * 288)) + ((((int)threadIdx.x) % 96) * 3)) + rx_outer_outer) + 64512)];
      if (((int)threadIdx.x) < 80) {
        kernel_shared[(((int)threadIdx.x) + 1456)] = kernel[(((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1456) / 96) * 4608)) + (rc_outer_outer * 288)) + ((((int)threadIdx.x) + 16) * 3)) + rx_outer_outer)];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
        for (int yy_c_outer_inner = 0; yy_c_outer_inner < 2; ++yy_c_outer_inner) {
          for (int xx_c_outer_inner = 0; xx_c_outer_inner < 7; ++xx_c_outer_inner) {
            for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
              conv2d_nchw_local[((yy_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 1792) + (rc_inner * 224)) + (((((int)threadIdx.x) % 14) >> 1) * 28)) + (yy_c_outer_inner * 14)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) / 14) * 96) + (rc_outer_inner * 24)) + (rc_inner * 3))]));
              conv2d_nchw_local[(((yy_c_outer_inner * 7) + xx_c_outer_inner) + 14)] = (conv2d_nchw_local[(((yy_c_outer_inner * 7) + xx_c_outer_inner) + 14)] + (pad_temp_shared[((((((rc_outer_inner * 1792) + (rc_inner * 224)) + (((((int)threadIdx.x) % 14) >> 1) * 28)) + (yy_c_outer_inner * 14)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 96) + (rc_outer_inner * 24)) + (rc_inner * 3)) + 768)]));
              conv2d_nchw_local[((yy_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[(((((((rc_outer_inner * 1792) + (rc_inner * 224)) + (((((int)threadIdx.x) % 14) >> 1) * 28)) + (yy_c_outer_inner * 14)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + 14)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 96) + (rc_outer_inner * 24)) + (rc_inner * 3)) + 1)]));
              conv2d_nchw_local[(((yy_c_outer_inner * 7) + xx_c_outer_inner) + 14)] = (conv2d_nchw_local[(((yy_c_outer_inner * 7) + xx_c_outer_inner) + 14)] + (pad_temp_shared[(((((((rc_outer_inner * 1792) + (rc_inner * 224)) + (((((int)threadIdx.x) % 14) >> 1) * 28)) + (yy_c_outer_inner * 14)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + 14)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 96) + (rc_outer_inner * 24)) + (rc_inner * 3)) + 769)]));
              conv2d_nchw_local[((yy_c_outer_inner * 7) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 7) + xx_c_outer_inner)] + (pad_temp_shared[(((((((rc_outer_inner * 1792) + (rc_inner * 224)) + (((((int)threadIdx.x) % 14) >> 1) * 28)) + (yy_c_outer_inner * 14)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + 28)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 96) + (rc_outer_inner * 24)) + (rc_inner * 3)) + 2)]));
              conv2d_nchw_local[(((yy_c_outer_inner * 7) + xx_c_outer_inner) + 14)] = (conv2d_nchw_local[(((yy_c_outer_inner * 7) + xx_c_outer_inner) + 14)] + (pad_temp_shared[(((((((rc_outer_inner * 1792) + (rc_inner * 224)) + (((((int)threadIdx.x) % 14) >> 1) * 28)) + (yy_c_outer_inner * 14)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + 28)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 96) + (rc_outer_inner * 24)) + (rc_inner * 3)) + 770)]));
            }
          }
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
      conv2d_nchw[(((((((int)blockIdx.x) * 3136) + ((((int)threadIdx.x) >> 1) * 28)) + (yy_inner * 14)) + ((((int)threadIdx.x) & 1) * 7)) + xx_inner)] = conv2d_nchw_local[((yy_inner * 7) + xx_inner)];
      conv2d_nchw[((((((((int)blockIdx.x) * 3136) + ((((int)threadIdx.x) >> 1) * 28)) + (yy_inner * 14)) + ((((int)threadIdx.x) & 1) * 7)) + xx_inner) + 1568)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 14)];
    }
  }
}


