
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(49) candidate0(float* __restrict__ tensor, float* __restrict__ data) {
  tensor[((((int)blockIdx.x) * 49) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 2; ++rv0) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      tensor[((((int)blockIdx.x) * 49) + ((int)threadIdx.x))] = max(tensor[((((int)blockIdx.x) * 49) + ((int)threadIdx.x))], data[(((((((((int)blockIdx.x) * 7) + (((int)threadIdx.x) / 7)) >> 4) * 448) + (rv0 * 224)) + ((((((int)blockIdx.x) * 49) + ((int)threadIdx.x)) % 112) * 2)) + rv1)]);
    }
  }
}


