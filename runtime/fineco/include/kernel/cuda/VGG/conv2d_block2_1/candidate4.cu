
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[128];
  __shared__ float pad_temp_shared[648];
  __shared__ float kernel_shared[2304];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 4; ++ff_c_outer_inner_init) {
    conv2d_nchw_local[(ff_c_outer_inner_init * 16)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 64)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 8)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 72)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 1)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 65)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 9)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 73)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 2)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 66)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 10)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 74)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 3)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 67)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 11)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 75)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 4)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 68)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 12)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 76)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 5)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 69)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 13)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 77)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 6)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 70)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 14)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 78)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 7)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 71)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 15)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_outer_inner_init * 16) + 79)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((int)threadIdx.x) / 18))) && (1 <= (((((int)blockIdx.x) % 7) * 16) + (((int)threadIdx.x) % 18)))) && ((((((int)blockIdx.x) % 7) * 16) + (((int)threadIdx.x) % 18)) < 113)) ? data[((((((rc_outer_outer * 25088) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) / 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) % 18)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 256)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 128) % 162) / 9))) && ((((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 128) % 162) / 9)) < 113)) && (1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 4) % 18)))) && ((((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 4) % 18)) < 113)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 256) / 324) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((((int)threadIdx.x) >> 1) + 128) % 162) / 9) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 4) % 18)) - 113)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 136) {
      pad_temp_shared[(((int)threadIdx.x) + 512)] = (((((((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 94) % 162) / 9)) < 113) && (1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 8) % 18)))) && ((((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 8) % 18)) < 113)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 512) / 324) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((((int)threadIdx.x) >> 1) + 94) % 162) / 9) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 8) % 18)) - 113)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 18) * 576) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) + 256) / 18) * 576) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) + 512) / 18) * 576) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) + 768) / 18) * 576) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) + 1024) / 18) * 576) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) + 1280) / 18) * 576) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) + 1536) / 18) * 576) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) + 1792) / 18) * 576) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) + 2048) / 18) * 576) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
        for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
          for (int yy_c_outer_inner = 0; yy_c_outer_inner < 4; ++yy_c_outer_inner) {
            conv2d_nchw_local[((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2))] + (pad_temp_shared[(((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 64)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 64)] + (pad_temp_shared[(((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1152)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 8)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 18)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 72)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 72)] + (pad_temp_shared[(((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1170)]));
            conv2d_nchw_local[((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2))] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 18)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 64)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 64)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 18)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1155)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 8)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 18)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 21)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 72)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 72)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 18)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1173)]));
            conv2d_nchw_local[((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2))] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 36)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 64)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 64)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 36)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1158)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 8)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 36)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 24)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 72)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 72)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 36)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1176)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 1)] * kernel_shared[(((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 65)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 65)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 1)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1152)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 9)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 9)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 1)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 18)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 73)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 73)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 1)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1170)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 19)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 65)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 65)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 19)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1155)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 9)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 9)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 19)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 21)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 73)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 73)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 19)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1173)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 37)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 65)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 65)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 37)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1158)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 9)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 9)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 37)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 24)]));
            conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 73)] = (conv2d_nchw_local[(((ff_c_outer_inner * 16) + (yy_c_outer_inner * 2)) + 73)] + (pad_temp_shared[((((((rc_outer_inner * 324) + (((((int)threadIdx.x) & 31) >> 3) * 72)) + (yy_c_outer_inner * 18)) + ((((int)threadIdx.x) & 7) * 2)) + rx_outer_inner) + 37)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 144) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + rx_outer_inner) + 1176)]));
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        conv2d_nchw[(((((((((((int)threadIdx.x) >> 5) * 100352) + (ff_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((int)threadIdx.x) & 31) >> 3) * 448)) + (yy_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 8) + (yy_inner * 2)) + xx_inner)];
        conv2d_nchw[((((((((((((int)threadIdx.x) >> 5) * 100352) + (ff_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((int)threadIdx.x) & 31) >> 3) * 448)) + (yy_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + xx_inner) + 802816)] = conv2d_nchw_local[((((ff_inner * 8) + (yy_inner * 2)) + xx_inner) + 64)];
      }
    }
  }
}


