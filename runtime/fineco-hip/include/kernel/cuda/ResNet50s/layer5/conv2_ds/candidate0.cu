#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(56) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[2];
  __shared__ float pad_temp_shared[360];
  __shared__ float kernel_shared[1152];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x) * 2)] = (((3 <= ((((int)threadIdx.x) * 2) % 45)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) * 2) % 3)))) ? data[((((((rc_outer_outer * 1568) + (((((int)threadIdx.x) * 2) / 45) * 196)) + ((((((int)threadIdx.x) * 2) % 45) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + ((((int)threadIdx.x) * 2) % 3)) - 15)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] = (((3 <= (((((int)threadIdx.x) * 2) + 1) % 45)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + (((((int)threadIdx.x) * 2) + 1) % 3)))) ? data[((((((rc_outer_outer * 1568) + ((((((int)threadIdx.x) * 2) + 1) / 45) * 196)) + (((((((int)threadIdx.x) * 2) + 1) % 45) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((((int)threadIdx.x) * 2) + 1) % 3)) - 15)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] = (((3 <= (((((int)threadIdx.x) * 2) + 22) % 45)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + (((((int)threadIdx.x) * 2) + 1) % 3)))) ? data[((((((rc_outer_outer * 1568) + ((((((int)threadIdx.x) * 2) + 112) / 45) * 196)) + (((((((int)threadIdx.x) * 2) + 22) % 45) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((((int)threadIdx.x) * 2) + 1) % 3)) - 15)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] = (((3 <= (((((int)threadIdx.x) * 2) + 23) % 45)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + (((((int)threadIdx.x) * 2) + 2) % 3)))) ? data[((((((rc_outer_outer * 1568) + ((((((int)threadIdx.x) * 2) + 113) / 45) * 196)) + (((((((int)threadIdx.x) * 2) + 23) % 45) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((((int)threadIdx.x) * 2) + 2) % 3)) - 15)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] = (((3 <= (((((int)threadIdx.x) * 2) + 44) % 45)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + (((((int)threadIdx.x) * 2) + 2) % 3)))) ? data[((((((rc_outer_outer * 1568) + ((((((int)threadIdx.x) * 2) + 224) / 45) * 196)) + (((((((int)threadIdx.x) * 2) + 44) % 45) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((((int)threadIdx.x) * 2) + 2) % 3)) - 15)] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] = (((3 <= ((((int)threadIdx.x) * 2) % 45)) && (1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) * 2) % 3)))) ? data[((((((rc_outer_outer * 1568) + (((((int)threadIdx.x) * 2) / 45) * 196)) + ((((((int)threadIdx.x) * 2) % 45) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + ((((int)threadIdx.x) * 2) % 3)) + 965)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 12) {
      pad_temp_shared[((((int)threadIdx.x) * 2) + 336)] = ((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) * 2) % 3))) ? data[((((((rc_outer_outer * 1568) + ((((((int)threadIdx.x) * 2) + 336) / 45) * 196)) + ((((((int)threadIdx.x) * 2) / 3) + 7) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + ((((int)threadIdx.x) * 2) % 3)) - 15)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 12) {
      pad_temp_shared[((((int)threadIdx.x) * 2) + 337)] = ((1 <= (((((int)blockIdx.x) % 7) * 2) + (((((int)threadIdx.x) * 2) + 1) % 3))) ? data[((((((rc_outer_outer * 1568) + ((((((int)threadIdx.x) * 2) + 337) / 45) * 196)) + (((((((int)threadIdx.x) * 2) + 22) % 45) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((((int)threadIdx.x) * 2) + 1) % 3)) - 15)] : 0.000000e+00f);
    }
    kernel_shared[(((int)threadIdx.x) * 3)] = kernel[(((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3))];
    kernel_shared[((((int)threadIdx.x) * 3) + 1)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 1)];
    kernel_shared[((((int)threadIdx.x) * 3) + 2)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 2)];
    kernel_shared[((((int)threadIdx.x) * 3) + 168)] = kernel[(((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 56) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3))];
    kernel_shared[((((int)threadIdx.x) * 3) + 169)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 56) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + 1)];
    kernel_shared[((((int)threadIdx.x) * 3) + 170)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 56) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + 2)];
    kernel_shared[((((int)threadIdx.x) * 3) + 336)] = kernel[(((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 112) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3))];
    kernel_shared[((((int)threadIdx.x) * 3) + 337)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 112) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + 1)];
    kernel_shared[((((int)threadIdx.x) * 3) + 338)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 112) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + 2)];
    kernel_shared[((((int)threadIdx.x) * 3) + 504)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 32256)];
    kernel_shared[((((int)threadIdx.x) * 3) + 505)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 32257)];
    kernel_shared[((((int)threadIdx.x) * 3) + 506)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 32258)];
    kernel_shared[((((int)threadIdx.x) * 3) + 672)] = kernel[(((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 224) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3))];
    kernel_shared[((((int)threadIdx.x) * 3) + 673)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 224) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + 1)];
    kernel_shared[((((int)threadIdx.x) * 3) + 674)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 224) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + 2)];
    kernel_shared[((((int)threadIdx.x) * 3) + 840)] = kernel[(((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 280) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3))];
    kernel_shared[((((int)threadIdx.x) * 3) + 841)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 280) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + 1)];
    kernel_shared[((((int)threadIdx.x) * 3) + 842)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + (((((int)threadIdx.x) + 280) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + 2)];
    if (((int)threadIdx.x) < 48) {
      kernel_shared[((((int)threadIdx.x) * 3) + 1008)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 64512)];
      kernel_shared[((((int)threadIdx.x) * 3) + 1009)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 64513)];
      kernel_shared[((((int)threadIdx.x) * 3) + 1010)] = kernel[((((((((int)blockIdx.x) / 7) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + 64514)];
    }
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) * 6)] * kernel_shared[((((int)threadIdx.x) / 7) * 144)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 1)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 2)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 45)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 9)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 46)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 10)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 47)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 11)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 90)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 18)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 91)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 19)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 92)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 20)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 135)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 27)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 136)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 28)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 137)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 29)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 180)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 36)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 181)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 37)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 182)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 38)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 225)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 45)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 226)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 46)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 227)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 47)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 270)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 54)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 271)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 55)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 272)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 56)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 315)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 63)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 316)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 64)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 317)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 65)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) * 6)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 72)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 73)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 74)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 45)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 81)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 46)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 82)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 47)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 83)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 90)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 90)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 91)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 91)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 92)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 92)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 135)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 99)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 136)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 100)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 137)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 101)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 180)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 108)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 181)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 109)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 182)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 110)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 225)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 117)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 226)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 118)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 227)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 119)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 270)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 126)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 271)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 127)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 272)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 128)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 315)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 135)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 316)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 136)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 317)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 137)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 3)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 3)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 4)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 4)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 5)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 5)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 48)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 12)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 49)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 13)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 50)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 14)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 93)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 21)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 94)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 22)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 95)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 23)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 138)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 30)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 139)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 31)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 140)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 32)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 183)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 39)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 184)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 40)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 185)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 41)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 228)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 48)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 229)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 49)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 230)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 50)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 273)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 57)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 274)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 58)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 275)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 59)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 318)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 66)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 319)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 67)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 320)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 68)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 3)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 75)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 4)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 76)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 5)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 77)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 48)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 84)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 49)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 85)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 50)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 86)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 93)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 93)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 94)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 94)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 95)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 95)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 138)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 102)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 139)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 103)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 140)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 104)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 183)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 111)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 184)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 112)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 185)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 113)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 228)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 120)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 229)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 121)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 230)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 122)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 273)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 129)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 274)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 130)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 275)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 131)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 318)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 138)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 319)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 139)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 320)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 140)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 6)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 6)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 7)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 7)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 8)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 8)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 51)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 15)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 52)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 16)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 53)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 17)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 96)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 24)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 97)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 25)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 98)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 26)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 141)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 33)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 142)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 34)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 143)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 35)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 186)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 42)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 187)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 43)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 188)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 44)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 231)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 51)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 232)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 52)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 233)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 53)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 276)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 60)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 277)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 61)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 278)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 62)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 321)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 69)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 322)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 70)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 323)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 71)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 6)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 78)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 7)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 79)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 8)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 80)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 51)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 87)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 52)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 88)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 53)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 89)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 96)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 96)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 97)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 97)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 98)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 98)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 141)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 105)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 142)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 106)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 143)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 107)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 186)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 114)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 187)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 115)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 188)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 116)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 231)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 123)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 232)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 124)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 233)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 125)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 276)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 132)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 277)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 133)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 278)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 134)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 321)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 141)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 322)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 142)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 7) * 6) + 323)] * kernel_shared[(((((int)threadIdx.x) / 7) * 144) + 143)]));
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    conv2d_nchw[((((((((int)blockIdx.x) / 7) * 784) + ((((int)threadIdx.x) / 7) * 98)) + (ff_inner * 49)) + ((((int)threadIdx.x) % 7) * 7)) + (((int)blockIdx.x) % 7))] = conv2d_nchw_local[ff_inner];
  }
}


