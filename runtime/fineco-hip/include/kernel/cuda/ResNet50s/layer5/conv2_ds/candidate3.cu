#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(56) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[14];
  __shared__ float pad_temp_shared[1800];
  __shared__ float kernel_shared[1152];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((15 <= ((int)threadIdx.x)) && (1 <= (((int)threadIdx.x) % 15))) ? data[((((rc_outer_outer * 1568) + ((((int)threadIdx.x) / 15) * 14)) + (((int)threadIdx.x) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 56)] = ((1 <= ((((int)threadIdx.x) + 11) % 15)) ? data[((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 56) / 15) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 112)] = ((1 <= ((((int)threadIdx.x) + 7) % 15)) ? data[((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 112) / 15) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 168)] = ((1 <= ((((int)threadIdx.x) + 3) % 15)) ? data[((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 168) / 15) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = (((15 <= ((((int)threadIdx.x) + 224) % 225)) && (1 <= ((((int)threadIdx.x) + 14) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 224) / 225) * 196)) + ((((((int)threadIdx.x) + 224) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 280)] = ((1 <= ((((int)threadIdx.x) + 10) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 280) / 225) * 196)) + ((((((int)threadIdx.x) + 55) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 336)] = ((1 <= ((((int)threadIdx.x) + 6) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 336) / 225) * 196)) + ((((((int)threadIdx.x) + 111) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 392)] = ((1 <= ((((int)threadIdx.x) + 2) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 392) / 225) * 196)) + ((((((int)threadIdx.x) + 167) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = (((15 <= ((((int)threadIdx.x) + 223) % 225)) && (1 <= ((((int)threadIdx.x) + 13) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 448) / 225) * 196)) + ((((((int)threadIdx.x) + 223) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 504)] = ((1 <= ((((int)threadIdx.x) + 9) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 504) / 225) * 196)) + ((((((int)threadIdx.x) + 54) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 560)] = ((1 <= ((((int)threadIdx.x) + 5) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 560) / 225) * 196)) + ((((((int)threadIdx.x) + 110) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 616)] = ((1 <= ((((int)threadIdx.x) + 1) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 616) / 225) * 196)) + ((((((int)threadIdx.x) + 166) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 672)] = (((15 <= ((((int)threadIdx.x) + 222) % 225)) && (1 <= ((((int)threadIdx.x) + 12) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 672) / 225) * 196)) + ((((((int)threadIdx.x) + 222) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 728)] = ((1 <= ((((int)threadIdx.x) + 8) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 728) / 225) * 196)) + ((((((int)threadIdx.x) + 53) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 784)] = ((1 <= ((((int)threadIdx.x) + 4) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 784) / 225) * 196)) + ((((((int)threadIdx.x) + 109) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 840)] = ((1 <= (((int)threadIdx.x) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 840) / 225) * 196)) + (((((int)threadIdx.x) / 15) + 11) * 14)) + (((int)threadIdx.x) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = (((15 <= ((((int)threadIdx.x) + 221) % 225)) && (1 <= ((((int)threadIdx.x) + 11) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 896) / 225) * 196)) + ((((((int)threadIdx.x) + 221) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 952)] = ((1 <= ((((int)threadIdx.x) + 7) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 952) / 225) * 196)) + ((((((int)threadIdx.x) + 52) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1008)] = ((1 <= ((((int)threadIdx.x) + 3) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1008) / 225) * 196)) + ((((((int)threadIdx.x) + 108) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1064)] = ((1 <= ((((int)threadIdx.x) + 14) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1064) / 225) * 196)) + ((((((int)threadIdx.x) + 164) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = (((15 <= ((((int)threadIdx.x) + 220) % 225)) && (1 <= ((((int)threadIdx.x) + 10) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1120) / 225) * 196)) + ((((((int)threadIdx.x) + 220) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = ((1 <= ((((int)threadIdx.x) + 6) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1176) / 225) * 196)) + ((((((int)threadIdx.x) + 51) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1232)] = ((1 <= ((((int)threadIdx.x) + 2) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1232) / 225) * 196)) + ((((((int)threadIdx.x) + 107) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1288)] = ((1 <= ((((int)threadIdx.x) + 13) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1288) / 225) * 196)) + ((((((int)threadIdx.x) + 163) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = (((15 <= ((((int)threadIdx.x) + 219) % 225)) && (1 <= ((((int)threadIdx.x) + 9) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1344) / 225) * 196)) + ((((((int)threadIdx.x) + 219) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1400)] = ((1 <= ((((int)threadIdx.x) + 5) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1400) / 225) * 196)) + ((((((int)threadIdx.x) + 50) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1456)] = ((1 <= ((((int)threadIdx.x) + 1) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1456) / 225) * 196)) + ((((((int)threadIdx.x) + 106) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1512)] = ((1 <= ((((int)threadIdx.x) + 12) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1512) / 225) * 196)) + ((((((int)threadIdx.x) + 162) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = (((15 <= ((((int)threadIdx.x) + 218) % 225)) && (1 <= ((((int)threadIdx.x) + 8) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1568) / 225) * 196)) + ((((((int)threadIdx.x) + 218) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1624)] = ((1 <= ((((int)threadIdx.x) + 4) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1624) / 225) * 196)) + ((((((int)threadIdx.x) + 49) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1680)] = ((1 <= (((int)threadIdx.x) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1680) / 225) * 196)) + (((((int)threadIdx.x) / 15) + 7) * 14)) + (((int)threadIdx.x) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1736)] = ((1 <= ((((int)threadIdx.x) + 11) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1736) / 225) * 196)) + ((((((int)threadIdx.x) + 161) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 15)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 8) {
      pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1792) / 225) * 196)) + ((((((int)threadIdx.x) + 217) % 225) / 15) * 14)) + (((int)threadIdx.x) + 7)) - 15)];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) * 73728) + (rc_outer_outer * 72)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 56)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 56) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 112) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 168)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 168) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 224) / 72) * 4608)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 8))];
    kernel_shared[(((int)threadIdx.x) + 280)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 280) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 336) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 392)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 392) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 448) / 72) * 4608)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 504)] = kernel[((((((int)blockIdx.x) * 73728) + (rc_outer_outer * 72)) + ((int)threadIdx.x)) + 32256)];
    kernel_shared[(((int)threadIdx.x) + 560)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 560) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 616)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 616) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 672) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 728)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 728) / 72) * 4608)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 8))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 784) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 840)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 840) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 896) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 952)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 952) / 72) * 4608)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[((((((int)blockIdx.x) * 73728) + (rc_outer_outer * 72)) + ((int)threadIdx.x)) + 64512)];
    kernel_shared[(((int)threadIdx.x) + 1064)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1064) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1120) / 72) * 4608)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 40))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int yy_c_outer_inner = 0; yy_c_outer_inner < 7; ++yy_c_outer_inner) {
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 72)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 73)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 74)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 225)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 9)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 225)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 81)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 226)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 10)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 226)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 82)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 227)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 11)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 227)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 83)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 450)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 18)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 450)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 90)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 451)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 19)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 451)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 91)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 452)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 20)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 452)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 92)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 675)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 27)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 675)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 99)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 676)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 28)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 676)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 100)]));
          conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 677)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 29)]));
          conv2d_nchw_local[(yy_c_outer_inner + 7)] = (conv2d_nchw_local[(yy_c_outer_inner + 7)] + (pad_temp_shared[(((((rc_outer_inner * 900) + (yy_c_outer_inner * 30)) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + 677)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + 101)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      conv2d_nchw[(((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 7) * 98)) + (ff_inner * 49)) + (yy_inner * 7)) + (((int)threadIdx.x) % 7))] = conv2d_nchw_local[((ff_inner * 7) + yy_inner)];
    }
  }
}


