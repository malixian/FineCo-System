#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[7];
  __shared__ float pad_temp_shared[648];
  __shared__ float kernel_shared[2304];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((((9 <= (((int)threadIdx.x) % 81)) && ((((int)threadIdx.x) % 81) < 72)) && (1 <= (((int)threadIdx.x) % 9))) && ((((int)threadIdx.x) % 9) < 8)) ? data[(((((rc_outer_outer * 392) + ((((int)threadIdx.x) / 81) * 49)) + (((((int)threadIdx.x) % 81) / 9) * 7)) + (((int)threadIdx.x) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = (((((9 <= ((((int)threadIdx.x) + 62) % 81)) && (((((int)threadIdx.x) + 62) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 8) % 9))) && (((((int)threadIdx.x) + 8) % 9) < 8)) ? data[(((((rc_outer_outer * 392) + (((((int)threadIdx.x) + 224) / 81) * 49)) + ((((((int)threadIdx.x) + 62) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 8) % 9)) - 8)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 200) {
      pad_temp_shared[(((int)threadIdx.x) + 448)] = (((((9 <= ((((int)threadIdx.x) + 43) % 81)) && (((((int)threadIdx.x) + 43) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 7) % 9))) && (((((int)threadIdx.x) + 7) % 9) < 8)) ? data[(((((rc_outer_outer * 392) + (((((int)threadIdx.x) + 448) / 81) * 49)) + ((((((int)threadIdx.x) + 43) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 7) % 9)) - 8)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) * 147456) + ((((int)threadIdx.x) / 72) * 4608)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 224) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 448) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 672) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 896) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 1120) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 1344) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 1568) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 1792) / 72) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((int)blockIdx.x) * 147456) + ((((int)threadIdx.x) / 72) * 4608)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 129024)];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((int)blockIdx.x) * 147456) + (((((int)threadIdx.x) + 2240) / 72) * 4608)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 8))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 9)] * kernel_shared[((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 27)] * kernel_shared[((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 36)] * kernel_shared[((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 45)] * kernel_shared[((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 54)] * kernel_shared[((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 9)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 27)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 36)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 45)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 54)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 63)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 27)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 36)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 45)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 54)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 63)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 72)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 81)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 9)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 90)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 9)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 99)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 9)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 108)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 9)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 117)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 9)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 126)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 9)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 135)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 9)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 90)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 12)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 99)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 12)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 108)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 12)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 117)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 12)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 126)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 12)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 135)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 12)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 144)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 12)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 99)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 15)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 108)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 15)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 117)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 15)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 126)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 15)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 135)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 15)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 144)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 15)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((rc_outer_inner * 162) + rx_outer_inner) + (((int)threadIdx.x) % 7)) + 153)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 72) + (rc_outer_inner * 18)) + rx_outer_inner) + 15)]));
      }
    }
  }
  conv2d_nchw[(((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7))] = conv2d_nchw_local[0];
  conv2d_nchw[((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 7)] = conv2d_nchw_local[1];
  conv2d_nchw[((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 14)] = conv2d_nchw_local[2];
  conv2d_nchw[((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 21)] = conv2d_nchw_local[3];
  conv2d_nchw[((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 28)] = conv2d_nchw_local[4];
  conv2d_nchw[((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 35)] = conv2d_nchw_local[5];
  conv2d_nchw[((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 42)] = conv2d_nchw_local[6];
}


