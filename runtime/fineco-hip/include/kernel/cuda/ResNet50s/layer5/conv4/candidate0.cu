#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(28) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[2];
  __shared__ float pad_temp_shared[448];
  __shared__ float kernel_shared[512];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    int2 _1 = make_int2(((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)))+(7*0), ((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)))+(7*1));
    *(float2*)(pad_temp_shared + (((int)threadIdx.x) * 2)) = make_float2(data[_1.x],data[_1.y]);
    int2 _2 = make_int2((((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 392))+(7*0), (((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 392))+(7*1));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 56)) = make_float2(data[_2.x],data[_2.y]);
    int2 _3 = make_int2((((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 784))+(7*0), (((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 784))+(7*1));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 112)) = make_float2(data[_3.x],data[_3.y]);
    int2 _4 = make_int2((((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 1176))+(7*0), (((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 1176))+(7*1));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 168)) = make_float2(data[_4.x],data[_4.y]);
    int2 _5 = make_int2((((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 1568))+(7*0), (((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 1568))+(7*1));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 224)) = make_float2(data[_5.x],data[_5.y]);
    int2 _6 = make_int2((((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 1960))+(7*0), (((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 1960))+(7*1));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 280)) = make_float2(data[_6.x],data[_6.y]);
    int2 _7 = make_int2((((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 2352))+(7*0), (((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 2352))+(7*1));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 336)) = make_float2(data[_7.x],data[_7.y]);
    int2 _8 = make_int2((((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 2744))+(7*0), (((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 14)) + (((int)blockIdx.x) % 7)) + 2744))+(7*1));
    *(float2*)(pad_temp_shared + ((((int)threadIdx.x) * 2) + 392)) = make_float2(data[_8.x],data[_8.y]);
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 64)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 28)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 64)) + ((int)threadIdx.x)) + 28)];
    kernel_shared[(((int)threadIdx.x) + 56)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 56) >> 6) * 2048)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63))];
    kernel_shared[(((int)threadIdx.x) + 84)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 84) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 20))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 112) >> 6) * 2048)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63))];
    kernel_shared[(((int)threadIdx.x) + 140)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 140) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 12))];
    kernel_shared[(((int)threadIdx.x) + 168)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 168) >> 6) * 2048)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63))];
    kernel_shared[(((int)threadIdx.x) + 196)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 196) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 4))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 224) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 32))];
    kernel_shared[(((int)threadIdx.x) + 252)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 252) >> 6) * 2048)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 60) & 63))];
    kernel_shared[(((int)threadIdx.x) + 280)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 280) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24))];
    kernel_shared[(((int)threadIdx.x) + 308)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 308) >> 6) * 2048)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 52) & 63))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 336) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 364)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 364) >> 6) * 2048)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 44) & 63))];
    kernel_shared[(((int)threadIdx.x) + 392)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 392) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8))];
    kernel_shared[(((int)threadIdx.x) + 420)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 420) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 36))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (rc_outer_outer * 64)) + ((int)threadIdx.x)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 476)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 476) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 28))];
    if (((int)threadIdx.x) < 8) {
      kernel_shared[(((int)threadIdx.x) + 504)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + (((((int)threadIdx.x) + 504) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 56))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 112) + (((int)threadIdx.x) % 7))] * kernel_shared[(((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16))]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((rc_outer_inner * 112) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 256)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 7)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 1)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 7)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 257)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 2)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 258)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 21)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 3)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 21)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 259)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 4)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 260)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 35)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 5)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 35)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 261)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 6)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 262)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 49)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 7)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 49)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 263)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 8)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 264)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 63)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 9)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 63)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 265)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 10)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 266)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 77)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 11)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 77)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 267)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 12)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 268)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 91)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 13)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 91)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 269)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 14)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 270)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 105)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 15)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (((int)threadIdx.x) % 7)) + 105)] * kernel_shared[((((((int)threadIdx.x) / 7) * 64) + (rc_outer_inner * 16)) + 271)]));
    }
  }
  conv2d_nchw[((((((int)blockIdx.x) / 7) * 392) + (((int)threadIdx.x) * 7)) + (((int)blockIdx.x) % 7))] = conv2d_nchw_local[0];
  conv2d_nchw[(((((((int)blockIdx.x) / 7) * 392) + (((int)threadIdx.x) * 7)) + (((int)blockIdx.x) % 7)) + 196)] = conv2d_nchw_local[1];
}


