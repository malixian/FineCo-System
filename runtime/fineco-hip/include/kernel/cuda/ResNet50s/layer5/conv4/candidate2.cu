#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[7];
  __shared__ float pad_temp_shared[448];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[(((rc_outer_outer * 3136) + (((int)threadIdx.x) * 7)) + (((int)blockIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = data[((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 7)) + (((int)blockIdx.x) % 7)) + 896)];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = data[((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 7)) + (((int)blockIdx.x) % 7)) + 1792)];
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[(((int)threadIdx.x) + 384)] = data[((((rc_outer_outer * 3136) + (((int)threadIdx.x) * 7)) + (((int)blockIdx.x) % 7)) + 2688)];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 262144) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4096)) + ((((int)threadIdx.x) >> 6) * 2048)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    }
    __syncthreads();
    for (int yy_c_outer_inner = 0; yy_c_outer_inner < 7; ++yy_c_outer_inner) {
      for (int rc_inner = 0; rc_inner < 64; ++rc_inner) {
        conv2d_nchw_local[yy_c_outer_inner] = (conv2d_nchw_local[yy_c_outer_inner] + (pad_temp_shared[((rc_inner * 7) + yy_c_outer_inner)] * kernel_shared[((((int)threadIdx.x) * 64) + rc_inner)]));
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
    conv2d_nchw[(((((((int)blockIdx.x) / 7) * 6272) + (((int)threadIdx.x) * 49)) + (yy_inner * 7)) + (((int)blockIdx.x) % 7))] = conv2d_nchw_local[yy_inner];
  }
}


