#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[7];
  __shared__ float pad_temp_shared[896];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 64)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 64) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 1) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 128) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 2) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 192)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 192) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 3) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 256) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 4) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 320)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 320) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 5) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 384)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 384) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 6) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 512)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 512) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 1) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 576)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 576) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 2) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 640)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 640) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 3) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 704)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 704) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 4) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 768)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 768) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 5) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 832)] = data[((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 832) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 6) % 7))];
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 128; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 1) * 2048)) + (rc_outer_outer * 128)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 1) * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 64; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(rc_outer_inner * 14)] * kernel_shared[((((int)threadIdx.x) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((rc_outer_inner * 14) + 1)] * kernel_shared[((((int)threadIdx.x) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_outer_inner * 14) + 2)] * kernel_shared[((((int)threadIdx.x) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((rc_outer_inner * 14) + 3)] * kernel_shared[((((int)threadIdx.x) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 14) + 4)] * kernel_shared[((((int)threadIdx.x) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((rc_outer_inner * 14) + 5)] * kernel_shared[((((int)threadIdx.x) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((rc_outer_inner * 14) + 6)] * kernel_shared[((((int)threadIdx.x) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 14) + 7)] * kernel_shared[(((((int)threadIdx.x) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((rc_outer_inner * 14) + 8)] * kernel_shared[(((((int)threadIdx.x) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_outer_inner * 14) + 9)] * kernel_shared[(((((int)threadIdx.x) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((rc_outer_inner * 14) + 10)] * kernel_shared[(((((int)threadIdx.x) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 14) + 11)] * kernel_shared[(((((int)threadIdx.x) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((rc_outer_inner * 14) + 12)] * kernel_shared[(((((int)threadIdx.x) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((rc_outer_inner * 14) + 13)] * kernel_shared[(((((int)threadIdx.x) * 128) + (rc_outer_inner * 2)) + 1)]));
    }
  }
  for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
    conv2d_nchw[(((((((int)blockIdx.x) / 7) * 3136) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + xx_inner)] = conv2d_nchw_local[xx_inner];
  }
}


