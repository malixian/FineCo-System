#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[7];
  __shared__ float pad_temp_shared[112];
  __shared__ float kernel_shared[8192];
  for (int xx_c_inner_init = 0; xx_c_inner_init < 7; ++xx_c_inner_init) {
    conv2d_nchw_local[xx_c_inner_init] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 112) {
      pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 784) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) / 7) * 262144) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16384)) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      for (int xx_c_inner = 0; xx_c_inner < 7; ++xx_c_inner) {
        conv2d_nchw_local[xx_c_inner] = (conv2d_nchw_local[xx_c_inner] + (pad_temp_shared[((rc_outer_inner * 7) + xx_c_inner)] * kernel_shared[((((int)threadIdx.x) * 16) + rc_outer_inner)]));
      }
    }
  }
  for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
    conv2d_nchw[(((((((int)blockIdx.x) / 7) * 25088) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + xx_inner)] = conv2d_nchw_local[xx_inner];
  }
}


