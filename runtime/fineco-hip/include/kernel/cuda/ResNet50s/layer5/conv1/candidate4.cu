#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[64];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) >> 2) * 196)) + ((((int)blockIdx.x) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 16384)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 24576)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 32768)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 40960)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 49152)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 65536)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 73728)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 81920)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 90112)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 98304)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 106496)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 114688)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 122880)];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 131072)];
    kernel_shared[(((int)threadIdx.x) + 2176)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 139264)];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 147456)];
    kernel_shared[(((int)threadIdx.x) + 2432)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 155648)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 163840)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 172032)];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 180224)];
    kernel_shared[(((int)threadIdx.x) + 2944)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 188416)];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 196608)];
    kernel_shared[(((int)threadIdx.x) + 3200)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 204800)];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 212992)];
    kernel_shared[(((int)threadIdx.x) + 3456)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 221184)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 229376)];
    kernel_shared[(((int)threadIdx.x) + 3712)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 237568)];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 245760)];
    kernel_shared[(((int)threadIdx.x) + 3968)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 253952)];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 262144)];
    kernel_shared[(((int)threadIdx.x) + 4224)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 270336)];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 278528)];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 286720)];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 294912)];
    kernel_shared[(((int)threadIdx.x) + 4736)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 303104)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 311296)];
    kernel_shared[(((int)threadIdx.x) + 4992)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 319488)];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 327680)];
    kernel_shared[(((int)threadIdx.x) + 5248)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 335872)];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 344064)];
    kernel_shared[(((int)threadIdx.x) + 5504)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 352256)];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 360448)];
    kernel_shared[(((int)threadIdx.x) + 5760)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 368640)];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 376832)];
    kernel_shared[(((int)threadIdx.x) + 6016)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 385024)];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 393216)];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 401408)];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 409600)];
    kernel_shared[(((int)threadIdx.x) + 6528)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 417792)];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 425984)];
    kernel_shared[(((int)threadIdx.x) + 6784)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 434176)];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 442368)];
    kernel_shared[(((int)threadIdx.x) + 7040)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 450560)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 458752)];
    kernel_shared[(((int)threadIdx.x) + 7296)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 466944)];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 475136)];
    kernel_shared[(((int)threadIdx.x) + 7552)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 483328)];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 491520)];
    kernel_shared[(((int)threadIdx.x) + 7808)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 499712)];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 507904)];
    kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[(((((((int)threadIdx.x) >> 4) * 1024) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 516096)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(rc_outer_inner * 16)] * kernel_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_outer_inner * 16) + 2)] * kernel_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(rc_outer_inner * 16)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2048)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((rc_outer_inner * 16) + 2)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2048)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(rc_outer_inner * 16)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4096)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((rc_outer_inner * 16) + 2)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4096)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(rc_outer_inner * 16)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6144)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 16) + 2)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6144)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 16) + 4)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_outer_inner * 16) + 6)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 16) + 4)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2049)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((rc_outer_inner * 16) + 6)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2049)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((rc_outer_inner * 16) + 4)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4097)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((rc_outer_inner * 16) + 6)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4097)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[((rc_outer_inner * 16) + 4)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6145)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 16) + 6)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6145)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 16) + 8)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_outer_inner * 16) + 10)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 16) + 8)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2050)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((rc_outer_inner * 16) + 10)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2050)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((rc_outer_inner * 16) + 8)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4098)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((rc_outer_inner * 16) + 10)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4098)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[((rc_outer_inner * 16) + 8)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6146)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 16) + 10)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6146)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 16) + 12)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((rc_outer_inner * 16) + 14)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 16) + 12)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2051)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((rc_outer_inner * 16) + 14)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2051)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((rc_outer_inner * 16) + 12)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4099)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((rc_outer_inner * 16) + 14)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4099)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[((rc_outer_inner * 16) + 12)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6147)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 16) + 14)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6147)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((rc_outer_inner * 16) + 1)] * kernel_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((rc_outer_inner * 16) + 3)] * kernel_shared[((((int)threadIdx.x) * 16) + (rc_outer_inner * 4))]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((rc_outer_inner * 16) + 1)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2048)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((rc_outer_inner * 16) + 3)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2048)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[((rc_outer_inner * 16) + 1)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4096)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[((rc_outer_inner * 16) + 3)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4096)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[((rc_outer_inner * 16) + 1)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6144)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[((rc_outer_inner * 16) + 3)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6144)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((rc_outer_inner * 16) + 5)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((rc_outer_inner * 16) + 7)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 1)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((rc_outer_inner * 16) + 5)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2049)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((rc_outer_inner * 16) + 7)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2049)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[((rc_outer_inner * 16) + 5)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4097)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[((rc_outer_inner * 16) + 7)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4097)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[((rc_outer_inner * 16) + 5)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6145)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[((rc_outer_inner * 16) + 7)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6145)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((rc_outer_inner * 16) + 9)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((rc_outer_inner * 16) + 11)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((rc_outer_inner * 16) + 9)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2050)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((rc_outer_inner * 16) + 11)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2050)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[((rc_outer_inner * 16) + 9)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4098)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[((rc_outer_inner * 16) + 11)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4098)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[((rc_outer_inner * 16) + 9)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6146)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[((rc_outer_inner * 16) + 11)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6146)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((rc_outer_inner * 16) + 13)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((rc_outer_inner * 16) + 15)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 3)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((rc_outer_inner * 16) + 13)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2051)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((rc_outer_inner * 16) + 15)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 2051)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[((rc_outer_inner * 16) + 13)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4099)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[((rc_outer_inner * 16) + 15)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 4099)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[((rc_outer_inner * 16) + 13)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6147)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[((rc_outer_inner * 16) + 15)] * kernel_shared[(((((int)threadIdx.x) * 16) + (rc_outer_inner * 4)) + 6147)]));
    }
  }
  for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
    conv2d_nchw[((((((int)threadIdx.x) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner)] = conv2d_nchw_local[xx_inner];
    conv2d_nchw[(((((((int)threadIdx.x) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner) + 14)] = conv2d_nchw_local[(xx_inner + 2)];
    conv2d_nchw[(((((((int)threadIdx.x) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner) + 25088)] = conv2d_nchw_local[(xx_inner + 4)];
    conv2d_nchw[(((((((int)threadIdx.x) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner) + 25102)] = conv2d_nchw_local[(xx_inner + 6)];
    conv2d_nchw[(((((((int)threadIdx.x) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner) + 50176)] = conv2d_nchw_local[(xx_inner + 8)];
    conv2d_nchw[(((((((int)threadIdx.x) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner) + 50190)] = conv2d_nchw_local[(xx_inner + 10)];
    conv2d_nchw[(((((((int)threadIdx.x) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner) + 75264)] = conv2d_nchw_local[(xx_inner + 12)];
    conv2d_nchw[(((((((int)threadIdx.x) * 196) + ((((int)blockIdx.x) / 7) * 28)) + ((((int)blockIdx.x) % 7) * 2)) + xx_inner) + 75278)] = conv2d_nchw_local[(xx_inner + 14)];
  }
}


