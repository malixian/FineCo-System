#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[1792];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = data[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 7840)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[(((((rc_outer_outer * 12544) + ((((int)threadIdx.x) / 28) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (((int)threadIdx.x) % 28)) + 10976)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 224) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 672) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 1120) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 21504)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 1568) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 2016) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 35840)];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 2464) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 43008)];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 2912) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 50176)];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 3360) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 3808) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 64512)];
    kernel_shared[(((int)threadIdx.x) + 4256)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 4256) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 71680)];
    kernel_shared[(((int)threadIdx.x) + 4704)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 4704) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 4928)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 78848)];
    kernel_shared[(((int)threadIdx.x) + 5152)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 5152) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 86016)];
    kernel_shared[(((int)threadIdx.x) + 5600)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 5600) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 5824)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 93184)];
    kernel_shared[(((int)threadIdx.x) + 6048)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 6048) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 100352)];
    kernel_shared[(((int)threadIdx.x) + 6496)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 6496) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 6720)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 107520)];
    kernel_shared[(((int)threadIdx.x) + 6944)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 6944) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 114688)];
    kernel_shared[(((int)threadIdx.x) + 7392)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 7392) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    kernel_shared[(((int)threadIdx.x) + 7616)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 121856)];
    kernel_shared[(((int)threadIdx.x) + 7840)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + (((((int)threadIdx.x) + 7840) >> 6) * 1024)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63))];
    if (((int)threadIdx.x) < 128) {
      kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 129024)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8))]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8))]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 29)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 57)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 85)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 113)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 141)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 169)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 197)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8))]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[(((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8))]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 43)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 71)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 99)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 127)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 155)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 183)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 211)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 64)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 64)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 65)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 29)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 65)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 66)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 57)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 66)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 67)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 85)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 67)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 68)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 113)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 68)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 69)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 141)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 69)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 70)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 169)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 70)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 71)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 197)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 71)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 64)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 64)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 65)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 43)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 65)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 66)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 71)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 66)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 67)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 99)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 67)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 68)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 127)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 68)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 69)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 155)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 69)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 70)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 183)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 70)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 71)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 211)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 71)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 128)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 128)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 129)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 29)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 129)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 130)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 57)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 130)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 131)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 85)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 131)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 132)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 113)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 132)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 133)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 141)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 133)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 134)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 169)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 134)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 135)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 197)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 135)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 128)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 128)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 129)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 43)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 129)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 130)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 71)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 130)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 131)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 99)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 131)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 132)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 127)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 132)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 133)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 155)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 133)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 134)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 183)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 134)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 135)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 211)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 135)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 192)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 192)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 193)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 29)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 193)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 194)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 57)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 194)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 195)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 85)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 195)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 196)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 113)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 196)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 197)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 141)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 197)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 198)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 169)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 198)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 199)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 197)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 199)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 192)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 192)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 193)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 43)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 193)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 194)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 71)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 194)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 195)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 99)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 195)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 196)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 127)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 196)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 197)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 155)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 197)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 198)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 183)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 198)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 199)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 224) + ((((int)threadIdx.x) % 7) * 2)) + 211)] * kernel_shared[((((((int)threadIdx.x) / 7) * 256) + (rc_outer_inner * 8)) + 199)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 7) * 784)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 4) + (yy_inner * 2)) + xx_inner)];
      }
    }
  }
}


