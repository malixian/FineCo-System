#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[32];
  __shared__ float pad_temp_shared[128];
  __shared__ float kernel_shared[128];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int xx_c_outer_inner_init = 0; xx_c_outer_inner_init < 2; ++xx_c_outer_inner_init) {
      for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
        for (int yy_c_inner_init = 0; yy_c_inner_init < 2; ++yy_c_inner_init) {
          for (int xx_c_inner_init = 0; xx_c_inner_init < 2; ++xx_c_inner_init) {
            conv2d_nchw_local[(((((ff_c_outer_inner_init * 16) + (ff_c_inner_init * 8)) + (yy_c_inner_init * 4)) + (xx_c_outer_inner_init * 2)) + xx_c_inner_init)] = 0.000000e+00f;
          }
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 6272) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 1) * 64) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int xx_c_outer_inner = 0; xx_c_outer_inner < 2; ++xx_c_outer_inner) {
          for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
            for (int yy_c_inner = 0; yy_c_inner < 2; ++yy_c_inner) {
              for (int xx_c_inner = 0; xx_c_inner < 2; ++xx_c_inner) {
                conv2d_nchw_local[(((((ff_c_outer_inner * 16) + (ff_c_inner * 8)) + (yy_c_inner * 4)) + (xx_c_outer_inner * 2)) + xx_c_inner)] = (conv2d_nchw_local[(((((ff_c_outer_inner * 16) + (ff_c_inner * 8)) + (yy_c_inner * 4)) + (xx_c_outer_inner * 2)) + xx_c_inner)] + (pad_temp_shared[((((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + (yy_c_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + (xx_c_outer_inner * 2)) + xx_c_inner)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 8) + (ff_c_outer_inner * 4)) + (ff_c_inner * 2)) + rc_outer_inner)]));
              }
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 4; ++xx_inner) {
        conv2d_nchw[(((((((((((int)threadIdx.x) >> 3) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 7) >> 1) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 8) + (yy_inner * 4)) + xx_inner)];
      }
    }
  }
}


