#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate7(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[4096];
  __shared__ float kernel_shared[2048];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x))] = data[(((((((rc_outer_outer * 200704) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 6272)) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 512)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1024)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1536)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 2048)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 2560)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 3072)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 4608)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 5120)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 5632)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 6144)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 6656)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[((((((((int)blockIdx.x) / 49) * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 7680)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 64; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 128)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[((((rc_outer_inner * 64) + (((((int)threadIdx.x) & 15) >> 3) * 32)) + (((int)threadIdx.x) & 7)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + rc_outer_inner) + 192)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
      conv2d_nchw[(((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 4) * 12544)) + (ff_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 3) * 224)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = conv2d_nchw_local[((ff_inner * 4) + yy_inner)];
    }
  }
}


