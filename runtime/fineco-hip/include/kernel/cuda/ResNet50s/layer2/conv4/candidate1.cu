#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[56];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[2048];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + (((((int)threadIdx.x) % 196) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 224) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 448) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 2) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 672) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 3) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 896) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1120) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1344) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 6) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + (((((int)threadIdx.x) % 196) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 1792) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2016) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 2) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2240) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 3) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2464) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2688) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 2912) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 6) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + (((((int)threadIdx.x) % 196) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3360) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3584) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 2) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 3808)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 3808) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 3) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 4032)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 4032) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 4256)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 4256) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 4480)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 4480) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 6) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + (((((int)threadIdx.x) % 196) / 28) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28)) + 75264)];
    pad_temp_shared[(((int)threadIdx.x) + 4928)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 4928) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 5152)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 5152) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 2) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 5376)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 5376) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 3) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 5600)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 5600) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 5824)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 5824) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 6048)] = data[((((((rc_outer_outer * 100352) + (((((int)threadIdx.x) + 6048) / 196) * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + ((((((int)threadIdx.x) / 28) + 6) % 7) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 28))];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5376)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8960)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12544)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) >> 5) * 256) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((rc_outer_outer * 32) + ((int)threadIdx.x)) + 16128)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + rc_outer_inner)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 32)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 64)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[((rc_outer_inner * 196) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
      conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[(((rc_outer_inner * 196) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + rc_outer_inner) + 96)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      conv2d_nchw[(((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 14))] = conv2d_nchw_local[((ff_inner * 7) + yy_inner)];
      conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 1) * 392)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 14)) + 14)] = conv2d_nchw_local[(((ff_inner * 7) + yy_inner) + 28)];
    }
  }
}


