#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[224];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[512];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
      for (int yy_c_inner_init = 0; yy_c_inner_init < 14; ++yy_c_inner_init) {
        conv2d_nchw_local[(((ff_c_outer_inner_init * 28) + (ff_c_inner_init * 14)) + yy_c_inner_init)] = 0.000000e+00f;
        conv2d_nchw_local[((((ff_c_outer_inner_init * 28) + (ff_c_inner_init * 14)) + yy_c_inner_init) + 56)] = 0.000000e+00f;
        conv2d_nchw_local[((((ff_c_outer_inner_init * 28) + (ff_c_inner_init * 14)) + yy_c_inner_init) + 112)] = 0.000000e+00f;
        conv2d_nchw_local[((((ff_c_outer_inner_init * 28) + (ff_c_inner_init * 14)) + yy_c_inner_init) + 168)] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 28; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + ((int)threadIdx.x))] = data[(((((rc_outer_outer * 25088) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 896)) + ((((int)threadIdx.x) / 14) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) >> 5)) < 16) {
        kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 224) + ((int)threadIdx.x))] = kernel[((((((((int)blockIdx.x) >> 2) * 4096) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1792)) + ((((int)threadIdx.x) >> 3) * 64)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
            for (int yy_c_inner = 0; yy_c_inner < 14; ++yy_c_inner) {
              conv2d_nchw_local[(((ff_c_outer_inner * 28) + (ff_c_inner * 14)) + yy_c_inner)] = (conv2d_nchw_local[(((ff_c_outer_inner * 28) + (ff_c_inner * 14)) + yy_c_inner)] + (pad_temp_shared[(((((rc_outer_inner * 1568) + (rc_inner * 784)) + (((((int)threadIdx.x) % 14) / 7) * 196)) + (yy_c_inner * 14)) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((((int)threadIdx.x) / 14) * 32) + (ff_c_outer_inner * 16)) + (ff_c_inner * 8)) + (rc_outer_inner * 2)) + rc_inner)]));
              conv2d_nchw_local[((((ff_c_outer_inner * 28) + (ff_c_inner * 14)) + yy_c_inner) + 56)] = (conv2d_nchw_local[((((ff_c_outer_inner * 28) + (ff_c_inner * 14)) + yy_c_inner) + 56)] + (pad_temp_shared[((((((rc_outer_inner * 1568) + (rc_inner * 784)) + (((((int)threadIdx.x) % 14) / 7) * 196)) + (yy_c_inner * 14)) + (((int)threadIdx.x) % 7)) + 7)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 32) + (ff_c_outer_inner * 16)) + (ff_c_inner * 8)) + (rc_outer_inner * 2)) + rc_inner)]));
              conv2d_nchw_local[((((ff_c_outer_inner * 28) + (ff_c_inner * 14)) + yy_c_inner) + 112)] = (conv2d_nchw_local[((((ff_c_outer_inner * 28) + (ff_c_inner * 14)) + yy_c_inner) + 112)] + (pad_temp_shared[((((((rc_outer_inner * 1568) + (rc_inner * 784)) + (((((int)threadIdx.x) % 14) / 7) * 196)) + (yy_c_inner * 14)) + (((int)threadIdx.x) % 7)) + 392)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 32) + (ff_c_outer_inner * 16)) + (ff_c_inner * 8)) + (rc_outer_inner * 2)) + rc_inner)]));
              conv2d_nchw_local[((((ff_c_outer_inner * 28) + (ff_c_inner * 14)) + yy_c_inner) + 168)] = (conv2d_nchw_local[((((ff_c_outer_inner * 28) + (ff_c_inner * 14)) + yy_c_inner) + 168)] + (pad_temp_shared[((((((rc_outer_inner * 1568) + (rc_inner * 784)) + (((((int)threadIdx.x) % 14) / 7) * 196)) + (yy_c_inner * 14)) + (((int)threadIdx.x) % 7)) + 399)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 32) + (ff_c_outer_inner * 16)) + (ff_c_inner * 8)) + (rc_outer_inner * 2)) + rc_inner)]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 14; ++yy_inner) {
      conv2d_nchw[((((((((((int)blockIdx.x) >> 2) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (ff_inner * 3136)) + (((((int)threadIdx.x) % 14) / 7) * 784)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 7))] = conv2d_nchw_local[((ff_inner * 14) + yy_inner)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (ff_inner * 3136)) + (((((int)threadIdx.x) % 14) / 7) * 784)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 7)) + 7)] = conv2d_nchw_local[(((ff_inner * 14) + yy_inner) + 56)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (ff_inner * 3136)) + (((((int)threadIdx.x) % 14) / 7) * 784)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 7)) + 1568)] = conv2d_nchw_local[(((ff_inner * 14) + yy_inner) + 112)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 200704) + ((((int)threadIdx.x) / 14) * 12544)) + (ff_inner * 3136)) + (((((int)threadIdx.x) % 14) / 7) * 784)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 7)) + 1575)] = conv2d_nchw_local[(((ff_inner * 14) + yy_inner) + 168)];
    }
  }
}


