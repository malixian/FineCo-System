#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[28];
  __shared__ float pad_temp_shared[1568];
  __shared__ float kernel_shared[2048];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 4; ++ff_c_outer_inner_init) {
    conv2d_nchw_local[ff_c_outer_inner_init] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_outer_inner_init + 4)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_outer_inner_init + 8)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_outer_inner_init + 12)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_outer_inner_init + 16)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_outer_inner_init + 20)] = 0.000000e+00f;
    conv2d_nchw_local[(ff_c_outer_inner_init + 24)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 50176) + ((((int)threadIdx.x) / 98) * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + (((((int)threadIdx.x) % 98) / 14) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 448) / 98) * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 896) / 98) * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 1) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    if (((int)threadIdx.x) < 224) {
      pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1344) / 98) * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + ((((((int)threadIdx.x) / 14) + 5) % 7) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 14))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 21504)];
    if (((int)threadIdx.x) < 256) {
      kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 28672)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          conv2d_nchw_local[ff_c_outer_inner] = (conv2d_nchw_local[ff_c_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 98)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + (((int)threadIdx.x) & 1))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 64) + (ff_c_outer_inner * 16)) + (rc_outer_inner * 8)) + rc_inner)]));
          conv2d_nchw_local[(ff_c_outer_inner + 4)] = (conv2d_nchw_local[(ff_c_outer_inner + 4)] + (pad_temp_shared[(((((rc_outer_inner * 784) + (rc_inner * 98)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + (((int)threadIdx.x) & 1)) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 64) + (ff_c_outer_inner * 16)) + (rc_outer_inner * 8)) + rc_inner)]));
          conv2d_nchw_local[(ff_c_outer_inner + 8)] = (conv2d_nchw_local[(ff_c_outer_inner + 8)] + (pad_temp_shared[(((((rc_outer_inner * 784) + (rc_inner * 98)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + (((int)threadIdx.x) & 1)) + 4)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 64) + (ff_c_outer_inner * 16)) + (rc_outer_inner * 8)) + rc_inner)]));
          conv2d_nchw_local[(ff_c_outer_inner + 12)] = (conv2d_nchw_local[(ff_c_outer_inner + 12)] + (pad_temp_shared[(((((rc_outer_inner * 784) + (rc_inner * 98)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + (((int)threadIdx.x) & 1)) + 6)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 64) + (ff_c_outer_inner * 16)) + (rc_outer_inner * 8)) + rc_inner)]));
          conv2d_nchw_local[(ff_c_outer_inner + 16)] = (conv2d_nchw_local[(ff_c_outer_inner + 16)] + (pad_temp_shared[(((((rc_outer_inner * 784) + (rc_inner * 98)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + (((int)threadIdx.x) & 1)) + 8)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 64) + (ff_c_outer_inner * 16)) + (rc_outer_inner * 8)) + rc_inner)]));
          conv2d_nchw_local[(ff_c_outer_inner + 20)] = (conv2d_nchw_local[(ff_c_outer_inner + 20)] + (pad_temp_shared[(((((rc_outer_inner * 784) + (rc_inner * 98)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + (((int)threadIdx.x) & 1)) + 10)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 64) + (ff_c_outer_inner * 16)) + (rc_outer_inner * 8)) + rc_inner)]));
          conv2d_nchw_local[(ff_c_outer_inner + 24)] = (conv2d_nchw_local[(ff_c_outer_inner + 24)] + (pad_temp_shared[(((((rc_outer_inner * 784) + (rc_inner * 98)) + (((((int)threadIdx.x) % 14) >> 1) * 14)) + (((int)threadIdx.x) & 1)) + 12)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 64) + (ff_c_outer_inner * 16)) + (rc_outer_inner * 8)) + rc_inner)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    conv2d_nchw[(((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + (((((int)threadIdx.x) % 14) >> 1) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) & 1))] = conv2d_nchw_local[ff_inner];
    conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + (((((int)threadIdx.x) % 14) >> 1) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) & 1)) + 2)] = conv2d_nchw_local[(ff_inner + 4)];
    conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + (((((int)threadIdx.x) % 14) >> 1) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) & 1)) + 4)] = conv2d_nchw_local[(ff_inner + 8)];
    conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + (((((int)threadIdx.x) % 14) >> 1) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) & 1)) + 6)] = conv2d_nchw_local[(ff_inner + 12)];
    conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + (((((int)threadIdx.x) % 14) >> 1) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) & 1)) + 8)] = conv2d_nchw_local[(ff_inner + 16)];
    conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + (((((int)threadIdx.x) % 14) >> 1) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) & 1)) + 10)] = conv2d_nchw_local[(ff_inner + 20)];
    conv2d_nchw[((((((((((int)threadIdx.x) / 14) * 12544) + (ff_inner * 3136)) + ((((int)blockIdx.x) >> 2) * 392)) + (((((int)threadIdx.x) % 14) >> 1) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) & 1)) + 12)] = conv2d_nchw_local[(ff_inner + 24)];
  }
}


