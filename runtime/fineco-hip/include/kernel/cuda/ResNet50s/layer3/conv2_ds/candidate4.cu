#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[4];
  __shared__ float pad_temp_shared[648];
  __shared__ float kernel_shared[9216];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((1 <= (((((int)blockIdx.x) / 7) * 8) + ((((int)threadIdx.x) % 81) / 9))) && (1 <= (((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) % 9)))) ? data[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 81) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) % 81) / 9) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) % 9)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 136) {
      pad_temp_shared[(((int)threadIdx.x) + 512)] = (((1 <= (((((int)blockIdx.x) / 7) * 8) + (((((int)threadIdx.x) + 26) % 81) / 9))) && (1 <= (((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 8) % 9)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 512) / 81) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + ((((((int)threadIdx.x) + 26) % 81) / 9) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) + 8) % 9)) - 57)] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x))] = kernel[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 3)) / 9) * 1152) + (rc_outer_outer * 72)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) % 72))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9))] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 2)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 4)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 6)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 1)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 3)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 5)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 7)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 2)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 4)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 6)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 324) + (rc_inner * 81)) + ((((int)threadIdx.x) & 3) * 18)) + (ry_inner * 9)) + 8)] * kernel_shared[((((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + (ry_inner * 3)) + 2)]));
        }
      }
    }
  }
  conv2d_nchw[(((((((int)threadIdx.x) >> 2) * 784) + ((((int)blockIdx.x) / 7) * 112)) + ((((int)threadIdx.x) & 3) * 28)) + ((((int)blockIdx.x) % 7) * 4))] = conv2d_nchw_local[0];
  conv2d_nchw[((((((((int)threadIdx.x) >> 2) * 784) + ((((int)blockIdx.x) / 7) * 112)) + ((((int)threadIdx.x) & 3) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 1)] = conv2d_nchw_local[1];
  conv2d_nchw[((((((((int)threadIdx.x) >> 2) * 784) + ((((int)blockIdx.x) / 7) * 112)) + ((((int)threadIdx.x) & 3) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 2)] = conv2d_nchw_local[2];
  conv2d_nchw[((((((((int)threadIdx.x) >> 2) * 784) + ((((int)blockIdx.x) / 7) * 112)) + ((((int)threadIdx.x) & 3) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + 3)] = conv2d_nchw_local[3];
}


