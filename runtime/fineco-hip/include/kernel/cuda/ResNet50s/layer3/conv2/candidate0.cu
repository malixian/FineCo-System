#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[4];
  __shared__ float pad_temp_shared[96];
  __shared__ float kernel_shared[1152];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      pad_temp_shared[(((int)threadIdx.x) * 3)] = ((((1 <= ((((((int)blockIdx.x) % 98) / 7) * 2) + ((((int)threadIdx.x) & 7) >> 1))) && (((((((int)blockIdx.x) % 98) / 7) * 2) + ((((int)threadIdx.x) & 7) >> 1)) < 29)) && (1 <= (((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) & 1) * 3)))) ? data[(((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (((((int)threadIdx.x) & 7) >> 1) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 3)) - 29)] : 0.000000e+00f);
      pad_temp_shared[((((int)threadIdx.x) * 3) + 1)] = (((1 <= ((((((int)blockIdx.x) % 98) / 7) * 2) + ((((int)threadIdx.x) & 7) >> 1))) && (((((((int)blockIdx.x) % 98) / 7) * 2) + ((((int)threadIdx.x) & 7) >> 1)) < 29)) ? data[(((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (((((int)threadIdx.x) & 7) >> 1) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 3)) - 28)] : 0.000000e+00f);
      pad_temp_shared[((((int)threadIdx.x) * 3) + 2)] = ((((1 <= ((((((int)blockIdx.x) % 98) / 7) * 2) + ((((int)threadIdx.x) & 7) >> 1))) && (((((((int)blockIdx.x) % 98) / 7) * 2) + ((((int)threadIdx.x) & 7) >> 1)) < 29)) && ((((((int)blockIdx.x) % 7) * 4) + ((((int)threadIdx.x) & 1) * 3)) < 27)) ? data[(((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) >> 3) * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (((((int)threadIdx.x) & 7) >> 1) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 3)) - 27)] : 0.000000e+00f);
    }
    *(float2*)(kernel_shared + (((int)threadIdx.x) * 2)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((int)threadIdx.x) / 18) * 1152)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) % 18) * 2)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 128)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((((int)threadIdx.x) * 2) + 128) / 36) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 2) + 20) % 36)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((((int)threadIdx.x) * 2) + 256) / 36) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 2) + 4) % 36)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 384)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((((int)threadIdx.x) * 2) + 384) / 36) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 2) + 24) % 36)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((((int)threadIdx.x) * 2) + 512) / 36) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 2) + 8) % 36)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 640)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((((int)threadIdx.x) * 2) + 640) / 36) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 2) + 28) % 36)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((((int)threadIdx.x) * 2) + 768) / 36) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 2) + 12) % 36)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 896)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((((int)threadIdx.x) * 2) + 896) / 36) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 2) + 32) % 36)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(kernel + (((((((int)blockIdx.x) / 98) * 36864) + ((((((int)threadIdx.x) * 2) + 1024) / 36) * 1152)) + (rc_outer_outer * 36)) + (((((int)threadIdx.x) * 2) + 16) % 36)));
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((int)threadIdx.x) & 3)] * kernel_shared[((((int)threadIdx.x) >> 2) * 72)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 6)] * kernel_shared[((((int)threadIdx.x) >> 2) * 72)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 3)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 3)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 6)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 18)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 6)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 24)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 9)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 9)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 12)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 12)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 15)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 42)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 15)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((int)threadIdx.x) & 3)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 36)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 36)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 39)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 39)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 42)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 18)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 42)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 24)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 45)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 45)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 48)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 48)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 51)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 42)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 51)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 1)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 1)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 4)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 13)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 4)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 13)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 7)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 19)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 7)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 25)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 10)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 10)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 13)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 13)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 16)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 43)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 16)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 37)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 37)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 40)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 13)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 40)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 13)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 43)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 19)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 43)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 25)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 46)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 46)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 49)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 49)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 52)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 43)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 52)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 2)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 2)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 5)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 14)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 5)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 14)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 8)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 8)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 26)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 11)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 11)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 14)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 38)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 14)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 38)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 17)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 44)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 17)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 38)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 38)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 41)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 14)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 41)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 14)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 44)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 44)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 26)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 47)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 47)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 50)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 38)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 50)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 38)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 53)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 44)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 53)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 48)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 18)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 54)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 18)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 54)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 21)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 60)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 21)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 60)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 24)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 66)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 24)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 72)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 27)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 78)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 27)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 78)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 30)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 84)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 30)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 84)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 33)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 90)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 33)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 48)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 54)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 54)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 54)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 54)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 57)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 60)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 57)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 60)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 60)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 66)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 60)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 72)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 63)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 78)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 63)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 78)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 66)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 84)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 66)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 84)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 69)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 90)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 69)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 49)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 19)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 55)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 19)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 55)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 22)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 61)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 22)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 61)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 25)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 67)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 25)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 73)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 28)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 79)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 28)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 79)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 31)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 85)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 31)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 85)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 34)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 91)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 34)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 49)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 55)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 55)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 55)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 55)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 58)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 61)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 58)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 61)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 61)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 67)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 61)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 73)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 64)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 79)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 79)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 67)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 85)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 67)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 85)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 70)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 91)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 70)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 50)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 20)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 56)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 20)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 56)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 23)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 62)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 23)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 62)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 26)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 68)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 26)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 74)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 29)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 80)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 29)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 80)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 32)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 86)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 32)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 86)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 35)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 92)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 35)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 50)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 56)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 56)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 56)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 56)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 59)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 62)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 59)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 62)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 62)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 68)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 62)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 74)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 65)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 80)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 65)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 80)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 68)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 86)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 68)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 86)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 71)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 92)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 72) + 71)]));
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[((((((((((int)blockIdx.x) / 98) * 25088) + ((((int)threadIdx.x) >> 2) * 1568)) + (ff_inner * 784)) + (((((int)blockIdx.x) % 98) / 7) * 56)) + (yy_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
    }
  }
}


