#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[2048];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 384)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 512)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 640)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 31360)];
    pad_temp_shared[(((int)threadIdx.x) + 768)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 37632)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 43904)];
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 1152)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 56448)];
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 62720)];
    pad_temp_shared[(((int)threadIdx.x) + 1408)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 68992)];
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 75264)];
    pad_temp_shared[(((int)threadIdx.x) + 1664)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 81536)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 87808)];
    pad_temp_shared[(((int)threadIdx.x) + 1920)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 4) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 94080)];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 512)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 1024)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 1536)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 2048)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 2560)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 3072)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 4608)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 5120)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 5632)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 6144)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 6656)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 7680)];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 2176)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 8704)];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 9216)];
    kernel_shared[(((int)threadIdx.x) + 2432)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 9728)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 10240)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 11264)];
    kernel_shared[(((int)threadIdx.x) + 2944)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 11776)];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 12288)];
    kernel_shared[(((int)threadIdx.x) + 3200)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 12800)];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 13312)];
    kernel_shared[(((int)threadIdx.x) + 3456)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 13824)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 3712)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 14848)];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 15360)];
    kernel_shared[(((int)threadIdx.x) + 3968)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 15872)];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 16384)];
    kernel_shared[(((int)threadIdx.x) + 4224)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 16896)];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 17408)];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 17920)];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 18432)];
    kernel_shared[(((int)threadIdx.x) + 4736)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 18944)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 19456)];
    kernel_shared[(((int)threadIdx.x) + 4992)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 19968)];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 20480)];
    kernel_shared[(((int)threadIdx.x) + 5248)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 20992)];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 21504)];
    kernel_shared[(((int)threadIdx.x) + 5504)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 22016)];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 22528)];
    kernel_shared[(((int)threadIdx.x) + 5760)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 23040)];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 23552)];
    kernel_shared[(((int)threadIdx.x) + 6016)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 24064)];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 24576)];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 25088)];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 25600)];
    kernel_shared[(((int)threadIdx.x) + 6528)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 26112)];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 26624)];
    kernel_shared[(((int)threadIdx.x) + 6784)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 27136)];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 27648)];
    kernel_shared[(((int)threadIdx.x) + 7040)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 28160)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 7296)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 29184)];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 29696)];
    kernel_shared[(((int)threadIdx.x) + 7552)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 30208)];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 30720)];
    kernel_shared[(((int)threadIdx.x) + 7808)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 31232)];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 31744)];
    kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[(((((((int)blockIdx.x) / 49) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 32256)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 128) + (((int)threadIdx.x) & 3))] * kernel_shared[(((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8))]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8))]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_outer_inner * 128) + (((int)threadIdx.x) & 3))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4096)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4096)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4097)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 24)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4097)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4098)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4098)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 56)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4099)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 56)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4099)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 72)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4100)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 72)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4100)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 88)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4101)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 88)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4101)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 104)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4102)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 104)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4102)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 120)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4103)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 120)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4103)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 4)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8))]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8))]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 4)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4096)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 12)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4096)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 20)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 28)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 1)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 20)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4097)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 28)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4097)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 36)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 44)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 2)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 36)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4098)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 44)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4098)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 52)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 60)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 3)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 52)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4099)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 60)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4099)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 68)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 76)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 68)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4100)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 76)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4100)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 84)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 92)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 5)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 84)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4101)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 92)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4101)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 100)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 108)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 6)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 100)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4102)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 108)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4102)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 116)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 124)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 7)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 116)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4103)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 3)) + 124)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 128) + (rc_outer_inner * 8)) + 4103)]));
    }
  }
  for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
    conv2d_nchw[(((((((((int)blockIdx.x) / 49) * 50176) + ((((int)threadIdx.x) >> 2) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (yy_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3))] = conv2d_nchw_local[yy_inner];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 50176) + ((((int)threadIdx.x) >> 2) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (yy_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 56)] = conv2d_nchw_local[(yy_inner + 2)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 50176) + ((((int)threadIdx.x) >> 2) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (yy_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 25088)] = conv2d_nchw_local[(yy_inner + 4)];
    conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 50176) + ((((int)threadIdx.x) >> 2) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + (yy_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 25144)] = conv2d_nchw_local[(yy_inner + 6)];
  }
}


