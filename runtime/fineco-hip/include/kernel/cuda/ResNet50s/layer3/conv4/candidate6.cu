#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate6(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[3136];
  __shared__ float kernel_shared[128];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[(((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x))];
    pad_temp_shared[(((int)threadIdx.x) + 196)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 392)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 588)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 980)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 3920)];
    pad_temp_shared[(((int)threadIdx.x) + 1176)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 1372)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 5488)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 1764)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 7056)];
    pad_temp_shared[(((int)threadIdx.x) + 1960)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 7840)];
    pad_temp_shared[(((int)threadIdx.x) + 2156)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 8624)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 2548)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 10192)];
    pad_temp_shared[(((int)threadIdx.x) + 2744)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 10976)];
    pad_temp_shared[(((int)threadIdx.x) + 2940)] = data[((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 196)) + ((int)threadIdx.x)) + 11760)];
    if (((int)threadIdx.x) < 128) {
      kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 98) * 2)] * kernel_shared[((((int)threadIdx.x) / 98) * 64)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1)] * kernel_shared[((((int)threadIdx.x) / 98) * 64)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) % 98) * 2)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 16)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 16)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 196)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 1)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 197)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 1)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 196)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 17)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 197)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 17)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((int)threadIdx.x) % 98) * 2)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 32)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 32)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((int)threadIdx.x) % 98) * 2)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 48)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 48)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 196)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 33)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 197)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 33)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 196)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 49)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 197)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 49)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 392)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 2)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 393)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 2)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 392)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 18)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 393)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 18)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 588)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 3)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 589)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 3)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 588)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 19)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 589)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 19)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 392)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 34)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 393)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 34)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 392)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 50)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 393)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 50)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 588)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 35)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 589)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 35)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 588)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 51)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 589)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 51)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 784)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 4)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 785)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 4)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 784)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 20)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 785)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 20)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 980)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 5)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 981)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 5)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 980)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 21)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 981)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 21)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 784)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 36)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 785)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 36)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 784)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 52)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 785)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 52)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 980)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 37)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 981)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 37)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 980)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 53)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 981)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 53)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1176)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 6)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1177)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 6)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1176)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 22)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1177)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 22)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1372)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 7)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1373)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 7)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1372)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 23)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1373)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 23)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1176)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 38)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1177)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 38)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1176)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 54)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1177)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 54)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1372)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 39)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1373)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 39)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1372)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 55)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1373)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 55)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1568)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 8)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1569)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 8)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1568)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 24)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1569)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 24)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1764)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 9)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1765)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 9)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1764)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 25)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1765)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 25)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1568)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 40)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1569)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 40)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1568)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 56)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1569)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 56)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1764)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 41)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1765)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 41)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1764)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 57)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1765)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 57)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1960)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 10)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1961)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 10)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1960)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 26)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1961)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 26)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2156)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 11)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2157)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 11)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2156)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 27)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2157)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 27)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1960)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 42)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1961)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 42)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1960)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 58)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 1961)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 58)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2156)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 43)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2157)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 43)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2156)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 59)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2157)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 59)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2352)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 12)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2353)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 12)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2352)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 28)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2353)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 28)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2548)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 13)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2549)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 13)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2548)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 29)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2549)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 29)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2352)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 44)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2353)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 44)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2352)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 60)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2353)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 60)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2548)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 45)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2549)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 45)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2548)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 61)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2549)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 61)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2744)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 14)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2745)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 14)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2744)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 30)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2745)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 30)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2940)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 15)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2941)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 15)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2940)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 31)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2941)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 31)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2744)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 46)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2745)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 46)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2744)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 62)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2745)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 62)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2940)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 47)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2941)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 47)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2940)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 63)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) % 98) * 2) + 2941)] * kernel_shared[(((((int)threadIdx.x) / 98) * 64) + 63)]));
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      conv2d_nchw[(((((((((int)blockIdx.x) >> 2) * 6272) + ((((int)threadIdx.x) / 98) * 3136)) + (ff_inner * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((int)threadIdx.x) % 98) * 2)) + xx_inner)] = conv2d_nchw_local[((ff_inner * 2) + xx_inner)];
    }
  }
}


