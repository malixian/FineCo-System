#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(98) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[32];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[512];
  for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
    conv2d_nchw_local[(ff_c_inner_init * 2)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 4)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 8)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 12)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 16)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 20)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 24)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 28)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 1)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 5)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 9)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 13)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 17)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 21)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 25)] = 0.000000e+00f;
    conv2d_nchw_local[((ff_c_inner_init * 2) + 29)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 98) + ((int)threadIdx.x))] = data[(((((((rc_outer_outer * 25088) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 1) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 1) * 196)) + ((((int)threadIdx.x) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 98)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + (((((int)threadIdx.x) + 98) >> 5) * 512)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 2) & 31))];
    kernel_shared[(((int)threadIdx.x) + 196)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + (((((int)threadIdx.x) + 196) >> 5) * 512)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 4) & 31))];
    kernel_shared[(((int)threadIdx.x) + 294)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + (((((int)threadIdx.x) + 294) >> 5) * 512)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 6) & 31))];
    kernel_shared[(((int)threadIdx.x) + 392)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + (((((int)threadIdx.x) + 392) >> 5) * 512)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) + 8) & 31))];
    if (((int)threadIdx.x) < 22) {
      kernel_shared[(((int)threadIdx.x) + 490)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + (((((int)threadIdx.x) + 490) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) + 10))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
          conv2d_nchw_local[(ff_c_inner * 2)] = (conv2d_nchw_local[(ff_c_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2))] * kernel_shared[(((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 4)] = (conv2d_nchw_local[((ff_c_inner * 2) + 4)] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2))] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 64)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 8)] = (conv2d_nchw_local[((ff_c_inner * 2) + 8)] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2))] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 128)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 12)] = (conv2d_nchw_local[((ff_c_inner * 2) + 12)] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2))] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 192)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 16)] = (conv2d_nchw_local[((ff_c_inner * 2) + 16)] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2))] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 256)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 20)] = (conv2d_nchw_local[((ff_c_inner * 2) + 20)] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2))] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 320)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 24)] = (conv2d_nchw_local[((ff_c_inner * 2) + 24)] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2))] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 384)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 28)] = (conv2d_nchw_local[((ff_c_inner * 2) + 28)] + (pad_temp_shared[(((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2))] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 448)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 1)] = (conv2d_nchw_local[((ff_c_inner * 2) + 1)] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2)) + 1)] * kernel_shared[(((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 5)] = (conv2d_nchw_local[((ff_c_inner * 2) + 5)] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2)) + 1)] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 64)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 9)] = (conv2d_nchw_local[((ff_c_inner * 2) + 9)] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2)) + 1)] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 128)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 13)] = (conv2d_nchw_local[((ff_c_inner * 2) + 13)] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2)) + 1)] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 192)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 17)] = (conv2d_nchw_local[((ff_c_inner * 2) + 17)] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2)) + 1)] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 256)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 21)] = (conv2d_nchw_local[((ff_c_inner * 2) + 21)] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2)) + 1)] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 320)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 25)] = (conv2d_nchw_local[((ff_c_inner * 2) + 25)] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2)) + 1)] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 384)]));
          conv2d_nchw_local[((ff_c_inner * 2) + 29)] = (conv2d_nchw_local[((ff_c_inner * 2) + 29)] + (pad_temp_shared[((((rc_outer_inner * 784) + (rc_inner * 196)) + (((int)threadIdx.x) * 2)) + 1)] * kernel_shared[((((ff_c_inner * 32) + (rc_outer_inner * 4)) + rc_inner) + 448)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      conv2d_nchw[((((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((int)threadIdx.x) / 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner)] = conv2d_nchw_local[((ff_inner * 2) + xx_inner)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((int)threadIdx.x) / 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 1568)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 4)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((int)threadIdx.x) / 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 3136)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 8)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((int)threadIdx.x) / 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 4704)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 12)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((int)threadIdx.x) / 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 6272)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 16)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((int)threadIdx.x) / 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 7840)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 20)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((int)threadIdx.x) / 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 9408)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 24)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 12544) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((int)threadIdx.x) / 7) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 10976)] = conv2d_nchw_local[(((ff_inner * 2) + xx_inner) + 28)];
    }
  }
}


