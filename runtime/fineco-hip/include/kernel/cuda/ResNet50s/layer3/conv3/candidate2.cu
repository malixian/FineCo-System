#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[64];
  __shared__ float pad_temp_shared[3584];
  __shared__ float kernel_shared[4096];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[56] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[57] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[58] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[59] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[60] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[61] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[62] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[63] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 7840)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 10976)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 14112)];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 15680)];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 17248)];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 20384)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 21952)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 23520)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 896)];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2688)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4480)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5376)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6272)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8064)];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8960)];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 9856)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 10752)];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 11648)];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12544)];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 13440)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 15232)];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 16128)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
        for (int yy_c_outer_inner = 0; yy_c_outer_inner < 2; ++yy_c_outer_inner) {
          conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] + (pad_temp_shared[((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8))]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8))]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] + (pad_temp_shared[((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2048)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 14)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2048)]));
          conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 112)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 1)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 126)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 1)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 112)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2049)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 126)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2049)]));
          conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 224)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 238)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 224)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2050)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 238)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2050)]));
          conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 336)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 3)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 350)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 3)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 336)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2051)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 350)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2051)]));
          conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 448)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 4)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 462)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 4)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 448)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2052)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 462)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2052)]));
          conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 560)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 5)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 574)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 5)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 560)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2053)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 574)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2053)]));
          conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 672)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 6)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 686)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 6)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 672)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2054)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 686)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2054)]));
          conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] = (conv2d_nchw_local[((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2))] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 784)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 7)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 798)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 7)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 32)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 784)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2055)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 48)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 798)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2055)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8))]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8))]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2048)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2048)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 113)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 1)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 127)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 1)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 113)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2049)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 127)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2049)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 225)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 239)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 225)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2050)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 239)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2050)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 337)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 3)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 351)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 3)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 337)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2051)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 351)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2051)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 449)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 4)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 463)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 4)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 449)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2052)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 463)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2052)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 561)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 5)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 575)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 5)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 561)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2053)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 575)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2053)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 673)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 6)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 687)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 6)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 673)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2054)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 687)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2054)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 785)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 7)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 799)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 7)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 33)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 785)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2055)]));
          conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (yy_c_outer_inner * 2)) + 49)] + (pad_temp_shared[(((((rc_outer_inner * 896) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + 799)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 128) + (ff_c_outer_inner * 32)) + (rc_outer_inner * 8)) + 2055)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        conv2d_nchw[(((((((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (ff_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 4) + (yy_inner * 2)) + xx_inner)];
        conv2d_nchw[((((((((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (ff_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 14)] = conv2d_nchw_local[((((ff_inner * 4) + (yy_inner * 2)) + xx_inner) + 16)];
        conv2d_nchw[((((((((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (ff_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 50176)] = conv2d_nchw_local[((((ff_inner * 4) + (yy_inner * 2)) + xx_inner) + 32)];
        conv2d_nchw[((((((((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (ff_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((int)threadIdx.x) % 14) / 7) * 56)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 7) * 2)) + xx_inner) + 50190)] = conv2d_nchw_local[((((ff_inner * 4) + (yy_inner * 2)) + xx_inner) + 48)];
      }
    }
  }
}


