#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[56];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[2048];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 196) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 224) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 2) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 448) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 672) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 6) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 896) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 8) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1120) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 10) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1344) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 12) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 196) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1792) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 2) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2016) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2240) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 6) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2464) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 8) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2688) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 10) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2912) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 12) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = data[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 196) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3360) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 2) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3584) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 3808)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3808) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 6) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 4032)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 4032) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 8) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 4256)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 4256) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 10) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 4480)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 4480) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 12) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = data[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (((((int)threadIdx.x) % 196) / 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 4928)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 4928) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 2) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 5152)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 5152) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 4) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 5376)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 5376) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 6) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 5600)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 5600) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 8) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 5824)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 5824) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 10) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    pad_temp_shared[(((int)threadIdx.x) + 6048)] = data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 6048) / 196) * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + ((((((int)threadIdx.x) / 14) + 12) % 14) * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 896)];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2688)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4480)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5376)];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6272)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) >> 2) * 8192) + ((((int)threadIdx.x) >> 5) * 128)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((int)blockIdx.x) >> 2) * 8192) + (rc_outer_outer * 32)) + ((int)threadIdx.x)) + 8064)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_outer_inner * 392) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[(((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2))]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[28] = (conv2d_nchw_local[28] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 294)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[29] = (conv2d_nchw_local[29] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 308)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 224)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[30] = (conv2d_nchw_local[30] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 322)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 238)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[31] = (conv2d_nchw_local[31] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 336)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 252)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[32] = (conv2d_nchw_local[32] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 350)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 266)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[33] = (conv2d_nchw_local[33] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 364)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 280)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[34] = (conv2d_nchw_local[34] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 378)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((rc_outer_inner * 392) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 32)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[35] = (conv2d_nchw_local[35] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 294)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[36] = (conv2d_nchw_local[36] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 308)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 224)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[37] = (conv2d_nchw_local[37] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 322)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 238)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[38] = (conv2d_nchw_local[38] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 336)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 252)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[39] = (conv2d_nchw_local[39] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 350)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 266)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[40] = (conv2d_nchw_local[40] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 364)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 280)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[41] = (conv2d_nchw_local[41] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 378)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 33)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((rc_outer_inner * 392) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[42] = (conv2d_nchw_local[42] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 294)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[43] = (conv2d_nchw_local[43] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 308)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 224)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[44] = (conv2d_nchw_local[44] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 322)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 238)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[45] = (conv2d_nchw_local[45] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 336)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 252)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[46] = (conv2d_nchw_local[46] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 350)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 266)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[47] = (conv2d_nchw_local[47] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 364)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 280)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[48] = (conv2d_nchw_local[48] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 378)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[((rc_outer_inner * 392) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 98)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 14)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 126)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 42)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 140)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 154)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 70)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 182)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 96)]));
      conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 196)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[49] = (conv2d_nchw_local[49] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 294)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 210)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[50] = (conv2d_nchw_local[50] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 308)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 224)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[51] = (conv2d_nchw_local[51] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 322)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 238)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[52] = (conv2d_nchw_local[52] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 336)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 252)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[53] = (conv2d_nchw_local[53] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 350)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 266)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[54] = (conv2d_nchw_local[54] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 364)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 280)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
      conv2d_nchw_local[55] = (conv2d_nchw_local[55] + (pad_temp_shared[(((rc_outer_inner * 392) + (((int)threadIdx.x) % 14)) + 378)] * kernel_shared[((((((int)threadIdx.x) / 14) * 128) + (rc_outer_inner * 2)) + 97)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      conv2d_nchw[((((((((((int)blockIdx.x) >> 2) * 50176) + ((((int)threadIdx.x) / 14) * 3136)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (yy_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14))] = conv2d_nchw_local[((ff_inner * 7) + yy_inner)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 2) * 50176) + ((((int)threadIdx.x) / 14) * 3136)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 3) >> 1) * 392)) + (yy_inner * 28)) + ((((int)blockIdx.x) & 1) * 14)) + (((int)threadIdx.x) % 14)) + 196)] = conv2d_nchw_local[(((ff_inner * 7) + yy_inner) + 28)];
    }
  }
}


