#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate7(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[4];
  __shared__ float pad_temp_shared[64];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) >> 2) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 16384)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 24576)];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 32768)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 40960)];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 49152)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 65536)];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 73728)];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 81920)];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 90112)];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 98304)];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 106496)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 114688)];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[((((((((int)blockIdx.x) / 49) * 131072) + ((((int)threadIdx.x) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 122880)];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((int)threadIdx.x) & 3)] * kernel_shared[((((int)threadIdx.x) >> 2) * 16)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((int)threadIdx.x) & 3)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2048)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((int)threadIdx.x) & 3)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4096)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((int)threadIdx.x) & 3)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6144)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 4)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 4)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2049)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 4)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4097)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 4)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6145)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2050)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4098)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6146)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2051)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4099)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6147)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2052)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4100)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6148)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2053)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4101)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6149)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 24)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 24)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2054)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 24)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4102)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 24)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6150)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 28)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 28)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2055)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 28)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4103)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 28)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6151)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2056)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4104)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6152)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2057)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4105)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6153)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2058)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4106)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6154)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 44)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 44)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2059)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 44)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4107)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 44)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6155)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 48)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 48)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2060)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 48)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4108)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 48)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6156)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 52)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 52)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2061)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 52)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4109)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 52)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6157)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 56)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 56)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2062)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 56)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4110)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 56)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6158)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 60)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 60)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 2063)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 60)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 4111)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[((((int)threadIdx.x) & 3) + 60)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 16) + 6159)]));
  }
  conv2d_nchw[(((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 2) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))] = conv2d_nchw_local[0];
  conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 2) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 25088)] = conv2d_nchw_local[1];
  conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 2) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 50176)] = conv2d_nchw_local[2];
  conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 2) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 75264)] = conv2d_nchw_local[3];
}


