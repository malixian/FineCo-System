#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[2];
  __shared__ float pad_temp_shared[200];
  __shared__ float kernel_shared[9216];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 40) {
      pad_temp_shared[(((int)threadIdx.x) * 5)] = (((1 <= ((((((int)blockIdx.x) % 49) / 7) * 4) + (((int)threadIdx.x) % 5))) && (1 <= (((int)blockIdx.x) % 7))) ? data[((((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 5) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 5) * 28)) + ((((int)blockIdx.x) % 7) * 4)) - 29)] : 0.000000e+00f);
      pad_temp_shared[((((int)threadIdx.x) * 5) + 1)] = ((1 <= ((((((int)blockIdx.x) % 49) / 7) * 4) + (((int)threadIdx.x) % 5))) ? data[((((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 5) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 5) * 28)) + ((((int)blockIdx.x) % 7) * 4)) - 28)] : 0.000000e+00f);
      pad_temp_shared[((((int)threadIdx.x) * 5) + 2)] = ((1 <= ((((((int)blockIdx.x) % 49) / 7) * 4) + (((int)threadIdx.x) % 5))) ? data[((((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 5) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 5) * 28)) + ((((int)blockIdx.x) % 7) * 4)) - 27)] : 0.000000e+00f);
      pad_temp_shared[((((int)threadIdx.x) * 5) + 3)] = ((1 <= ((((((int)blockIdx.x) % 49) / 7) * 4) + (((int)threadIdx.x) % 5))) ? data[((((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 5) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 5) * 28)) + ((((int)blockIdx.x) % 7) * 4)) - 26)] : 0.000000e+00f);
      pad_temp_shared[((((int)threadIdx.x) * 5) + 4)] = ((1 <= ((((((int)blockIdx.x) % 49) / 7) * 4) + (((int)threadIdx.x) % 5))) ? data[((((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 5) * 784)) + (((((int)blockIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 5) * 28)) + ((((int)blockIdx.x) % 7) * 4)) - 25)] : 0.000000e+00f);
    }
    *(float4*)(kernel_shared + (((int)threadIdx.x) * 4)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((int)threadIdx.x) / 18) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 18) * 4)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((((int)threadIdx.x) * 4) + 1024) / 72) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) * 4) + 16) % 72)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((((int)threadIdx.x) * 4) + 2048) / 72) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) * 4) + 32) % 72)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((((int)threadIdx.x) * 4) + 3072) / 72) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) * 4) + 48) % 72)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 4096)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((((int)threadIdx.x) * 4) + 4096) / 72) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) * 4) + 64) % 72)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 5120)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((((int)threadIdx.x) * 4) + 5120) / 72) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) * 4) + 8) % 72)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 6144)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((((int)threadIdx.x) * 4) + 6144) / 72) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) * 4) + 24) % 72)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 7168)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((((int)threadIdx.x) * 4) + 7168) / 72) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) * 4) + 40) % 72)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 8192)) = *(float4*)(kernel + (((((((int)blockIdx.x) / 49) * 294912) + ((((((int)threadIdx.x) * 4) + 8192) / 72) * 2304)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) * 4) + 56) % 72)));
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2))] * kernel_shared[((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9))]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2))] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4608)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 1)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4609)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 2)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 2)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 2)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4610)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 5)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 3)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 5)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4611)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 6)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 6)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4612)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 7)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 5)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 7)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4613)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 10)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 6)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 10)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4614)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 11)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 7)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 11)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4615)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 12)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 8)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (rc_inner * 25)) + (((((int)threadIdx.x) & 3) >> 1) * 10)) + ((((int)threadIdx.x) & 1) * 2)) + 12)] * kernel_shared[(((((((int)threadIdx.x) >> 2) * 72) + (rc_outer_inner * 36)) + (rc_inner * 9)) + 4616)]));
      }
    }
  }
  conv2d_nchw[(((((((((int)blockIdx.x) / 49) * 25088) + ((((int)threadIdx.x) >> 2) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))] = conv2d_nchw_local[0];
  conv2d_nchw[((((((((((int)blockIdx.x) / 49) * 25088) + ((((int)threadIdx.x) >> 2) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (((((int)threadIdx.x) & 3) >> 1) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 12544)] = conv2d_nchw_local[1];
}


