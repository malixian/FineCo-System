#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[50];
  __shared__ float kernel_shared[4608];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 50) {
      pad_temp_shared[((int)threadIdx.x)] = (((1 <= (((((int)blockIdx.x) / 7) * 4) + ((((int)threadIdx.x) % 25) / 5))) && (1 <= (((((int)blockIdx.x) % 7) * 4) + (((int)threadIdx.x) % 5)))) ? data[(((((((rc_outer_outer * 1568) + ((((int)threadIdx.x) / 25) * 784)) + ((((int)blockIdx.x) / 7) * 112)) + (((((int)threadIdx.x) % 25) / 5) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) % 5)) - 29)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 18) * 2304) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[(((((((int)threadIdx.x) + 128) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) + 256) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[(((((((int)threadIdx.x) + 384) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) + 512) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[(((((((int)threadIdx.x) + 640) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) + 768) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)threadIdx.x) + 896) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) + 1024) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[(((((((int)threadIdx.x) / 18) * 2304) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18)) + 147456)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) + 1280) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[(((((((int)threadIdx.x) + 1408) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) + 1536) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[(((((((int)threadIdx.x) + 1664) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) + 1792) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[(((((((int)threadIdx.x) + 1920) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) + 2048) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2176)] = kernel[(((((((int)threadIdx.x) + 2176) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((int)threadIdx.x) / 18) * 2304) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18)) + 294912)];
    kernel_shared[(((int)threadIdx.x) + 2432)] = kernel[(((((((int)threadIdx.x) + 2432) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) + 2560) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((int)threadIdx.x) + 2688) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)threadIdx.x) + 2816) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2944)] = kernel[(((((((int)threadIdx.x) + 2944) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) + 3072) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3200)] = kernel[(((((((int)threadIdx.x) + 3200) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)threadIdx.x) + 3328) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3456)] = kernel[(((((((int)threadIdx.x) / 18) * 2304) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18)) + 442368)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) + 3584) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3712)] = kernel[(((((((int)threadIdx.x) + 3712) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)threadIdx.x) + 3840) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3968)] = kernel[(((((((int)threadIdx.x) + 3968) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) + 4096) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4224)] = kernel[(((((((int)threadIdx.x) + 4224) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)threadIdx.x) + 4352) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[(((((((int)threadIdx.x) + 4480) / 18) * 2304) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 1) * 2)] * kernel_shared[((((int)threadIdx.x) >> 1) * 36)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((((int)threadIdx.x) & 1) * 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2304)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[((((int)threadIdx.x) >> 1) * 36)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2304)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 5)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 5)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2307)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 15)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 3)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 15)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2307)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2310)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 6)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2310)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 25)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 25)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2313)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 9)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2313)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2316)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 12)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2316)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2319)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 45)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 15)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 45)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2319)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 1) * 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[((((int)threadIdx.x) & 1) * 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2322)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 18)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2322)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 5)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 5)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2325)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 15)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 21)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 15)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2325)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2328)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 24)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2328)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 25)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 25)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2331)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 27)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2331)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2334)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 30)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2334)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2337)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 45)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 33)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 45)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2337)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2305)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 1)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2305)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2308)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 4)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2308)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2311)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 21)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 7)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 21)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2311)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 26)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 26)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2314)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 10)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2314)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2317)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 41)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 13)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 41)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2317)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2320)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 46)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 16)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 46)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2320)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2323)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 19)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2323)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2326)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 22)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2326)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2329)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 21)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 25)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 21)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2329)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 26)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 26)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2332)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 28)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2332)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2335)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 41)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 31)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 41)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2335)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2338)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 46)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 34)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 46)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2338)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2306)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2306)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2309)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 17)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 5)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 17)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2309)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2312)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 22)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 8)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 22)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2312)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 27)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 27)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2315)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 11)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2315)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2318)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 42)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 14)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 42)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2318)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
    conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2321)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 47)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 17)]));
    conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 47)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2321)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2324)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 20)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2324)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2327)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 17)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 23)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 17)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2327)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2330)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 22)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 26)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 22)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2330)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 27)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 27)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2333)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 29)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2333)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2336)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 42)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 32)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 42)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2336)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
    conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2339)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 47)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 35)]));
    conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 47)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 36) + 2339)]));
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[(((((((((int)threadIdx.x) >> 1) * 392) + (ff_inner * 196)) + ((((int)blockIdx.x) / 7) * 28)) + (yy_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
      conv2d_nchw[((((((((((int)threadIdx.x) >> 1) * 392) + (ff_inner * 196)) + ((((int)blockIdx.x) / 7) * 28)) + (yy_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 25088)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 4)];
    }
  }
}


