#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate5(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[3584];
  __shared__ float kernel_shared[1024];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 9408)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 15680)];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 112) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((int)threadIdx.x) % 112)) + 21952)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    if (((int)threadIdx.x) < 128) {
      kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 16384) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 32; ++rc_inner) {
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((rc_inner * 112) + (((int)threadIdx.x) % 28))] * kernel_shared[(((((int)threadIdx.x) / 28) * 32) + rc_inner)]));
      conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[((rc_inner * 112) + (((int)threadIdx.x) % 28))] * kernel_shared[((((((int)threadIdx.x) / 28) * 32) + rc_inner) + 512)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((rc_inner * 112) + (((int)threadIdx.x) % 28)) + 28)] * kernel_shared[(((((int)threadIdx.x) / 28) * 32) + rc_inner)]));
      conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((rc_inner * 112) + (((int)threadIdx.x) % 28)) + 28)] * kernel_shared[((((((int)threadIdx.x) / 28) * 32) + rc_inner) + 512)]));
      conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((rc_inner * 112) + (((int)threadIdx.x) % 28)) + 56)] * kernel_shared[(((((int)threadIdx.x) / 28) * 32) + rc_inner)]));
      conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((rc_inner * 112) + (((int)threadIdx.x) % 28)) + 56)] * kernel_shared[((((((int)threadIdx.x) / 28) * 32) + rc_inner) + 512)]));
      conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((rc_inner * 112) + (((int)threadIdx.x) % 28)) + 84)] * kernel_shared[(((((int)threadIdx.x) / 28) * 32) + rc_inner)]));
      conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((rc_inner * 112) + (((int)threadIdx.x) % 28)) + 84)] * kernel_shared[((((((int)threadIdx.x) / 28) * 32) + rc_inner) + 512)]));
    }
  }
  for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
    conv2d_nchw[((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 28) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (yy_inner * 28)) + (((int)threadIdx.x) % 28))] = conv2d_nchw_local[yy_inner];
    conv2d_nchw[(((((((((int)blockIdx.x) / 7) * 25088) + ((((int)threadIdx.x) / 28) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (yy_inner * 28)) + (((int)threadIdx.x) % 28)) + 12544)] = conv2d_nchw_local[(yy_inner + 4)];
  }
}


