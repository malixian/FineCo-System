#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[32];
  __shared__ float pad_temp_shared[512];
  __shared__ float kernel_shared[8192];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = data[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = data[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 384)] = data[(((((((rc_outer_outer * 25088) + ((((int)threadIdx.x) >> 4) * 784)) + ((((int)blockIdx.x) / 7) * 112)) + (((((int)threadIdx.x) & 15) >> 2) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3)) + 18816)];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 10240)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 18432)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 20480)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 22528)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 26624)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 30720)];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 32768)];
    kernel_shared[(((int)threadIdx.x) + 2176)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 34816)];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 36864)];
    kernel_shared[(((int)threadIdx.x) + 2432)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 38912)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 40960)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 43008)];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 45056)];
    kernel_shared[(((int)threadIdx.x) + 2944)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 47104)];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 49152)];
    kernel_shared[(((int)threadIdx.x) + 3200)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 51200)];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 53248)];
    kernel_shared[(((int)threadIdx.x) + 3456)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 55296)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 3712)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 59392)];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 61440)];
    kernel_shared[(((int)threadIdx.x) + 3968)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 63488)];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 65536)];
    kernel_shared[(((int)threadIdx.x) + 4224)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 67584)];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 69632)];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 71680)];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 73728)];
    kernel_shared[(((int)threadIdx.x) + 4736)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 75776)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 77824)];
    kernel_shared[(((int)threadIdx.x) + 4992)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 79872)];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 81920)];
    kernel_shared[(((int)threadIdx.x) + 5248)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 83968)];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 86016)];
    kernel_shared[(((int)threadIdx.x) + 5504)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 88064)];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 90112)];
    kernel_shared[(((int)threadIdx.x) + 5760)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 92160)];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 94208)];
    kernel_shared[(((int)threadIdx.x) + 6016)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 96256)];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 98304)];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 100352)];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 102400)];
    kernel_shared[(((int)threadIdx.x) + 6528)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 104448)];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 106496)];
    kernel_shared[(((int)threadIdx.x) + 6784)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 108544)];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 110592)];
    kernel_shared[(((int)threadIdx.x) + 7040)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 112640)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 114688)];
    kernel_shared[(((int)threadIdx.x) + 7296)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 116736)];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 118784)];
    kernel_shared[(((int)threadIdx.x) + 7552)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 120832)];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 122880)];
    kernel_shared[(((int)threadIdx.x) + 7808)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 124928)];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 126976)];
    kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[(((((((int)threadIdx.x) >> 5) * 512) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 129024)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int yy_c_outer_inner = 0; yy_c_outer_inner < 2; ++yy_c_outer_inner) {
        for (int xx_c_outer_inner = 0; xx_c_outer_inner < 2; ++xx_c_outer_inner) {
          conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner)] * kernel_shared[(((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8))]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 32)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 64)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 96)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 128)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 160)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 192)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] + (pad_temp_shared[(((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 224)]));
          conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 1)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 33)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 65)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 97)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 129)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 161)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 193)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 16)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 225)]));
          conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 2)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 34)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 66)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 98)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 130)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 162)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 194)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 226)]));
          conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 3)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 35)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 67)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 99)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 131)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 163)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 195)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 48)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 227)]));
          conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 4)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 36)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 68)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 100)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 132)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 164)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 196)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 228)]));
          conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 5)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 37)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 69)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 101)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 133)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 165)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 197)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 80)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 229)]));
          conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 6)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 38)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 70)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 102)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 134)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 166)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 198)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 230)]));
          conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + xx_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 7)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 39)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 8)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 71)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 12)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 103)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 16)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 135)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 167)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 24)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 199)]));
          conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] = (conv2d_nchw_local[(((yy_c_outer_inner * 2) + xx_c_outer_inner) + 28)] + (pad_temp_shared[((((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (yy_c_outer_inner * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_c_outer_inner) + 112)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 256) + (rc_outer_inner * 8)) + 231)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        conv2d_nchw[(((((((((((int)threadIdx.x) >> 2) * 6272) + (ff_inner * 784)) + ((((int)blockIdx.x) / 7) * 112)) + (((((int)threadIdx.x) & 3) >> 1) * 56)) + (yy_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 4) + (yy_inner * 2)) + xx_inner)];
      }
    }
  }
}


