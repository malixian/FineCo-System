#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[28];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[2048];
  for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 7; ++yy_c_outer_inner_init) {
    conv2d_nchw_local[(yy_c_outer_inner_init * 2)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 2) + 14)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 2) + 1)] = 0.000000e+00f;
    conv2d_nchw_local[((yy_c_outer_inner_init * 2) + 15)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + (((int)threadIdx.x) % 196))];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 448) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 2) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 896) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 4) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1344) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 6) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1792) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 1) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2240) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 3) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2688) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 5) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + (((int)threadIdx.x) % 196)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3584) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 2) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 4032)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 4032) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 4) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 4480)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 4480) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 6) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 4928)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 4928) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 1) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 5376)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 5376) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 3) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 5824)] = data[(((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 5824) / 196) * 784)) + ((((int)blockIdx.x) & 3) * 196)) + ((((((int)threadIdx.x) / 28) + 5) % 7) * 28)) + (((int)threadIdx.x) % 28))];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 21504)];
    if (((int)threadIdx.x) < 256) {
      kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 5) * 512)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672)];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
      for (int yy_c_outer_inner = 0; yy_c_outer_inner < 7; ++yy_c_outer_inner) {
        conv2d_nchw_local[(yy_c_outer_inner * 2)] = (conv2d_nchw_local[(yy_c_outer_inner * 2)] + (pad_temp_shared[(((rc_outer_inner * 196) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[(((((int)threadIdx.x) / 14) * 64) + rc_outer_inner)]));
        conv2d_nchw_local[((yy_c_outer_inner * 2) + 14)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + 14)] + (pad_temp_shared[(((rc_outer_inner * 196) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 32)]));
        conv2d_nchw_local[((yy_c_outer_inner * 2) + 1)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + 1)] + (pad_temp_shared[((((rc_outer_inner * 196) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 14) * 64) + rc_outer_inner)]));
        conv2d_nchw_local[((yy_c_outer_inner * 2) + 15)] = (conv2d_nchw_local[((yy_c_outer_inner * 2) + 15)] + (pad_temp_shared[((((rc_outer_inner * 196) + (yy_c_outer_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + rc_outer_inner) + 32)]));
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        conv2d_nchw[((((((((((int)blockIdx.x) >> 2) * 50176) + ((((int)threadIdx.x) / 14) * 1568)) + (ff_inner * 784)) + ((((int)blockIdx.x) & 3) * 196)) + (yy_inner * 28)) + ((((int)threadIdx.x) % 14) * 2)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 14) + (yy_inner * 2)) + xx_inner)];
      }
    }
  }
}


