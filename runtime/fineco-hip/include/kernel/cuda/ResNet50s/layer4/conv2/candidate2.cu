#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[896];
  __shared__ float kernel_shared[3072];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) % 56) / 14))) && ((((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) % 56) / 14)) < 15)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 14)))) && ((rx_outer_outer + (((int)threadIdx.x) % 14)) < 15)) ? data[((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) / 56) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + rx_outer_outer) + (((int)threadIdx.x) % 56)) - 15)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 224)] = (((((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) % 56) / 14))) && ((((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) % 56) / 14)) < 15)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 14)))) && ((rx_outer_outer + (((int)threadIdx.x) % 14)) < 15)) ? data[((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) / 56) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + rx_outer_outer) + (((int)threadIdx.x) % 56)) + 769)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 448)] = (((((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) % 56) / 14))) && ((((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) % 56) / 14)) < 15)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 14)))) && ((rx_outer_outer + (((int)threadIdx.x) % 14)) < 15)) ? data[((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) / 56) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + rx_outer_outer) + (((int)threadIdx.x) % 56)) + 1553)] : 0.000000e+00f);
      pad_temp_shared[(((int)threadIdx.x) + 672)] = (((((1 <= (((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) % 56) / 14))) && ((((((int)blockIdx.x) % 7) * 2) + ((((int)threadIdx.x) % 56) / 14)) < 15)) && (1 <= (rx_outer_outer + (((int)threadIdx.x) % 14)))) && ((rx_outer_outer + (((int)threadIdx.x) % 14)) < 15)) ? data[((((((rc_outer_outer * 3136) + ((((int)threadIdx.x) / 56) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + rx_outer_outer) + (((int)threadIdx.x) % 56)) + 2337)] : 0.000000e+00f);
      kernel_shared[((int)threadIdx.x)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + ((((int)threadIdx.x) / 48) * 2304)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 224) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 32) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 448) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 16) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((((int)blockIdx.x) / 7) * 147456) + ((((int)threadIdx.x) / 48) * 2304)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer) + 32256)];
      kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 896) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 32) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 1120) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 16) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((((int)blockIdx.x) / 7) * 147456) + ((((int)threadIdx.x) / 48) * 2304)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer) + 64512)];
      kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 1568) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 32) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 1792) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 16) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((((int)blockIdx.x) / 7) * 147456) + ((((int)threadIdx.x) / 48) * 2304)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer) + 96768)];
      kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 2240) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 32) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 2464) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 16) % 48) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((((int)blockIdx.x) / 7) * 147456) + ((((int)threadIdx.x) / 48) * 2304)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) % 48) * 3)) + rx_outer_outer) + 129024)];
      if (((int)threadIdx.x) < 160) {
        kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[((((((((int)blockIdx.x) / 7) * 147456) + (((((int)threadIdx.x) + 2912) / 48) * 2304)) + (rc_outer_outer * 144)) + (((((int)threadIdx.x) + 32) % 48) * 3)) + rx_outer_outer)];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
        for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
          for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
            for (int yy_c_outer_inner = 0; yy_c_outer_inner < 2; ++yy_c_outer_inner) {
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[(((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 1536)]));
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 3)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 56)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 1539)]));
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 6)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 112)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 1542)]));
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 9)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 168)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 1545)]));
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 224)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 12)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 224)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 1548)]));
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 280)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 15)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 280)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 1551)]));
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 336)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 18)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 336)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 1554)]));
              conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] = (conv2d_nchw_local[((ff_c_outer_inner * 2) + yy_c_outer_inner)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 392)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 21)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 2) + yy_c_outer_inner) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 448) + (yy_c_outer_inner * 14)) + (ry_outer_inner * 14)) + (((int)threadIdx.x) % 14)) + 392)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 96) + (ff_c_outer_inner * 48)) + (rc_outer_inner * 24)) + ry_outer_inner) + 1557)]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[(((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
      conv2d_nchw[((((((((((int)blockIdx.x) / 7) * 12544) + ((((int)threadIdx.x) / 14) * 392)) + (ff_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (yy_inner * 14)) + (((int)threadIdx.x) % 14)) + 6272)] = conv2d_nchw_local[(((ff_inner * 2) + yy_inner) + 4)];
    }
  }
}


