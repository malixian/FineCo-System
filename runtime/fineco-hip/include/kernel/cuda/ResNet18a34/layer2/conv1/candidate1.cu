#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[56];
  __shared__ float pad_temp_shared[3840];
  __shared__ float kernel_shared[2304];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((1 <= ((((((int)blockIdx.x) & 7) >> 1) * 14) + (((int)threadIdx.x) / 30))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + (((int)threadIdx.x) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + (((int)threadIdx.x) % 30)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + ((((int)threadIdx.x) / 30) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = (((1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 14) % 30))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 14) % 30)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((int)threadIdx.x) + 224) / 30) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 14) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = (((((1 <= ((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 224) % 240) / 15))) && (((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 224) % 240) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 28) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 28) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 448) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 224) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 28) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 672)] = (((1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 12) % 30))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 12) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 672) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 96) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 12) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = (((((1 <= ((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 208) % 240) / 15))) && (((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 208) % 240) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 26) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 26) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 896) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 208) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 26) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = (((1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 10) % 30))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 10) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1120) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 80) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 10) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = (((((1 <= ((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 192) % 240) / 15))) && (((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 192) % 240) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 24) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 24) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1344) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 192) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 24) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = (((1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 8) % 30))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 8) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1568) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 64) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 8) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((((1 <= ((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 176) % 240) / 15))) && (((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 176) % 240) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 22) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 22) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1792) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 176) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 22) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = (((1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 6) % 30))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 6) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2016) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 48) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 6) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = (((((1 <= ((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 160) % 240) / 15))) && (((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 160) % 240) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 20) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 20) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2240) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 160) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 20) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = (((1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 4) % 30))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 4) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2464) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 32) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 4) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = (((((1 <= ((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 144) % 240) / 15))) && (((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 144) % 240) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 18) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 18) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2688) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 144) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 18) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = (((1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 2) % 30))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 2) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2912) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 16) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 2) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = ((((((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 128) % 240) / 15)) < 57) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 16) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 16) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3136) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 128) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 16) % 30)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = ((((1 <= ((((((int)blockIdx.x) & 7) >> 1) * 14) + (((int)threadIdx.x) / 30))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + (((int)threadIdx.x) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + (((int)threadIdx.x) % 30)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + ((((int)threadIdx.x) / 30) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 30)) + 21895)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = (((1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 14) % 30))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 14) % 30)) < 57)) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3584) / 480) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 112) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 14) % 30)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 32) {
      pad_temp_shared[(((int)threadIdx.x) + 3808)] = ((((((((((int)blockIdx.x) & 7) >> 1) * 14) + ((((((int)threadIdx.x) >> 1) + 224) % 240) / 15)) < 57) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 28) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 28) % 30)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((((int)threadIdx.x) >> 1) + 224) % 240) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 28) % 30)) + 21895)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 224) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 448) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 672) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 896) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1120) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1344) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1568) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 1792) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) >> 3) * 18432) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 16128)];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[(((((((int)blockIdx.x) >> 3) * 18432) + (((((int)threadIdx.x) + 2240) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 8))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
          for (int xx_c_outer_inner = 0; xx_c_outer_inner < 7; ++xx_c_outer_inner) {
            conv2d_nchw_local[xx_c_outer_inner] = (conv2d_nchw_local[xx_c_outer_inner] + (pad_temp_shared[((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw_local[(xx_c_outer_inner + 14)] = (conv2d_nchw_local[(xx_c_outer_inner + 14)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 14)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw_local[(xx_c_outer_inner + 28)] = (conv2d_nchw_local[(xx_c_outer_inner + 28)] + (pad_temp_shared[((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 1152)]));
            conv2d_nchw_local[(xx_c_outer_inner + 42)] = (conv2d_nchw_local[(xx_c_outer_inner + 42)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 14)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 1152)]));
            conv2d_nchw_local[(xx_c_outer_inner + 7)] = (conv2d_nchw_local[(xx_c_outer_inner + 7)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 30)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw_local[(xx_c_outer_inner + 21)] = (conv2d_nchw_local[(xx_c_outer_inner + 21)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 44)] * kernel_shared[(((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw_local[(xx_c_outer_inner + 35)] = (conv2d_nchw_local[(xx_c_outer_inner + 35)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 30)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 1152)]));
            conv2d_nchw_local[(xx_c_outer_inner + 49)] = (conv2d_nchw_local[(xx_c_outer_inner + 49)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 44)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 1152)]));
            conv2d_nchw_local[xx_c_outer_inner] = (conv2d_nchw_local[xx_c_outer_inner] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 480)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
            conv2d_nchw_local[(xx_c_outer_inner + 14)] = (conv2d_nchw_local[(xx_c_outer_inner + 14)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 494)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
            conv2d_nchw_local[(xx_c_outer_inner + 28)] = (conv2d_nchw_local[(xx_c_outer_inner + 28)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 480)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 1161)]));
            conv2d_nchw_local[(xx_c_outer_inner + 42)] = (conv2d_nchw_local[(xx_c_outer_inner + 42)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 494)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 1161)]));
            conv2d_nchw_local[(xx_c_outer_inner + 7)] = (conv2d_nchw_local[(xx_c_outer_inner + 7)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 510)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
            conv2d_nchw_local[(xx_c_outer_inner + 21)] = (conv2d_nchw_local[(xx_c_outer_inner + 21)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 524)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
            conv2d_nchw_local[(xx_c_outer_inner + 35)] = (conv2d_nchw_local[(xx_c_outer_inner + 35)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 510)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 1161)]));
            conv2d_nchw_local[(xx_c_outer_inner + 49)] = (conv2d_nchw_local[(xx_c_outer_inner + 49)] + (pad_temp_shared[(((((((rc_outer_inner * 960) + (((((int)threadIdx.x) % 14) >> 1) * 60)) + (ry_outer_inner * 30)) + ((((int)threadIdx.x) & 1) * 7)) + xx_c_outer_inner) + rx_outer_inner) + 524)] * kernel_shared[((((((((int)threadIdx.x) / 14) * 72) + (rc_outer_inner * 18)) + (ry_outer_inner * 3)) + rx_outer_inner) + 1161)]));
          }
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 3) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((int)threadIdx.x) % 14) >> 1) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) & 1) * 7)) + xx_inner)] = conv2d_nchw_local[((yy_inner * 7) + xx_inner)];
      conv2d_nchw[((((((((((((int)blockIdx.x) >> 3) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((int)threadIdx.x) % 14) >> 1) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) & 1) * 7)) + xx_inner) + 14)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 14)];
      conv2d_nchw[((((((((((((int)blockIdx.x) >> 3) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((int)threadIdx.x) % 14) >> 1) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) & 1) * 7)) + xx_inner) + 50176)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 28)];
      conv2d_nchw[((((((((((((int)blockIdx.x) >> 3) * 100352) + ((((int)threadIdx.x) / 14) * 3136)) + (((((int)blockIdx.x) & 7) >> 1) * 784)) + (((((int)threadIdx.x) % 14) >> 1) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) & 1) * 7)) + xx_inner) + 50190)] = conv2d_nchw_local[(((yy_inner * 7) + xx_inner) + 42)];
    }
  }
}


