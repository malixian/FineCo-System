
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate7(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[640];
  __shared__ float kernel_shared[3072];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 4; ++ff_c_outer_inner_init) {
    for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 2; ++yy_c_outer_inner_init) {
      for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
        conv2d_nchw_local[(((ff_c_outer_inner_init * 4) + (ff_c_inner_init * 2)) + yy_c_outer_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x))] = (((((1 <= (((((int)blockIdx.x) / 14) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + (((int)threadIdx.x) >> 2)) % 10))) && ((((((int)blockIdx.x) / 14) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + (((int)threadIdx.x) >> 2)) % 10)) < 57)) && (1 <= ((((((int)blockIdx.x) % 14) * 4) + rx_outer_outer) + (((int)threadIdx.x) & 3)))) && (((((((int)blockIdx.x) % 14) * 4) + rx_outer_outer) + (((int)threadIdx.x) & 3)) < 57)) ? data[((((((((rc_outer_outer * 50176) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) / 5) * 3136)) + ((((int)blockIdx.x) / 14) * 448)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + (((int)threadIdx.x) >> 2)) % 10) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + rx_outer_outer) + (((int)threadIdx.x) & 3)) - 57)] : 0.000000e+00f);
      }
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
        kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x))] = kernel[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + (((int)threadIdx.x) >> 4)) / 3) * 576) + (rc_outer_outer * 144)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) % 48) * 3)) + rx_outer_outer)];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
        for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
          for (int yy_c_outer_inner = 0; yy_c_outer_inner < 2; ++yy_c_outer_inner) {
            for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
              for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
                for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
                  conv2d_nchw_local[(((ff_c_outer_inner * 4) + (ff_c_inner * 2)) + yy_c_outer_inner)] = (conv2d_nchw_local[(((ff_c_outer_inner * 4) + (ff_c_inner * 2)) + yy_c_outer_inner)] + (pad_temp_shared[((((((rc_outer_inner * 80) + (rc_inner * 40)) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + (yy_c_outer_inner * 4)) + (ry_inner * 4)) + (((int)threadIdx.x) & 3))] * kernel_shared[(((((((((int)threadIdx.x) >> 4) * 384) + (ff_c_outer_inner * 96)) + (ff_c_inner * 48)) + (rc_outer_inner * 6)) + (rc_inner * 3)) + ry_inner)]));
                }
              }
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv2d_nchw[((((((((((int)threadIdx.x) >> 4) * 25088) + (ff_inner * 3136)) + ((((int)blockIdx.x) / 14) * 448)) + (((((int)threadIdx.x) & 15) >> 2) * 112)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 14) * 4)) + (((int)threadIdx.x) & 3))] = conv2d_nchw_local[((ff_inner * 2) + yy_inner)];
    }
  }
}


