
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[16];
  __shared__ float pad_temp_shared[1600];
  __shared__ float kernel_shared[9216];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)blockIdx.x) / 7) * 8) + ((((int)threadIdx.x) % 100) / 10))) && ((((((int)blockIdx.x) / 7) * 8) + ((((int)threadIdx.x) % 100) / 10)) < 57)) && (1 <= (((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) % 10)))) && ((((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) % 10)) < 57)) ? data[(((((((rc_outer_outer * 50176) + ((((int)threadIdx.x) / 100) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) % 100) / 10) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) % 10)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 256)] = (((((1 <= (((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 28) % 50) / 5))) && ((((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 28) % 50) / 5)) < 57)) && (1 <= (((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 6) % 10)))) && ((((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 6) % 10)) < 57)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 256) / 100) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 28) % 50) / 5) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) + 6) % 10)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 512)] = (((((1 <= (((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 6) % 50) / 5))) && ((((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 6) % 50) / 5)) < 57)) && (1 <= (((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 2) % 10)))) && ((((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 2) % 10)) < 57)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 512) / 100) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 6) % 50) / 5) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) + 2) % 10)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 768)] = (((((1 <= (((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 34) % 50) / 5))) && ((((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 34) % 50) / 5)) < 57)) && (1 <= (((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 8) % 10)))) && ((((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 8) % 10)) < 57)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 768) / 100) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 34) % 50) / 5) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) + 8) % 10)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((((1 <= (((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 12) % 50) / 5))) && ((((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 12) % 50) / 5)) < 57)) && (1 <= (((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 4) % 10)))) && ((((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 4) % 10)) < 57)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1024) / 100) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 12) % 50) / 5) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) + 4) % 10)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = (((((1 <= (((((int)blockIdx.x) / 7) * 8) + (((((int)threadIdx.x) / 10) + 8) % 10))) && ((((((int)blockIdx.x) / 7) * 8) + (((((int)threadIdx.x) / 10) + 8) % 10)) < 57)) && (1 <= (((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) % 10)))) && ((((((int)blockIdx.x) % 7) * 8) + (((int)threadIdx.x) % 10)) < 57)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1280) / 100) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + ((((((int)threadIdx.x) / 10) + 8) % 10) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) % 10)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[(((int)threadIdx.x) + 1536)] = (((((((((int)blockIdx.x) / 7) * 8) + ((((((int)threadIdx.x) >> 1) + 18) % 50) / 5)) < 57) && (1 <= (((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 6) % 10)))) && ((((((int)blockIdx.x) % 7) * 8) + ((((int)threadIdx.x) + 6) % 10)) < 57)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1536) / 100) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((((int)threadIdx.x) >> 1) + 18) % 50) / 5) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) + 6) % 10)) - 57)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 144) * 576) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) + 256) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) + 512) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) + 768) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) + 1024) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 16) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) + 1280) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) + 1536) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) + 1792) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) + 2048) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((int)threadIdx.x) / 144) * 576) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144)) + 9216)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) + 2560) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)threadIdx.x) + 2816) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) + 3072) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)threadIdx.x) + 3328) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 16) % 144))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) + 3584) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)threadIdx.x) + 3840) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) + 4096) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)threadIdx.x) + 4352) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[(((((((int)threadIdx.x) / 144) * 576) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144)) + 18432)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[(((((((int)threadIdx.x) + 4864) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)threadIdx.x) + 5120) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[(((((((int)threadIdx.x) + 5376) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)threadIdx.x) + 5632) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 16) % 144))];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[(((((((int)threadIdx.x) + 5888) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)threadIdx.x) + 6144) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[(((((((int)threadIdx.x) + 6400) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)threadIdx.x) + 6656) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[(((((((int)threadIdx.x) / 144) * 576) + (rc_outer_outer * 144)) + (((int)threadIdx.x) % 144)) + 27648)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)threadIdx.x) + 7168) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[(((((((int)threadIdx.x) + 7424) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)threadIdx.x) + 7680) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[(((((((int)threadIdx.x) + 7936) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 16) % 144))];
    kernel_shared[(((int)threadIdx.x) + 8192)] = kernel[(((((((int)threadIdx.x) + 8192) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 8448)] = kernel[(((((((int)threadIdx.x) + 8448) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 8704)] = kernel[(((((((int)threadIdx.x) + 8704) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 8960)] = kernel[(((((((int)threadIdx.x) + 8960) / 144) * 576) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 32) % 144))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner)]));
        conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4608)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 1)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner)]));
        conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 1)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4608)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 10)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner)]));
        conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 10)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4608)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 11)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner)]));
        conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 11)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4608)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 20)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner)]));
        conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 20)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4608)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 21)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner)]));
        conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 21)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4608)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 30)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner)]));
        conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 30)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4608)]));
        conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 31)] * kernel_shared[((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner)]));
        conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 31)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4608)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 10)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 10)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4611)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 11)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 11)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4611)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 20)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 20)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4611)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 21)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 21)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4611)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 30)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 30)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4611)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 31)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 31)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4611)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4611)]));
        conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 41)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 3)]));
        conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 41)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4611)]));
        conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 20)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 20)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4614)]));
        conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 21)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 21)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4614)]));
        conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 30)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 30)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4614)]));
        conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 31)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 31)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4614)]));
        conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 40)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4614)]));
        conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 41)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 41)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4614)]));
        conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 50)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 50)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4614)]));
        conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 51)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 6)]));
        conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((((rc_outer_inner * 100) + (((((int)threadIdx.x) & 7) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 2)) + rx_outer_inner) + 51)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 144) + (rc_outer_inner * 9)) + rx_outer_inner) + 4614)]));
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
      conv2d_nchw[((((((((((int)threadIdx.x) >> 3) * 3136) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 7) >> 2) * 224)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + xx_inner)] = conv2d_nchw_local[((yy_inner * 2) + xx_inner)];
      conv2d_nchw[(((((((((((int)threadIdx.x) >> 3) * 3136) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 7) >> 2) * 224)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + xx_inner) + 100352)] = conv2d_nchw_local[(((yy_inner * 2) + xx_inner) + 8)];
    }
  }
}


