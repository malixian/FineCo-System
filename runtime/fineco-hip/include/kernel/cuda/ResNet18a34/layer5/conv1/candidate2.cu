#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[7];
  __shared__ float pad_temp_shared[1296];
  __shared__ float kernel_shared[2304];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((((9 <= (((int)threadIdx.x) % 81)) && ((((int)threadIdx.x) % 81) < 72)) && (1 <= (((int)threadIdx.x) % 9))) && ((((int)threadIdx.x) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + ((((int)threadIdx.x) / 81) * 49)) + (((((int)threadIdx.x) % 81) / 9) * 7)) + (((int)threadIdx.x) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 112)] = (((((9 <= ((((int)threadIdx.x) + 31) % 81)) && (((((int)threadIdx.x) + 31) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 4) % 9))) && (((((int)threadIdx.x) + 4) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 112) / 81) * 49)) + ((((((int)threadIdx.x) + 31) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 4) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = (((((9 <= ((((int)threadIdx.x) + 62) % 81)) && (((((int)threadIdx.x) + 62) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 8) % 9))) && (((((int)threadIdx.x) + 8) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 224) / 81) * 49)) + ((((((int)threadIdx.x) + 62) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 8) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 336)] = (((((9 <= ((((int)threadIdx.x) + 12) % 81)) && (((((int)threadIdx.x) + 12) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 3) % 9))) && (((((int)threadIdx.x) + 3) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 336) / 81) * 49)) + ((((((int)threadIdx.x) + 12) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 3) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = (((((9 <= ((((int)threadIdx.x) + 43) % 81)) && (((((int)threadIdx.x) + 43) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 7) % 9))) && (((((int)threadIdx.x) + 7) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 448) / 81) * 49)) + ((((((int)threadIdx.x) + 43) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 7) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 560)] = (((((9 <= ((((int)threadIdx.x) + 74) % 81)) && (((((int)threadIdx.x) + 74) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 2) % 9))) && (((((int)threadIdx.x) + 2) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 560) / 81) * 49)) + ((((((int)threadIdx.x) + 74) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 2) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 672)] = (((((9 <= ((((int)threadIdx.x) + 24) % 81)) && (((((int)threadIdx.x) + 24) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 6) % 9))) && (((((int)threadIdx.x) + 6) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 672) / 81) * 49)) + ((((((int)threadIdx.x) + 24) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 6) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 784)] = (((((9 <= ((((int)threadIdx.x) + 55) % 81)) && (((((int)threadIdx.x) + 55) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 1) % 9))) && (((((int)threadIdx.x) + 1) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 784) / 81) * 49)) + ((((((int)threadIdx.x) + 55) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 1) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = (((((9 <= ((((int)threadIdx.x) + 5) % 81)) && (((((int)threadIdx.x) + 5) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 5) % 9))) && (((((int)threadIdx.x) + 5) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 896) / 81) * 49)) + ((((((int)threadIdx.x) + 5) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 5) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1008)] = (((((1 <= (((((int)threadIdx.x) / 9) + 4) % 9)) && (((((int)threadIdx.x) + 36) % 81) < 72)) && (1 <= (((int)threadIdx.x) % 9))) && ((((int)threadIdx.x) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 1008) / 81) * 49)) + ((((((int)threadIdx.x) / 9) + 4) % 9) * 7)) + (((int)threadIdx.x) % 9)) - 8)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = (((((9 <= ((((int)threadIdx.x) + 67) % 81)) && (((((int)threadIdx.x) + 67) % 81) < 72)) && (1 <= ((((int)threadIdx.x) + 4) % 9))) && (((((int)threadIdx.x) + 4) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 1120) / 81) * 49)) + ((((((int)threadIdx.x) + 67) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 4) % 9)) - 8)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[(((int)threadIdx.x) + 1232)] = ((((((int)threadIdx.x) < 55) && (1 <= ((((int)threadIdx.x) + 8) % 9))) && (((((int)threadIdx.x) + 8) % 9) < 8)) ? data[(((((rc_outer_outer * 784) + (((((int)threadIdx.x) + 1232) / 81) * 49)) + ((((((int)threadIdx.x) + 17) % 81) / 9) * 7)) + ((((int)threadIdx.x) + 8) % 9)) - 8)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) * 73728) + (rc_outer_outer * 144)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 112) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 224) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 336) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 448) / 144) * 4608)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 560)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 560) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 672) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 784) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 896) / 144) * 4608)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) + 32))];
    kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[((((((int)blockIdx.x) * 73728) + (rc_outer_outer * 144)) + ((int)threadIdx.x)) + 32256)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1120) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1232)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1232) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 80) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1344) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 48) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1456)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1456) / 144) * 4608)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1568) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 128) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1680)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1680) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 96) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1792) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 64) % 144))];
    kernel_shared[(((int)threadIdx.x) + 1904)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1904) / 144) * 4608)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) + 32))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((int)blockIdx.x) * 73728) + (rc_outer_outer * 144)) + ((int)threadIdx.x)) + 64512)];
    kernel_shared[(((int)threadIdx.x) + 2128)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 2128) / 144) * 4608)) + (rc_outer_outer * 144)) + ((((int)threadIdx.x) + 112) % 144))];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 2240) / 144) * 4608)) + (rc_outer_outer * 144)) + (((int)threadIdx.x) + 80))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9))] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 1)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 2)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 3)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 4)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 5)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 6)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3))]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 3)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 4)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 5)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 6)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 7)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 1)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 3)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 4)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 5)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 6)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 7)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 2)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 648) + (rc_inner * 81)) + (ry_outer_inner * 9)) + ((((int)threadIdx.x) % 7) * 9)) + 8)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 72)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + 2)]));
        }
      }
    }
  }
  for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
    conv2d_nchw[(((((int)blockIdx.x) * 784) + (((int)threadIdx.x) * 7)) + xx_inner)] = conv2d_nchw_local[xx_inner];
  }
}


