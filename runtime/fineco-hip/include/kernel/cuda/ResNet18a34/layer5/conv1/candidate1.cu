#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(448) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[2];
  __shared__ float pad_temp_shared[72];
  __shared__ float kernel_shared[3072];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      __syncthreads();
      if (((int)threadIdx.x) < 72) {
        pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((int)threadIdx.x) % 9)) && ((((int)threadIdx.x) % 9) < 8)) && (1 <= (rx_outer_outer + (((int)blockIdx.x) % 7)))) && ((rx_outer_outer + (((int)blockIdx.x) % 7)) < 8)) ? data[((((((rc_outer_outer * 392) + ((((int)threadIdx.x) / 9) * 49)) + ((((int)threadIdx.x) % 9) * 7)) + rx_outer_outer) + (((int)blockIdx.x) % 7)) - 8)] : 0.000000e+00f);
      }
      kernel_shared[((int)threadIdx.x)] = kernel[((((((((int)blockIdx.x) / 7) * 589824) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((((int)blockIdx.x) / 7) * 589824) + (((((int)threadIdx.x) + 448) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 589824) + (((((int)threadIdx.x) + 896) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((((int)blockIdx.x) / 7) * 589824) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + rx_outer_outer) + 258048)];
      kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 7) * 589824) + (((((int)threadIdx.x) + 1792) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 16) % 24) * 3)) + rx_outer_outer)];
      kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((((int)blockIdx.x) / 7) * 589824) + (((((int)threadIdx.x) + 2240) / 24) * 4608)) + (rc_outer_outer * 72)) + (((((int)threadIdx.x) + 8) % 24) * 3)) + rx_outer_outer)];
      if (((int)threadIdx.x) < 384) {
        kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((((int)blockIdx.x) / 7) * 589824) + ((((int)threadIdx.x) / 24) * 4608)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) % 24) * 3)) + rx_outer_outer) + 516096)];
      }
      __syncthreads();
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((int)threadIdx.x) % 7)] * kernel_shared[((((int)threadIdx.x) / 7) * 48)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 1)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 2)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 9)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 3)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 10)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 4)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 11)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 5)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 18)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 6)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 19)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 7)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 20)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 8)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 27)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 9)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 28)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 10)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 29)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 11)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 36)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 12)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 37)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 13)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 38)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 14)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 45)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 15)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 46)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 16)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 47)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 17)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 54)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 18)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 55)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 19)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 56)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 20)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 63)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 21)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 64)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 22)]));
      conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 65)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 23)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((int)threadIdx.x) % 7)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 24)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 25)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 26)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 9)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 27)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 10)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 28)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 11)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 29)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 18)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 30)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 19)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 31)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 20)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 32)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 27)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 33)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 28)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 34)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 29)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 35)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 36)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 36)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 37)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 37)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 38)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 38)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 45)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 39)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 46)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 40)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 47)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 41)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 54)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 42)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 55)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 43)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 56)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 44)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 63)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 45)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 64)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 46)]));
      conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[((((int)threadIdx.x) % 7) + 65)] * kernel_shared[(((((int)threadIdx.x) / 7) * 48) + 47)]));
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    conv2d_nchw[((((((((int)blockIdx.x) / 7) * 6272) + ((((int)threadIdx.x) / 7) * 98)) + (ff_inner * 49)) + ((((int)threadIdx.x) % 7) * 7)) + (((int)blockIdx.x) % 7))] = conv2d_nchw_local[ff_inner];
  }
}


