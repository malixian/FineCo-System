
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate7(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[1];
  __shared__ float pad_temp_shared[36];
  __shared__ float kernel_shared[9216];
  conv2d_nchw_local[0] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 36) {
      pad_temp_shared[((int)threadIdx.x)] = (((1 <= ((((((int)blockIdx.x) % 49) / 7) * 2) + ((((int)threadIdx.x) % 9) / 3))) && (1 <= (((((int)blockIdx.x) % 7) * 2) + (((int)threadIdx.x) % 3)))) ? data[(((((((rc_outer_outer * 784) + ((((int)threadIdx.x) / 9) * 196)) + (((((int)blockIdx.x) % 49) / 7) * 28)) + (((((int)threadIdx.x) % 9) / 3) * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) % 3)) - 15)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + ((((int)threadIdx.x) / 36) * 2304)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 256) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 4) % 36))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 512) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 768) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 12) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 1024) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 16) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 1280) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 20) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 1536) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 24) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 1792) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 28) % 36))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 2048) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 32) % 36))];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[((((((((int)blockIdx.x) / 49) * 589824) + ((((int)threadIdx.x) / 36) * 2304)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36)) + 147456)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 2560) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 4) % 36))];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 2816) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 3072) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 12) % 36))];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 3328) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 16) % 36))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 3584) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 20) % 36))];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 3840) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 24) % 36))];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 4096) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 28) % 36))];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 4352) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 32) % 36))];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[((((((((int)blockIdx.x) / 49) * 589824) + ((((int)threadIdx.x) / 36) * 2304)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36)) + 294912)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 4864) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 4) % 36))];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 5120) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 5376) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 12) % 36))];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 5632) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 16) % 36))];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 5888) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 20) % 36))];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 6144) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 24) % 36))];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 6400) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 28) % 36))];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 6656) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 32) % 36))];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[((((((((int)blockIdx.x) / 49) * 589824) + ((((int)threadIdx.x) / 36) * 2304)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36)) + 442368)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 7168) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 4) % 36))];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 7424) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 7680) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 12) % 36))];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 7936) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 16) % 36))];
    kernel_shared[(((int)threadIdx.x) + 8192)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 8192) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 20) % 36))];
    kernel_shared[(((int)threadIdx.x) + 8448)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 8448) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 24) % 36))];
    kernel_shared[(((int)threadIdx.x) + 8704)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 8704) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 28) % 36))];
    kernel_shared[(((int)threadIdx.x) + 8960)] = kernel[(((((((int)blockIdx.x) / 49) * 589824) + (((((int)threadIdx.x) + 8960) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 32) % 36))];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[0] * kernel_shared[(((int)threadIdx.x) * 36)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[1] * kernel_shared[((((int)threadIdx.x) * 36) + 1)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[2] * kernel_shared[((((int)threadIdx.x) * 36) + 2)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[3] * kernel_shared[((((int)threadIdx.x) * 36) + 3)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[4] * kernel_shared[((((int)threadIdx.x) * 36) + 4)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[5] * kernel_shared[((((int)threadIdx.x) * 36) + 5)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[6] * kernel_shared[((((int)threadIdx.x) * 36) + 6)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[7] * kernel_shared[((((int)threadIdx.x) * 36) + 7)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[8] * kernel_shared[((((int)threadIdx.x) * 36) + 8)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[9] * kernel_shared[((((int)threadIdx.x) * 36) + 9)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[10] * kernel_shared[((((int)threadIdx.x) * 36) + 10)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[11] * kernel_shared[((((int)threadIdx.x) * 36) + 11)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[12] * kernel_shared[((((int)threadIdx.x) * 36) + 12)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[13] * kernel_shared[((((int)threadIdx.x) * 36) + 13)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[14] * kernel_shared[((((int)threadIdx.x) * 36) + 14)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[15] * kernel_shared[((((int)threadIdx.x) * 36) + 15)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[16] * kernel_shared[((((int)threadIdx.x) * 36) + 16)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[17] * kernel_shared[((((int)threadIdx.x) * 36) + 17)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[18] * kernel_shared[((((int)threadIdx.x) * 36) + 18)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[19] * kernel_shared[((((int)threadIdx.x) * 36) + 19)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[20] * kernel_shared[((((int)threadIdx.x) * 36) + 20)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[21] * kernel_shared[((((int)threadIdx.x) * 36) + 21)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[22] * kernel_shared[((((int)threadIdx.x) * 36) + 22)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[23] * kernel_shared[((((int)threadIdx.x) * 36) + 23)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[24] * kernel_shared[((((int)threadIdx.x) * 36) + 24)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[25] * kernel_shared[((((int)threadIdx.x) * 36) + 25)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[26] * kernel_shared[((((int)threadIdx.x) * 36) + 26)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[27] * kernel_shared[((((int)threadIdx.x) * 36) + 27)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[28] * kernel_shared[((((int)threadIdx.x) * 36) + 28)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[29] * kernel_shared[((((int)threadIdx.x) * 36) + 29)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[30] * kernel_shared[((((int)threadIdx.x) * 36) + 30)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[31] * kernel_shared[((((int)threadIdx.x) * 36) + 31)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[32] * kernel_shared[((((int)threadIdx.x) * 36) + 32)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[33] * kernel_shared[((((int)threadIdx.x) * 36) + 33)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[34] * kernel_shared[((((int)threadIdx.x) * 36) + 34)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[35] * kernel_shared[((((int)threadIdx.x) * 36) + 35)]));
  }
  conv2d_nchw[((((((int)blockIdx.x) / 49) * 12544) + (((int)threadIdx.x) * 49)) + (((int)blockIdx.x) % 49))] = conv2d_nchw_local[0];
}


