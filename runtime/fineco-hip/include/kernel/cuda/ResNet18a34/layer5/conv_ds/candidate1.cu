#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[14];
  __shared__ float pad_temp_shared[1800];
  __shared__ float kernel_shared[2304];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((15 <= ((int)threadIdx.x)) && (1 <= (((int)threadIdx.x) % 15))) ? data[((((rc_outer_outer * 1568) + ((((int)threadIdx.x) / 15) * 14)) + (((int)threadIdx.x) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 112)] = ((1 <= ((((int)threadIdx.x) + 7) % 15)) ? data[((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 112) / 15) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = (((15 <= ((((int)threadIdx.x) + 224) % 225)) && (1 <= ((((int)threadIdx.x) + 14) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 224) / 225) * 196)) + ((((((int)threadIdx.x) + 224) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 336)] = ((1 <= ((((int)threadIdx.x) + 6) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 336) / 225) * 196)) + ((((((int)threadIdx.x) + 111) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = (((15 <= ((((int)threadIdx.x) + 223) % 225)) && (1 <= ((((int)threadIdx.x) + 13) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 448) / 225) * 196)) + ((((((int)threadIdx.x) + 223) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 560)] = ((1 <= ((((int)threadIdx.x) + 5) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 560) / 225) * 196)) + ((((((int)threadIdx.x) + 110) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 672)] = (((15 <= ((((int)threadIdx.x) + 222) % 225)) && (1 <= ((((int)threadIdx.x) + 12) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 672) / 225) * 196)) + ((((((int)threadIdx.x) + 222) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 784)] = ((1 <= ((((int)threadIdx.x) + 4) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 784) / 225) * 196)) + ((((((int)threadIdx.x) + 109) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = (((15 <= ((((int)threadIdx.x) + 221) % 225)) && (1 <= ((((int)threadIdx.x) + 11) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 896) / 225) * 196)) + ((((((int)threadIdx.x) + 221) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1008)] = ((1 <= ((((int)threadIdx.x) + 3) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1008) / 225) * 196)) + ((((((int)threadIdx.x) + 108) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = (((15 <= ((((int)threadIdx.x) + 220) % 225)) && (1 <= ((((int)threadIdx.x) + 10) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1120) / 225) * 196)) + ((((((int)threadIdx.x) + 220) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1232)] = ((1 <= ((((int)threadIdx.x) + 2) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1232) / 225) * 196)) + ((((((int)threadIdx.x) + 107) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = (((15 <= ((((int)threadIdx.x) + 219) % 225)) && (1 <= ((((int)threadIdx.x) + 9) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1344) / 225) * 196)) + ((((((int)threadIdx.x) + 219) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1456)] = ((1 <= ((((int)threadIdx.x) + 1) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1456) / 225) * 196)) + ((((((int)threadIdx.x) + 106) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = (((15 <= ((((int)threadIdx.x) + 218) % 225)) && (1 <= ((((int)threadIdx.x) + 8) % 15))) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1568) / 225) * 196)) + ((((((int)threadIdx.x) + 218) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 15)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1680)] = ((1 <= (((int)threadIdx.x) % 15)) ? data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1680) / 225) * 196)) + (((((int)threadIdx.x) / 15) + 7) * 14)) + (((int)threadIdx.x) % 15)) - 15)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 8) {
      pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((rc_outer_outer * 1568) + (((((int)threadIdx.x) + 1792) / 225) * 196)) + ((((((int)threadIdx.x) + 217) % 225) / 15) * 14)) + (((int)threadIdx.x) + 7)) - 15)];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) / 72) * 2304)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 112) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 224) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 336) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 448) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 560)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 560) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 672) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 784) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 896) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[(((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) / 72) * 2304)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 32256)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1120) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1232)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1232) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1344) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1456)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1456) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1568) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1680)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1680) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1792) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1904)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 1904) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[(((((((int)blockIdx.x) * 73728) + ((((int)threadIdx.x) / 72) * 2304)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 64512)];
    kernel_shared[(((int)threadIdx.x) + 2128)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 2128) / 72) * 2304)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((int)blockIdx.x) * 73728) + (((((int)threadIdx.x) + 2240) / 72) * 2304)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 8))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 30)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 60)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 90)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 120)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 150)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 180)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 225)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 255)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 285)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 315)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 345)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 375)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 405)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 9)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 450)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 18)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 480)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 18)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 510)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 18)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 540)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 18)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 570)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 18)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 600)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 18)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 630)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 18)]));
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 675)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 27)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 705)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 27)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 735)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 27)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 765)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 27)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 795)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 27)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 825)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 27)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 855)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 27)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 72)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 30)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 72)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 60)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 72)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 90)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 72)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 120)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 72)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 150)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 72)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 180)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 72)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 225)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 81)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 255)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 81)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 285)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 81)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 315)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 81)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 345)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 81)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 375)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 81)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 405)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 81)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 450)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 90)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 480)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 90)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 510)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 90)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 540)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 90)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 570)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 90)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 600)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 90)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 630)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 90)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 675)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 99)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 705)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 99)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 735)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 99)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 765)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 99)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 795)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 99)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 825)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 99)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 900) + (ry_outer_inner * 15)) + ((((int)threadIdx.x) % 7) * 2)) + rx_outer_inner) + 855)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 36)) + (ry_outer_inner * 3)) + rx_outer_inner) + 99)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      conv2d_nchw[(((((((int)blockIdx.x) * 1568) + ((((int)threadIdx.x) / 7) * 98)) + (ff_inner * 49)) + (yy_inner * 7)) + (((int)threadIdx.x) % 7))] = conv2d_nchw_local[((ff_inner * 7) + yy_inner)];
    }
  }
}


