#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[56];
  __shared__ float pad_temp_shared[3480];
  __shared__ float kernel_shared[4608];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((int)threadIdx.x) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + (((int)threadIdx.x) % 15)))) ? data[((((((rc_outer_outer * 25088) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((int)threadIdx.x) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 112)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 7) % 15))) ? data[((((((rc_outer_outer * 25088) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + (((((int)threadIdx.x) + 112) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 14) % 15))) ? data[((((((rc_outer_outer * 25088) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + (((((int)threadIdx.x) + 224) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 336)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 336) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 6) % 15)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 336) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 336) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 13) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 13) % 15)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 448) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 13) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 560)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 5) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 560) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 125) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 672)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 12) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 672) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 237) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 784)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 349) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 4) % 15)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 784) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 349) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 11) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 896) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 26) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1008)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 3) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1008) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 138) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 10) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1120) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 250) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1232)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 362) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 2) % 15)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1232) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 362) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 9) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1344) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 39) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1456)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 1) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1456) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 151) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 8) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1568) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 263) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1680)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) / 15) + 25) % 29))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + (((int)threadIdx.x) % 15)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1680) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) / 15) + 25) % 29) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 7) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1792) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 52) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1904)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 14) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1904) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 164) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 6) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2016) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 276) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2128)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 388) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 13) % 15)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2128) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 388) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 5) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2240) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 65) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 12) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2352) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 177) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 4) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2464) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 289) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2576)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 401) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 11) % 15)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2576) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 401) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 3) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2688) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 78) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2800)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 10) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2800) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 190) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 2) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2912) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 302) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3024)] = (((1 <= ((((((int)blockIdx.x) & 7) >> 2) * 28) + (((((int)threadIdx.x) + 414) % 435) / 15))) && (1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 9) % 15)))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3024) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 414) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 1) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3136) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 91) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3248)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 8) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3248) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 203) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = ((1 <= (((((int)blockIdx.x) & 3) * 14) + (((int)threadIdx.x) % 15))) ? data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3360) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + (((((int)threadIdx.x) / 15) + 21) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) % 15)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 8) {
      pad_temp_shared[(((int)threadIdx.x) + 3472)] = data[(((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 3472) / 435) * 3136)) + (((((int)blockIdx.x) & 7) >> 2) * 1568)) + ((((((int)threadIdx.x) + 427) % 435) / 15) * 56)) + ((((int)blockIdx.x) & 3) * 14)) + (((int)threadIdx.x) + 7)) - 57)];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 112) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 224) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 336) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 448) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 560)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 560) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 672) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 784) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 896) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[((((((((int)blockIdx.x) >> 3) * 36864) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 8064)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 1120) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1232)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 1232) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 1344) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1456)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 1456) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 1568) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1680)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 1680) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 1792) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1904)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 1904) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) >> 3) * 36864) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 16128)];
    kernel_shared[(((int)threadIdx.x) + 2128)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 2128) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 2240) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2352)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 2352) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 2464) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2576)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 2576) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 2688) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2800)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 2800) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 2912) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3024)] = kernel[((((((((int)blockIdx.x) >> 3) * 36864) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 24192)];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 3136) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3248)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 3248) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 3360) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3472)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 3472) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 3584) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3696)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 3696) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 3808) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3920)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 3920) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[((((((((int)blockIdx.x) >> 3) * 36864) + ((((int)threadIdx.x) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 32256)];
    kernel_shared[(((int)threadIdx.x) + 4144)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 4144) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4256)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 4256) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4368)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 4368) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 4480) / 72) * 576)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    if (((int)threadIdx.x) < 16) {
      kernel_shared[(((int)threadIdx.x) + 4592)] = kernel[(((((((int)blockIdx.x) >> 3) * 36864) + (((((int)threadIdx.x) + 4592) / 72) * 576)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 56))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
          for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
            for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
              conv2d_nchw_local[(ff_c_outer_inner * 7)] = (conv2d_nchw_local[(ff_c_outer_inner * 7)] + (pad_temp_shared[(((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner)] * kernel_shared[(((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 14)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 14)] + (pad_temp_shared[(((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 1152)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 28)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 28)] + (pad_temp_shared[(((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 42)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 42)] + (pad_temp_shared[(((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 3456)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 1)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 2)] * kernel_shared[(((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 15)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 15)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 2)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 1152)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 29)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 29)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 2)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 43)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 43)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 2)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 3456)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 2)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 4)] * kernel_shared[(((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 16)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 16)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 4)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 1152)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 30)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 30)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 4)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 44)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 44)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 4)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 3456)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 3)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 6)] * kernel_shared[(((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 17)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 17)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 6)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 1152)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 31)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 31)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 6)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 45)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 45)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 6)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 3456)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 4)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 8)] * kernel_shared[(((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 18)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 18)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 8)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 1152)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 32)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 32)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 8)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 46)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 46)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 8)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 3456)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 5)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 5)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 10)] * kernel_shared[(((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 19)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 19)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 10)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 1152)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 33)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 33)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 10)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 47)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 47)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 10)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 3456)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 6)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 6)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 12)] * kernel_shared[(((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 20)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 20)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 12)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 1152)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 34)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 34)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 12)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
              conv2d_nchw_local[((ff_c_outer_inner * 7) + 48)] = (conv2d_nchw_local[((ff_c_outer_inner * 7) + 48)] + (pad_temp_shared[((((((rc_outer_inner * 870) + (rc_inner * 435)) + ((((int)threadIdx.x) % 14) * 30)) + (ry_outer_inner * 15)) + rx_inner) + 12)] * kernel_shared[((((((((((int)threadIdx.x) / 14) * 144) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 3456)]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
      conv2d_nchw[((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) / 14) * 1568)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) % 14) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + xx_inner)] = conv2d_nchw_local[((ff_inner * 7) + xx_inner)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) / 14) * 1568)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) % 14) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + xx_inner) + 12544)] = conv2d_nchw_local[(((ff_inner * 7) + xx_inner) + 14)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) / 14) * 1568)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) % 14) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + xx_inner) + 25088)] = conv2d_nchw_local[(((ff_inner * 7) + xx_inner) + 28)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) / 14) * 1568)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 7) >> 2) * 392)) + ((((int)threadIdx.x) % 14) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + xx_inner) + 37632)] = conv2d_nchw_local[(((ff_inner * 7) + xx_inner) + 42)];
    }
  }
}


