
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[28];
  __shared__ float pad_temp_shared[648];
  __shared__ float kernel_shared[4608];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 81) / 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 81) / 9)) < 29)) && (1 <= (((((int)blockIdx.x) & 3) * 7) + (((int)threadIdx.x) % 9)))) && ((((((int)blockIdx.x) & 3) * 7) + (((int)threadIdx.x) % 9)) < 29)) ? data[(((((((rc_outer_outer * 6272) + ((((int)threadIdx.x) / 81) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + (((((int)threadIdx.x) % 81) / 9) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 9)) - 29)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 112)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 31) % 81) / 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 31) % 81) / 9)) < 29)) && (1 <= (((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 4) % 9)))) && ((((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 4) % 9)) < 29)) ? data[(((((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 112) / 81) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) + 31) % 81) / 9) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + ((((int)threadIdx.x) + 4) % 9)) - 29)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 62) % 81) / 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 62) % 81) / 9)) < 29)) && (1 <= (((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 8) % 9)))) && ((((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 8) % 9)) < 29)) ? data[(((((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 224) / 81) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) + 62) % 81) / 9) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + ((((int)threadIdx.x) + 8) % 9)) - 29)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 336)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 12) % 81) / 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 12) % 81) / 9)) < 29)) && (1 <= (((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 3) % 9)))) && ((((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 3) % 9)) < 29)) ? data[(((((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 336) / 81) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) + 12) % 81) / 9) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + ((((int)threadIdx.x) + 3) % 9)) - 29)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 43) % 81) / 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 43) % 81) / 9)) < 29)) && (1 <= (((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 7) % 9)))) && ((((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 7) % 9)) < 29)) ? data[(((((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 448) / 81) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) + 43) % 81) / 9) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + ((((int)threadIdx.x) + 7) % 9)) - 29)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 88) {
      pad_temp_shared[(((int)threadIdx.x) + 560)] = (((((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 74) % 81) / 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) + 74) % 81) / 9)) < 29)) && (1 <= (((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 2) % 9)))) && ((((((int)blockIdx.x) & 3) * 7) + ((((int)threadIdx.x) + 2) % 9)) < 29)) ? data[(((((((rc_outer_outer * 6272) + (((((int)threadIdx.x) + 560) / 81) * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + ((((((int)threadIdx.x) + 74) % 81) / 9) * 28)) + ((((int)blockIdx.x) & 3) * 7)) + ((((int)threadIdx.x) + 2) % 9)) - 29)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 112) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 224) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 336) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 448) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 560)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 560) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 672) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 784) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 896) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[((((((((int)blockIdx.x) >> 4) * 73728) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 16128)];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 1120) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1232)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 1232) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 1344) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1456)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 1456) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 1568) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1680)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 1680) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 1792) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1904)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 1904) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) >> 4) * 73728) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 32256)];
    kernel_shared[(((int)threadIdx.x) + 2128)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 2128) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 2240) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2352)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 2352) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 2464) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2576)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 2576) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 2688) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2800)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 2800) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 2912) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3024)] = kernel[((((((((int)blockIdx.x) >> 4) * 73728) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 48384)];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 3136) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3248)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 3248) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 3360) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3472)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 3472) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 3584) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3696)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 3696) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 3808) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3920)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 3920) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[((((((((int)blockIdx.x) >> 4) * 73728) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 64512)];
    kernel_shared[(((int)threadIdx.x) + 4144)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 4144) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4256)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 4256) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4368)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 4368) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 4480) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    if (((int)threadIdx.x) < 16) {
      kernel_shared[(((int)threadIdx.x) + 4592)] = kernel[(((((((int)blockIdx.x) >> 4) * 73728) + (((((int)threadIdx.x) + 4592) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) + 56))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7))] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
          conv2d_nchw_local[14] = (conv2d_nchw_local[14] + (pad_temp_shared[((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
          conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 9)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
          conv2d_nchw_local[15] = (conv2d_nchw_local[15] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 9)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
          conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
          conv2d_nchw_local[16] = (conv2d_nchw_local[16] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
          conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 27)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
          conv2d_nchw_local[17] = (conv2d_nchw_local[17] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 27)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
          conv2d_nchw_local[4] = (conv2d_nchw_local[4] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 36)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
          conv2d_nchw_local[18] = (conv2d_nchw_local[18] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 36)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
          conv2d_nchw_local[5] = (conv2d_nchw_local[5] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 45)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
          conv2d_nchw_local[19] = (conv2d_nchw_local[19] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 45)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
          conv2d_nchw_local[6] = (conv2d_nchw_local[6] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 54)] * kernel_shared[(((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
          conv2d_nchw_local[20] = (conv2d_nchw_local[20] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 54)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2304)]));
          conv2d_nchw_local[7] = (conv2d_nchw_local[7] + (pad_temp_shared[((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
          conv2d_nchw_local[21] = (conv2d_nchw_local[21] + (pad_temp_shared[((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7))] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2376)]));
          conv2d_nchw_local[8] = (conv2d_nchw_local[8] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 9)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
          conv2d_nchw_local[22] = (conv2d_nchw_local[22] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 9)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2376)]));
          conv2d_nchw_local[9] = (conv2d_nchw_local[9] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
          conv2d_nchw_local[23] = (conv2d_nchw_local[23] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 18)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2376)]));
          conv2d_nchw_local[10] = (conv2d_nchw_local[10] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 27)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
          conv2d_nchw_local[24] = (conv2d_nchw_local[24] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 27)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2376)]));
          conv2d_nchw_local[11] = (conv2d_nchw_local[11] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 36)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
          conv2d_nchw_local[25] = (conv2d_nchw_local[25] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 36)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2376)]));
          conv2d_nchw_local[12] = (conv2d_nchw_local[12] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 45)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
          conv2d_nchw_local[26] = (conv2d_nchw_local[26] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 45)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2376)]));
          conv2d_nchw_local[13] = (conv2d_nchw_local[13] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 54)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
          conv2d_nchw_local[27] = (conv2d_nchw_local[27] + (pad_temp_shared[(((((rc_outer_inner * 81) + (ry_outer_inner * 9)) + rx_inner) + (((int)threadIdx.x) % 7)) + 54)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 144) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 2376)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      conv2d_nchw[((((((((((int)blockIdx.x) >> 4) * 50176) + ((((int)threadIdx.x) / 7) * 1568)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + (yy_inner * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7))] = conv2d_nchw_local[((ff_inner * 7) + yy_inner)];
      conv2d_nchw[(((((((((((int)blockIdx.x) >> 4) * 50176) + ((((int)threadIdx.x) / 7) * 1568)) + (ff_inner * 784)) + (((((int)blockIdx.x) & 15) >> 2) * 196)) + (yy_inner * 28)) + ((((int)blockIdx.x) & 3) * 7)) + (((int)threadIdx.x) % 7)) + 25088)] = conv2d_nchw_local[(((ff_inner * 7) + yy_inner) + 14)];
    }
  }
}


