
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[4];
  __shared__ float pad_temp_shared[50];
  __shared__ float kernel_shared[4608];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 50) {
      pad_temp_shared[((int)threadIdx.x)] = (((1 <= (((((int)blockIdx.x) / 7) * 4) + ((((int)threadIdx.x) % 25) / 5))) && (1 <= (((((int)blockIdx.x) % 7) * 4) + (((int)threadIdx.x) % 5)))) ? data[(((((((rc_outer_outer * 1568) + ((((int)threadIdx.x) / 25) * 784)) + ((((int)blockIdx.x) / 7) * 112)) + (((((int)threadIdx.x) % 25) / 5) * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) % 5)) - 29)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 18) * 1152) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) + 256) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) + 512) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) + 768) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) + 1024) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) + 1280) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) + 1536) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) + 1792) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)threadIdx.x) + 2048) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[(((((((int)threadIdx.x) / 18) * 1152) + (rc_outer_outer * 18)) + (((int)threadIdx.x) % 18)) + 147456)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)threadIdx.x) + 2560) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 4) % 18))];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)threadIdx.x) + 2816) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 8) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)threadIdx.x) + 3072) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 12) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)threadIdx.x) + 3328) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 16) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)threadIdx.x) + 3584) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 2) % 18))];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)threadIdx.x) + 3840) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 6) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)threadIdx.x) + 4096) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 10) % 18))];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)threadIdx.x) + 4352) / 18) * 1152) + (rc_outer_outer * 18)) + ((((int)threadIdx.x) + 14) % 18))];
    __syncthreads();
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[((((int)threadIdx.x) & 1) * 2)] * kernel_shared[((((int)threadIdx.x) >> 1) * 18)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[((((int)threadIdx.x) & 1) * 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2304)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 1)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 1)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2305)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 2)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2306)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 5)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 3)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 5)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2307)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 4)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 6)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2308)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 5)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 7)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2309)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 6)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2310)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 7)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2311)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 8)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2312)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 25)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 9)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 25)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2313)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 26)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 10)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 26)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2314)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 27)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 11)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 27)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2315)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 12)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 30)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2316)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 13)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 31)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2317)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 14)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2318)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 15)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2319)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 16)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2320)]));
    conv2d_nchw_local[0] = (conv2d_nchw_local[0] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 17)]));
    conv2d_nchw_local[2] = (conv2d_nchw_local[2] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2321)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[((((int)threadIdx.x) >> 1) * 18)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 10)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2304)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 1)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 11)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2305)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 12)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2306)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 15)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 3)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 15)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2307)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 4)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 16)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2308)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 17)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 5)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 17)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2309)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 6)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 20)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2310)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 21)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 7)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 21)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2311)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 22)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 8)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 22)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2312)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 9)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 35)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2313)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 10)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 36)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2314)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 11)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 37)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2315)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 12)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2316)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 41)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 13)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 41)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2317)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 42)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 14)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 42)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2318)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 45)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 15)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 45)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2319)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 46)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 16)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 46)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2320)]));
    conv2d_nchw_local[1] = (conv2d_nchw_local[1] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 47)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 17)]));
    conv2d_nchw_local[3] = (conv2d_nchw_local[3] + (pad_temp_shared[(((((int)threadIdx.x) & 1) * 2) + 47)] * kernel_shared[(((((int)threadIdx.x) >> 1) * 18) + 2321)]));
  }
  for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
    conv2d_nchw[((((((((int)threadIdx.x) >> 1) * 196) + ((((int)blockIdx.x) / 7) * 28)) + (yy_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1))] = conv2d_nchw_local[yy_inner];
    conv2d_nchw[(((((((((int)threadIdx.x) >> 1) * 196) + ((((int)blockIdx.x) / 7) * 28)) + (yy_inner * 14)) + ((((int)blockIdx.x) % 7) * 2)) + (((int)threadIdx.x) & 1)) + 25088)] = conv2d_nchw_local[(yy_inner + 2)];
  }
}


