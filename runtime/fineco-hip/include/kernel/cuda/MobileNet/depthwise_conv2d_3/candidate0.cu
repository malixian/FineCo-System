#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[4];
  __shared__ float PaddedInput_shared[1440];
  __shared__ float kernel_shared[72];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((((1 <= ((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((int)threadIdx.x) % 180) / 30))) && (((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((int)threadIdx.x) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + (((int)threadIdx.x) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + (((int)threadIdx.x) % 30)) < 57)) ? Input[((((((((((int)blockIdx.x) / 28) * 25088) + ((((int)threadIdx.x) / 180) * 3136)) + (((((int)blockIdx.x) % 28) >> 1) * 224)) + (((((int)threadIdx.x) % 180) / 30) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + (((int)threadIdx.x) % 30)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 224)] = (((((1 <= ((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 22) % 90) / 15))) && (((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 22) % 90) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 14) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 14) % 30)) < 57)) ? Input[((((((((((int)blockIdx.x) / 28) * 25088) + (((((int)threadIdx.x) + 224) / 180) * 3136)) + (((((int)blockIdx.x) % 28) >> 1) * 224)) + (((((((int)threadIdx.x) >> 1) + 22) % 90) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 14) % 30)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = (((((1 <= ((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 44) % 90) / 15))) && (((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 44) % 90) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 28) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 28) % 30)) < 57)) ? Input[((((((((((int)blockIdx.x) / 28) * 25088) + (((((int)threadIdx.x) + 448) / 180) * 3136)) + (((((int)blockIdx.x) % 28) >> 1) * 224)) + (((((((int)threadIdx.x) >> 1) + 44) % 90) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 28) % 30)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 672)] = (((((1 <= ((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 66) % 90) / 15))) && (((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 66) % 90) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 12) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 12) % 30)) < 57)) ? Input[((((((((((int)blockIdx.x) / 28) * 25088) + (((((int)threadIdx.x) + 672) / 180) * 3136)) + (((((int)blockIdx.x) % 28) >> 1) * 224)) + (((((((int)threadIdx.x) >> 1) + 66) % 90) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 12) % 30)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((((1 <= ((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 88) % 90) / 15))) && (((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 88) % 90) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 26) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 26) % 30)) < 57)) ? Input[((((((((((int)blockIdx.x) / 28) * 25088) + (((((int)threadIdx.x) + 896) / 180) * 3136)) + (((((int)blockIdx.x) % 28) >> 1) * 224)) + (((((((int)threadIdx.x) >> 1) + 88) % 90) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 26) % 30)) - 57)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1120)] = (((((1 <= ((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 20) % 90) / 15))) && (((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 20) % 90) / 15)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 10) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 10) % 30)) < 57)) ? Input[((((((((((int)blockIdx.x) / 28) * 25088) + (((((int)threadIdx.x) + 1120) / 180) * 3136)) + (((((int)blockIdx.x) % 28) >> 1) * 224)) + (((((((int)threadIdx.x) >> 1) + 20) % 90) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 10) % 30)) - 57)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 96) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = ((((((((((int)blockIdx.x) % 28) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 42) % 90) / 15)) < 57) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 24) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((int)threadIdx.x) + 24) % 30)) < 57)) ? Input[((((((((((int)blockIdx.x) / 28) * 25088) + (((((int)threadIdx.x) + 1344) / 180) * 3136)) + (((((int)blockIdx.x) % 28) >> 1) * 224)) + (((((((int)threadIdx.x) >> 1) + 42) % 90) / 15) * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) + 24) % 30)) - 57)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 72) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) / 28) * 72) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2))] * kernel_shared[(((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3))]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 30)] * kernel_shared[(((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3))]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3))]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 31)] * kernel_shared[(((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3))]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3)) + 1)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 31)] * kernel_shared[((((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3)) + 1)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 2)] * kernel_shared[((((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3)) + 1)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 32)] * kernel_shared[((((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3)) + 1)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 2)] * kernel_shared[((((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3)) + 2)]));
    DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 32)] * kernel_shared[((((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3)) + 2)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 3)] * kernel_shared[((((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3)) + 2)]));
    DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 180) + (((((int)threadIdx.x) % 28) / 14) * 60)) + (di_outer_inner * 30)) + ((((int)threadIdx.x) % 14) * 2)) + 33)] * kernel_shared[((((((int)threadIdx.x) / 28) * 9) + (di_outer_inner * 3)) + 2)]));
  }
  for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
    for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
      compute[(((((((((((int)blockIdx.x) / 28) * 25088) + ((((int)threadIdx.x) / 28) * 3136)) + (((((int)blockIdx.x) % 28) >> 1) * 224)) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) & 1) * 28)) + ((((int)threadIdx.x) % 14) * 2)) + i3_inner)] = max(DepthwiseConv2d[((i2_inner * 2) + i3_inner)], 0.000000e+00f);
    }
  }
}


