#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(196) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[128];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[1024];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 4; ++ff_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 4; ++xx_inner_init) {
        conv2d_nchw[(((ff_outer_inner_init * 16) + (ff_inner_init * 4)) + xx_inner_init)] = 0.000000e+00f;
        conv2d_nchw[((((ff_outer_inner_init * 16) + (ff_inner_init * 4)) + xx_inner_init) + 32)] = 0.000000e+00f;
        conv2d_nchw[((((ff_outer_inner_init * 16) + (ff_inner_init * 4)) + xx_inner_init) + 64)] = 0.000000e+00f;
        conv2d_nchw[((((ff_outer_inner_init * 16) + (ff_inner_init * 4)) + xx_inner_init) + 96)] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 196) + ((int)threadIdx.x))] = Input[(((((rc_outer_outer * 50176) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 1) * 3136)) + ((((int)blockIdx.x) & 7) * 392)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 1) * 196)) + ((int)threadIdx.x))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 49) + (((int)threadIdx.x) >> 2)) < 256) {
        kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 196) + ((int)threadIdx.x))] = kernel[(((((((int)blockIdx.x) >> 3) * 8192) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 49) + (((int)threadIdx.x) >> 2)) >> 2) * 128)) + (rc_outer_outer * 16)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 196) + ((int)threadIdx.x)) & 15))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
          for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
            for (int xx_inner = 0; xx_inner < 4; ++xx_inner) {
              conv2d_nchw[(((ff_outer_inner * 16) + (ff_inner * 4)) + xx_inner)] = (conv2d_nchw[(((ff_outer_inner * 16) + (ff_inner * 4)) + xx_inner)] + (pad_temp_shared[((((rc_outer_inner * 1568) + (rc_inner * 392)) + ((((int)threadIdx.x) % 98) * 4)) + xx_inner)] * kernel_shared[((((((((int)threadIdx.x) / 98) * 128) + (ff_outer_inner * 64)) + (ff_inner * 16)) + (rc_outer_inner * 4)) + rc_inner)]));
              conv2d_nchw[((((ff_outer_inner * 16) + (ff_inner * 4)) + xx_inner) + 32)] = (conv2d_nchw[((((ff_outer_inner * 16) + (ff_inner * 4)) + xx_inner) + 32)] + (pad_temp_shared[((((rc_outer_inner * 1568) + (rc_inner * 392)) + ((((int)threadIdx.x) % 98) * 4)) + xx_inner)] * kernel_shared[(((((((((int)threadIdx.x) / 98) * 128) + (ff_outer_inner * 64)) + (ff_inner * 16)) + (rc_outer_inner * 4)) + rc_inner) + 256)]));
              conv2d_nchw[((((ff_outer_inner * 16) + (ff_inner * 4)) + xx_inner) + 64)] = (conv2d_nchw[((((ff_outer_inner * 16) + (ff_inner * 4)) + xx_inner) + 64)] + (pad_temp_shared[((((rc_outer_inner * 1568) + (rc_inner * 392)) + ((((int)threadIdx.x) % 98) * 4)) + xx_inner)] * kernel_shared[(((((((((int)threadIdx.x) / 98) * 128) + (ff_outer_inner * 64)) + (ff_inner * 16)) + (rc_outer_inner * 4)) + rc_inner) + 512)]));
              conv2d_nchw[((((ff_outer_inner * 16) + (ff_inner * 4)) + xx_inner) + 96)] = (conv2d_nchw[((((ff_outer_inner * 16) + (ff_inner * 4)) + xx_inner) + 96)] + (pad_temp_shared[((((rc_outer_inner * 1568) + (rc_inner * 392)) + ((((int)threadIdx.x) % 98) * 4)) + xx_inner)] * kernel_shared[(((((((((int)threadIdx.x) / 98) * 128) + (ff_outer_inner * 64)) + (ff_inner * 16)) + (rc_outer_inner * 4)) + rc_inner) + 768)]));
            }
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    for (int i3_inner = 0; i3_inner < 4; ++i3_inner) {
      compute[(((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 98) * 25088)) + (i1_inner * 3136)) + ((((int)blockIdx.x) & 7) * 392)) + ((((int)threadIdx.x) % 98) * 4)) + i3_inner)] = max(conv2d_nchw[((i1_inner * 4) + i3_inner)], 0.000000e+00f);
      compute[((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 98) * 25088)) + (i1_inner * 3136)) + ((((int)blockIdx.x) & 7) * 392)) + ((((int)threadIdx.x) % 98) * 4)) + i3_inner) + 50176)] = max(conv2d_nchw[(((i1_inner * 4) + i3_inner) + 32)], 0.000000e+00f);
      compute[((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 98) * 25088)) + (i1_inner * 3136)) + ((((int)blockIdx.x) & 7) * 392)) + ((((int)threadIdx.x) % 98) * 4)) + i3_inner) + 100352)] = max(conv2d_nchw[(((i1_inner * 4) + i3_inner) + 64)], 0.000000e+00f);
      compute[((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 98) * 25088)) + (i1_inner * 3136)) + ((((int)blockIdx.x) & 7) * 392)) + ((((int)threadIdx.x) % 98) * 4)) + i3_inner) + 150528)] = max(conv2d_nchw[(((i1_inner * 4) + i3_inner) + 96)], 0.000000e+00f);
    }
  }
}


