#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long

extern "C" __global__ void __launch_bounds__(448) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[32];
  __shared__ float pad_temp_shared[2025];
  __shared__ float kernel_shared[288];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    conv2d_nchw[(ff_outer_inner_init * 4)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 8)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 16)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 24)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 1)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 9)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 17)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 25)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 2)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 10)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 18)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 26)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 3)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 11)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 19)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 27)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 3; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((1 <= ((((int)blockIdx.x) * 8) + (((int)threadIdx.x) / 225))) && (1 <= (((int)threadIdx.x) % 225))) ? data[(((((rc_outer_outer * 50176) + (((int)blockIdx.x) * 1792)) + ((((int)threadIdx.x) / 225) * 224)) + (((int)threadIdx.x) % 225)) - 225)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = ((1 <= ((((int)threadIdx.x) + 223) % 225)) ? data[(((((rc_outer_outer * 50176) + (((int)blockIdx.x) * 1792)) + (((((int)threadIdx.x) + 448) / 225) * 224)) + ((((int)threadIdx.x) + 223) % 225)) - 225)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = ((1 <= ((((int)threadIdx.x) + 221) % 225)) ? data[(((((rc_outer_outer * 50176) + (((int)blockIdx.x) * 1792)) + (((((int)threadIdx.x) + 896) / 225) * 224)) + ((((int)threadIdx.x) + 221) % 225)) - 225)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = ((1 <= ((((int)threadIdx.x) + 219) % 225)) ? data[(((((rc_outer_outer * 50176) + (((int)blockIdx.x) * 1792)) + (((((int)threadIdx.x) + 1344) / 225) * 224)) + ((((int)threadIdx.x) + 219) % 225)) - 225)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 233) {
      pad_temp_shared[(((int)threadIdx.x) + 1792)] = ((1 <= ((((int)threadIdx.x) + 217) % 225)) ? data[(((((rc_outer_outer * 50176) + (((int)blockIdx.x) * 1792)) + (((((int)threadIdx.x) + 1792) / 225) * 224)) + ((((int)threadIdx.x) + 217) % 225)) - 225)] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 288) {
      kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 9) * 27) + (rc_outer_outer * 9)) + (((int)threadIdx.x) % 9))];
    }
    __syncthreads();
    for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
          for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
            conv2d_nchw[((ff_outer_inner * 4) + (yy_outer_inner * 2))] = (conv2d_nchw[((ff_outer_inner * 4) + (yy_outer_inner * 2))] + (pad_temp_shared[((((yy_outer_inner * 900) + (ry_outer_inner * 225)) + ((((int)threadIdx.x) % 112) * 2)) + rx_inner)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 18) + (ff_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
            conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 8)] = (conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 8)] + (pad_temp_shared[((((yy_outer_inner * 900) + (ry_outer_inner * 225)) + ((((int)threadIdx.x) % 112) * 2)) + rx_inner)] * kernel_shared[((((((((int)threadIdx.x) / 112) * 18) + (ff_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
            conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 16)] = (conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 16)] + (pad_temp_shared[((((yy_outer_inner * 900) + (ry_outer_inner * 225)) + ((((int)threadIdx.x) % 112) * 2)) + rx_inner)] * kernel_shared[((((((((int)threadIdx.x) / 112) * 18) + (ff_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 144)]));
            conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 24)] = (conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 24)] + (pad_temp_shared[((((yy_outer_inner * 900) + (ry_outer_inner * 225)) + ((((int)threadIdx.x) % 112) * 2)) + rx_inner)] * kernel_shared[((((((((int)threadIdx.x) / 112) * 18) + (ff_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 216)]));
            conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 1)] = (conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 1)] + (pad_temp_shared[(((((yy_outer_inner * 900) + (ry_outer_inner * 225)) + ((((int)threadIdx.x) % 112) * 2)) + rx_inner) + 450)] * kernel_shared[(((((((int)threadIdx.x) / 112) * 18) + (ff_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
            conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 9)] = (conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 9)] + (pad_temp_shared[(((((yy_outer_inner * 900) + (ry_outer_inner * 225)) + ((((int)threadIdx.x) % 112) * 2)) + rx_inner) + 450)] * kernel_shared[((((((((int)threadIdx.x) / 112) * 18) + (ff_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 72)]));
            conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 17)] = (conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 17)] + (pad_temp_shared[(((((yy_outer_inner * 900) + (ry_outer_inner * 225)) + ((((int)threadIdx.x) % 112) * 2)) + rx_inner) + 450)] * kernel_shared[((((((((int)threadIdx.x) / 112) * 18) + (ff_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 144)]));
            conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 25)] = (conv2d_nchw[(((ff_outer_inner * 4) + (yy_outer_inner * 2)) + 25)] + (pad_temp_shared[(((((yy_outer_inner * 900) + (ry_outer_inner * 225)) + ((((int)threadIdx.x) % 112) * 2)) + rx_inner) + 450)] * kernel_shared[((((((((int)threadIdx.x) / 112) * 18) + (ff_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_inner) + 216)]));
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
      compute[((((((((int)threadIdx.x) / 112) * 25088) + (i1_inner * 12544)) + (((int)blockIdx.x) * 448)) + (i2_inner * 112)) + (((int)threadIdx.x) % 112))] = max((conv2d_nchw[((i1_inner * 4) + i2_inner)] + bias[(((((int)threadIdx.x) / 112) * 2) + i1_inner)]), 0.000000e+00f);
      compute[(((((((((int)threadIdx.x) / 112) * 25088) + (i1_inner * 12544)) + (((int)blockIdx.x) * 448)) + (i2_inner * 112)) + (((int)threadIdx.x) % 112)) + 100352)] = max((conv2d_nchw[(((i1_inner * 4) + i2_inner) + 8)] + bias[((((((int)threadIdx.x) / 112) * 2) + i1_inner) + 8)]), 0.000000e+00f);
      compute[(((((((((int)threadIdx.x) / 112) * 25088) + (i1_inner * 12544)) + (((int)blockIdx.x) * 448)) + (i2_inner * 112)) + (((int)threadIdx.x) % 112)) + 200704)] = max((conv2d_nchw[(((i1_inner * 4) + i2_inner) + 16)] + bias[((((((int)threadIdx.x) / 112) * 2) + i1_inner) + 16)]), 0.000000e+00f);
      compute[(((((((((int)threadIdx.x) / 112) * 25088) + (i1_inner * 12544)) + (((int)blockIdx.x) * 448)) + (i2_inner * 112)) + (((int)threadIdx.x) % 112)) + 301056)] = max((conv2d_nchw[(((i1_inner * 4) + i2_inner) + 24)] + bias[((((((int)threadIdx.x) / 112) * 2) + i1_inner) + 24)]), 0.000000e+00f);
    }
  }
}


