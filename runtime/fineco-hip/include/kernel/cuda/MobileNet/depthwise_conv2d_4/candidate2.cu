#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) candidate2(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[4];
  __shared__ float PaddedInput_shared[10368];
  __shared__ float kernel_shared[1152];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 4; ++i_outer_inner_init) {
    DepthwiseConv2d[i_outer_inner_init] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 21; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 7)) < 81) {
      PaddedInput_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x))] = (((1 <= (((((int)blockIdx.x) / 7) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) % 81) / 9))) && (1 <= (((((int)blockIdx.x) % 7) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) % 9)))) ? Input[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) / 81) * 3136) + ((((int)blockIdx.x) / 7) * 448)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) % 81) / 9) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) % 9)) - 57)] : 0.000000e+00f);
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 4) + (((int)threadIdx.x) >> 7)) < 9) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 512) + ((int)threadIdx.x))] = kernel[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 512) + ((int)threadIdx.x))];
    }
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    for (int i_outer_inner = 0; i_outer_inner < 4; ++i_outer_inner) {
      for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
        DepthwiseConv2d[i_outer_inner] = (DepthwiseConv2d[i_outer_inner] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 81) + (i_outer_inner * 18)) + (di_outer_inner * 9)) + ((((int)threadIdx.x) & 3) * 2)) + dj_inner)] * kernel_shared[((((((int)threadIdx.x) >> 2) * 9) + (di_outer_inner * 3)) + dj_inner)]));
      }
    }
  }
  for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
    compute[((((((((int)threadIdx.x) >> 2) * 784) + ((((int)blockIdx.x) / 7) * 112)) + (i2_inner * 28)) + ((((int)blockIdx.x) % 7) * 4)) + (((int)threadIdx.x) & 3))] = max(DepthwiseConv2d[i2_inner], 0.000000e+00f);
  }
}


