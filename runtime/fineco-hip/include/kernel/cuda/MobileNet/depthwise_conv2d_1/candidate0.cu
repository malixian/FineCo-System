#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[4];
  __shared__ float PaddedInput_shared[1392];
  __shared__ float kernel_shared[36];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = ((((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 4) + (((int)threadIdx.x) / 58))) && (1 <= (((((int)blockIdx.x) & 1) * 56) + (((int)threadIdx.x) % 58)))) && ((((((int)blockIdx.x) & 1) * 56) + (((int)threadIdx.x) % 58)) < 113)) ? Input[(((((((((int)blockIdx.x) / 56) * 50176) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + ((((int)threadIdx.x) / 58) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + (((int)threadIdx.x) % 58)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 224)] = (((((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 112) % 174) / 29))) && (((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 112) % 174) / 29)) < 113)) && (1 <= (((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 50) % 58)))) && ((((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 50) % 58)) < 113)) ? Input[((((((((((int)blockIdx.x) / 56) * 50176) + (((((int)threadIdx.x) + 224) / 348) * 12544)) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + (((((((int)threadIdx.x) >> 1) + 112) % 174) / 29) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) + 50) % 58)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = ((((((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 50) % 174) / 29)) < 113) && (1 <= (((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 42) % 58)))) && ((((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 42) % 58)) < 113)) ? Input[((((((((((int)blockIdx.x) / 56) * 50176) + (((((int)threadIdx.x) + 448) / 348) * 12544)) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + (((((((int)threadIdx.x) >> 1) + 50) % 174) / 29) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) + 42) % 58)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 672)] = (((((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 162) % 174) / 29))) && (((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 162) % 174) / 29)) < 113)) && (1 <= (((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 34) % 58)))) && ((((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 34) % 58)) < 113)) ? Input[((((((((((int)blockIdx.x) / 56) * 50176) + (((((int)threadIdx.x) + 672) / 348) * 12544)) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + (((((((int)threadIdx.x) >> 1) + 162) % 174) / 29) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) + 34) % 58)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((((1 <= ((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 100) % 174) / 29))) && (((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 100) % 174) / 29)) < 113)) && (1 <= (((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 26) % 58)))) && ((((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 26) % 58)) < 113)) ? Input[((((((((((int)blockIdx.x) / 56) * 50176) + (((((int)threadIdx.x) + 896) / 348) * 12544)) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + (((((((int)threadIdx.x) >> 1) + 100) % 174) / 29) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) + 26) % 58)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1120)] = ((((((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 38) % 174) / 29)) < 113) && (1 <= (((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 18) % 58)))) && ((((((int)blockIdx.x) & 1) * 56) + ((((int)threadIdx.x) + 18) % 58)) < 113)) ? Input[((((((((((int)blockIdx.x) / 56) * 50176) + (((((int)threadIdx.x) + 1120) / 348) * 12544)) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + (((((((int)threadIdx.x) >> 1) + 38) % 174) / 29) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) + 18) % 58)) - 113)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 48) {
    PaddedInput_shared[(((int)threadIdx.x) + 1344)] = (((((((((int)blockIdx.x) % 56) >> 1) * 4) + ((((((int)threadIdx.x) >> 1) + 150) % 174) / 29)) < 113) && ((((((int)blockIdx.x) & 1) * 56) + (((int)threadIdx.x) + 10)) < 113)) ? Input[((((((((((int)blockIdx.x) / 56) * 50176) + (((((int)threadIdx.x) + 1344) / 348) * 12544)) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + (((((((int)threadIdx.x) >> 1) + 150) % 174) / 29) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + (((int)threadIdx.x) + 10)) - 113)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 36) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) / 56) * 36) + ((int)threadIdx.x))];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2))] * kernel_shared[((((int)threadIdx.x) / 56) * 9)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 116)] * kernel_shared[((((int)threadIdx.x) / 56) * 9)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 117)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 1)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 2)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 118)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 1)] * kernel_shared[((((int)threadIdx.x) / 56) * 9)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 117)] * kernel_shared[((((int)threadIdx.x) / 56) * 9)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 1)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 118)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 1)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 3)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 119)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 2)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 58)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 3)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 174)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 3)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 59)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 4)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 175)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 4)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 60)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 5)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 176)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 5)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 59)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 3)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 175)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 3)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 60)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 4)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 176)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 4)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 61)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 5)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 177)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 5)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 116)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 6)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 232)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 6)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 117)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 7)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 233)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 7)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 118)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 8)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 234)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 117)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 6)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 233)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 6)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 118)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 7)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 234)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 7)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 119)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 8)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[(((((((int)threadIdx.x) / 56) * 348) + (((((int)threadIdx.x) % 56) / 28) * 58)) + ((((int)threadIdx.x) % 28) * 2)) + 235)] * kernel_shared[(((((int)threadIdx.x) / 56) * 9) + 8)]));
  for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
    compute[((((((((((int)blockIdx.x) / 56) * 50176) + ((((int)threadIdx.x) / 56) * 12544)) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + (((((int)threadIdx.x) % 56) / 28) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) % 28) * 2)) + i3_inner)] = max(DepthwiseConv2d[i3_inner], 0.000000e+00f);
    compute[(((((((((((int)blockIdx.x) / 56) * 50176) + ((((int)threadIdx.x) / 56) * 12544)) + (((((int)blockIdx.x) % 56) >> 1) * 448)) + (((((int)threadIdx.x) % 56) / 28) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((int)threadIdx.x) % 28) * 2)) + i3_inner) + 224)] = max(DepthwiseConv2d[(i3_inner + 2)], 0.000000e+00f);
  }
}


