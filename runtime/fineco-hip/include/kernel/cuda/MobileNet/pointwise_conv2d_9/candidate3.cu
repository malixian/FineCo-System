#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate3(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[7];
  __shared__ float pad_temp_shared[448];
  __shared__ float kernel_shared[8192];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[((((rc_outer_outer * 3136) + ((((int)threadIdx.x) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + (((int)threadIdx.x) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = Input[((((rc_outer_outer * 3136) + (((((int)threadIdx.x) + 128) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 2) % 7))];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = Input[((((rc_outer_outer * 3136) + (((((int)threadIdx.x) + 256) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 4) % 7))];
    if (((int)threadIdx.x) < 64) {
      pad_temp_shared[(((int)threadIdx.x) + 384)] = Input[((((rc_outer_outer * 3136) + (((((int)threadIdx.x) + 384) / 7) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + ((((int)threadIdx.x) + 6) % 7))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 2048)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 6144)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 10240)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 12288)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 16384)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 18432)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 20480)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 22528)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 24576)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 26624)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 30720)];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 32768)];
    kernel_shared[(((int)threadIdx.x) + 2176)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 34816)];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 36864)];
    kernel_shared[(((int)threadIdx.x) + 2432)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 38912)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 40960)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 43008)];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 45056)];
    kernel_shared[(((int)threadIdx.x) + 2944)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 47104)];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 49152)];
    kernel_shared[(((int)threadIdx.x) + 3200)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 51200)];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 53248)];
    kernel_shared[(((int)threadIdx.x) + 3456)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 55296)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 3712)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 59392)];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 61440)];
    kernel_shared[(((int)threadIdx.x) + 3968)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 63488)];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 65536)];
    kernel_shared[(((int)threadIdx.x) + 4224)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 67584)];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 69632)];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 71680)];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 73728)];
    kernel_shared[(((int)threadIdx.x) + 4736)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 75776)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 77824)];
    kernel_shared[(((int)threadIdx.x) + 4992)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 79872)];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 81920)];
    kernel_shared[(((int)threadIdx.x) + 5248)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 83968)];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 86016)];
    kernel_shared[(((int)threadIdx.x) + 5504)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 88064)];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 90112)];
    kernel_shared[(((int)threadIdx.x) + 5760)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 92160)];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 94208)];
    kernel_shared[(((int)threadIdx.x) + 6016)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 96256)];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 98304)];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 100352)];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 102400)];
    kernel_shared[(((int)threadIdx.x) + 6528)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 104448)];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 106496)];
    kernel_shared[(((int)threadIdx.x) + 6784)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 108544)];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 110592)];
    kernel_shared[(((int)threadIdx.x) + 7040)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 112640)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 114688)];
    kernel_shared[(((int)threadIdx.x) + 7296)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 116736)];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 118784)];
    kernel_shared[(((int)threadIdx.x) + 7552)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 120832)];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 122880)];
    kernel_shared[(((int)threadIdx.x) + 7808)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 124928)];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 126976)];
    kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[((((((((int)blockIdx.x) / 7) * 131072) + ((((int)threadIdx.x) >> 6) * 1024)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 129024)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((rc_outer_inner * 112) + (rc_inner * 7))] * kernel_shared[(((((int)threadIdx.x) * 64) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 7)) + 1)] * kernel_shared[(((((int)threadIdx.x) * 64) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 7)) + 2)] * kernel_shared[(((((int)threadIdx.x) * 64) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 7)) + 3)] * kernel_shared[(((((int)threadIdx.x) * 64) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 7)) + 4)] * kernel_shared[(((((int)threadIdx.x) * 64) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 7)) + 5)] * kernel_shared[(((((int)threadIdx.x) * 64) + (rc_outer_inner * 16)) + rc_inner)]));
        conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 112) + (rc_inner * 7)) + 6)] * kernel_shared[(((((int)threadIdx.x) * 64) + (rc_outer_inner * 16)) + rc_inner)]));
      }
    }
  }
  compute[((((((int)blockIdx.x) / 7) * 6272) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7))] = max(conv2d_nchw[0], 0.000000e+00f);
  compute[(((((((int)blockIdx.x) / 7) * 6272) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + 1)] = max(conv2d_nchw[1], 0.000000e+00f);
  compute[(((((((int)blockIdx.x) / 7) * 6272) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + 2)] = max(conv2d_nchw[2], 0.000000e+00f);
  compute[(((((((int)blockIdx.x) / 7) * 6272) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + 3)] = max(conv2d_nchw[3], 0.000000e+00f);
  compute[(((((((int)blockIdx.x) / 7) * 6272) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + 4)] = max(conv2d_nchw[4], 0.000000e+00f);
  compute[(((((((int)blockIdx.x) / 7) * 6272) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + 5)] = max(conv2d_nchw[5], 0.000000e+00f);
  compute[(((((((int)blockIdx.x) / 7) * 6272) + (((int)threadIdx.x) * 49)) + ((((int)blockIdx.x) % 7) * 7)) + 6)] = max(conv2d_nchw[6], 0.000000e+00f);
}


