#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(392) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[16];
  __shared__ float PaddedInput_shared[8192];
  __shared__ float kernel_shared[288];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  DepthwiseConv2d[4] = 0.000000e+00f;
  DepthwiseConv2d[5] = 0.000000e+00f;
  DepthwiseConv2d[6] = 0.000000e+00f;
  DepthwiseConv2d[7] = 0.000000e+00f;
  DepthwiseConv2d[8] = 0.000000e+00f;
  DepthwiseConv2d[9] = 0.000000e+00f;
  DepthwiseConv2d[10] = 0.000000e+00f;
  DepthwiseConv2d[11] = 0.000000e+00f;
  DepthwiseConv2d[12] = 0.000000e+00f;
  DepthwiseConv2d[13] = 0.000000e+00f;
  DepthwiseConv2d[14] = 0.000000e+00f;
  DepthwiseConv2d[15] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((((16 <= (((int)threadIdx.x) & 255)) && ((((int)threadIdx.x) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + ((((int)threadIdx.x) >> 8) * 196)) + (((((int)threadIdx.x) & 255) >> 4) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 392)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 17) & 31)) && (((((int)threadIdx.x) + 136) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 392) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 17) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 784)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 1) & 15)) && (((((int)threadIdx.x) + 16) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 784) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 1) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1176)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 19) & 31)) && (((((int)threadIdx.x) + 152) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 1176) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 19) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1568)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 2) & 15)) && (((((int)threadIdx.x) + 32) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 1568) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 2) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1960)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 21) & 31)) && (((((int)threadIdx.x) + 168) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 1960) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 21) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2352)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 3) & 15)) && (((((int)threadIdx.x) + 48) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 2352) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 3) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2744)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 23) & 31)) && (((((int)threadIdx.x) + 184) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 2744) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 23) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3136)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 4) & 15)) && (((((int)threadIdx.x) + 64) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 3136) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 4) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3528)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 25) & 31)) && (((((int)threadIdx.x) + 200) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 3528) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 25) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3920)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 5) & 15)) && (((((int)threadIdx.x) + 80) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 3920) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 5) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4312)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 27) & 31)) && (((((int)threadIdx.x) + 216) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 4312) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 27) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 4704)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 6) & 15)) && (((((int)threadIdx.x) + 96) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 4704) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 6) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5096)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 29) & 31)) && (((((int)threadIdx.x) + 232) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 5096) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 29) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5488)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 7) & 15)) && (((((int)threadIdx.x) + 112) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 5488) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 7) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 5880)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 31) & 31)) && (((((int)threadIdx.x) + 248) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 5880) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 31) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6272)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 8) & 15)) && (((((int)threadIdx.x) + 128) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 6272) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 8) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 6664)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 1) & 31)) && (((((int)threadIdx.x) + 8) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 6664) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 1) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 7056)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 9) & 15)) && (((((int)threadIdx.x) + 144) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 7056) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 9) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 7448)] = (((((2 <= (((((int)threadIdx.x) >> 3) + 3) & 31)) && (((((int)threadIdx.x) + 24) & 255) < 240)) && (1 <= ((((int)threadIdx.x) + 8) & 15))) && (((((int)threadIdx.x) + 8) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 7448) >> 8) * 196)) + (((((((int)threadIdx.x) >> 3) + 3) & 31) >> 1) * 14)) + ((((int)threadIdx.x) + 8) & 15)) - 15)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 352) {
    PaddedInput_shared[(((int)threadIdx.x) + 7840)] = (((((1 <= (((((int)threadIdx.x) >> 4) + 10) & 15)) && (((((int)threadIdx.x) + 160) & 255) < 240)) && (1 <= (((int)threadIdx.x) & 15))) && ((((int)threadIdx.x) & 15) < 15)) ? Input[(((((((int)blockIdx.x) * 6272) + (((((int)threadIdx.x) + 7840) >> 8) * 196)) + ((((((int)threadIdx.x) >> 4) + 10) & 15) * 14)) + (((int)threadIdx.x) & 15)) - 15)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 288) {
    kernel_shared[((int)threadIdx.x)] = kernel[((((int)blockIdx.x) * 288) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int c_outer_inner = 0; c_outer_inner < 8; ++c_outer_inner) {
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[(((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9))]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9))]));
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 1)]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 1)]));
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 2)]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 3)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 2)]));
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 16)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 3)]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 17)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 3)]));
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 17)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 4)]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 18)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 4)]));
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 18)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 5)]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 19)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 5)]));
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 32)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 6)]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 33)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 6)]));
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 33)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 7)]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 34)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 7)]));
    DepthwiseConv2d[(c_outer_inner * 2)] = (DepthwiseConv2d[(c_outer_inner * 2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 34)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 8)]));
    DepthwiseConv2d[((c_outer_inner * 2) + 1)] = (DepthwiseConv2d[((c_outer_inner * 2) + 1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 98) * 2048) + (c_outer_inner * 256)) + (((((int)threadIdx.x) % 98) / 7) * 16)) + ((((int)threadIdx.x) % 7) * 2)) + 35)] * kernel_shared[((((((int)threadIdx.x) / 98) * 72) + (c_outer_inner * 9)) + 8)]));
  }
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
      compute[(((((((int)blockIdx.x) * 6272) + ((((int)threadIdx.x) / 98) * 1568)) + (i1_inner * 196)) + ((((int)threadIdx.x) % 98) * 2)) + i3_inner)] = max(DepthwiseConv2d[((i1_inner * 2) + i3_inner)], 0.000000e+00f);
    }
  }
}


