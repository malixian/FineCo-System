#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate3(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[7];
  __shared__ float PaddedInput_shared[3600];
  __shared__ float kernel_shared[144];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  DepthwiseConv2d[2] = 0.000000e+00f;
  DepthwiseConv2d[3] = 0.000000e+00f;
  DepthwiseConv2d[4] = 0.000000e+00f;
  DepthwiseConv2d[5] = 0.000000e+00f;
  DepthwiseConv2d[6] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((15 <= ((int)threadIdx.x)) && (1 <= (((int)threadIdx.x) % 15))) ? Input[((((((int)blockIdx.x) * 3136) + ((((int)threadIdx.x) / 15) * 14)) + (((int)threadIdx.x) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 112)] = ((1 <= ((((int)threadIdx.x) + 7) % 15)) ? Input[((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 112) / 15) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 224)] = (((15 <= ((((int)threadIdx.x) + 224) % 225)) && (1 <= ((((int)threadIdx.x) + 14) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 224) / 225) * 196)) + ((((((int)threadIdx.x) + 224) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 336)] = ((1 <= ((((int)threadIdx.x) + 6) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 336) / 225) * 196)) + ((((((int)threadIdx.x) + 111) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = (((15 <= ((((int)threadIdx.x) + 223) % 225)) && (1 <= ((((int)threadIdx.x) + 13) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 448) / 225) * 196)) + ((((((int)threadIdx.x) + 223) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 560)] = ((1 <= ((((int)threadIdx.x) + 5) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 560) / 225) * 196)) + ((((((int)threadIdx.x) + 110) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 672)] = (((15 <= ((((int)threadIdx.x) + 222) % 225)) && (1 <= ((((int)threadIdx.x) + 12) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 672) / 225) * 196)) + ((((((int)threadIdx.x) + 222) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 784)] = ((1 <= ((((int)threadIdx.x) + 4) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 784) / 225) * 196)) + ((((((int)threadIdx.x) + 109) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((15 <= ((((int)threadIdx.x) + 221) % 225)) && (1 <= ((((int)threadIdx.x) + 11) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 896) / 225) * 196)) + ((((((int)threadIdx.x) + 221) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1008)] = ((1 <= ((((int)threadIdx.x) + 3) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1008) / 225) * 196)) + ((((((int)threadIdx.x) + 108) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1120)] = (((15 <= ((((int)threadIdx.x) + 220) % 225)) && (1 <= ((((int)threadIdx.x) + 10) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1120) / 225) * 196)) + ((((((int)threadIdx.x) + 220) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1232)] = ((1 <= ((((int)threadIdx.x) + 2) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1232) / 225) * 196)) + ((((((int)threadIdx.x) + 107) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1344)] = (((15 <= ((((int)threadIdx.x) + 219) % 225)) && (1 <= ((((int)threadIdx.x) + 9) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1344) / 225) * 196)) + ((((((int)threadIdx.x) + 219) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1456)] = ((1 <= ((((int)threadIdx.x) + 1) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1456) / 225) * 196)) + ((((((int)threadIdx.x) + 106) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1568)] = (((15 <= ((((int)threadIdx.x) + 218) % 225)) && (1 <= ((((int)threadIdx.x) + 8) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1568) / 225) * 196)) + ((((((int)threadIdx.x) + 218) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1680)] = ((1 <= (((int)threadIdx.x) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1680) / 225) * 196)) + (((((int)threadIdx.x) / 15) + 7) * 14)) + (((int)threadIdx.x) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1792)] = (((15 <= ((((int)threadIdx.x) + 217) % 225)) && (1 <= ((((int)threadIdx.x) + 7) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1792) / 225) * 196)) + ((((((int)threadIdx.x) + 217) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1904)] = ((1 <= ((((int)threadIdx.x) + 14) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 1904) / 225) * 196)) + ((((((int)threadIdx.x) + 104) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2016)] = (((15 <= ((((int)threadIdx.x) + 216) % 225)) && (1 <= ((((int)threadIdx.x) + 6) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2016) / 225) * 196)) + ((((((int)threadIdx.x) + 216) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 6) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2128)] = ((1 <= ((((int)threadIdx.x) + 13) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2128) / 225) * 196)) + ((((((int)threadIdx.x) + 103) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 13) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2240)] = (((15 <= ((((int)threadIdx.x) + 215) % 225)) && (1 <= ((((int)threadIdx.x) + 5) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2240) / 225) * 196)) + ((((((int)threadIdx.x) + 215) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 5) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2352)] = ((1 <= ((((int)threadIdx.x) + 12) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2352) / 225) * 196)) + ((((((int)threadIdx.x) + 102) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 12) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2464)] = (((15 <= ((((int)threadIdx.x) + 214) % 225)) && (1 <= ((((int)threadIdx.x) + 4) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2464) / 225) * 196)) + ((((((int)threadIdx.x) + 214) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 4) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2576)] = ((1 <= ((((int)threadIdx.x) + 11) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2576) / 225) * 196)) + ((((((int)threadIdx.x) + 101) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 11) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2688)] = (((15 <= ((((int)threadIdx.x) + 213) % 225)) && (1 <= ((((int)threadIdx.x) + 3) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2688) / 225) * 196)) + ((((((int)threadIdx.x) + 213) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 3) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2800)] = ((1 <= ((((int)threadIdx.x) + 10) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2800) / 225) * 196)) + ((((((int)threadIdx.x) + 100) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 10) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 2912)] = (((15 <= ((((int)threadIdx.x) + 212) % 225)) && (1 <= ((((int)threadIdx.x) + 2) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 2912) / 225) * 196)) + ((((((int)threadIdx.x) + 212) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 2) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3024)] = ((1 <= ((((int)threadIdx.x) + 9) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3024) / 225) * 196)) + ((((((int)threadIdx.x) + 99) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 9) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3136)] = (((15 <= ((((int)threadIdx.x) + 211) % 225)) && (1 <= ((((int)threadIdx.x) + 1) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3136) / 225) * 196)) + ((((((int)threadIdx.x) + 211) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 1) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3248)] = ((1 <= ((((int)threadIdx.x) + 8) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3248) / 225) * 196)) + ((((((int)threadIdx.x) + 98) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 8) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3360)] = (((1 <= (((((int)threadIdx.x) / 15) + 14) % 15)) && (1 <= (((int)threadIdx.x) % 15))) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3360) / 225) * 196)) + ((((((int)threadIdx.x) / 15) + 14) % 15) * 14)) + (((int)threadIdx.x) % 15)) - 15)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 3472)] = ((1 <= ((((int)threadIdx.x) + 7) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3472) / 225) * 196)) + ((((((int)threadIdx.x) + 97) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 7) % 15)) - 15)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 16) {
    PaddedInput_shared[(((int)threadIdx.x) + 3584)] = ((1 <= ((((int)threadIdx.x) + 14) % 15)) ? Input[(((((((int)blockIdx.x) * 3136) + (((((int)threadIdx.x) + 3584) / 225) * 196)) + ((((((int)threadIdx.x) + 209) % 225) / 15) * 14)) + ((((int)threadIdx.x) + 14) % 15)) - 15)] : 0.000000e+00f);
  }
  kernel_shared[((int)threadIdx.x)] = kernel[((((int)blockIdx.x) * 144) + ((int)threadIdx.x))];
  if (((int)threadIdx.x) < 32) {
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[(((((int)blockIdx.x) * 144) + ((int)threadIdx.x)) + 112)];
  }
  __syncthreads();
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2))] * kernel_shared[((((int)threadIdx.x) / 7) * 9)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 30)] * kernel_shared[((((int)threadIdx.x) / 7) * 9)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 60)] * kernel_shared[((((int)threadIdx.x) / 7) * 9)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 90)] * kernel_shared[((((int)threadIdx.x) / 7) * 9)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 120)] * kernel_shared[((((int)threadIdx.x) / 7) * 9)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 150)] * kernel_shared[((((int)threadIdx.x) / 7) * 9)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 180)] * kernel_shared[((((int)threadIdx.x) / 7) * 9)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 1)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 1)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 31)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 1)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 61)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 1)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 91)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 1)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 121)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 1)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 151)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 1)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 181)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 1)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 2)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 32)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 2)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 62)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 2)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 92)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 2)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 122)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 2)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 152)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 2)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 182)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 2)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 15)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 3)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 45)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 3)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 75)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 3)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 105)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 3)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 135)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 3)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 165)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 3)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 195)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 3)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 16)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 4)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 46)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 4)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 76)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 4)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 106)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 4)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 136)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 4)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 166)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 4)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 196)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 4)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 17)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 5)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 47)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 5)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 77)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 5)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 107)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 5)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 137)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 5)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 167)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 5)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 197)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 5)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 30)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 6)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 60)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 6)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 90)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 6)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 120)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 6)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 150)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 6)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 180)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 6)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 210)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 6)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 31)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 7)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 61)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 7)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 91)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 7)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 121)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 7)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 151)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 7)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 181)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 7)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 211)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 7)]));
  DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 32)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 8)]));
  DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 62)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 8)]));
  DepthwiseConv2d[2] = (DepthwiseConv2d[2] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 92)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 8)]));
  DepthwiseConv2d[3] = (DepthwiseConv2d[3] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 122)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 8)]));
  DepthwiseConv2d[4] = (DepthwiseConv2d[4] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 152)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 8)]));
  DepthwiseConv2d[5] = (DepthwiseConv2d[5] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 182)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 8)]));
  DepthwiseConv2d[6] = (DepthwiseConv2d[6] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 225) + ((((int)threadIdx.x) % 7) * 2)) + 212)] * kernel_shared[(((((int)threadIdx.x) / 7) * 9) + 8)]));
  compute[(((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7))] = max(DepthwiseConv2d[0], 0.000000e+00f);
  compute[((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 7)] = max(DepthwiseConv2d[1], 0.000000e+00f);
  compute[((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 14)] = max(DepthwiseConv2d[2], 0.000000e+00f);
  compute[((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 21)] = max(DepthwiseConv2d[3], 0.000000e+00f);
  compute[((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 28)] = max(DepthwiseConv2d[4], 0.000000e+00f);
  compute[((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 35)] = max(DepthwiseConv2d[5], 0.000000e+00f);
  compute[((((((int)blockIdx.x) * 784) + ((((int)threadIdx.x) / 7) * 49)) + (((int)threadIdx.x) % 7)) + 42)] = max(DepthwiseConv2d[6], 0.000000e+00f);
}


