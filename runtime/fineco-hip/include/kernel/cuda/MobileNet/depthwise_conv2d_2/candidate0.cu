#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate0(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float DepthwiseConv2d[2];
  __shared__ float PaddedInput_shared[1938];
  __shared__ float kernel_shared[18];
  DepthwiseConv2d[0] = 0.000000e+00f;
  DepthwiseConv2d[1] = 0.000000e+00f;
  PaddedInput_shared[((int)threadIdx.x)] = (((1 <= ((((((int)blockIdx.x) % 14) / 7) * 56) + (((int)threadIdx.x) / 17))) && (1 <= (((((int)blockIdx.x) % 7) * 16) + (((int)threadIdx.x) % 17)))) ? Input[(((((((((int)blockIdx.x) / 14) * 25088) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + ((((int)threadIdx.x) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) % 17)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 224)] = ((1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 3) % 17))) ? Input[(((((((((int)blockIdx.x) / 14) * 25088) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + (((((int)threadIdx.x) + 224) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 3) % 17)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 448)] = ((1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 6) % 17))) ? Input[(((((((((int)blockIdx.x) / 14) * 25088) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + (((((int)threadIdx.x) + 448) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 6) % 17)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 672)] = ((1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 9) % 17))) ? Input[(((((((((int)blockIdx.x) / 14) * 25088) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + (((((int)threadIdx.x) + 672) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 9) % 17)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 896)] = (((1 <= ((((((int)blockIdx.x) % 14) / 7) * 56) + (((((int)threadIdx.x) + 896) % 969) / 17))) && (1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 12) % 17)))) ? Input[((((((((((int)blockIdx.x) / 14) * 25088) + (((((int)threadIdx.x) + 896) / 969) * 12544)) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + ((((((int)threadIdx.x) + 896) % 969) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 12) % 17)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1120)] = ((1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 15) % 17))) ? Input[((((((((((int)blockIdx.x) / 14) * 25088) + (((((int)threadIdx.x) + 1120) / 969) * 12544)) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + ((((((int)threadIdx.x) + 151) % 969) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 15) % 17)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1344)] = ((1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 1) % 17))) ? Input[((((((((((int)blockIdx.x) / 14) * 25088) + (((((int)threadIdx.x) + 1344) / 969) * 12544)) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + ((((((int)threadIdx.x) + 375) % 969) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 1) % 17)) - 113)] : 0.000000e+00f);
  PaddedInput_shared[(((int)threadIdx.x) + 1568)] = ((1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 4) % 17))) ? Input[((((((((((int)blockIdx.x) / 14) * 25088) + (((((int)threadIdx.x) + 1568) / 969) * 12544)) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + ((((((int)threadIdx.x) + 599) % 969) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 4) % 17)) - 113)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 146) {
    PaddedInput_shared[(((int)threadIdx.x) + 1792)] = ((1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 7) % 17))) ? Input[((((((((((int)blockIdx.x) / 14) * 25088) + (((((int)threadIdx.x) + 1792) / 969) * 12544)) + (((((int)blockIdx.x) % 14) / 7) * 6272)) + ((((((int)threadIdx.x) + 823) % 969) / 17) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 7) % 17)) - 113)] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 18) {
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) / 14) * 18) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[(((((((int)threadIdx.x) / 112) * 969) + (((((int)threadIdx.x) % 112) >> 2) * 34)) + (di_outer_inner * 17)) + ((((int)threadIdx.x) & 3) * 4))] * kernel_shared[(((((int)threadIdx.x) / 112) * 9) + (di_outer_inner * 3))]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 969) + (((((int)threadIdx.x) % 112) >> 2) * 34)) + (di_outer_inner * 17)) + ((((int)threadIdx.x) & 3) * 4)) + 2)] * kernel_shared[(((((int)threadIdx.x) / 112) * 9) + (di_outer_inner * 3))]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 969) + (((((int)threadIdx.x) % 112) >> 2) * 34)) + (di_outer_inner * 17)) + ((((int)threadIdx.x) & 3) * 4)) + 1)] * kernel_shared[((((((int)threadIdx.x) / 112) * 9) + (di_outer_inner * 3)) + 1)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 969) + (((((int)threadIdx.x) % 112) >> 2) * 34)) + (di_outer_inner * 17)) + ((((int)threadIdx.x) & 3) * 4)) + 3)] * kernel_shared[((((((int)threadIdx.x) / 112) * 9) + (di_outer_inner * 3)) + 1)]));
    DepthwiseConv2d[0] = (DepthwiseConv2d[0] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 969) + (((((int)threadIdx.x) % 112) >> 2) * 34)) + (di_outer_inner * 17)) + ((((int)threadIdx.x) & 3) * 4)) + 2)] * kernel_shared[((((((int)threadIdx.x) / 112) * 9) + (di_outer_inner * 3)) + 2)]));
    DepthwiseConv2d[1] = (DepthwiseConv2d[1] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 969) + (((((int)threadIdx.x) % 112) >> 2) * 34)) + (di_outer_inner * 17)) + ((((int)threadIdx.x) & 3) * 4)) + 4)] * kernel_shared[((((((int)threadIdx.x) / 112) * 9) + (di_outer_inner * 3)) + 2)]));
  }
  for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
    compute[((((((((((int)blockIdx.x) / 14) * 6272) + ((((int)threadIdx.x) / 112) * 3136)) + (((((int)blockIdx.x) % 14) / 7) * 1568)) + (((((int)threadIdx.x) % 112) >> 2) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + i3_inner)] = max(DepthwiseConv2d[i3_inner], 0.000000e+00f);
  }
}


