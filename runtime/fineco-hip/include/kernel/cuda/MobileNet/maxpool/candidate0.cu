#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) candidate0(float* __restrict__ data, float* __restrict__ tensor) {
  float normal_reduce_temp0[1];
  float red_buf0[1];
  __shared__ float tensor1[1];
  normal_reduce_temp0[0] = 0.000000e+00f;
  normal_reduce_temp0[0] = (normal_reduce_temp0[0] + data[((((int)blockIdx.x) * 49) + ((int)threadIdx.x))]);
  if (((int)threadIdx.x) < 17) {
    normal_reduce_temp0[0] = (normal_reduce_temp0[0] + data[(((((int)blockIdx.x) * 49) + ((int)threadIdx.x)) + 32)]);
  }
  uint mask[1];
  float t0[1];
  red_buf0[0] = normal_reduce_temp0[0];
  // mask[0] = __activemask();                              // ! HIP 中没有 __activemask()
  // t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
  t0[0] = __shfl_down(red_buf0[0], 16, 64);                 // ! HIP 中没有 __shfl_down_sync
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down(red_buf0[0], 8, 64);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down(red_buf0[0], 4, 64);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down(red_buf0[0], 2, 64);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down(red_buf0[0], 1, 64);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  red_buf0[0] = __shfl(red_buf0[0], 0, 64);                 // ! HIP 中没有 __shfl_sync
  tensor1[0] = red_buf0[0];
  __syncthreads();
  if (((int)threadIdx.x) < 1) {
    tensor[(((int)blockIdx.x) + ((int)threadIdx.x))] = (tensor1[((int)threadIdx.x)] * 2.040816e-02f);
  }
}


