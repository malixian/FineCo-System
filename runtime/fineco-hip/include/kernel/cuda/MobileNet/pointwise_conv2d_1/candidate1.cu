#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate1(float* __restrict__ Input, float* __restrict__ kernel, float* __restrict__ compute) {
  float conv2d_nchw[224];
  __shared__ float pad_temp_shared[3136];
  __shared__ float kernel_shared[256];
  for (int ff_inner_init = 0; ff_inner_init < 4; ++ff_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 14; ++yy_inner_init) {
      conv2d_nchw[((ff_inner_init * 14) + yy_inner_init)] = 0.000000e+00f;
      conv2d_nchw[(((ff_inner_init * 14) + yy_inner_init) + 56)] = 0.000000e+00f;
      conv2d_nchw[(((ff_inner_init * 14) + yy_inner_init) + 112)] = 0.000000e+00f;
      conv2d_nchw[(((ff_inner_init * 14) + yy_inner_init) + 168)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = Input[(((((rc_outer_outer * 50176) + ((((int)blockIdx.x) >> 2) * 3136)) + ((((int)threadIdx.x) / 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = Input[((((((rc_outer_outer * 50176) + ((((int)blockIdx.x) >> 2) * 3136)) + ((((int)threadIdx.x) / 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28)) + 896)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = Input[((((((rc_outer_outer * 50176) + ((((int)blockIdx.x) >> 2) * 3136)) + ((((int)threadIdx.x) / 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28)) + 1792)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 672) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + ((((((int)threadIdx.x) / 28) + 24) % 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 896) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 4) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1120) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 12) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1344) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 20) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = Input[((((((rc_outer_outer * 50176) + ((((int)blockIdx.x) >> 2) * 3136)) + ((((int)threadIdx.x) / 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1792) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 8) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2016) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 16) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2240) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + ((((((int)threadIdx.x) / 28) + 24) % 28) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2464) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 4) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2688) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 12) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = Input[((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 2912) / 784) * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) / 28) + 20) * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    if (((int)threadIdx.x) < 32) {
      kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 1792)];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
      for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
        for (int yy_inner = 0; yy_inner < 14; ++yy_inner) {
          conv2d_nchw[((ff_inner * 14) + yy_inner)] = (conv2d_nchw[((ff_inner * 14) + yy_inner)] + (pad_temp_shared[((((rc_inner * 784) + (((((int)threadIdx.x) % 56) / 28) * 392)) + (yy_inner * 28)) + (((int)threadIdx.x) % 28))] * kernel_shared[((((((int)threadIdx.x) / 56) * 16) + (ff_inner * 4)) + rc_inner)]));
          conv2d_nchw[(((ff_inner * 14) + yy_inner) + 56)] = (conv2d_nchw[(((ff_inner * 14) + yy_inner) + 56)] + (pad_temp_shared[((((rc_inner * 784) + (((((int)threadIdx.x) % 56) / 28) * 392)) + (yy_inner * 28)) + (((int)threadIdx.x) % 28))] * kernel_shared[(((((((int)threadIdx.x) / 56) * 16) + (ff_inner * 4)) + rc_inner) + 64)]));
          conv2d_nchw[(((ff_inner * 14) + yy_inner) + 112)] = (conv2d_nchw[(((ff_inner * 14) + yy_inner) + 112)] + (pad_temp_shared[((((rc_inner * 784) + (((((int)threadIdx.x) % 56) / 28) * 392)) + (yy_inner * 28)) + (((int)threadIdx.x) % 28))] * kernel_shared[(((((((int)threadIdx.x) / 56) * 16) + (ff_inner * 4)) + rc_inner) + 128)]));
          conv2d_nchw[(((ff_inner * 14) + yy_inner) + 168)] = (conv2d_nchw[(((ff_inner * 14) + yy_inner) + 168)] + (pad_temp_shared[((((rc_inner * 784) + (((((int)threadIdx.x) % 56) / 28) * 392)) + (yy_inner * 28)) + (((int)threadIdx.x) % 28))] * kernel_shared[(((((((int)threadIdx.x) / 56) * 16) + (ff_inner * 4)) + rc_inner) + 192)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 14; ++i2_inner) {
      compute[((((((((((int)threadIdx.x) / 56) * 50176) + (i1_inner * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) % 56) / 28) * 1568)) + (i2_inner * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28))] = max(conv2d_nchw[((i1_inner * 14) + i2_inner)], 0.000000e+00f);
      compute[(((((((((((int)threadIdx.x) / 56) * 50176) + (i1_inner * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) % 56) / 28) * 1568)) + (i2_inner * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28)) + 200704)] = max(conv2d_nchw[(((i1_inner * 14) + i2_inner) + 56)], 0.000000e+00f);
      compute[(((((((((((int)threadIdx.x) / 56) * 50176) + (i1_inner * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) % 56) / 28) * 1568)) + (i2_inner * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28)) + 401408)] = max(conv2d_nchw[(((i1_inner * 14) + i2_inner) + 112)], 0.000000e+00f);
      compute[(((((((((((int)threadIdx.x) / 56) * 50176) + (i1_inner * 12544)) + ((((int)blockIdx.x) >> 2) * 3136)) + (((((int)threadIdx.x) % 56) / 28) * 1568)) + (i2_inner * 112)) + ((((int)blockIdx.x) & 3) * 28)) + (((int)threadIdx.x) % 28)) + 602112)] = max(conv2d_nchw[(((i1_inner * 14) + i2_inner) + 168)], 0.000000e+00f);
    }
  }
}


