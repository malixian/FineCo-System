#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(224) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[224];
  __shared__ float pad_temp_shared[3712];
  __shared__ float kernel_shared[2304];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int xx_c_outer_inner_init = 0; xx_c_outer_inner_init < 8; ++xx_c_outer_inner_init) {
      conv2d_nchw_local[((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7))] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 112)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 1)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 113)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 2)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 114)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 3)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 115)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 4)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 116)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 5)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 117)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 6)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_outer_inner_init * 56) + (xx_c_outer_inner_init * 7)) + 118)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((1 <= (((((int)blockIdx.x) & 3) * 14) + (((int)threadIdx.x) / 58))) && (1 <= (((int)threadIdx.x) % 58))) && ((((int)threadIdx.x) % 58) < 57)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 784)) + ((((int)threadIdx.x) / 58) * 56)) + (((int)threadIdx.x) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 224)] = (((1 <= ((((int)threadIdx.x) + 50) % 58)) && (((((int)threadIdx.x) + 50) % 58) < 57)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 784)) + (((((int)threadIdx.x) + 224) / 58) * 56)) + ((((int)threadIdx.x) + 50) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 448)] = (((1 <= ((((int)threadIdx.x) + 42) % 58)) && (((((int)threadIdx.x) + 42) % 58) < 57)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 784)) + (((((int)threadIdx.x) + 448) / 58) * 56)) + ((((int)threadIdx.x) + 42) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 672)] = (((((((((int)blockIdx.x) & 3) * 14) + ((((int)threadIdx.x) + 672) / 58)) < 57) && (1 <= ((((int)threadIdx.x) + 34) % 58))) && (((((int)threadIdx.x) + 34) % 58) < 57)) ? data[(((((rc_outer_outer * 12544) + ((((int)blockIdx.x) & 3) * 784)) + (((((int)threadIdx.x) + 672) / 58) * 56)) + ((((int)threadIdx.x) + 34) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 896)] = (((((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((((int)threadIdx.x) >> 1) + 448) % 464) / 29))) && ((((((int)blockIdx.x) & 3) * 14) + ((((((int)threadIdx.x) >> 1) + 448) % 464) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 26) % 58))) && (((((int)threadIdx.x) + 26) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 896) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 448) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 26) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = (((1 <= ((((int)threadIdx.x) + 18) % 58)) && (((((int)threadIdx.x) + 18) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1120) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 96) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 18) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = (((1 <= ((((int)threadIdx.x) + 10) % 58)) && (((((int)threadIdx.x) + 10) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1344) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 208) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 10) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = (((1 <= ((((int)threadIdx.x) + 2) % 58)) && (((((int)threadIdx.x) + 2) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1568) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 320) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 2) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((((int)threadIdx.x) >> 1) + 432) % 464) / 29))) && ((((((int)blockIdx.x) & 3) * 14) + ((((((int)threadIdx.x) >> 1) + 432) % 464) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 52) % 58))) && (((((int)threadIdx.x) + 52) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 1792) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 432) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 52) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = (((1 <= ((((int)threadIdx.x) + 44) % 58)) && (((((int)threadIdx.x) + 44) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 2016) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 80) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 44) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = (((1 <= ((((int)threadIdx.x) + 36) % 58)) && (((((int)threadIdx.x) + 36) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 2240) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 192) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 36) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = (((1 <= ((((int)threadIdx.x) + 28) % 58)) && (((((int)threadIdx.x) + 28) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 2464) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 304) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 28) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = (((((1 <= (((((int)blockIdx.x) & 3) * 14) + ((((((int)threadIdx.x) >> 1) + 416) % 464) / 29))) && ((((((int)blockIdx.x) & 3) * 14) + ((((((int)threadIdx.x) >> 1) + 416) % 464) / 29)) < 57)) && (1 <= ((((int)threadIdx.x) + 20) % 58))) && (((((int)threadIdx.x) + 20) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 2688) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 416) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 20) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = (((1 <= ((((int)threadIdx.x) + 12) % 58)) && (((((int)threadIdx.x) + 12) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 2912) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 64) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 12) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = (((1 <= ((((int)threadIdx.x) + 4) % 58)) && (((((int)threadIdx.x) + 4) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 3136) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 176) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 4) % 58)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = (((1 <= ((((int)threadIdx.x) + 54) % 58)) && (((((int)threadIdx.x) + 54) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 3360) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 288) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 54) % 58)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 128) {
      pad_temp_shared[(((int)threadIdx.x) + 3584)] = (((((((((int)blockIdx.x) & 3) * 14) + ((((((int)threadIdx.x) >> 1) + 400) % 464) / 29)) < 57) && (1 <= ((((int)threadIdx.x) + 46) % 58))) && (((((int)threadIdx.x) + 46) % 58) < 57)) ? data[((((((rc_outer_outer * 12544) + (((((int)threadIdx.x) + 3584) / 928) * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + (((((((int)threadIdx.x) >> 1) + 400) % 464) / 29) * 56)) + ((((int)threadIdx.x) + 46) % 58)) - 57)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + ((((int)threadIdx.x) / 36) * 2304)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 224) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 448) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 16) % 36))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 672) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 24) % 36))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 896) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 32) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 1120) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 4) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 1344) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 12) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 1568) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 20) % 36))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 1792) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 28) % 36))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((((int)blockIdx.x) >> 2) * 147456) + ((((int)threadIdx.x) / 36) * 2304)) + (rc_outer_outer * 36)) + (((int)threadIdx.x) % 36)) + 129024)];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[(((((((int)blockIdx.x) >> 2) * 147456) + (((((int)threadIdx.x) + 2240) / 36) * 2304)) + (rc_outer_outer * 36)) + ((((int)threadIdx.x) + 8) % 36))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
          for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
            for (int xx_c_outer_inner = 0; xx_c_outer_inner < 8; ++xx_c_outer_inner) {
              conv2d_nchw_local[((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7))] = (conv2d_nchw_local[((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7))] + (pad_temp_shared[(((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 112)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 112)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 406)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 1)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 1)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 1)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 113)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 113)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 407)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 2)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 2)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 2)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 114)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 114)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 408)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 3)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 3)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 3)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 115)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 115)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 409)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 4)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 4)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 4)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 116)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 116)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 410)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 5)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 5)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 5)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 117)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 117)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 411)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 6)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 6)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 6)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
              conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 118)] = (conv2d_nchw_local[(((ff_c_outer_inner * 56) + (xx_c_outer_inner * 7)) + 118)] + (pad_temp_shared[((((((rc_outer_inner * 928) + (ry_outer_inner * 58)) + ((((int)threadIdx.x) % 7) * 58)) + (xx_c_outer_inner * 7)) + rx_outer_inner) + 412)] * kernel_shared[((((((((int)threadIdx.x) / 7) * 72) + (ff_c_outer_inner * 36)) + (rc_outer_inner * 9)) + (ry_outer_inner * 3)) + rx_outer_inner)]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int xx_inner = 0; xx_inner < 56; ++xx_inner) {
      conv2d_nchw[(((((((((int)blockIdx.x) >> 2) * 200704) + ((((int)threadIdx.x) / 7) * 6272)) + (ff_inner * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + ((((int)threadIdx.x) % 7) * 56)) + xx_inner)] = conv2d_nchw_local[((ff_inner * 56) + xx_inner)];
      conv2d_nchw[((((((((((int)blockIdx.x) >> 2) * 200704) + ((((int)threadIdx.x) / 7) * 6272)) + (ff_inner * 3136)) + ((((int)blockIdx.x) & 3) * 784)) + ((((int)threadIdx.x) % 7) * 56)) + xx_inner) + 392)] = conv2d_nchw_local[(((ff_inner * 56) + xx_inner) + 112)];
    }
  }
}


