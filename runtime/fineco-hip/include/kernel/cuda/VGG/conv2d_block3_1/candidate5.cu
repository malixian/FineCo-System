#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate5(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[56];
  __shared__ float pad_temp_shared[2784];
  __shared__ float kernel_shared[4608];
  conv2d_nchw_local[0] = 0.000000e+00f;
  conv2d_nchw_local[1] = 0.000000e+00f;
  conv2d_nchw_local[2] = 0.000000e+00f;
  conv2d_nchw_local[3] = 0.000000e+00f;
  conv2d_nchw_local[4] = 0.000000e+00f;
  conv2d_nchw_local[5] = 0.000000e+00f;
  conv2d_nchw_local[6] = 0.000000e+00f;
  conv2d_nchw_local[7] = 0.000000e+00f;
  conv2d_nchw_local[8] = 0.000000e+00f;
  conv2d_nchw_local[9] = 0.000000e+00f;
  conv2d_nchw_local[10] = 0.000000e+00f;
  conv2d_nchw_local[11] = 0.000000e+00f;
  conv2d_nchw_local[12] = 0.000000e+00f;
  conv2d_nchw_local[13] = 0.000000e+00f;
  conv2d_nchw_local[14] = 0.000000e+00f;
  conv2d_nchw_local[15] = 0.000000e+00f;
  conv2d_nchw_local[16] = 0.000000e+00f;
  conv2d_nchw_local[17] = 0.000000e+00f;
  conv2d_nchw_local[18] = 0.000000e+00f;
  conv2d_nchw_local[19] = 0.000000e+00f;
  conv2d_nchw_local[20] = 0.000000e+00f;
  conv2d_nchw_local[21] = 0.000000e+00f;
  conv2d_nchw_local[22] = 0.000000e+00f;
  conv2d_nchw_local[23] = 0.000000e+00f;
  conv2d_nchw_local[24] = 0.000000e+00f;
  conv2d_nchw_local[25] = 0.000000e+00f;
  conv2d_nchw_local[26] = 0.000000e+00f;
  conv2d_nchw_local[27] = 0.000000e+00f;
  conv2d_nchw_local[28] = 0.000000e+00f;
  conv2d_nchw_local[29] = 0.000000e+00f;
  conv2d_nchw_local[30] = 0.000000e+00f;
  conv2d_nchw_local[31] = 0.000000e+00f;
  conv2d_nchw_local[32] = 0.000000e+00f;
  conv2d_nchw_local[33] = 0.000000e+00f;
  conv2d_nchw_local[34] = 0.000000e+00f;
  conv2d_nchw_local[35] = 0.000000e+00f;
  conv2d_nchw_local[36] = 0.000000e+00f;
  conv2d_nchw_local[37] = 0.000000e+00f;
  conv2d_nchw_local[38] = 0.000000e+00f;
  conv2d_nchw_local[39] = 0.000000e+00f;
  conv2d_nchw_local[40] = 0.000000e+00f;
  conv2d_nchw_local[41] = 0.000000e+00f;
  conv2d_nchw_local[42] = 0.000000e+00f;
  conv2d_nchw_local[43] = 0.000000e+00f;
  conv2d_nchw_local[44] = 0.000000e+00f;
  conv2d_nchw_local[45] = 0.000000e+00f;
  conv2d_nchw_local[46] = 0.000000e+00f;
  conv2d_nchw_local[47] = 0.000000e+00f;
  conv2d_nchw_local[48] = 0.000000e+00f;
  conv2d_nchw_local[49] = 0.000000e+00f;
  conv2d_nchw_local[50] = 0.000000e+00f;
  conv2d_nchw_local[51] = 0.000000e+00f;
  conv2d_nchw_local[52] = 0.000000e+00f;
  conv2d_nchw_local[53] = 0.000000e+00f;
  conv2d_nchw_local[54] = 0.000000e+00f;
  conv2d_nchw_local[55] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((6 <= ((int)threadIdx.x)) && (1 <= (((((int)blockIdx.x) % 14) * 4) + (((int)threadIdx.x) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + (((int)threadIdx.x) % 6)) < 57)) ? data[(((((rc_outer_outer * 25088) + ((((int)threadIdx.x) / 6) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + (((int)threadIdx.x) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 256)] = (((((3 <= (((((int)threadIdx.x) >> 1) + 128) % 174)) && (((((int)threadIdx.x) + 256) % 348) < 342)) && (1 <= (((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 4) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 4) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 256) / 348) * 3136)) + (((((((int)threadIdx.x) >> 1) + 128) % 174) / 3) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + ((((int)threadIdx.x) + 4) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 512)] = (((((3 <= (((((int)threadIdx.x) >> 1) + 82) % 174)) && (((((int)threadIdx.x) + 164) % 348) < 342)) && (1 <= (((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 512) / 348) * 3136)) + (((((((int)threadIdx.x) >> 1) + 82) % 174) / 3) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + ((((int)threadIdx.x) + 2) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 768)] = (((1 <= (((((int)blockIdx.x) % 14) * 4) + (((int)threadIdx.x) % 6))) && ((((((int)blockIdx.x) % 14) * 4) + (((int)threadIdx.x) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 768) / 348) * 3136)) + (((((int)threadIdx.x) / 6) + 12) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + (((int)threadIdx.x) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((((3 <= (((((int)threadIdx.x) >> 1) + 164) % 174)) && (((((int)threadIdx.x) + 328) % 348) < 342)) && (1 <= (((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 4) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 4) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1024) / 348) * 3136)) + (((((((int)threadIdx.x) >> 1) + 164) % 174) / 3) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + ((((int)threadIdx.x) + 4) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = (((((3 <= (((((int)threadIdx.x) >> 1) + 118) % 174)) && (((((int)threadIdx.x) + 236) % 348) < 342)) && (1 <= (((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1280) / 348) * 3136)) + (((((((int)threadIdx.x) >> 1) + 118) % 174) / 3) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + ((((int)threadIdx.x) + 2) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = (((((1 <= (((((int)threadIdx.x) / 6) + 24) % 58)) && (((((int)threadIdx.x) + 144) % 348) < 342)) && (1 <= (((((int)blockIdx.x) % 14) * 4) + (((int)threadIdx.x) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + (((int)threadIdx.x) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1536) / 348) * 3136)) + ((((((int)threadIdx.x) / 6) + 24) % 58) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + (((int)threadIdx.x) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((1 <= (((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 4) % 6))) && ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 4) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 1792) / 348) * 3136)) + (((((((int)threadIdx.x) >> 1) + 26) % 174) / 3) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + ((((int)threadIdx.x) + 4) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2048)] = (((((3 <= (((((int)threadIdx.x) >> 1) + 154) % 174)) && (((((int)threadIdx.x) + 308) % 348) < 342)) && (1 <= (((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 2) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2048) / 348) * 3136)) + (((((((int)threadIdx.x) >> 1) + 154) % 174) / 3) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + ((((int)threadIdx.x) + 2) % 6)) - 57)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 2304)] = (((((1 <= (((((int)threadIdx.x) / 6) + 36) % 58)) && (((((int)threadIdx.x) + 216) % 348) < 342)) && (1 <= (((((int)blockIdx.x) % 14) * 4) + (((int)threadIdx.x) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + (((int)threadIdx.x) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2304) / 348) * 3136)) + ((((((int)threadIdx.x) / 6) + 36) % 58) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + (((int)threadIdx.x) % 6)) - 57)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 224) {
      pad_temp_shared[(((int)threadIdx.x) + 2560)] = ((((((int)threadIdx.x) < 218) && (1 <= (((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 4) % 6)))) && ((((((int)blockIdx.x) % 14) * 4) + ((((int)threadIdx.x) + 4) % 6)) < 57)) ? data[((((((rc_outer_outer * 25088) + (((((int)threadIdx.x) + 2560) / 348) * 3136)) + (((((((int)threadIdx.x) >> 1) + 62) % 174) / 3) * 56)) + ((((int)blockIdx.x) % 14) * 4)) + ((((int)threadIdx.x) + 4) % 6)) - 57)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72))];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 256) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 512) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 768) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 1024) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 1280) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 1536) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 1792) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 2048) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[((((((((int)blockIdx.x) / 14) * 73728) + ((((int)threadIdx.x) / 72) * 1152)) + (rc_outer_outer * 72)) + (((int)threadIdx.x) % 72)) + 36864)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 2560) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 40) % 72))];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 2816) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 8) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 3072) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 48) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 3328) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 16) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 3584) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 56) % 72))];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 3840) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 24) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 4096) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 64) % 72))];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[(((((((int)blockIdx.x) / 14) * 73728) + (((((int)threadIdx.x) + 4352) / 72) * 1152)) + (rc_outer_outer * 72)) + ((((int)threadIdx.x) + 32) % 72))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 4; ++ff_c_outer_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          conv2d_nchw_local[(ff_c_outer_inner * 14)] = (conv2d_nchw_local[(ff_c_outer_inner * 14)] + (pad_temp_shared[((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2))] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 1)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 6)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 3)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 7)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 4)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 12)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 5)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 5)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 13)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 6)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 6)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 18)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 7)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 7)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 19)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 8)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 24)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 9)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 9)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 25)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 10)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 30)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 11)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 11)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 31)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 12)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 12)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 36)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 13)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 13)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 37)] * kernel_shared[(((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3))]));
          conv2d_nchw_local[(ff_c_outer_inner * 14)] = (conv2d_nchw_local[(ff_c_outer_inner * 14)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 1)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 7)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 3)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 8)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 4)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 13)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 5)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 5)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 14)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 6)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 6)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 19)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 7)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 7)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 20)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 8)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 25)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 9)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 9)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 26)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 10)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 31)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 11)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 11)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 32)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 12)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 12)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 37)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 13)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 13)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 38)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 1)]));
          conv2d_nchw_local[(ff_c_outer_inner * 14)] = (conv2d_nchw_local[(ff_c_outer_inner * 14)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 2)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 1)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 3)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 2)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 8)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 3)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 3)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 9)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 4)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 14)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 5)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 5)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 15)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 6)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 6)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 20)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 7)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 7)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 21)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 8)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 26)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 9)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 9)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 27)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 10)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 32)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 11)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 11)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 33)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 12)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 12)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 38)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
          conv2d_nchw_local[((ff_c_outer_inner * 14) + 13)] = (conv2d_nchw_local[((ff_c_outer_inner * 14) + 13)] + (pad_temp_shared[(((((rc_outer_inner * 348) + (((((int)threadIdx.x) & 15) >> 1) * 42)) + (ry_inner * 6)) + ((((int)threadIdx.x) & 1) * 2)) + 39)] * kernel_shared[((((((((int)threadIdx.x) >> 4) * 288) + (ff_c_outer_inner * 72)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + 2)]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
        conv2d_nchw[(((((((((((int)blockIdx.x) / 14) * 200704) + ((((int)threadIdx.x) >> 4) * 12544)) + (ff_inner * 3136)) + (((((int)threadIdx.x) & 15) >> 1) * 392)) + (yy_inner * 56)) + ((((int)blockIdx.x) % 14) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 14) + (yy_inner * 2)) + xx_inner)];
      }
    }
  }
}


