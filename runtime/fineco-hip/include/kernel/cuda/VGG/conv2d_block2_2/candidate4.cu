#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv2d_nchw) {
  float conv2d_nchw_local[128];
  __shared__ float pad_temp_shared[1296];
  __shared__ float kernel_shared[4608];
  for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 4; ++yy_c_outer_inner_init) {
    for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
      conv2d_nchw_local[((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8))] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 64)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 1)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 65)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 2)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 66)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 3)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 67)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 4)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 68)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 5)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 69)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 6)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 70)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 7)] = 0.000000e+00f;
      conv2d_nchw_local[(((ff_c_inner_init * 32) + (yy_c_outer_inner_init * 8)) + 71)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = ((((1 <= (((((int)blockIdx.x) / 7) * 16) + (((int)threadIdx.x) / 18))) && (1 <= (((((int)blockIdx.x) % 7) * 16) + (((int)threadIdx.x) % 18)))) && ((((((int)blockIdx.x) % 7) * 16) + (((int)threadIdx.x) % 18)) < 113)) ? data[((((((rc_outer_outer * 50176) + ((((int)blockIdx.x) / 7) * 1792)) + ((((int)threadIdx.x) / 18) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) % 18)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 256)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 128) % 162) / 9))) && ((((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 128) % 162) / 9)) < 113)) && (1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 4) % 18)))) && ((((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 4) % 18)) < 113)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 256) / 324) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((((int)threadIdx.x) >> 1) + 128) % 162) / 9) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 4) % 18)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 512)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 94) % 162) / 9))) && ((((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 94) % 162) / 9)) < 113)) && (1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 8) % 18)))) && ((((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 8) % 18)) < 113)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 512) / 324) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((((int)threadIdx.x) >> 1) + 94) % 162) / 9) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 8) % 18)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 768)] = (((((1 <= (((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 60) % 162) / 9))) && ((((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 60) % 162) / 9)) < 113)) && (1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 12) % 18)))) && ((((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 12) % 18)) < 113)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 768) / 324) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((((int)threadIdx.x) >> 1) + 60) % 162) / 9) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 12) % 18)) - 113)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((((((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 26) % 162) / 9)) < 113) && (1 <= (((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 16) % 18)))) && ((((((int)blockIdx.x) % 7) * 16) + ((((int)threadIdx.x) + 16) % 18)) < 113)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1024) / 324) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((((int)threadIdx.x) >> 1) + 26) % 162) / 9) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) + 16) % 18)) - 113)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 16) {
      pad_temp_shared[(((int)threadIdx.x) + 1280)] = ((((((((int)blockIdx.x) / 7) * 16) + ((((((int)threadIdx.x) >> 1) + 154) % 162) / 9)) < 113) && ((((((int)blockIdx.x) % 7) * 16) + (((int)threadIdx.x) + 2)) < 113)) ? data[(((((((rc_outer_outer * 50176) + (((((int)threadIdx.x) + 1280) / 324) * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((((int)threadIdx.x) >> 1) + 154) % 162) / 9) * 112)) + ((((int)blockIdx.x) % 7) * 16)) + (((int)threadIdx.x) + 2)) - 113)] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 2)) / 9) * 1152) + (rc_outer_outer * 36)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x)) % 36))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ry_outer_inner = 0; ry_outer_inner < 3; ++ry_outer_inner) {
        for (int yy_c_outer_inner = 0; yy_c_outer_inner < 4; ++yy_c_outer_inner) {
          for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
            for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
              for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
                conv2d_nchw_local[((ff_c_inner * 32) + (yy_c_outer_inner * 8))] = (conv2d_nchw_local[((ff_c_inner * 32) + (yy_c_outer_inner * 8))] + (pad_temp_shared[(((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 64)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 64)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 8)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 1)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 1)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 1)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 65)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 65)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 9)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 2)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 2)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 2)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 66)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 66)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 10)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 3)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 3)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 3)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 67)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 67)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 11)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 4)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 4)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 18)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 68)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 68)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 26)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 5)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 5)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 19)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 69)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 69)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 27)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 6)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 6)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 20)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 70)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 70)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 28)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 7)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 7)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 21)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
                conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 71)] = (conv2d_nchw_local[(((ff_c_inner * 32) + (yy_c_outer_inner * 8)) + 71)] + (pad_temp_shared[((((((((rc_outer_inner * 648) + (rc_inner * 324)) + (((((int)threadIdx.x) & 3) >> 1) * 144)) + (yy_c_outer_inner * 36)) + (ry_outer_inner * 18)) + ((((int)threadIdx.x) & 1) * 4)) + rx_inner) + 29)] * kernel_shared[(((((((((int)threadIdx.x) >> 2) * 72) + (ff_c_inner * 36)) + (rc_outer_inner * 18)) + (rc_inner * 9)) + (ry_outer_inner * 3)) + rx_inner)]));
              }
            }
          }
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
    for (int yy_inner = 0; yy_inner < 8; ++yy_inner) {
      for (int xx_inner = 0; xx_inner < 4; ++xx_inner) {
        conv2d_nchw[(((((((((((int)threadIdx.x) >> 2) * 25088) + (ff_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((int)threadIdx.x) & 3) >> 1) * 896)) + (yy_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + xx_inner)] = conv2d_nchw_local[(((ff_inner * 32) + (yy_inner * 4)) + xx_inner)];
        conv2d_nchw[((((((((((((int)threadIdx.x) >> 2) * 25088) + (ff_inner * 12544)) + ((((int)blockIdx.x) / 7) * 1792)) + (((((int)threadIdx.x) & 3) >> 1) * 896)) + (yy_inner * 112)) + ((((int)blockIdx.x) % 7) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + xx_inner) + 8)] = conv2d_nchw_local[((((ff_inner * 32) + (yy_inner * 4)) + xx_inner) + 64)];
      }
    }
  }
}


