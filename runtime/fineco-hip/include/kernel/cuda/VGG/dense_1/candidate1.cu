#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate1(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[1];
  __shared__ float data_shared[32];
  __shared__ float weight_shared[4096];
  T_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 784; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      data_shared[((int)threadIdx.x)] = data[((k_outer_outer * 32) + ((int)threadIdx.x))];
    }
    weight_shared[((int)threadIdx.x)] = weight[((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    weight_shared[(((int)threadIdx.x) + 128)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 100352)];
    weight_shared[(((int)threadIdx.x) + 256)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 200704)];
    weight_shared[(((int)threadIdx.x) + 384)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 301056)];
    weight_shared[(((int)threadIdx.x) + 512)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 401408)];
    weight_shared[(((int)threadIdx.x) + 640)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 501760)];
    weight_shared[(((int)threadIdx.x) + 768)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 602112)];
    weight_shared[(((int)threadIdx.x) + 896)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 702464)];
    weight_shared[(((int)threadIdx.x) + 1024)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 802816)];
    weight_shared[(((int)threadIdx.x) + 1152)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 903168)];
    weight_shared[(((int)threadIdx.x) + 1280)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1003520)];
    weight_shared[(((int)threadIdx.x) + 1408)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1103872)];
    weight_shared[(((int)threadIdx.x) + 1536)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1204224)];
    weight_shared[(((int)threadIdx.x) + 1664)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1304576)];
    weight_shared[(((int)threadIdx.x) + 1792)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1404928)];
    weight_shared[(((int)threadIdx.x) + 1920)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1505280)];
    weight_shared[(((int)threadIdx.x) + 2048)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1605632)];
    weight_shared[(((int)threadIdx.x) + 2176)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1705984)];
    weight_shared[(((int)threadIdx.x) + 2304)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1806336)];
    weight_shared[(((int)threadIdx.x) + 2432)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1906688)];
    weight_shared[(((int)threadIdx.x) + 2560)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2007040)];
    weight_shared[(((int)threadIdx.x) + 2688)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2107392)];
    weight_shared[(((int)threadIdx.x) + 2816)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2207744)];
    weight_shared[(((int)threadIdx.x) + 2944)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2308096)];
    weight_shared[(((int)threadIdx.x) + 3072)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2408448)];
    weight_shared[(((int)threadIdx.x) + 3200)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2508800)];
    weight_shared[(((int)threadIdx.x) + 3328)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2609152)];
    weight_shared[(((int)threadIdx.x) + 3456)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2709504)];
    weight_shared[(((int)threadIdx.x) + 3584)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2809856)];
    weight_shared[(((int)threadIdx.x) + 3712)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2910208)];
    weight_shared[(((int)threadIdx.x) + 3840)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3010560)];
    weight_shared[(((int)threadIdx.x) + 3968)] = weight[(((((((int)blockIdx.x) * 3211264) + ((((int)threadIdx.x) >> 5) * 25088)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3110912)];
    __syncthreads();
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[0] * weight_shared[(((int)threadIdx.x) * 32)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[1] * weight_shared[((((int)threadIdx.x) * 32) + 1)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[2] * weight_shared[((((int)threadIdx.x) * 32) + 2)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[3] * weight_shared[((((int)threadIdx.x) * 32) + 3)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[4] * weight_shared[((((int)threadIdx.x) * 32) + 4)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[5] * weight_shared[((((int)threadIdx.x) * 32) + 5)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[6] * weight_shared[((((int)threadIdx.x) * 32) + 6)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[7] * weight_shared[((((int)threadIdx.x) * 32) + 7)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[8] * weight_shared[((((int)threadIdx.x) * 32) + 8)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[9] * weight_shared[((((int)threadIdx.x) * 32) + 9)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[10] * weight_shared[((((int)threadIdx.x) * 32) + 10)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[11] * weight_shared[((((int)threadIdx.x) * 32) + 11)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[12] * weight_shared[((((int)threadIdx.x) * 32) + 12)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[13] * weight_shared[((((int)threadIdx.x) * 32) + 13)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[14] * weight_shared[((((int)threadIdx.x) * 32) + 14)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[15] * weight_shared[((((int)threadIdx.x) * 32) + 15)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[16] * weight_shared[((((int)threadIdx.x) * 32) + 16)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[17] * weight_shared[((((int)threadIdx.x) * 32) + 17)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[18] * weight_shared[((((int)threadIdx.x) * 32) + 18)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[19] * weight_shared[((((int)threadIdx.x) * 32) + 19)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[20] * weight_shared[((((int)threadIdx.x) * 32) + 20)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[21] * weight_shared[((((int)threadIdx.x) * 32) + 21)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[22] * weight_shared[((((int)threadIdx.x) * 32) + 22)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[23] * weight_shared[((((int)threadIdx.x) * 32) + 23)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[24] * weight_shared[((((int)threadIdx.x) * 32) + 24)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[25] * weight_shared[((((int)threadIdx.x) * 32) + 25)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[26] * weight_shared[((((int)threadIdx.x) * 32) + 26)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[27] * weight_shared[((((int)threadIdx.x) * 32) + 27)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[28] * weight_shared[((((int)threadIdx.x) * 32) + 28)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[29] * weight_shared[((((int)threadIdx.x) * 32) + 29)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[30] * weight_shared[((((int)threadIdx.x) * 32) + 30)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[31] * weight_shared[((((int)threadIdx.x) * 32) + 31)]));
  }
  compute[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))]);
}


