#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(250) candidate0(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[1];
  __shared__ float data_shared[8];
  __shared__ float weight_shared[2000];
  T_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 2) {
      *(float4*)(data_shared + (((int)threadIdx.x) * 4)) = *(float4*)(data + ((k_outer_outer * 8) + (((int)threadIdx.x) * 4)));
    }
    weight_shared[((int)threadIdx.x)] = weight[((((((int)blockIdx.x) * 1024000) + ((((int)threadIdx.x) >> 3) * 4096)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    weight_shared[(((int)threadIdx.x) + 250)] = weight[((((((int)blockIdx.x) * 1024000) + (((((int)threadIdx.x) + 250) >> 3) * 4096)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7))];
    weight_shared[(((int)threadIdx.x) + 500)] = weight[((((((int)blockIdx.x) * 1024000) + (((((int)threadIdx.x) + 500) >> 3) * 4096)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7))];
    weight_shared[(((int)threadIdx.x) + 750)] = weight[((((((int)blockIdx.x) * 1024000) + (((((int)threadIdx.x) + 750) >> 3) * 4096)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7))];
    weight_shared[(((int)threadIdx.x) + 1000)] = weight[(((((((int)blockIdx.x) * 1024000) + ((((int)threadIdx.x) >> 3) * 4096)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 512000)];
    weight_shared[(((int)threadIdx.x) + 1250)] = weight[((((((int)blockIdx.x) * 1024000) + (((((int)threadIdx.x) + 1250) >> 3) * 4096)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7))];
    weight_shared[(((int)threadIdx.x) + 1500)] = weight[((((((int)blockIdx.x) * 1024000) + (((((int)threadIdx.x) + 1500) >> 3) * 4096)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7))];
    weight_shared[(((int)threadIdx.x) + 1750)] = weight[((((((int)blockIdx.x) * 1024000) + (((((int)threadIdx.x) + 1750) >> 3) * 4096)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7))];
    __syncthreads();
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[0] * weight_shared[(((int)threadIdx.x) * 8)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[1] * weight_shared[((((int)threadIdx.x) * 8) + 1)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[2] * weight_shared[((((int)threadIdx.x) * 8) + 2)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[3] * weight_shared[((((int)threadIdx.x) * 8) + 3)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[4] * weight_shared[((((int)threadIdx.x) * 8) + 4)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[5] * weight_shared[((((int)threadIdx.x) * 8) + 5)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[6] * weight_shared[((((int)threadIdx.x) * 8) + 6)]));
    T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[7] * weight_shared[((((int)threadIdx.x) * 8) + 7)]));
  }
  compute[((((int)blockIdx.x) * 250) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 250) + ((int)threadIdx.x))]);
}


