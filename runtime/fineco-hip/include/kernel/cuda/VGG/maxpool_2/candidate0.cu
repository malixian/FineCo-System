#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) candidate0(float* __restrict__ tensor, float* __restrict__ data) {
  tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 2; ++rv0) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], data[(((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) / 7) * 224) + (rv0 * 112)) + ((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 56) * 2)) + rv1)]);
    }
  }
}


