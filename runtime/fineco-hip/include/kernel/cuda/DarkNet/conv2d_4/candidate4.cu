#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate4(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[16];
  __shared__ float pad_temp_shared[4096];
  __shared__ float kernel_shared[2048];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[15] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 384)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 512)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 640)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 31360)];
    pad_temp_shared[(((int)threadIdx.x) + 768)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 37632)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 43904)];
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 1152)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 56448)];
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 62720)];
    pad_temp_shared[(((int)threadIdx.x) + 1408)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 68992)];
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 75264)];
    pad_temp_shared[(((int)threadIdx.x) + 1664)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 81536)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 87808)];
    pad_temp_shared[(((int)threadIdx.x) + 1920)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 94080)];
    pad_temp_shared[(((int)threadIdx.x) + 2048)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 100352)];
    pad_temp_shared[(((int)threadIdx.x) + 2176)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 106624)];
    pad_temp_shared[(((int)threadIdx.x) + 2304)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 112896)];
    pad_temp_shared[(((int)threadIdx.x) + 2432)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 119168)];
    pad_temp_shared[(((int)threadIdx.x) + 2560)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 125440)];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 131712)];
    pad_temp_shared[(((int)threadIdx.x) + 2816)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 137984)];
    pad_temp_shared[(((int)threadIdx.x) + 2944)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 144256)];
    pad_temp_shared[(((int)threadIdx.x) + 3072)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 150528)];
    pad_temp_shared[(((int)threadIdx.x) + 3200)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 156800)];
    pad_temp_shared[(((int)threadIdx.x) + 3328)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 163072)];
    pad_temp_shared[(((int)threadIdx.x) + 3456)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 169344)];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 175616)];
    pad_temp_shared[(((int)threadIdx.x) + 3712)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 181888)];
    pad_temp_shared[(((int)threadIdx.x) + 3840)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 188160)];
    pad_temp_shared[(((int)threadIdx.x) + 3968)] = data[(((((((rc_outer_outer * 200704) + ((((int)threadIdx.x) >> 6) * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 194432)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 256)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 512)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 768)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1024)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1280)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1536)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1792)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 2048)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 2304)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 2560)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 2816)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 3072)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 3328)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[((((((((int)blockIdx.x) / 49) * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 3840)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2))]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2))]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2))]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[(((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2))]));
      conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 1)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[(((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 64)]));
      conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 65)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[(((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 128)]));
      conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 128)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 128)]));
      conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 128)]));
      conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 129)]));
      conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 129)]));
      conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 129)]));
      conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 129)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[(((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 192)]));
      conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 192)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 8)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 192)]));
      conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 40)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 192)]));
      conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 64)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 193)]));
      conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 96)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 193)]));
      conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 72)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 193)]));
      conv2d_nchw[15] = (conv2d_nchw[15] + (pad_temp_shared[((((rc_outer_inner * 128) + (((((int)threadIdx.x) & 15) >> 3) * 16)) + (((int)threadIdx.x) & 7)) + 104)] * kernel_shared[((((((int)threadIdx.x) >> 4) * 256) + (rc_outer_inner * 2)) + 193)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[(((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 4) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = max((conv2d_nchw[((i1_inner * 2) + i2_inner)] + bias[((((((int)blockIdx.x) / 49) * 32) + ((((int)threadIdx.x) >> 4) * 4)) + i1_inner)]), 0.000000e+00f);
      compute[((((((((((((int)blockIdx.x) / 49) * 100352) + ((((int)threadIdx.x) >> 4) * 12544)) + (i1_inner * 3136)) + (((((int)blockIdx.x) % 49) / 7) * 448)) + (((((int)threadIdx.x) & 15) >> 3) * 112)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 224)] = max((conv2d_nchw[(((i1_inner * 2) + i2_inner) + 8)] + bias[((((((int)blockIdx.x) / 49) * 32) + ((((int)threadIdx.x) >> 4) * 4)) + i1_inner)]), 0.000000e+00f);
    }
  }
}


