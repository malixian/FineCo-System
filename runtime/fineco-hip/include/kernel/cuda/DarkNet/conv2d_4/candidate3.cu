#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[32];
  __shared__ float pad_temp_shared[2048];
  __shared__ float kernel_shared[2048];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    conv2d_nchw[(ff_outer_inner_init * 4)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 8)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 16)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 24)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 2)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 10)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 18)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 26)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 1)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 9)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 17)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 25)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 3)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 11)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 19)] = 0.000000e+00f;
    conv2d_nchw[((ff_outer_inner_init * 4) + 27)] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))];
    pad_temp_shared[(((int)threadIdx.x) + 128)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 6272)];
    pad_temp_shared[(((int)threadIdx.x) + 256)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 12544)];
    pad_temp_shared[(((int)threadIdx.x) + 384)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 18816)];
    pad_temp_shared[(((int)threadIdx.x) + 512)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 25088)];
    pad_temp_shared[(((int)threadIdx.x) + 640)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 31360)];
    pad_temp_shared[(((int)threadIdx.x) + 768)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 37632)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 43904)];
    pad_temp_shared[(((int)threadIdx.x) + 1024)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 50176)];
    pad_temp_shared[(((int)threadIdx.x) + 1152)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 56448)];
    pad_temp_shared[(((int)threadIdx.x) + 1280)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 62720)];
    pad_temp_shared[(((int)threadIdx.x) + 1408)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 68992)];
    pad_temp_shared[(((int)threadIdx.x) + 1536)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 75264)];
    pad_temp_shared[(((int)threadIdx.x) + 1664)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 81536)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 87808)];
    pad_temp_shared[(((int)threadIdx.x) + 1920)] = data[(((((((rc_outer_outer * 100352) + ((((int)threadIdx.x) >> 6) * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (((((int)threadIdx.x) & 63) >> 3) * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 94080)];
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 512)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1024)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 1536)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2560)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3072)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 3584)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4608)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5120)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 5632)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6656)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[(((((((int)threadIdx.x) >> 5) * 128) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 7680)];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
          for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
            conv2d_nchw[((ff_outer_inner * 4) + yy_outer_inner)] = (conv2d_nchw[((ff_outer_inner * 4) + yy_outer_inner)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 64)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 4)) + rc_inner)]));
            conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 8)] = (conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 256) + (rc_inner * 64)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 4)) + rc_inner)]));
            conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 16)] = (conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 16)] + (pad_temp_shared[(((((rc_outer_inner * 256) + (rc_inner * 64)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 4)) + rc_inner)]));
            conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 24)] = (conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 24)] + (pad_temp_shared[(((((rc_outer_inner * 256) + (rc_inner * 64)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[(((((((int)threadIdx.x) >> 3) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 4)) + rc_inner)]));
            conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 2)] = (conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 2)] + (pad_temp_shared[((((rc_outer_inner * 256) + (rc_inner * 64)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7))] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 4)) + rc_inner) + 32)]));
            conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 10)] = (conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 256) + (rc_inner * 64)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7)) + 16)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 4)) + rc_inner) + 32)]));
            conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 18)] = (conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 18)] + (pad_temp_shared[(((((rc_outer_inner * 256) + (rc_inner * 64)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7)) + 32)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 4)) + rc_inner) + 32)]));
            conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 26)] = (conv2d_nchw[(((ff_outer_inner * 4) + yy_outer_inner) + 26)] + (pad_temp_shared[(((((rc_outer_inner * 256) + (rc_inner * 64)) + (yy_outer_inner * 8)) + (((int)threadIdx.x) & 7)) + 48)] * kernel_shared[((((((((int)threadIdx.x) >> 3) * 128) + (ff_outer_inner * 64)) + (rc_outer_inner * 4)) + rc_inner) + 32)]));
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[(((((((((int)threadIdx.x) >> 3) * 12544) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7))] = max((conv2d_nchw[((i1_inner * 2) + i2_inner)] + bias[(((((int)threadIdx.x) >> 3) * 4) + i1_inner)]), 0.000000e+00f);
      compute[((((((((((int)threadIdx.x) >> 3) * 12544) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 112)] = max((conv2d_nchw[(((i1_inner * 2) + i2_inner) + 8)] + bias[(((((int)threadIdx.x) >> 3) * 4) + i1_inner)]), 0.000000e+00f);
      compute[((((((((((int)threadIdx.x) >> 3) * 12544) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 224)] = max((conv2d_nchw[(((i1_inner * 2) + i2_inner) + 16)] + bias[(((((int)threadIdx.x) >> 3) * 4) + i1_inner)]), 0.000000e+00f);
      compute[((((((((((int)threadIdx.x) >> 3) * 12544) + (i1_inner * 3136)) + ((((int)blockIdx.x) / 7) * 448)) + (i2_inner * 56)) + ((((int)blockIdx.x) % 7) * 8)) + (((int)threadIdx.x) & 7)) + 336)] = max((conv2d_nchw[(((i1_inner * 2) + i2_inner) + 24)] + bias[(((((int)threadIdx.x) >> 3) * 4) + i1_inner)]), 0.000000e+00f);
    }
  }
}


