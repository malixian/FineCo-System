#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[4];
  __shared__ float pad_temp_shared[896];
  __shared__ float kernel_shared[512];
  for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
      conv2d_nchw[((ff_inner_init * 2) + yy_inner_init)] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      *(float4*)(pad_temp_shared + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4))) = *(float4*)(data + (((((rc_outer_outer * 6272) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3136)) + ((((int)threadIdx.x) / 7) * 196)) + ((((int)blockIdx.x) % 7) * 28)) + ((((int)threadIdx.x) % 7) * 4)));
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) >> 4)) < 32) {
        kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 112) + ((int)threadIdx.x))] = kernel[(((((((int)blockIdx.x) / 7) * 8192) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) >> 4)) >> 1) * 512)) + (rc_outer_outer * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 112) + ((int)threadIdx.x)) & 31))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
      for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
        for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
          conv2d_nchw[((ff_inner * 2) + yy_inner)] = (conv2d_nchw[((ff_inner * 2) + yy_inner)] + (pad_temp_shared[(((rc_outer_inner * 28) + (yy_inner * 14)) + (((int)threadIdx.x) % 14))] * kernel_shared[((((((int)threadIdx.x) / 14) * 64) + (ff_inner * 32)) + rc_outer_inner)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[(((((((((int)blockIdx.x) / 7) * 3136) + ((((int)threadIdx.x) / 14) * 392)) + (i1_inner * 196)) + ((((int)blockIdx.x) % 7) * 28)) + (i2_inner * 14)) + (((int)threadIdx.x) % 14))] = max((conv2d_nchw[((i1_inner * 2) + i2_inner)] + bias[((((((int)blockIdx.x) / 7) * 16) + ((((int)threadIdx.x) / 14) * 2)) + i1_inner)]), 0.000000e+00f);
    }
  }
}


