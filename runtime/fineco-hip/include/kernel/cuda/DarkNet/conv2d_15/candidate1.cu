#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[14];
  __shared__ float pad_temp_shared[6272];
  __shared__ float kernel_shared[4096];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = data[((rc_outer_outer * 6272) + ((int)threadIdx.x))];
    pad_temp_shared[(((int)threadIdx.x) + 112)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 112)];
    pad_temp_shared[(((int)threadIdx.x) + 224)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 224)];
    pad_temp_shared[(((int)threadIdx.x) + 336)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 336)];
    pad_temp_shared[(((int)threadIdx.x) + 448)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 448)];
    pad_temp_shared[(((int)threadIdx.x) + 560)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 560)];
    pad_temp_shared[(((int)threadIdx.x) + 672)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 672)];
    pad_temp_shared[(((int)threadIdx.x) + 784)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 784)];
    pad_temp_shared[(((int)threadIdx.x) + 896)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 896)];
    pad_temp_shared[(((int)threadIdx.x) + 1008)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1008)];
    pad_temp_shared[(((int)threadIdx.x) + 1120)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1120)];
    pad_temp_shared[(((int)threadIdx.x) + 1232)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1232)];
    pad_temp_shared[(((int)threadIdx.x) + 1344)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1344)];
    pad_temp_shared[(((int)threadIdx.x) + 1456)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1456)];
    pad_temp_shared[(((int)threadIdx.x) + 1568)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1568)];
    pad_temp_shared[(((int)threadIdx.x) + 1680)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1680)];
    pad_temp_shared[(((int)threadIdx.x) + 1792)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1792)];
    pad_temp_shared[(((int)threadIdx.x) + 1904)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 1904)];
    pad_temp_shared[(((int)threadIdx.x) + 2016)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2016)];
    pad_temp_shared[(((int)threadIdx.x) + 2128)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2128)];
    pad_temp_shared[(((int)threadIdx.x) + 2240)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2240)];
    pad_temp_shared[(((int)threadIdx.x) + 2352)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2352)];
    pad_temp_shared[(((int)threadIdx.x) + 2464)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2464)];
    pad_temp_shared[(((int)threadIdx.x) + 2576)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2576)];
    pad_temp_shared[(((int)threadIdx.x) + 2688)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2688)];
    pad_temp_shared[(((int)threadIdx.x) + 2800)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2800)];
    pad_temp_shared[(((int)threadIdx.x) + 2912)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 2912)];
    pad_temp_shared[(((int)threadIdx.x) + 3024)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3024)];
    pad_temp_shared[(((int)threadIdx.x) + 3136)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3136)];
    pad_temp_shared[(((int)threadIdx.x) + 3248)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3248)];
    pad_temp_shared[(((int)threadIdx.x) + 3360)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3360)];
    pad_temp_shared[(((int)threadIdx.x) + 3472)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3472)];
    pad_temp_shared[(((int)threadIdx.x) + 3584)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3584)];
    pad_temp_shared[(((int)threadIdx.x) + 3696)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3696)];
    pad_temp_shared[(((int)threadIdx.x) + 3808)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3808)];
    pad_temp_shared[(((int)threadIdx.x) + 3920)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 3920)];
    pad_temp_shared[(((int)threadIdx.x) + 4032)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4032)];
    pad_temp_shared[(((int)threadIdx.x) + 4144)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4144)];
    pad_temp_shared[(((int)threadIdx.x) + 4256)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4256)];
    pad_temp_shared[(((int)threadIdx.x) + 4368)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4368)];
    pad_temp_shared[(((int)threadIdx.x) + 4480)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4480)];
    pad_temp_shared[(((int)threadIdx.x) + 4592)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4592)];
    pad_temp_shared[(((int)threadIdx.x) + 4704)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4704)];
    pad_temp_shared[(((int)threadIdx.x) + 4816)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4816)];
    pad_temp_shared[(((int)threadIdx.x) + 4928)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 4928)];
    pad_temp_shared[(((int)threadIdx.x) + 5040)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5040)];
    pad_temp_shared[(((int)threadIdx.x) + 5152)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5152)];
    pad_temp_shared[(((int)threadIdx.x) + 5264)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5264)];
    pad_temp_shared[(((int)threadIdx.x) + 5376)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5376)];
    pad_temp_shared[(((int)threadIdx.x) + 5488)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5488)];
    pad_temp_shared[(((int)threadIdx.x) + 5600)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5600)];
    pad_temp_shared[(((int)threadIdx.x) + 5712)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5712)];
    pad_temp_shared[(((int)threadIdx.x) + 5824)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5824)];
    pad_temp_shared[(((int)threadIdx.x) + 5936)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 5936)];
    pad_temp_shared[(((int)threadIdx.x) + 6048)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 6048)];
    pad_temp_shared[(((int)threadIdx.x) + 6160)] = data[(((rc_outer_outer * 6272) + ((int)threadIdx.x)) + 6160)];
    kernel_shared[((int)threadIdx.x)] = kernel[(((((int)blockIdx.x) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 112)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 112) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 224) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 336)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 336) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 448) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 560)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 560) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 672)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 672) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 784)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 784) >> 7) * 1024)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((((int)blockIdx.x) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 7168)];
    kernel_shared[(((int)threadIdx.x) + 1008)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1008) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1120)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1120) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1232)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1232) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1344)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1344) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1456)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1456) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1568)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1568) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 1680)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1680) >> 7) * 1024)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((((int)blockIdx.x) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 14336)];
    kernel_shared[(((int)threadIdx.x) + 1904)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 1904) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2016)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2016) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2128)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2128) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2240)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2240) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2352)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2352) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2464)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2464) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2576)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2576) >> 7) * 1024)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((((int)blockIdx.x) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 21504)];
    kernel_shared[(((int)threadIdx.x) + 2800)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2800) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 2912)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 2912) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3024)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3024) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3136)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3136) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 64) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3248)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3248) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 48) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3360)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3360) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 32) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3472)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3472) >> 7) * 1024)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 16))];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((((int)blockIdx.x) * 32768) + (rc_outer_outer * 128)) + ((int)threadIdx.x)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 3696)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3696) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 112) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3808)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3808) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 96) & 127))];
    kernel_shared[(((int)threadIdx.x) + 3920)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 3920) >> 7) * 1024)) + (rc_outer_outer * 128)) + ((((int)threadIdx.x) + 80) & 127))];
    if (((int)threadIdx.x) < 64) {
      kernel_shared[(((int)threadIdx.x) + 4032)] = kernel[((((((int)blockIdx.x) * 32768) + (((((int)threadIdx.x) + 4032) >> 7) * 1024)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) + 64))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[(((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner)] * kernel_shared[(((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32))]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[(((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2048)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 49)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 1)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 49)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2049)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 98)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2050)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 147)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 3)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 147)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2051)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 196)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 4)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 196)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2052)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 245)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 5)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 245)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2053)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 294)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 6)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 294)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2054)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 343)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 7)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 343)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2055)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 392)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 8)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 392)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2056)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 441)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 9)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 441)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2057)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 490)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 10)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 490)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2058)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 539)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 11)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 539)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2059)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 588)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 12)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 588)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2060)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 637)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 13)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 637)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2061)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 686)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 14)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 686)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2062)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 735)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 15)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 735)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2063)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 784)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 16)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 784)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2064)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 833)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 17)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 833)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2065)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 882)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 18)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 882)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2066)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 931)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 19)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 931)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2067)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 980)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 20)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 980)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2068)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1029)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 21)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1029)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2069)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1078)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 22)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1078)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2070)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1127)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 23)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1127)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2071)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1176)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 24)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1176)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2072)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1225)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 25)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1225)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2073)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1274)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 26)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1274)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2074)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1323)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 27)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1323)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2075)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1372)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 28)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1372)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2076)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1421)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 29)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1421)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2077)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1470)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 30)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1470)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2078)]));
        conv2d_nchw[xx_outer_inner] = (conv2d_nchw[xx_outer_inner] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1519)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 31)]));
        conv2d_nchw[(xx_outer_inner + 7)] = (conv2d_nchw[(xx_outer_inner + 7)] + (pad_temp_shared[((((rc_outer_inner * 1568) + ((((int)threadIdx.x) % 7) * 7)) + xx_outer_inner) + 1519)] * kernel_shared[((((((int)threadIdx.x) / 7) * 128) + (rc_outer_inner * 32)) + 2079)]));
      }
    }
  }
  for (int i3_inner = 0; i3_inner < 7; ++i3_inner) {
    compute[(((((int)blockIdx.x) * 1568) + (((int)threadIdx.x) * 7)) + i3_inner)] = max((conv2d_nchw[i3_inner] + bias[((((int)blockIdx.x) * 32) + (((int)threadIdx.x) / 7))]), 0.000000e+00f);
    compute[((((((int)blockIdx.x) * 1568) + (((int)threadIdx.x) * 7)) + i3_inner) + 784)] = max((conv2d_nchw[(i3_inner + 7)] + bias[(((((int)blockIdx.x) * 32) + (((int)threadIdx.x) / 7)) + 16)]), 0.000000e+00f);
  }
}


