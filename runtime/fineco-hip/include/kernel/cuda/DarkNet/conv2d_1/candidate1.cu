#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[224];
  __shared__ float pad_temp_shared[6156];
  __shared__ float kernel_shared[864];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 8; ++ff_inner_init) {
      conv2d_nchw[((ff_outer_inner_init * 56) + (ff_inner_init * 7))] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 112)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 1)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 113)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 2)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 114)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 3)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 115)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 4)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 116)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 5)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 117)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 6)] = 0.000000e+00f;
      conv2d_nchw[(((ff_outer_inner_init * 56) + (ff_inner_init * 7)) + 118)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 25; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 2)) < 1539) {
      pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x))] = (((((1 <= (((((int)blockIdx.x) / 14) * 112) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) >> 1)) % 1026) / 9))) && ((((((int)blockIdx.x) / 14) * 112) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) >> 1)) % 1026) / 9)) < 225)) && (1 <= (((((int)blockIdx.x) % 14) * 16) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x)) % 18)))) && ((((((int)blockIdx.x) % 14) * 16) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x)) % 18)) < 225)) ? data[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 2)) / 513) * 50176) + ((((int)blockIdx.x) / 14) * 25088)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) >> 1)) % 1026) / 9) * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 256) + ((int)threadIdx.x)) % 18)) - 225)] : 0.000000e+00f);
    }
  }
  kernel_shared[((int)threadIdx.x)] = kernel[((int)threadIdx.x)];
  kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((int)threadIdx.x) + 256)];
  kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((int)threadIdx.x) + 512)];
  if (((int)threadIdx.x) < 96) {
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((int)threadIdx.x) + 768)];
  }
  __syncthreads();
  for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
    for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
            conv2d_nchw[((ff_outer_inner * 56) + (ff_inner * 7))] = (conv2d_nchw[((ff_outer_inner * 56) + (ff_inner * 7))] + (pad_temp_shared[(((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15))] * kernel_shared[(((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 112)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 112)] + (pad_temp_shared[(((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15))] * kernel_shared[((((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 432)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 1)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 1)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 18)] * kernel_shared[(((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 113)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 113)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 18)] * kernel_shared[((((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 432)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 2)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 2)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 36)] * kernel_shared[(((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 114)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 114)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 36)] * kernel_shared[((((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 432)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 3)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 3)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 54)] * kernel_shared[(((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 115)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 115)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 54)] * kernel_shared[((((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 432)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 4)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 4)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 72)] * kernel_shared[(((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 116)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 116)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 72)] * kernel_shared[((((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 432)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 5)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 5)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 90)] * kernel_shared[(((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 117)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 117)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 90)] * kernel_shared[((((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 432)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 6)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 6)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 108)] * kernel_shared[(((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner)]));
            conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 118)] = (conv2d_nchw[(((ff_outer_inner * 56) + (ff_inner * 7)) + 118)] + (pad_temp_shared[((((((rc_inner * 2052) + ((((int)threadIdx.x) >> 4) * 126)) + (ry_inner * 18)) + rx_outer_inner) + (((int)threadIdx.x) & 15)) + 108)] * kernel_shared[((((((ff_outer_inner * 216) + (ff_inner * 27)) + (rc_inner * 9)) + (ry_inner * 3)) + rx_outer_inner) + 432)]));
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 16; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 7; ++i2_inner) {
      compute[((((((i1_inner * 50176) + ((((int)blockIdx.x) / 14) * 25088)) + ((((int)threadIdx.x) >> 4) * 1568)) + (i2_inner * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((int)threadIdx.x) & 15))] = max((conv2d_nchw[((i1_inner * 7) + i2_inner)] + bias[i1_inner]), 0.000000e+00f);
      compute[(((((((i1_inner * 50176) + ((((int)blockIdx.x) / 14) * 25088)) + ((((int)threadIdx.x) >> 4) * 1568)) + (i2_inner * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((int)threadIdx.x) & 15)) + 802816)] = max((conv2d_nchw[(((i1_inner * 7) + i2_inner) + 112)] + bias[(i1_inner + 16)]), 0.000000e+00f);
    }
  }
}


