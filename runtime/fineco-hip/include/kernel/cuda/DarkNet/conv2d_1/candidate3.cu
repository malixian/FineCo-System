#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) candidate3(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[128];
  __shared__ float pad_temp_shared[3468];
  __shared__ float kernel_shared[864];
  for (int ff_inner_init = 0; ff_inner_init < 4; ++ff_inner_init) {
    conv2d_nchw[(ff_inner_init * 16)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 64)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 1)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 65)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 2)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 66)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 3)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 67)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 4)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 68)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 5)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 69)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 6)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 70)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 7)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 71)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 8)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 72)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 9)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 73)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 10)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 74)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 11)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 75)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 12)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 76)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 13)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 77)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 14)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 78)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 15)] = 0.000000e+00f;
    conv2d_nchw[((ff_inner_init * 16) + 79)] = 0.000000e+00f;
  }
  pad_temp_shared[((int)threadIdx.x)] = ((((1 <= (((((int)blockIdx.x) / 7) * 32) + (((int)threadIdx.x) / 34))) && (1 <= (((((int)blockIdx.x) % 7) * 32) + (((int)threadIdx.x) % 34)))) && ((((((int)blockIdx.x) % 7) * 32) + (((int)threadIdx.x) % 34)) < 225)) ? data[((((((((int)blockIdx.x) / 7) * 7168) + ((((int)threadIdx.x) / 34) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + (((int)threadIdx.x) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 256)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 18) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 18) % 34)) < 225)) ? data[((((((((int)blockIdx.x) / 7) * 7168) + (((((int)threadIdx.x) + 256) / 34) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 18) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 512)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 2) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 2) % 34)) < 225)) ? data[((((((((int)blockIdx.x) / 7) * 7168) + (((((int)threadIdx.x) + 512) / 34) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 2) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 768)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 20) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 20) % 34)) < 225)) ? data[((((((((int)blockIdx.x) / 7) * 7168) + (((((int)threadIdx.x) + 768) / 34) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 20) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 1024)] = (((((1 <= (((((int)blockIdx.x) / 7) * 32) + ((((((int)threadIdx.x) >> 1) + 512) % 578) / 17))) && ((((((int)blockIdx.x) / 7) * 32) + ((((((int)threadIdx.x) >> 1) + 512) % 578) / 17)) < 225)) && (1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 4) % 34)))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 4) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 1024) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 512) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 4) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 1280)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 22) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 22) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 1280) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 62) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 22) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 1536)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 6) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 6) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 1536) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 190) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 6) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 1792)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 24) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 24) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 1792) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 318) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 24) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 2048)] = (((((((((int)blockIdx.x) / 7) * 32) + ((((((int)threadIdx.x) >> 1) + 446) % 578) / 17)) < 225) && (1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 8) % 34)))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 8) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 2048) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 446) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 8) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 2304)] = (((((1 <= (((((int)blockIdx.x) / 7) * 32) + ((((((int)threadIdx.x) >> 1) + 574) % 578) / 17))) && ((((((int)blockIdx.x) / 7) * 32) + ((((((int)threadIdx.x) >> 1) + 574) % 578) / 17)) < 225)) && (1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 26) % 34)))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 26) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 2304) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 574) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 26) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 2560)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 10) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 10) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 2560) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 124) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 10) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 2816)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 28) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 28) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 2816) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 252) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 28) % 34)) - 225)] : 0.000000e+00f);
  pad_temp_shared[(((int)threadIdx.x) + 3072)] = (((1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 12) % 34))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 12) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 3072) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 380) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 12) % 34)) - 225)] : 0.000000e+00f);
  if (((int)threadIdx.x) < 140) {
    pad_temp_shared[(((int)threadIdx.x) + 3328)] = (((((((((int)blockIdx.x) / 7) * 32) + ((((((int)threadIdx.x) >> 1) + 508) % 578) / 17)) < 225) && (1 <= (((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 30) % 34)))) && ((((((int)blockIdx.x) % 7) * 32) + ((((int)threadIdx.x) + 30) % 34)) < 225)) ? data[((((((((((int)threadIdx.x) + 3328) / 1156) * 50176) + ((((int)blockIdx.x) / 7) * 7168)) + (((((((int)threadIdx.x) >> 1) + 508) % 578) / 17) * 224)) + ((((int)blockIdx.x) % 7) * 32)) + ((((int)threadIdx.x) + 30) % 34)) - 225)] : 0.000000e+00f);
  }
  kernel_shared[((int)threadIdx.x)] = kernel[((int)threadIdx.x)];
  kernel_shared[(((int)threadIdx.x) + 256)] = kernel[(((int)threadIdx.x) + 256)];
  kernel_shared[(((int)threadIdx.x) + 512)] = kernel[(((int)threadIdx.x) + 512)];
  if (((int)threadIdx.x) < 96) {
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[(((int)threadIdx.x) + 768)];
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
    for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
      for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
        for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
          conv2d_nchw[(ff_inner * 16)] = (conv2d_nchw[(ff_inner * 16)] + (pad_temp_shared[((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31))] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 64)] = (conv2d_nchw[((ff_inner * 16) + 64)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 544)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 1)] = (conv2d_nchw[((ff_inner * 16) + 1)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 34)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 65)] = (conv2d_nchw[((ff_inner * 16) + 65)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 578)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 2)] = (conv2d_nchw[((ff_inner * 16) + 2)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 68)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 66)] = (conv2d_nchw[((ff_inner * 16) + 66)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 612)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 3)] = (conv2d_nchw[((ff_inner * 16) + 3)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 102)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 67)] = (conv2d_nchw[((ff_inner * 16) + 67)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 646)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 4)] = (conv2d_nchw[((ff_inner * 16) + 4)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 136)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 68)] = (conv2d_nchw[((ff_inner * 16) + 68)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 680)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 5)] = (conv2d_nchw[((ff_inner * 16) + 5)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 170)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 69)] = (conv2d_nchw[((ff_inner * 16) + 69)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 714)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 6)] = (conv2d_nchw[((ff_inner * 16) + 6)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 204)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 70)] = (conv2d_nchw[((ff_inner * 16) + 70)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 748)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 7)] = (conv2d_nchw[((ff_inner * 16) + 7)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 238)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 71)] = (conv2d_nchw[((ff_inner * 16) + 71)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 782)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 8)] = (conv2d_nchw[((ff_inner * 16) + 8)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 272)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 72)] = (conv2d_nchw[((ff_inner * 16) + 72)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 816)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 9)] = (conv2d_nchw[((ff_inner * 16) + 9)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 306)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 73)] = (conv2d_nchw[((ff_inner * 16) + 73)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 850)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 10)] = (conv2d_nchw[((ff_inner * 16) + 10)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 340)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 74)] = (conv2d_nchw[((ff_inner * 16) + 74)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 884)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 11)] = (conv2d_nchw[((ff_inner * 16) + 11)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 374)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 75)] = (conv2d_nchw[((ff_inner * 16) + 75)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 918)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 12)] = (conv2d_nchw[((ff_inner * 16) + 12)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 408)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 76)] = (conv2d_nchw[((ff_inner * 16) + 76)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 952)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 13)] = (conv2d_nchw[((ff_inner * 16) + 13)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 442)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 77)] = (conv2d_nchw[((ff_inner * 16) + 77)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 986)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 14)] = (conv2d_nchw[((ff_inner * 16) + 14)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 476)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 78)] = (conv2d_nchw[((ff_inner * 16) + 78)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 1020)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 15)] = (conv2d_nchw[((ff_inner * 16) + 15)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 510)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
          conv2d_nchw[((ff_inner * 16) + 79)] = (conv2d_nchw[((ff_inner * 16) + 79)] + (pad_temp_shared[(((((rc_outer_inner * 1156) + (ry_inner * 34)) + rx_inner) + (((int)threadIdx.x) & 31)) + 1054)] * kernel_shared[((((((((int)threadIdx.x) >> 5) * 108) + (ff_inner * 27)) + (rc_outer_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 16; ++i2_inner) {
      compute[(((((((((int)threadIdx.x) >> 5) * 200704) + (i1_inner * 50176)) + ((((int)blockIdx.x) / 7) * 7168)) + (i2_inner * 224)) + ((((int)blockIdx.x) % 7) * 32)) + (((int)threadIdx.x) & 31))] = max((conv2d_nchw[((i1_inner * 16) + i2_inner)] + bias[(((((int)threadIdx.x) >> 5) * 4) + i1_inner)]), 0.000000e+00f);
      compute[((((((((((int)threadIdx.x) >> 5) * 200704) + (i1_inner * 50176)) + ((((int)blockIdx.x) / 7) * 7168)) + (i2_inner * 224)) + ((((int)blockIdx.x) % 7) * 32)) + (((int)threadIdx.x) & 31)) + 3584)] = max((conv2d_nchw[(((i1_inner * 16) + i2_inner) + 64)] + bias[(((((int)threadIdx.x) >> 5) * 4) + i1_inner)]), 0.000000e+00f);
    }
  }
}


