#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) candidate2(float* __restrict__ I, float* __restrict__ W, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[1];
  __shared__ float I_shared[96];
  __shared__ float W_shared[3072];
  T_batch_matmul_NT_local[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 24) {
      *(float4*)(I_shared + (((int)threadIdx.x) * 4)) = *(float4*)(I + ((((((int)blockIdx.x) / 24) * 768) + (k_outer_outer * 96)) + (((int)threadIdx.x) * 4)));
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 96; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      W_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 32) + ((int)threadIdx.x))] = W[((((((((int)blockIdx.x) % 24) * 24576) + ((ax0_ax1_fused_ax2_fused_outer_outer / 3) * 768)) + (k_outer_outer * 96)) + ((ax0_ax1_fused_ax2_fused_outer_outer % 3) * 32)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 48; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        T_batch_matmul_NT_local[0] = (T_batch_matmul_NT_local[0] + (I_shared[((k_outer_inner * 2) + k_inner)] * W_shared[(((((int)threadIdx.x) * 96) + (k_outer_inner * 2)) + k_inner)]));
      }
    }
  }
  T_batch_matmul_NT[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = T_batch_matmul_NT_local[0];
}


