#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(5) candidate0(float* __restrict__ Q, float* __restrict__ K, float* __restrict__ T_divide) {
  float T_batch_matmul_NT[1];
  __shared__ float Q_shared[24];
  __shared__ float K_shared[120];
  T_batch_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    Q_shared[((int)threadIdx.x)] = Q[(((((int)blockIdx.x) * 768) + (k_outer_outer * 24)) + ((int)threadIdx.x))];
    Q_shared[(((int)threadIdx.x) + 5)] = Q[((((((int)blockIdx.x) * 768) + (k_outer_outer * 24)) + ((int)threadIdx.x)) + 5)];
    Q_shared[(((int)threadIdx.x) + 10)] = Q[((((((int)blockIdx.x) * 768) + (k_outer_outer * 24)) + ((int)threadIdx.x)) + 10)];
    Q_shared[(((int)threadIdx.x) + 15)] = Q[((((((int)blockIdx.x) * 768) + (k_outer_outer * 24)) + ((int)threadIdx.x)) + 15)];
    if (((int)threadIdx.x) < 4) {
      Q_shared[(((int)threadIdx.x) + 20)] = Q[((((((int)blockIdx.x) * 768) + (k_outer_outer * 24)) + ((int)threadIdx.x)) + 20)];
    }
    K_shared[((int)threadIdx.x)] = K[((k_outer_outer * 24) + ((int)threadIdx.x))];
    K_shared[(((int)threadIdx.x) + 5)] = K[(((k_outer_outer * 24) + ((int)threadIdx.x)) + 5)];
    K_shared[(((int)threadIdx.x) + 10)] = K[(((k_outer_outer * 24) + ((int)threadIdx.x)) + 10)];
    K_shared[(((int)threadIdx.x) + 15)] = K[(((k_outer_outer * 24) + ((int)threadIdx.x)) + 15)];
    K_shared[(((int)threadIdx.x) + 20)] = K[(((((((int)threadIdx.x) + 20) / 24) * 768) + (k_outer_outer * 24)) + ((((int)threadIdx.x) + 20) % 24))];
    K_shared[(((int)threadIdx.x) + 25)] = K[(((((((int)threadIdx.x) + 25) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 1))];
    K_shared[(((int)threadIdx.x) + 30)] = K[(((((((int)threadIdx.x) + 30) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 6))];
    K_shared[(((int)threadIdx.x) + 35)] = K[(((((((int)threadIdx.x) + 35) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 11))];
    K_shared[(((int)threadIdx.x) + 40)] = K[(((((((int)threadIdx.x) + 40) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 16))];
    K_shared[(((int)threadIdx.x) + 45)] = K[(((((((int)threadIdx.x) + 45) / 24) * 768) + (k_outer_outer * 24)) + ((((int)threadIdx.x) + 21) % 24))];
    K_shared[(((int)threadIdx.x) + 50)] = K[(((((((int)threadIdx.x) + 50) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 2))];
    K_shared[(((int)threadIdx.x) + 55)] = K[(((((((int)threadIdx.x) + 55) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 7))];
    K_shared[(((int)threadIdx.x) + 60)] = K[(((((((int)threadIdx.x) + 60) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 12))];
    K_shared[(((int)threadIdx.x) + 65)] = K[(((((((int)threadIdx.x) + 65) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 17))];
    K_shared[(((int)threadIdx.x) + 70)] = K[(((((((int)threadIdx.x) + 70) / 24) * 768) + (k_outer_outer * 24)) + ((((int)threadIdx.x) + 22) % 24))];
    K_shared[(((int)threadIdx.x) + 75)] = K[(((((((int)threadIdx.x) + 75) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 3))];
    K_shared[(((int)threadIdx.x) + 80)] = K[(((((((int)threadIdx.x) + 80) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 8))];
    K_shared[(((int)threadIdx.x) + 85)] = K[(((((((int)threadIdx.x) + 85) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 13))];
    K_shared[(((int)threadIdx.x) + 90)] = K[(((((((int)threadIdx.x) + 90) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 18))];
    K_shared[(((int)threadIdx.x) + 95)] = K[(((((((int)threadIdx.x) + 95) / 24) * 768) + (k_outer_outer * 24)) + ((((int)threadIdx.x) + 23) % 24))];
    K_shared[(((int)threadIdx.x) + 100)] = K[(((((((int)threadIdx.x) + 100) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 4))];
    K_shared[(((int)threadIdx.x) + 105)] = K[(((((((int)threadIdx.x) + 105) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 9))];
    K_shared[(((int)threadIdx.x) + 110)] = K[(((((((int)threadIdx.x) + 110) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 14))];
    K_shared[(((int)threadIdx.x) + 115)] = K[(((((((int)threadIdx.x) + 115) / 24) * 768) + (k_outer_outer * 24)) + (((int)threadIdx.x) + 19))];
    __syncthreads();
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[0] * K_shared[(((int)threadIdx.x) * 24)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[1] * K_shared[((((int)threadIdx.x) * 24) + 1)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[2] * K_shared[((((int)threadIdx.x) * 24) + 2)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[3] * K_shared[((((int)threadIdx.x) * 24) + 3)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[4] * K_shared[((((int)threadIdx.x) * 24) + 4)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[5] * K_shared[((((int)threadIdx.x) * 24) + 5)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[6] * K_shared[((((int)threadIdx.x) * 24) + 6)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[7] * K_shared[((((int)threadIdx.x) * 24) + 7)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[8] * K_shared[((((int)threadIdx.x) * 24) + 8)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[9] * K_shared[((((int)threadIdx.x) * 24) + 9)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[10] * K_shared[((((int)threadIdx.x) * 24) + 10)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[11] * K_shared[((((int)threadIdx.x) * 24) + 11)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[12] * K_shared[((((int)threadIdx.x) * 24) + 12)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[13] * K_shared[((((int)threadIdx.x) * 24) + 13)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[14] * K_shared[((((int)threadIdx.x) * 24) + 14)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[15] * K_shared[((((int)threadIdx.x) * 24) + 15)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[16] * K_shared[((((int)threadIdx.x) * 24) + 16)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[17] * K_shared[((((int)threadIdx.x) * 24) + 17)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[18] * K_shared[((((int)threadIdx.x) * 24) + 18)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[19] * K_shared[((((int)threadIdx.x) * 24) + 19)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[20] * K_shared[((((int)threadIdx.x) * 24) + 20)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[21] * K_shared[((((int)threadIdx.x) * 24) + 21)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[22] * K_shared[((((int)threadIdx.x) * 24) + 22)]));
    T_batch_matmul_NT[0] = (T_batch_matmul_NT[0] + (Q_shared[23] * K_shared[((((int)threadIdx.x) * 24) + 23)]));
  }
  T_divide[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))] = (T_batch_matmul_NT[0] * 1.613743e-02f);
}


