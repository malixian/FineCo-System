#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(156) candidate2(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[2];
  __shared__ float pad_temp_shared[1080];
  __shared__ float kernel_shared[5184];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[((int)threadIdx.x)] = (((((1 <= (((((int)threadIdx.x) % 45) / 15) + (((int)blockIdx.x) % 13))) && ((((((int)threadIdx.x) % 45) / 15) + (((int)blockIdx.x) % 13)) < 14)) && (1 <= (((int)threadIdx.x) % 15))) && ((((int)threadIdx.x) % 15) < 14)) ? data[((((((rc_outer_outer * 4056) + ((((int)threadIdx.x) / 45) * 169)) + (((((int)threadIdx.x) % 45) / 15) * 13)) + ((((int)blockIdx.x) % 13) * 13)) + (((int)threadIdx.x) % 15)) - 14)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 156)] = (((((1 <= (((((((int)threadIdx.x) / 3) + 7) % 15) / 5) + (((int)blockIdx.x) % 13))) && ((((((((int)threadIdx.x) / 3) + 7) % 15) / 5) + (((int)blockIdx.x) % 13)) < 14)) && (1 <= ((((int)threadIdx.x) + 6) % 15))) && (((((int)threadIdx.x) + 6) % 15) < 14)) ? data[((((((rc_outer_outer * 4056) + (((((int)threadIdx.x) + 156) / 45) * 169)) + (((((((int)threadIdx.x) / 3) + 7) % 15) / 5) * 13)) + ((((int)blockIdx.x) % 13) * 13)) + ((((int)threadIdx.x) + 6) % 15)) - 14)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 312)] = (((((1 <= (((((((int)threadIdx.x) / 3) + 14) % 15) / 5) + (((int)blockIdx.x) % 13))) && ((((((((int)threadIdx.x) / 3) + 14) % 15) / 5) + (((int)blockIdx.x) % 13)) < 14)) && (1 <= ((((int)threadIdx.x) + 12) % 15))) && (((((int)threadIdx.x) + 12) % 15) < 14)) ? data[((((((rc_outer_outer * 4056) + (((((int)threadIdx.x) + 312) / 45) * 169)) + (((((((int)threadIdx.x) / 3) + 14) % 15) / 5) * 13)) + ((((int)blockIdx.x) % 13) * 13)) + ((((int)threadIdx.x) + 12) % 15)) - 14)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 468)] = (((((1 <= (((((((int)threadIdx.x) / 3) + 6) % 15) / 5) + (((int)blockIdx.x) % 13))) && ((((((((int)threadIdx.x) / 3) + 6) % 15) / 5) + (((int)blockIdx.x) % 13)) < 14)) && (1 <= ((((int)threadIdx.x) + 3) % 15))) && (((((int)threadIdx.x) + 3) % 15) < 14)) ? data[((((((rc_outer_outer * 4056) + (((((int)threadIdx.x) + 468) / 45) * 169)) + (((((((int)threadIdx.x) / 3) + 6) % 15) / 5) * 13)) + ((((int)blockIdx.x) % 13) * 13)) + ((((int)threadIdx.x) + 3) % 15)) - 14)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 624)] = (((((1 <= (((((((int)threadIdx.x) / 3) + 13) % 15) / 5) + (((int)blockIdx.x) % 13))) && ((((((((int)threadIdx.x) / 3) + 13) % 15) / 5) + (((int)blockIdx.x) % 13)) < 14)) && (1 <= ((((int)threadIdx.x) + 9) % 15))) && (((((int)threadIdx.x) + 9) % 15) < 14)) ? data[((((((rc_outer_outer * 4056) + (((((int)threadIdx.x) + 624) / 45) * 169)) + (((((((int)threadIdx.x) / 3) + 13) % 15) / 5) * 13)) + ((((int)blockIdx.x) % 13) * 13)) + ((((int)threadIdx.x) + 9) % 15)) - 14)] : 0.000000e+00f);
    pad_temp_shared[(((int)threadIdx.x) + 780)] = (((((1 <= ((((int)blockIdx.x) % 13) + (((((int)threadIdx.x) / 15) + 1) % 3))) && (((((int)blockIdx.x) % 13) + (((((int)threadIdx.x) / 15) + 1) % 3)) < 14)) && (1 <= (((int)threadIdx.x) % 15))) && ((((int)threadIdx.x) % 15) < 14)) ? data[((((((rc_outer_outer * 4056) + (((((int)threadIdx.x) + 780) / 45) * 169)) + ((((int)blockIdx.x) % 13) * 13)) + ((((((int)threadIdx.x) / 15) + 1) % 3) * 13)) + (((int)threadIdx.x) % 15)) - 14)] : 0.000000e+00f);
    if (((int)threadIdx.x) < 144) {
      pad_temp_shared[(((int)threadIdx.x) + 936)] = (((((1 <= (((((((int)threadIdx.x) / 3) + 12) % 15) / 5) + (((int)blockIdx.x) % 13))) && ((((((((int)threadIdx.x) / 3) + 12) % 15) / 5) + (((int)blockIdx.x) % 13)) < 14)) && (1 <= ((((int)threadIdx.x) + 6) % 15))) && (((((int)threadIdx.x) + 6) % 15) < 14)) ? data[((((((rc_outer_outer * 4056) + (((((int)threadIdx.x) + 936) / 45) * 169)) + (((((((int)threadIdx.x) / 3) + 12) % 15) / 5) * 13)) + ((((int)blockIdx.x) % 13) * 13)) + ((((int)threadIdx.x) + 6) % 15)) - 14)] : 0.000000e+00f);
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((((int)blockIdx.x) / 13) * 41472) + (rc_outer_outer * 216)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 156)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 156) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 52) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 312)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 312) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 32) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 468)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 468) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 12) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 624)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 624) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 64) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 780)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 780) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 44) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 936)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 936) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 24) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 1092)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 1092) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 4) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 1248)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 1248) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 56) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 1404)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 1404) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 36) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 1560)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 1560) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 16) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 1716)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 1716) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 68) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 1872)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 1872) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 48) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 2028)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 2028) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 28) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 2184)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 2184) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 8) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 2340)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 2340) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 60) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 2496)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 2496) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 40) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 2652)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 2652) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 20) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 2808)] = kernel[(((((((int)blockIdx.x) / 13) * 41472) + (rc_outer_outer * 216)) + ((int)threadIdx.x)) + 22464)];
    kernel_shared[(((int)threadIdx.x) + 2964)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 2964) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 52) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 3120)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 3120) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 32) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 3276)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 3276) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 12) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 3432)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 3432) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 64) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 3588)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 3588) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 44) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 3744)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 3744) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 24) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 3900)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 3900) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 4) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 4056)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 4056) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 56) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 4212)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 4212) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 36) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 4368)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 4368) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 16) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 4524)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 4524) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 68) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 4680)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 4680) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 48) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 4836)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 4836) / 216) * 1728)) + (rc_outer_outer * 216)) + ((((((int)threadIdx.x) / 3) + 28) % 72) * 3)) + (((int)threadIdx.x) % 3))];
    kernel_shared[(((int)threadIdx.x) + 4992)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 4992) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 8) * 3)) + (((int)threadIdx.x) % 3))];
    if (((int)threadIdx.x) < 36) {
      kernel_shared[(((int)threadIdx.x) + 5148)] = kernel[((((((((int)blockIdx.x) / 13) * 41472) + (((((int)threadIdx.x) + 5148) / 216) * 1728)) + (rc_outer_outer * 216)) + (((((int)threadIdx.x) / 3) + 60) * 3)) + (((int)threadIdx.x) % 3))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 6; ++rc_outer_inner) {
      for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13))] * kernel_shared[((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 15)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 3)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 30)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 6)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 45)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 9)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 60)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 12)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 75)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 15)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 90)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 18)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 105)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 21)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 120)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 24)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 135)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 27)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 150)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 30)]));
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 165)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 33)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[(((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13))] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 216)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 15)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 219)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 30)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 222)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 45)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 225)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 60)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 228)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 75)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 231)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 90)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 234)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 105)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 237)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 120)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 240)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 135)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 243)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 150)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 246)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((rc_outer_inner * 180) + rx_outer_inner) + (((int)threadIdx.x) % 13)) + 165)] * kernel_shared[(((((((int)threadIdx.x) / 13) * 432) + (rc_outer_inner * 36)) + rx_outer_inner) + 249)]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    compute[((((((((int)blockIdx.x) / 13) * 4056) + ((((int)threadIdx.x) / 13) * 338)) + (i1_inner * 169)) + ((((int)blockIdx.x) % 13) * 13)) + (((int)threadIdx.x) % 13))] = max((conv2d_nchw[i1_inner] + bias[((((((int)blockIdx.x) / 13) * 24) + ((((int)threadIdx.x) / 13) * 2)) + i1_inner)]), 0.000000e+00f);
  }
}


