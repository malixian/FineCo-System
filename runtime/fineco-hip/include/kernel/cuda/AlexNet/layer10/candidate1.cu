#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) candidate1(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[1];
  __shared__ float data_shared[256];
  __shared__ float weight_shared[8192];
  T_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    *(float2*)(data_shared + (((int)threadIdx.x) * 2)) = *(float2*)(data + ((k_outer_outer * 256) + (((int)threadIdx.x) * 2)));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 64)) = *(float2*)(data + (((k_outer_outer * 256) + (((int)threadIdx.x) * 2)) + 64));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 128)) = *(float2*)(data + (((k_outer_outer * 256) + (((int)threadIdx.x) * 2)) + 128));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 192)) = *(float2*)(data + (((k_outer_outer * 256) + (((int)threadIdx.x) * 2)) + 192));
    *(float4*)(weight_shared + (((int)threadIdx.x) * 4)) = *(float4*)(weight + (((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 128)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 128));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 2048));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 384)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 2048));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 4096));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 640)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 4096));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 6144));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 896)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 6144));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 8192));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1152)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 8192));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 10240));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1408)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 10240));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 12288));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1664)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 12288));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 14336));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1920)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 14336));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 16384));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2176)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 16384));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2304)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 18432));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2432)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 18432));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 20480));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2688)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 20480));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2816)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 22528));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2944)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 22528));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 24576));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3200)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 24576));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3328)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 26624));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3456)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 26624));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 28672));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3712)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 28672));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3840)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 30720));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3968)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 30720));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4096)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 32768));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4224)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 32768));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4352)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 34816));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4480)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 34816));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4608)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 36864));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4736)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 36864));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4864)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 38912));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 4992)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 38912));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5120)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 40960));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5248)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 40960));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5376)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 43008));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5504)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 43008));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5632)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 45056));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5760)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 45056));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 5888)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 47104));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6016)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 47104));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6144)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 49152));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6272)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 49152));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6400)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 51200));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6528)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 51200));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6656)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 53248));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6784)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 53248));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 6912)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 55296));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7040)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 55296));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7168)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 57344));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7296)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 57344));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7424)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 59392));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7552)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 59392));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7680)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 61440));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7808)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 61440));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 7936)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 63488));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 8064)) = *(float4*)(weight + ((((((int)blockIdx.x) * 65536) + (k_outer_outer * 256)) + ((((int)threadIdx.x) * 4) + 128)) + 63488));
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[(k_outer_inner * 64)] * weight_shared[((((int)threadIdx.x) * 256) + (k_outer_inner * 64))]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 1)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 1)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 2)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 2)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 3)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 3)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 4)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 4)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 5)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 5)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 6)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 6)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 7)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 7)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 8)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 8)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 9)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 9)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 10)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 10)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 11)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 11)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 12)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 12)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 13)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 13)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 14)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 14)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 15)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 15)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 16)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 16)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 17)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 17)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 18)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 18)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 19)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 19)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 20)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 20)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 21)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 21)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 22)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 22)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 23)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 23)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 24)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 24)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 25)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 25)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 26)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 26)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 27)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 27)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 28)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 28)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 29)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 29)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 30)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 30)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 31)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 31)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 32)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 32)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 33)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 33)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 34)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 34)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 35)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 35)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 36)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 36)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 37)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 37)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 38)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 38)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 39)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 39)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 40)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 40)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 41)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 41)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 42)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 42)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 43)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 43)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 44)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 44)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 45)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 45)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 46)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 46)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 47)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 47)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 48)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 48)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 49)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 49)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 50)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 50)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 51)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 51)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 52)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 52)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 53)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 53)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 54)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 54)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 55)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 55)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 56)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 56)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 57)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 57)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 58)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 58)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 59)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 59)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 60)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 60)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 61)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 61)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 62)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 62)]));
      T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + 63)] * weight_shared[(((((int)threadIdx.x) * 256) + (k_outer_inner * 64)) + 63)]));
    }
  }
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}


