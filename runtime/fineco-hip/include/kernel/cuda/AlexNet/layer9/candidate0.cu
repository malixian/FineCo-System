#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(4) candidate0(float* __restrict__ data, float* __restrict__ weight, float* __restrict__ compute, float* __restrict__ bias) {
  float T_matmul_NT[1];
  __shared__ float data_shared[512];
  __shared__ float weight_shared[2048];
  T_matmul_NT[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 9; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(data_shared + ((ax0_ax1_fused_outer_outer * 16) + (((int)threadIdx.x) * 4))) = *(float4*)(data + (((k_outer_outer * 512) + (ax0_ax1_fused_outer_outer * 16)) + (((int)threadIdx.x) * 4)));
    }
    for (int ax0_ax1_fused_outer_outer1 = 0; ax0_ax1_fused_outer_outer1 < 128; ++ax0_ax1_fused_outer_outer1) {
      *(float4*)(weight_shared + ((ax0_ax1_fused_outer_outer1 * 16) + (((int)threadIdx.x) * 4))) = *(float4*)(weight + (((((((int)blockIdx.x) * 18432) + ((ax0_ax1_fused_outer_outer1 >> 5) * 4608)) + (k_outer_outer * 512)) + ((ax0_ax1_fused_outer_outer1 & 31) * 16)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 64; ++k_inner) {
        T_matmul_NT[0] = (T_matmul_NT[0] + (data_shared[((k_outer_inner * 64) + k_inner)] * weight_shared[(((((int)threadIdx.x) * 512) + (k_outer_inner * 64)) + k_inner)]));
      }
    }
  }
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (T_matmul_NT[0] + bias[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}


