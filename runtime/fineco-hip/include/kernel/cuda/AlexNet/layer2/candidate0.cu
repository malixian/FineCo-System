#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(54) candidate0(float* __restrict__ tensor, float* __restrict__ data) {
  tensor[((((int)blockIdx.x) * 54) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      tensor[((((int)blockIdx.x) * 54) + ((int)threadIdx.x))] = max(tensor[((((int)blockIdx.x) * 54) + ((int)threadIdx.x))], data[((((((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) / 27)) / 27) * 3025) + ((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) / 27)) % 27) * 110)) + (rv0 * 55)) + ((((int)threadIdx.x) % 27) * 2)) + rv1)]);
    }
  }
}
