#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(220) candidate1(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute, float* __restrict__ bias) {
  float conv2d_nchw[15];
  __shared__ float pad_temp_shared[6129];
  __shared__ float kernel_shared[1452];
  conv2d_nchw[0] = 0.000000e+00f;
  conv2d_nchw[5] = 0.000000e+00f;
  conv2d_nchw[10] = 0.000000e+00f;
  conv2d_nchw[1] = 0.000000e+00f;
  conv2d_nchw[6] = 0.000000e+00f;
  conv2d_nchw[11] = 0.000000e+00f;
  conv2d_nchw[2] = 0.000000e+00f;
  conv2d_nchw[7] = 0.000000e+00f;
  conv2d_nchw[12] = 0.000000e+00f;
  conv2d_nchw[3] = 0.000000e+00f;
  conv2d_nchw[8] = 0.000000e+00f;
  conv2d_nchw[13] = 0.000000e+00f;
  conv2d_nchw[4] = 0.000000e+00f;
  conv2d_nchw[9] = 0.000000e+00f;
  conv2d_nchw[14] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 3; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 28; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 220) + ((int)threadIdx.x)) < 6129) {
        pad_temp_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 220) + ((int)threadIdx.x))] = (((((1 <= (((((int)blockIdx.x) % 11) * 10) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 220) + ((int)threadIdx.x)) / 454))) && ((((((int)blockIdx.x) % 11) * 10) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 220) + ((int)threadIdx.x)) / 454)) < 113)) && (2 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 220) + ((int)threadIdx.x)) % 227))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 220) + ((int)threadIdx.x)) % 227) < 226)) ? data[(((((rc_outer_outer * 50176) + ((((int)blockIdx.x) % 11) * 4480)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 220) + ((int)threadIdx.x)) / 227) * 224)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 220) + ((int)threadIdx.x)) % 227)) - 450)] : 0.000000e+00f);
      }
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((((((int)blockIdx.x) / 11) * 4356) + ((((int)threadIdx.x) / 121) * 363)) + (rc_outer_outer * 121)) + (((int)threadIdx.x) % 121))];
    kernel_shared[(((int)threadIdx.x) + 220)] = kernel[((((((((int)blockIdx.x) / 11) * 4356) + (((((int)threadIdx.x) + 220) / 121) * 363)) + (rc_outer_outer * 121)) + ((((((int)threadIdx.x) / 11) + 9) % 11) * 11)) + (((int)threadIdx.x) % 11))];
    kernel_shared[(((int)threadIdx.x) + 440)] = kernel[((((((((int)blockIdx.x) / 11) * 4356) + (((((int)threadIdx.x) + 440) / 121) * 363)) + (rc_outer_outer * 121)) + ((((((int)threadIdx.x) / 11) + 7) % 11) * 11)) + (((int)threadIdx.x) % 11))];
    kernel_shared[(((int)threadIdx.x) + 660)] = kernel[((((((((int)blockIdx.x) / 11) * 4356) + (((((int)threadIdx.x) + 660) / 121) * 363)) + (rc_outer_outer * 121)) + ((((((int)threadIdx.x) / 11) + 5) % 11) * 11)) + (((int)threadIdx.x) % 11))];
    kernel_shared[(((int)threadIdx.x) + 880)] = kernel[((((((((int)blockIdx.x) / 11) * 4356) + (((((int)threadIdx.x) + 880) / 121) * 363)) + (rc_outer_outer * 121)) + ((((((int)threadIdx.x) / 11) + 3) % 11) * 11)) + (((int)threadIdx.x) % 11))];
    kernel_shared[(((int)threadIdx.x) + 1100)] = kernel[((((((((int)blockIdx.x) / 11) * 4356) + (((((int)threadIdx.x) + 1100) / 121) * 363)) + (rc_outer_outer * 121)) + ((((((int)threadIdx.x) / 11) + 1) % 11) * 11)) + (((int)threadIdx.x) % 11))];
    if (((int)threadIdx.x) < 132) {
      kernel_shared[(((int)threadIdx.x) + 1320)] = kernel[((((((((int)blockIdx.x) / 11) * 4356) + (((((int)threadIdx.x) + 1320) / 121) * 363)) + (rc_outer_outer * 121)) + ((((((int)threadIdx.x) / 11) + 10) % 11) * 11)) + (((int)threadIdx.x) % 11))];
    }
    __syncthreads();
    for (int rx_outer_inner = 0; rx_outer_inner < 11; ++rx_outer_inner) {
      for (int ry_inner = 0; ry_inner < 11; ++ry_inner) {
        conv2d_nchw[0] = (conv2d_nchw[0] + (pad_temp_shared[(((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner)] * kernel_shared[((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner)]));
        conv2d_nchw[5] = (conv2d_nchw[5] + (pad_temp_shared[(((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 484)]));
        conv2d_nchw[10] = (conv2d_nchw[10] + (pad_temp_shared[(((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 968)]));
        conv2d_nchw[1] = (conv2d_nchw[1] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 908)] * kernel_shared[((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner)]));
        conv2d_nchw[6] = (conv2d_nchw[6] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 908)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 484)]));
        conv2d_nchw[11] = (conv2d_nchw[11] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 908)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 968)]));
        conv2d_nchw[2] = (conv2d_nchw[2] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 1816)] * kernel_shared[((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner)]));
        conv2d_nchw[7] = (conv2d_nchw[7] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 1816)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 484)]));
        conv2d_nchw[12] = (conv2d_nchw[12] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 1816)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 968)]));
        conv2d_nchw[3] = (conv2d_nchw[3] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 2724)] * kernel_shared[((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner)]));
        conv2d_nchw[8] = (conv2d_nchw[8] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 2724)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 484)]));
        conv2d_nchw[13] = (conv2d_nchw[13] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 2724)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 968)]));
        conv2d_nchw[4] = (conv2d_nchw[4] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 3632)] * kernel_shared[((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner)]));
        conv2d_nchw[9] = (conv2d_nchw[9] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 3632)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 484)]));
        conv2d_nchw[14] = (conv2d_nchw[14] + (pad_temp_shared[((((ry_inner * 227) + ((((int)threadIdx.x) % 55) * 4)) + rx_outer_inner) + 3632)] * kernel_shared[(((((((int)threadIdx.x) / 55) * 121) + (ry_inner * 11)) + rx_outer_inner) + 968)]));
      }
    }
  }
  for (int i2_inner = 0; i2_inner < 5; ++i2_inner) {
    compute[((((((((int)blockIdx.x) / 11) * 36300) + ((((int)threadIdx.x) / 55) * 3025)) + ((((int)blockIdx.x) % 11) * 275)) + (i2_inner * 55)) + (((int)threadIdx.x) % 55))] = max((conv2d_nchw[i2_inner] + bias[(((((int)blockIdx.x) / 11) * 12) + (((int)threadIdx.x) / 55))]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 11) * 36300) + ((((int)threadIdx.x) / 55) * 3025)) + ((((int)blockIdx.x) % 11) * 275)) + (i2_inner * 55)) + (((int)threadIdx.x) % 55)) + 12100)] = max((conv2d_nchw[(i2_inner + 5)] + bias[((((((int)blockIdx.x) / 11) * 12) + (((int)threadIdx.x) / 55)) + 4)]), 0.000000e+00f);
    compute[(((((((((int)blockIdx.x) / 11) * 36300) + ((((int)threadIdx.x) / 55) * 3025)) + ((((int)blockIdx.x) % 11) * 275)) + (i2_inner * 55)) + (((int)threadIdx.x) % 55)) + 24200)] = max((conv2d_nchw[(i2_inner + 10)] + bias[((((((int)blockIdx.x) / 11) * 12) + (((int)threadIdx.x) / 55)) + 8)]), 0.000000e+00f);
  }
}


